
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <unistd.h>

__global__ void kernel(int * inout, const int N)
{
    int gid = threadIdx.x + blockIdx.x * blockDim.x;
    int gstride = gridDim.x * blockDim.x;

   for (; gid < N; gid+= gstride) inout[gid] *= 2;
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main(void)
{
    const int N = 2<<20, sz = N * sizeof(int);

    int * inputs, * outputs, * _inout;

    gpuErrchk( hipHostMalloc((void **)&inputs, sz, hipHostMallocDefault) );
    gpuErrchk( hipHostMalloc((void **)&outputs, sz, hipHostMallocDefault) );
    gpuErrchk( hipMalloc((void **)&_inout, sz) );

    for(int i=0; i<N; i++) { inputs[i] = i; outputs[i] = 0; }

    hipStream_t stream[2];
    for (int i = 0; i < 2; i++)
        gpuErrchk( hipStreamCreate(&stream[i]) );

    gpuErrchk( hipMemcpyAsync(_inout, inputs, sz, hipMemcpyHostToDevice, stream[1]) );

    kernel<<<128, 128, 0, stream[1]>>>(_inout, N);
    gpuErrchk(hipPeekAtLastError());

    gpuErrchk( hipMemcpyAsync(outputs, _inout, sz, hipMemcpyDeviceToHost, stream[1]) );

    for(int i = 0; i < 2; i++)
        gpuErrchk( hipStreamDestroy(stream[i]) );

    hipDeviceSynchronize();
    //sleep(1); // remove the sleep and see what happens....

    for(int i = 0; i < N; i++)
        assert( (2 * inputs[i]) == outputs[i] );

    hipDeviceReset();

    return 0;
}