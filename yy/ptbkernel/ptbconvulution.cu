#include <stdio.h>
#include <stdlib.h>
#include<hip/hip_runtime.h>

#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)
#define BLOCKX 9
#define BLOCKY 9
#define THREADX 9
#define THREADY 9
#define COREX 9
#define COREY 9
#define RESIZEBLOCKX 45
#define RESIZETHREADX 36
#define ITERATION ((BLOCKX*BLOCKY*THREADX*THREADY-1)/(RESIZEBLOCKX*RESIZETHREADX)+1)
#define LEFT (BLOCKX*BLOCKY*THREADX*THREADY - ITERATION*RESIZEBLOCKX*RESIZETHREADX)
#define CHECK(res) if(res!=hipSuccess){exit(-1);}
void check(hipError_t err)
    {
    const char * errorStr;
    errorStr = hipGetErrorString(err);
    //printf("checkCudaErrors()  error = %04d %s\n",err, errorStr);
    printf("checkCudaErrors()  error =  %s\n", errorStr);
    }
//this code is to test the ptb way yy suggests

#define GPU_RETURN_STATUS(cmd) \
{ \
    hipError_t result = cmd; \
    if (result != hipSuccess) { \
        std::cout << #cmd " error, return code:" << result << " | " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); \
    } \
}

// These are the inline versions for all of the SDK helper functions
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    const char *errorStr = NULL;
    errorStr = hipGetErrorString(err);
    fprintf(stderr,
            "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, "
            "line %i.\n",
            err, errorStr, file, line);
    exit(EXIT_FAILURE);
  }
}


__global__ void convolutionkernel(float** photo,float**** temp,float** convolutioncore,float** result) {
    //confirm the element
    int newx = blockIdx.x;
    int newy = blockIdx.y;

    //get the data based on the threadIdx.x and threadIdx.y
    int thx = threadIdx.x;
    int thy = threadIdx.y;

    //caculate(COREX * COREY thread respectively by each thread)

    temp[newy][newx][thy][thx] = photo[newy + thy][newx + thx] * convolutioncore[thy][thx];

    __syncthreads();

    //get the final result by one thread

    if (thx == 0 && thy == 0){
    for(int i = 0;i < COREY;i++){
        for(int j = 0;j < COREX;j++){
            result[newy][newx] +=temp[newy][newx][i][j];
            }
        }
    }

}

__global__ void resizeconvolutionkernel(float** photo,float**** temp,float** convolutioncore,float** result) {
    int oldx = blockIdx.x;
    //int oldy = blockIdx.y;

    //get the data based on the threadIdx.x and threadIdx.y
    int oldthx = threadIdx.x;
    //int oldthy = threadIdx.y;
    int index = oldx*RESIZETHREADX + oldthx;
    int newy = 0.0;
    int newx = 0.0;
    int thy = 0.0;
    int thx = 0.0;
    for(int i=0;i<ITERATION;i++)
    {
        if(i!=(ITERATION-1))
        {

            index = i*RESIZETHREADX*RESIZEBLOCKX +oldx*RESIZETHREADX + oldthx;
            newy = index/BLOCKX*COREX*COREY;
            newx = (index-newy*(BLOCKX*COREX*COREY))/COREX*COREY;
            thy = index - (newy*(BLOCKX*COREX*COREY) -newx*(COREX*COREY)/COREY);
            thx = index - (newy*(BLOCKX*COREX*COREY) -newx*(COREX*COREY) - thy*COREY);

            //caculate(COREX * COREY thread respectively by each thread)

            temp[newy][newx][thy][thx] = photo[newy + thy][newx + thx] * convolutioncore[thy][thx];

            __syncthreads();

            //get the final result by one thread

            if (thx == 0 && thy == 0){
            for(int i = 0;i < COREY;i++){
                for(int j = 0;j < COREX;j++){
                    result[newy][newx] +=temp[newy][newx][i][j];
                    }
                }
            }
        }
        else
        {
            if(index<LEFT){
                //index 3279  newy 4  newx 4
                index = i*RESIZETHREADX*RESIZEBLOCKX +oldx*RESIZETHREADX + oldthx;
                newy = index/BLOCKX*COREX*COREY;
                newx = (index-newy*(BLOCKX*COREX*COREY))/COREX*COREY;
                thy = (index - newy*(BLOCKX*COREX*COREY) -newx*(COREX*COREY))/COREY;
                thx = index - newy*(BLOCKX*COREX*COREY) -newx*(COREX*COREY) - thy*COREY;

                //caculate(COREX * COREY thread respectively by each thread)

                temp[newy][newx][thy][thx] = photo[newy + thy][newx + thx] * convolutioncore[thy][thx];

                __syncthreads();

                //get the final result by one thread

                if (thx == 0 && thy == 0){
                for(int i = 0;i < COREY;i++){
                    for(int j = 0;j < COREX;j++){
                        result[newy][newx] +=temp[newy][newx][i][j];
                        }
                    }
                }
            }
        }
    }
}




void run_kernel() {
    //device variable
    float **dphoto2 = NULL;
    float *dphoto1 = NULL;
    float **dconvolutioncore2 = NULL;
    float *dconvolutioncore1 = NULL;
    float **dresult2 = NULL;
    float *dresult1 = NULL;
    float ****dtemp4 = NULL;
    float ***dtemp3 = NULL;
    float **dtemp2 = NULL;
    float *dtemp1 = NULL;

    //Host variable
    float **hphoto2 = NULL;
    float *hphoto1 = NULL;
    float **hconvolutioncore2 = NULL;
    float *hconvolutioncore1 = NULL;
    float **hresult2 = NULL;
    float *hresult1 = NULL;
    /*
    float ****htemp4 = NULL;
    float ***htemp3 = NULL;
    float **htemp2 = NULL;
    float *htemp1 = NULL;
    */
    float*** htemp4[BLOCKY];
    float** htemp3[BLOCKY][BLOCKX];
    float* htemp2[BLOCKY][BLOCKX][COREY];
    float htemp1[BLOCKY][BLOCKX][COREY][COREX];

	hipError_t res;

    //test
    int ite = ITERATION;
    printf("Iteration:%d \n",ite);

    //manage dphoto
	res = hipMalloc((void**)(&dphoto2), (BLOCKY+COREY-1)*sizeof(float*));CHECK(res)
	res = hipMalloc((void**)(&dphoto1), (BLOCKY+COREY-1)*(BLOCKX+COREX-1)*sizeof(float));CHECK(res)
	printf("103 \n");
	hphoto2 = (float**)malloc((BLOCKY+COREY-1)*(BLOCKX+COREX-1)*sizeof(float*));
	hphoto1 = (float*)malloc((BLOCKY+COREY-1)*(BLOCKX+COREX-1)*sizeof(float));
	 for (int r = 0; r < (BLOCKY+COREY-1) ; r++)
	{
		hphoto2[r] = dphoto1 + r * (BLOCKX+COREX-1);
	}
	for (int r = 0; r < ((BLOCKY+COREY-1)*(BLOCKX+COREX-1)); r++)
	{
		hphoto1[r] = 2.0;
	}
	res = hipMemcpy((void*)(dphoto2), (void*)(hphoto2), (BLOCKY+COREY-1)*sizeof(float*), hipMemcpyHostToDevice);CHECK(res)
    res = hipMemcpy((void*)(dphoto1), (void*)(hphoto1), ((BLOCKY+COREY-1)*(BLOCKX+COREX-1))*sizeof(float), hipMemcpyHostToDevice);CHECK(res)
    printf("116 \n");
    //manage dconvolutioncore
	res = hipMalloc((void**)(&dconvolutioncore2), COREY*sizeof(float*));CHECK(res)
	res = hipMalloc((void**)(&dconvolutioncore1), COREY*COREX*sizeof(float));CHECK(res)
	printf("120 \n");
	hconvolutioncore2 = (float**)malloc(COREY*sizeof(float*));
	hconvolutioncore1 = (float*)malloc(COREY*COREX*sizeof(float));
	for (int r = 0; r < COREY; r++)
	{
		hconvolutioncore2[r] = dconvolutioncore1 + r * COREX;
	}
	for (int r = 0; r < COREY*COREX; r++)
	{
		hconvolutioncore1[r] = 3.0;
	}
	res = hipMemcpy((void*)(dconvolutioncore2), (void*)(hconvolutioncore2), COREY*sizeof(float*), hipMemcpyHostToDevice);CHECK(res)
    res = hipMemcpy((void*)(dconvolutioncore1), (void*)(hconvolutioncore1), COREY*COREX*sizeof(float), hipMemcpyHostToDevice);CHECK(res)

    //manage dresult
	res = hipMalloc((void**)(&dresult2), BLOCKY*sizeof(float*));CHECK(res)
	res = hipMalloc((void**)(&dresult1), BLOCKY*BLOCKX*sizeof(float));CHECK(res)
	hresult2 = (float**)malloc(BLOCKY*sizeof(float*));
	hresult1 = (float*)malloc(BLOCKY*BLOCKX*sizeof(float));
	 for (int r = 0; r < BLOCKY; r++)
	{
		hresult2[r] = dresult1 + r * BLOCKX;
	}
	for (int r = 0; r < BLOCKY*BLOCKX; r++)
	{
		hresult1[r] = 0.0;
	}
	res = hipMemcpy((void*)(dresult2), (void*)(hresult2), BLOCKY*sizeof(float*), hipMemcpyHostToDevice);CHECK(res)
    res = hipMemcpy((void*)(dresult1), (void*)(hresult1), BLOCKY*BLOCKX*sizeof(float), hipMemcpyHostToDevice);CHECK(res)
    printf("149 \n");

    //manage dtemp
	res = hipMalloc((void**)(&dtemp1), BLOCKY*BLOCKX*COREY*COREX*sizeof(float));CHECK(res)
	res = hipMalloc((void**)(&dtemp2), BLOCKY*BLOCKX*COREY*sizeof(float*));CHECK(res)
	res = hipMalloc((void**)(&dtemp3), BLOCKY*BLOCKX*sizeof(float**));CHECK(res)
	res = hipMalloc((void**)(&dtemp4), BLOCKY*sizeof(float***));CHECK(res)

    /*
	htemp1 = (float*)malloc(BLOCKY*BLOCKX*COREY*COREX*sizeof(float));
	htemp2 = (float**)malloc(BLOCKY*BLOCKX*COREY*sizeof(float*));
	htemp3 = (float***)malloc(BLOCKY*BLOCKX*sizeof(float**));
	htemp4 = (float****)malloc(BLOCKY*sizeof(float***));
    */

	for(int h=0;h<BLOCKY;h++)
	{
	    htemp4[h] = dtemp3 + h*BLOCKX;
	        for(int i=0;i<BLOCKX;i++)
	        {
	        htemp3[h][i] = dtemp2 + h*BLOCKX*COREY + i*COREY;
	            for(int j=0;j<COREY;j++){
	                htemp2[h][i][j] = dtemp1+ h*BLOCKX*COREY*COREX + i*COREY*COREX+j* COREX;
                }
            }
    }

	res = hipMemcpy((void*)(dtemp4), (void*)(htemp4), BLOCKY*sizeof(float***), hipMemcpyHostToDevice);CHECK(res)
	res = hipMemcpy((void*)(dtemp3), (void*)(htemp3), BLOCKY*BLOCKX*sizeof(float**), hipMemcpyHostToDevice);CHECK(res)
	res = hipMemcpy((void*)(dtemp2), (void*)(htemp2), BLOCKY*BLOCKX*COREY*sizeof(float*), hipMemcpyHostToDevice);CHECK(res)
	printf("179 \n");

	dim3 dimBlock(COREX,COREY);
	dim3 dimGrid(BLOCKX,BLOCKY);
    printf("183 \n");
    //convolutionkernel<<<dimGrid, dimBlock>>>(dphoto2,dtemp4,dconvolutioncore2,dresult2);
    resizeconvolutionkernel<<<RESIZEBLOCKX, RESIZETHREADX>>>(dphoto2,dtemp4,dconvolutioncore2,dresult2);
    printf("185 \n");
	res = hipMemcpy((void*)(hphoto1), (void*)(dresult1), BLOCKY*BLOCKX*sizeof(float), hipMemcpyDeviceToHost);
	//prinf（"err: %d \n",res);
	check(res);
    printf("189 \n");
	for (int r = 0; r < BLOCKY; r++)
	{
		printf("\ncolum %d ",r);
		for (int c = 0; c < BLOCKX; c++)
		{
			printf("%f ", hphoto1[r*BLOCKX+c]);
		}
	}
    printf("196 \n");
}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        printf("args num error! argc:%d", argc);
        exit(1);
    }
    int gpu_no = atoi(argv[1]);
    checkCudaErrors(hipSetDevice(gpu_no));
	run_kernel();
	return 0;
}

