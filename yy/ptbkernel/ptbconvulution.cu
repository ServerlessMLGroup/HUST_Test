#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)
#define BLOCKX 9
#define BLOCKY 9
#define THREADX 9
#define THREADY 9
#define COREX 9
#define COREY 9
#define RESIZEBLOCKX 20
#define RESIZETHREADX 30
#define ITERATION ((BLOCKX*BLOCKY*THREADX*THREADY-1)/(RESIZEBLOCKX*RESIZETHREADX)+1)
#define LEFT (BLOCKX*BLOCKY*THREADX*THREADY - (ITERATION-1)*RESIZEBLOCKX*RESIZETHREADX)
#define CHECK(res) if(res!=hipSuccess){exit(-1);}
void check(hipError_t err)
    {
    const char * errorStr;
    errorStr = hipGetErrorString(err);
    //printf("checkCudaErrors()  error = %04d %s\n",err, errorStr);
    printf("checkCudaErrors()  error =  %s\n", errorStr);
    }
//this code is to test the ptb way yy suggests

#define GPU_RETURN_STATUS(cmd) \
{ \
    hipError_t result = cmd; \
    if (result != hipSuccess) { \
        std::cout << #cmd " error, return code:" << result << " | " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); \
    } \
}

// These are the inline versions for all of the SDK helper functions
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    const char *errorStr = NULL;

    errorStr = hipGetErrorString(err);
    fprintf(stderr,
            "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, "
            "line %i.\n",
            err, errorStr, file, line);
    exit(EXIT_FAILURE);
  }
}

//diy concolution kernel
__global__ void convolutionkernel(float** photo,float**** temp,float** convolutioncore,float** result) {
    //confirm the element
    int newx = blockIdx.x;
    int newy = blockIdx.y;

    for(){
    //get the data based on the threadIdx.x and threadIdx.y
    int thx = threadIdx.x;
    int thy = threadIdx.y;

    //caculate(COREX * COREY thread respectively by each thread)
    temp[newy][newx][thy][thx] = photo[newy + thy][newx + thx] * convolutioncore[thy][thx];
    __syncthreads();

    //get the final result by one thread
    if (thx == 0 && thy == 0)
    for(int i = 0;i < COREY;i++){
        for(int j = 0;j < COREX;j++){
            result[newy][newx] +=temp[newy][newx][i][j];
            }
        }
    }
    }
}

__global__ void resizeconvolutionkernel(float** photo,float**** temp,float** convolutioncore,float** result) {
//preparation
    //get the blockidx and threadidx
    int oldx = blockIdx.x;
    int oldthx = threadIdx.x;

    //calculate the offset of the theread in the grid
    int offset = oldx*RESIZETHREADX + oldthx;

    //create index to get the virtual offset in the virtual grid fuether
    int index = offset;

    //new y ,new x ,thy ,thx represent the virtual blockidx.y blockidx.x threadx.y threadx.x
    int newy = 0;
    int newx = 0;
    int thy = 0;
    int thx = 0;

    for(int i=0;i<ITERATION;i++)
    {
        //the last time should be limited because (RESIZEBLOCKX*RESIZETHREADX)may not able to be
        //divided exactly by BLOCKX*BLOCKY*THREADX*THREADY
        if(i!=(ITERATION-1))
        {
            //get the new y ,new x ,thy ,thx
            index = i*RESIZETHREADX*RESIZEBLOCKX +offset;
            newy = index / (BLOCKX*COREX*COREY);
            newx = (index - newy * (BLOCKX*COREX*COREY))/(COREX*COREY);
            thy = (index - newy*(BLOCKX*COREX*COREY) -newx*(COREX*COREY))/COREX;
            thx = index - newy*(BLOCKX*COREX*COREY) -newx*(COREX*COREY) - thy*COREX;

            //caculate(COREX * COREY thread respectively by each thread)
            temp[newy][newx][thy][thx] = photo[newy + thy][newx + thx] * convolutioncore[thy][thx];
        }
        else
        {
            if(offset<LEFT){
                //should be limited within the index
                index = i*RESIZETHREADX*RESIZEBLOCKX +oldx*RESIZETHREADX + oldthx;
                newy = index/(BLOCKX*COREX*COREY);
                newx = (index-newy*(BLOCKX*COREX*COREY))/(COREX*COREY);
                thy = (index - newy*(BLOCKX*COREX*COREY) -newx*(COREX*COREY))/COREX;
                thx = index - newy*(BLOCKX*COREX*COREY) -newx*(COREX*COREY) - thy*COREY;
                //caculate(COREX * COREY thread respectively by each thread)
                temp[newy][newx][thy][thx] = photo[newy + thy][newx + thx] * convolutioncore[thy][thx];
            }
        }
    }

    //syncronize the threads
    __syncthreads();

    //get the result
    for(int i=0;i<ITERATION;i++)
    {
        if(i!=(ITERATION-1))
        {
            //get the final result by one thread
            index = i*RESIZETHREADX*RESIZEBLOCKX +oldx*RESIZETHREADX + oldthx;
            newy = index/(BLOCKX*COREX*COREY);
            newx = (index-newy*(BLOCKX*COREX*COREY))/(COREX*COREY);
            thy = (index - newy*(BLOCKX*COREX*COREY) -newx*(COREX*COREY))/COREX;
            thx = index - newy*(BLOCKX*COREX*COREY) -newx*(COREX*COREY) - thy*COREY;
            //get the final result by thread 0
            if (thx == 0 && thy == 0){
            for(int i = 0;i < COREY;i++){
                for(int j = 0;j < COREX;j++){
                    result[newy][newx] +=temp[newy][newx][i][j];
                    }
                }
            }
        }
        else
        {
            if(offset<LEFT){
                index = i*RESIZETHREADX*RESIZEBLOCKX +oldx*RESIZETHREADX + oldthx;
                newy = index/(BLOCKX*COREX*COREY);
                newx = (index-newy*(BLOCKX*COREX*COREY))/(COREX*COREY);
                thy = (index - newy*(BLOCKX*COREX*COREY) -newx*(COREX*COREY))/COREX;
                thx = index - newy*(BLOCKX*COREX*COREY) -newx*(COREX*COREY) - thy*COREY;
                //get the final result by one thread
                if (thx == 0 && thy == 0){
                for(int i = 0;i < COREY;i++){
                    for(int j = 0;j < COREX;j++){
                        result[newy][newx] +=temp[newy][newx][i][j];
                        }
                    }
                }
            }
        }
    }
}




void run_kernel() {
    //device variable  dphoto:the old matrix  dconvolutioncore:as the name
    //dtemp:four dimesional ,for each node in the photo, save the temporory result of concolution
    //dresult:save the result
    float **dphoto2 = NULL;
    float *dphoto1 = NULL;
    float **dconvolutioncore2 = NULL;
    float *dconvolutioncore1 = NULL;
    float **dresult2 = NULL;
    float *dresult1 = NULL;
    float ****dtemp4 = NULL;
    float ***dtemp3 = NULL;
    float **dtemp2 = NULL;
    float *dtemp1 = NULL;

    //Host variable ,just as above
    float **hphoto2 = NULL;
    float *hphoto1 = NULL;
    float **hconvolutioncore2 = NULL;
    float *hconvolutioncore1 = NULL;
    float **hresult2 = NULL;
    float *hresult1 = NULL;
    float*** htemp4[BLOCKY];
    float** htemp3[BLOCKY][BLOCKX];
    float* htemp2[BLOCKY][BLOCKX][COREY];
    float htemp1[BLOCKY][BLOCKX][COREY][COREX];

	hipError_t res;

    //test ITERATION and LEFT
    int ite = ITERATION;
    int lef = LEFT;
    printf("Iteration:%d \n",ite);
    printf("Left:%d \n",lef);

    //manage dphoto
	res = hipMalloc((void**)(&dphoto2), (BLOCKY+COREY-1)*sizeof(float*));CHECK(res)
	res = hipMalloc((void**)(&dphoto1), (BLOCKY+COREY-1)*(BLOCKX+COREX-1)*sizeof(float));CHECK(res)
	hphoto2 = (float**)malloc((BLOCKY+COREY-1)*(BLOCKX+COREX-1)*sizeof(float*));
	hphoto1 = (float*)malloc((BLOCKY+COREY-1)*(BLOCKX+COREX-1)*sizeof(float));
	//manage two dimesion array
	for (int r = 0; r < (BLOCKY+COREY-1) ; r++)
	{
		hphoto2[r] = dphoto1 + r * (BLOCKX+COREX-1);
	}
	//assign the data to the photo
	for (int r = 0; r < ((BLOCKY+COREY-1)*(BLOCKX+COREX-1)); r++)
	{
		hphoto1[r] = 2.0 ;
	}
	res = hipMemcpy((void*)(dphoto2), (void*)(hphoto2), (BLOCKY+COREY-1)*sizeof(float*), hipMemcpyHostToDevice);CHECK(res)
    res = hipMemcpy((void*)(dphoto1), (void*)(hphoto1), ((BLOCKY+COREY-1)*(BLOCKX+COREX-1))*sizeof(float), hipMemcpyHostToDevice);CHECK(res)

    //manage dconvolutioncore
	res = hipMalloc((void**)(&dconvolutioncore2), COREY*sizeof(float*));CHECK(res)
	res = hipMalloc((void**)(&dconvolutioncore1), COREY*COREX*sizeof(float));CHECK(res)
	hconvolutioncore2 = (float**)malloc(COREY*sizeof(float*));
	hconvolutioncore1 = (float*)malloc(COREY*COREX*sizeof(float));
	//manage two dimesion array
	for (int r = 0; r < COREY; r++)
	{
		hconvolutioncore2[r] = dconvolutioncore1 + r * COREX;
	}
	//assign the data to the core
	for (int r = 0; r < COREY*COREX; r++)
	{
		hconvolutioncore1[r] = 3.0;
	}
	res = hipMemcpy((void*)(dconvolutioncore2), (void*)(hconvolutioncore2), COREY*sizeof(float*), hipMemcpyHostToDevice);CHECK(res)
    res = hipMemcpy((void*)(dconvolutioncore1), (void*)(hconvolutioncore1), COREY*COREX*sizeof(float), hipMemcpyHostToDevice);CHECK(res)

    //manage dresult
	res = hipMalloc((void**)(&dresult2), BLOCKY*sizeof(float*));CHECK(res)
	res = hipMalloc((void**)(&dresult1), BLOCKY*BLOCKX*sizeof(float));CHECK(res)
	hresult2 = (float**)malloc(BLOCKY*sizeof(float*));
	hresult1 = (float*)malloc(BLOCKY*BLOCKX*sizeof(float));
	//manage two dimesion array
	for (int r = 0; r < BLOCKY; r++)
	{
		hresult2[r] = dresult1 + r * BLOCKX;
	}
	//assign the data to the result
	for (int r = 0; r < BLOCKY*BLOCKX; r++)
	{
		hresult1[r] = 0.0;
	}
	res = hipMemcpy((void*)(dresult2), (void*)(hresult2), BLOCKY*sizeof(float*), hipMemcpyHostToDevice);CHECK(res)
    res = hipMemcpy((void*)(dresult1), (void*)(hresult1), BLOCKY*BLOCKX*sizeof(float), hipMemcpyHostToDevice);CHECK(res)

    //manage dtemp
	res = hipMalloc((void**)(&dtemp1), BLOCKY*BLOCKX*COREY*COREX*sizeof(float));CHECK(res)
	res = hipMalloc((void**)(&dtemp2), BLOCKY*BLOCKX*COREY*sizeof(float*));CHECK(res)
	res = hipMalloc((void**)(&dtemp3), BLOCKY*BLOCKX*sizeof(float**));CHECK(res)
	res = hipMalloc((void**)(&dtemp4), BLOCKY*sizeof(float***));CHECK(res)
    //manage four dimesion array,malloc is fininshed in line 190
	for(int h=0;h<BLOCKY;h++)
	{
	    htemp4[h] = dtemp3 + h*BLOCKX;
	        for(int i=0;i<BLOCKX;i++)
	        {
	        htemp3[h][i] = dtemp2 + h*BLOCKX*COREY + i*COREY;
	            for(int j=0;j<COREY;j++){
	                htemp2[h][i][j] = dtemp1+ h*BLOCKX*COREY*COREX + i*COREY*COREX+j* COREX;
                }
            }
    }
	res = hipMemcpy((void*)(dtemp4), (void*)(htemp4), BLOCKY*sizeof(float***), hipMemcpyHostToDevice);CHECK(res)
	res = hipMemcpy((void*)(dtemp3), (void*)(htemp3), BLOCKY*BLOCKX*sizeof(float**), hipMemcpyHostToDevice);CHECK(res)
	res = hipMemcpy((void*)(dtemp2), (void*)(htemp2), BLOCKY*BLOCKX*COREY*sizeof(float*), hipMemcpyHostToDevice);CHECK(res)

    //normal way
	dim3 dimBlock(COREX,COREY);
	dim3 dimGrid(BLOCKX,BLOCKY);
    convolutionkernel<<<dimGrid, dimBlock>>>(dphoto2,dtemp4,dconvolutioncore2,dresult2);

    //ptb way
    //resizeconvolutionkernel<<<RESIZEBLOCKX, RESIZETHREADX>>>(dphoto2,dtemp4,dconvolutioncore2,dresult2);

    //Get the result
	res = hipMemcpy((void*)(hphoto1), (void*)(dresult1), BLOCKY*BLOCKX*sizeof(float), hipMemcpyDeviceToHost);
	check(res);
	for (int r = 0; r < BLOCKY; r++)
	{
		printf("\ncolum %d ",r);
		for (int c = 0; c < BLOCKX; c++)
		{
			printf("%f ", hphoto1[r*BLOCKX+c]);
		}
	}
    printf("\n");
}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        printf("args num error! argc:%d", argc);
        exit(1);
    }
    int gpu_no = atoi(argv[1]);
    checkCudaErrors(hipSetDevice(gpu_no));
	run_kernel();
	return 0;
}

