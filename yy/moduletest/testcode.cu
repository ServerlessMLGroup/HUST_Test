#include "hip/hip_runtime.h"
#include <iostream>
#include <thread>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "unistd.h"
#include <thread>
#include <mutex>
#include <random>
#include <ctime>
#include <time.h>
using namespace std;

__global__ void testkernel(float n1,float n2) {
    float n3;
    for(int i=0;i<100000;i++)
    {
    n3 =n1/n2;
    }
}



//yy add
//yy add
void thread1(hipCtx_t ctx)
{
   //1.cpu bundle
   /*
   cpu_set_t mask;
   CPU_ZERO(&mask);
   CPU_SET(16, &mask); //指定该线程使用的CPU
   if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0)
   {
           perror("pthread_setaffinity_np");
   }
   */

   //2.create new context?
   /*
   hipSetDevice(2);
   //here,maybe just hipSetDevice can make change
   hipCtx_t tempcont;
   hipDevice_t dev;

   err = hipCtxGetDevice(&dev);
   if(err)
   {
       std::cout<<"Can't get device, err" << err<<std::endl;
   }
   err = hipCtxCreate(&tempcont,hipDeviceScheduleYield,dev);
   if(err)
   {
       std::cout<<"Can't create Context, err" << err << std::endl;
   }
   hipCtxGetCurrent(pctx);
   std::cout<<"new context"<<*pctx<<std::endl;
   */

   //3.push old context and load
   int err;
   hipCtx_t* pctx;
   err=hipCtxPushCurrent(ctx);
   if(err){
   std::cout<<"Push Context ERR! "<<err<<std::endl;
   }
   hipCtxGetCurrent(pctx);
   std::cout<<"set context"<<*pctx<<std::endl;

   size_t now=0;
   size_t total=0;
   hipMemGetInfo(&now,&total);
   std::cout<<"1 Size before"<<now<<std::endl;
   hipModule_t mod1,mod2,mod3,mod4,mod5,mod6;
   hipModuleLoad(&mod1, "/home/wuhao/HUST_Test/yy/moduletest/temp1.ptx");
   hipModuleLoad(&mod2, "/home/wuhao/HUST_Test/yy/moduletest/temp2.ptx");
   hipModuleLoad(&mod3, "/home/wuhao/HUST_Test/yy/moduletest/temp3.ptx");
   hipModuleLoad(&mod4, "/home/wuhao/HUST_Test/yy/moduletest/temp4.ptx");
   hipModuleLoad(&mod5, "/home/wuhao/HUST_Test/yy/moduletest/temp5.ptx");
   hipModuleLoad(&mod6, "/home/wuhao/HUST_Test/yy/moduletest/temp6.ptx");
   hipMemGetInfo(&now,&total);
   std::cout<<"1 Size after"<<now<<std::endl;
}

void thread2(hipCtx_t ctx)
{
   //1.cpu bundle
   /*
   cpu_set_t mask;
   CPU_ZERO(&mask);
   CPU_SET(16, &mask); //指定该线程使用的CPU
   if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0)
   {
           perror("pthread_setaffinity_np");
   }
   */

   //2.create new context?
   /*
   hipSetDevice(2);
   //here,maybe just hipSetDevice can make change
   hipCtx_t tempcont;
   hipDevice_t dev;

   err = hipCtxGetDevice(&dev);
   if(err)
   {
       std::cout<<"Can't get device, err" << err<<std::endl;
   }
   err = hipCtxCreate(&tempcont,hipDeviceScheduleYield,dev);
   if(err)
   {
       std::cout<<"Can't create Context, err" << err << std::endl;
   }
   hipCtxGetCurrent(pctx);
   std::cout<<"new context"<<*pctx<<std::endl;
   */


   int err;
   hipCtx_t* pctx;
   err=hipCtxPushCurrent(ctx);
   if(err){
   std::cout<<"Push Context ERR! "<<err<<std::endl;
   }
   hipCtxGetCurrent(pctx);
   std::cout<<"set context"<<*pctx<<std::endl;

   size_t now=0;
   size_t total=0;
   hipMemGetInfo(&now,&total);
   std::cout<<"2 Size before"<<now<<std::endl;
   hipModule_t mod1,mod2,mod3,mod4,mod5,mod6;
   hipModuleLoad(&mod1, "/home/wuhao/HUST_Test/yy/moduletest/temp7.ptx");
   hipModuleLoad(&mod2, "/home/wuhao/HUST_Test/yy/moduletest/temp8.ptx");
   hipModuleLoad(&mod3, "/home/wuhao/HUST_Test/yy/moduletest/temm9.ptx");
   hipModuleLoad(&mod4, "/home/wuhao/HUST_Test/yy/moduletest/temp10.ptx");
   hipModuleLoad(&mod5, "/home/wuhao/HUST_Test/yy/moduletest/temp11.ptx");
   hipModuleLoad(&mod6, "/home/wuhao/HUST_Test/yy/moduletest/temp12.ptx");
   hipMemGetInfo(&now,&total);
   std::cout<<"2 Size after"<<now<<std::endl;
}

int main()
{
    hipInit(0);
    hipSetDevice(2);


    //1.create context
    hipCtx_t cont1;
    hipCtx_t cont2;
    hipDevice_t dev;
    int err;
    err = hipCtxGetDevice(&dev);
    if(err)
    {
        std::cout<<"Can't get device, err" << err<<std::endl;
        return 0;
    }
    err = hipCtxCreate(&cont1,hipDeviceScheduleYield,dev);
    if(err)
    {
        std::cout<<"Can't create Context, err" << err << std::endl;
        return 0;
    }
    err = hipCtxCreate(&cont2,hipDeviceScheduleYield,dev);
    if(err)
    {
        std::cout<<"Can't create Context, err" << err << std::endl;
        return 0;
    }

    //1.1 kernel?
    testkernel<<<20, 128>>>(1.0,2.0);

    //1.2 data transfer?
    hipStream_t firststream;
    hipStreamCreateWithFlags(&firststream,0);


    float* cpudata;
    hipDeviceptr_t* gpudata;
    size_t size = 5*1024*1024;
    hipMemAllocHost((void**)(&cpudata),size);
    //hipMalloc(gpudata, size);
    /*
    for(int i=0;i<5*1024*1024;i++)
    {
        cpudata[i]=1.0;
    }
    */
    //hipMemcpyHtoDAsync((hipDeviceptr_t)(*gpudata),cpudata,size,firststream);
    hipStreamSynchronize(firststream);


    //2.test in the mom thread
    size_t now=0;
    size_t total=0;
    hipMemGetInfo(&now,&total);
    std::cout<<"Size now"<<now<<std::endl;

    hipModule_t mod1,mod2,mod3,mod4,mod5,mod6;
    hipModuleLoad(&mod1, "/home/wuhao/HUST_Test/yy/moduletest/temp1.ptx");
    hipModuleLoad(&mod2, "/home/wuhao/HUST_Test/yy/moduletest/temp2.ptx");
    hipModuleLoad(&mod3, "/home/wuhao/HUST_Test/yy/moduletest/temp3.ptx");
    hipModuleLoad(&mod4, "/home/wuhao/HUST_Test/yy/moduletest/temp4.ptx");
    hipModuleLoad(&mod5, "/home/wuhao/HUST_Test/yy/moduletest/temp5.ptx");
    hipModuleLoad(&mod6, "/home/wuhao/HUST_Test/yy/moduletest/temp6.ptx");

    hipMemGetInfo(&now,&total);
    std::cout<<"Size now"<<now<<std::endl;

    //3.test in two child thread
    /*
    thread first=thread(thread1,cont1);
    thread second=thread(thread2,cont1);
    first.join();
    second.join();
    */

    return 0;
}
