#include "hip/hip_runtime.h"
#include <iostream>
#include <thread>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "unistd.h"
#include <thread>
#include <mutex>
#include <random>
#include <ctime>
#include <time.h>
using namespace std;

__global__ void testkernel(float n1,float n2) {
    float n3;
    for(int i=0;i<100000;i++)
    {
    n3 =n1/n2;
    }
}



//yy add
//yy add
void thread1(hipCtx_t ctx)
{
   //1.cpu bundle
   /*
   cpu_set_t mask;
   CPU_ZERO(&mask);
   CPU_SET(16, &mask); //指定该线程使用的CPU
   if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0)
   {
           perror("pthread_setaffinity_np");
   }
   */

   //2.create new context?
   /*
   hipSetDevice(2);
   //here,maybe just hipSetDevice can make change
   hipCtx_t tempcont;
   hipDevice_t dev;

   err = hipCtxGetDevice(&dev);
   if(err)
   {
       std::cout<<"Can't get device, err" << err<<std::endl;
   }
   err = hipCtxCreate(&tempcont,hipDeviceScheduleYield,dev);
   if(err)
   {
       std::cout<<"Can't create Context, err" << err << std::endl;
   }
   hipCtxGetCurrent(pctx);
   std::cout<<"new context"<<*pctx<<std::endl;
   */

   //3.push old context and load
   int err;
   hipCtx_t* pctx;
   err=hipCtxPushCurrent(ctx);
   if(err){
   std::cout<<"Push Context ERR! "<<err<<std::endl;
   }
   hipCtxGetCurrent(pctx);
   std::cout<<"set context"<<*pctx<<std::endl;

   size_t now=0;
   size_t total=0;
   hipMemGetInfo(&now,&total);
   std::cout<<"1 Size before"<<now<<std::endl;
   hipModule_t mod1,mod2,mod3,mod4,mod5,mod6;
   hipModuleLoad(&mod1, "/home/wuhao/HUST_Test/yy/moduletest/temp1.ptx");
   hipModuleLoad(&mod2, "/home/wuhao/HUST_Test/yy/moduletest/temp2.ptx");
   hipModuleLoad(&mod3, "/home/wuhao/HUST_Test/yy/moduletest/temp3.ptx");
   hipModuleLoad(&mod4, "/home/wuhao/HUST_Test/yy/moduletest/temp4.ptx");
   hipModuleLoad(&mod5, "/home/wuhao/HUST_Test/yy/moduletest/temp5.ptx");
   hipModuleLoad(&mod6, "/home/wuhao/HUST_Test/yy/moduletest/temp6.ptx");
   hipMemGetInfo(&now,&total);
   std::cout<<"1 Size after"<<now<<std::endl;
}

void thread2(hipCtx_t ctx)
{
   //1.cpu bundle
   /*
   cpu_set_t mask;
   CPU_ZERO(&mask);
   CPU_SET(16, &mask); //指定该线程使用的CPU
   if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0)
   {
           perror("pthread_setaffinity_np");
   }
   */

   //2.create new context?
   /*
   hipSetDevice(2);
   //here,maybe just hipSetDevice can make change
   hipCtx_t tempcont;
   hipDevice_t dev;

   err = hipCtxGetDevice(&dev);
   if(err)
   {
       std::cout<<"Can't get device, err" << err<<std::endl;
   }
   err = hipCtxCreate(&tempcont,hipDeviceScheduleYield,dev);
   if(err)
   {
       std::cout<<"Can't create Context, err" << err << std::endl;
   }
   hipCtxGetCurrent(pctx);
   std::cout<<"new context"<<*pctx<<std::endl;
   */


   int err;
   hipCtx_t* pctx;
   err=hipCtxPushCurrent(ctx);
   if(err){
   std::cout<<"Push Context ERR! "<<err<<std::endl;
   }
   hipCtxGetCurrent(pctx);
   std::cout<<"set context"<<*pctx<<std::endl;

   size_t now=0;
   size_t total=0;
   hipMemGetInfo(&now,&total);
   std::cout<<"2 Size before"<<now<<std::endl;
   hipModule_t mod1,mod2,mod3,mod4,mod5,mod6;
   hipModuleLoad(&mod1, "/home/wuhao/HUST_Test/yy/moduletest/temp7.ptx");
   hipModuleLoad(&mod2, "/home/wuhao/HUST_Test/yy/moduletest/temp8.ptx");
   hipModuleLoad(&mod3, "/home/wuhao/HUST_Test/yy/moduletest/temm9.ptx");
   hipModuleLoad(&mod4, "/home/wuhao/HUST_Test/yy/moduletest/temp10.ptx");
   hipModuleLoad(&mod5, "/home/wuhao/HUST_Test/yy/moduletest/temp11.ptx");
   hipModuleLoad(&mod6, "/home/wuhao/HUST_Test/yy/moduletest/temp12.ptx");
   hipMemGetInfo(&now,&total);
   std::cout<<"2 Size after"<<now<<std::endl;
}

int main()
{
    hipInit(0);
    hipSetDevice(1);


    //1.create context
    hipCtx_t cont1;
    hipCtx_t cont2;
    hipDevice_t dev;
    int err;
    err = hipCtxGetDevice(&dev);
    if(err)
    {
        std::cout<<"Can't get device, err" << err<<std::endl;
        return 0;
    }
    err = hipCtxCreate(&cont1,hipDeviceScheduleYield,dev);
    if(err)
    {
        std::cout<<"Can't create Context, err" << err << std::endl;
        return 0;
    }
    err = hipCtxCreate(&cont2,hipDeviceScheduleYield,dev);
    if(err)
    {
        std::cout<<"Can't create Context, err" << err << std::endl;
        return 0;
    }

    /*
    //1.1 kernel?
    testkernel<<<20, 128>>>(1.0,2.0);

    //1.2 data transfer?

    hipStream_t firststream;
    hipStreamCreateWithFlags(&firststream,0);


    float* cpudata;
    hipDeviceptr_t gpudata;
    size_t size = 5*1024*1024;
    hipMemAllocHost((void**)(&cpudata),size);
    hipMalloc((hipDeviceptr_t*)(&gpudata), size);


    for(int i=0;i<(5*1024*1024/4);i++)
    {
        //cpudata[i]=1.0;
    }


    hipMemcpyHtoDAsync((hipDeviceptr_t)(gpudata),cpudata,size,firststream);
    hipStreamSynchronize(firststream);
    */

    //2.test in the mom thread
    size_t now=0;
    size_t total=0;
    hipMemGetInfo(&now,&total);
    std::cout<<"Size now"<<now<<std::endl;

    hipModule_t mod1,mod2,mod3,mod4,mod5,mod6;
    hipModuleLoad(&mod1, "/home/wuhao/HUST_Test/yy/moduletest/temp1.ptx");
    /*
    hipModuleLoad(&mod2, "/home/wuhao/HUST_Test/yy/moduletest/temp2.ptx");
    hipModuleLoad(&mod3, "/home/wuhao/HUST_Test/yy/moduletest/temp3.ptx");
    hipModuleLoad(&mod4, "/home/wuhao/HUST_Test/yy/moduletest/temp4.ptx");
    hipModuleLoad(&mod5, "/home/wuhao/HUST_Test/yy/moduletest/temp5.ptx");
    hipModuleLoad(&mod6, "/home/wuhao/HUST_Test/yy/moduletest/temp6.ptx");
    */

    hipMemGetInfo(&now,&total);
    std::cout<<"Size now"<<now<<std::endl;

    // 3. load cuda kernels
    hipFunction_t kernel;
    int result=hipModuleGetFunction(&kernel, mod1, "fused_add_10_kernel0");
    std::cout<<"result "<<result<<std::endl;


    //yy add stream
    hipStream_t firststream;
    hipStreamCreateWithFlags(&firststream,0);


    size_t storage_size1 = 602112;
    hipDeviceptr_t device_ptr1;
    std::vector<char> temp1;
    temp1.resize(storage_size1, 0);
    hipMalloc((hipDeviceptr_t*)&device_ptr1, storage_size1);
    hipMemcpyHtoD(device_ptr1, temp1.data(), storage_size1);

    size_t storage_size2 = 602112;
    hipDeviceptr_t device_ptr2;
    std::vector<char> temp2;
    temp2.resize(storage_size2, 0);
    hipMalloc((hipDeviceptr_t*)&device_ptr2, storage_size2);
    hipMemcpyHtoD(device_ptr2, temp2.data(), storage_size2);

    size_t storage_size3 = 12;
    hipDeviceptr_t device_ptr3;
    std::vector<char> temp3;
    temp3.resize(storage_size3, 0);
    hipMalloc((hipDeviceptr_t*)&device_ptr3, storage_size3);
    hipMemcpyHtoD(device_ptr3, temp3.data(), storage_size3);

    std::vector<hipDeviceptr_t*> kernel_arg;
    kernel_arg.push_back(&device_ptr1);
    kernel_arg.push_back(&device_ptr2);
    kernel_arg.push_back(&device_ptr3);

    hipMemGetInfo(&now,&total);
    std::cout<<"Size now 3  "<<now<<std::endl;

    hipModuleLaunchKernel(kernel,
        147, 1, 1,
        1024, 1, 1,
        0, firststream, (void **)kernel_arg.data(), 0 // raw_args是json中指示的storage的下标
    );

    hipMemGetInfo(&now,&total);
    std::cout<<"Size now 4  "<<now<<std::endl;

    //4.test in two child thread
    /*
    thread first=thread(thread1,cont1);
    thread second=thread(thread2,cont1);
    first.join();
    second.join();
    */

    return 0;
}
