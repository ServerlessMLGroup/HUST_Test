#include "hip/hip_runtime.h"
#include <iostream>
#include <thread>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "unistd.h"
#include <thread>
#include <mutex>
#include <random>
#include <ctime>
#include <time.h>
using namespace std;

//yy add
//yy add
void thread1(hipCtx_t ctx)
{
   /*
   hipSetDevice(2);
   hipCtx_t tempcont;
   hipDevice_t dev;
   int err;
   err = hipCtxGetDevice(&dev);
   if(err)
   {
       std::cout<<"Can't get device, err" << err<<std::endl;
   }
   err = hipCtxCreate(&tempcont,hipDeviceScheduleYield,dev);
   if(err)
   {
       std::cout<<"Can't create Context, err" << err << std::endl;
   }
   */

   int err;
   hipModule_t mod;
   hipModuleLoad(&mod, "/home/wuhao/HUST_Test/djx/json2kernel/resource/resnet18.ptx");
   sleep(1);


   err=hipCtxPushCurrent(ctx);
   if(err){
   std::cout<<"Push Context ERR! "<<err<<std::endl;
   }

   hipModule_t mod1,mod2,mod3,mod4,mod5,mod6;
   hipModuleLoad(&mod1, "/home/wuhao/HUST_Test/djx/json2kernel/resource/resnet18.ptx");
   hipModuleLoad(&mod2, "/home/wuhao/HUST_Test/djx/json2kernel/resource/resnet18.ptx");
   hipModuleLoad(&mod3, "/home/wuhao/HUST_Test/djx/json2kernel/resource/resnet18.ptx");
   hipModuleLoad(&mod4, "/home/wuhao/HUST_Test/djx/json2kernel/resource/resnet18.ptx");
   hipModuleLoad(&mod5, "/home/wuhao/HUST_Test/djx/json2kernel/resource/resnet18.ptx");
   hipModuleLoad(&mod6, "/home/wuhao/HUST_Test/djx/json2kernel/resource/resnet18.ptx");
}
// add fininshed


int main()
{
    hipInit(0);
    hipSetDevice(2);
    //clock for collection

    //yy change
    hipCtx_t cont1;
    hipDevice_t dev;
    int err;
    err = hipCtxGetDevice(&dev);
    if(err)
    {
        std::cout<<"Can't get device, err" << err<<std::endl;
        return 0;
    }
    err = hipCtxCreate(&cont1,hipDeviceScheduleYield,dev);
    if(err)
    {
        std::cout<<"Can't create Context, err" << err << std::endl;
        return 0;
    }

    thread first=thread(thread1,cont1);
    //thread second=thread(thread1,cont1);
    first.join();
    //second.join();

    return 0;
}
