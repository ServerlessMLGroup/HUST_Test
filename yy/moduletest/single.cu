#include "hip/hip_runtime.h"
#include <iostream>
#include <thread>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "unistd.h"
#include <thread>
#include <mutex>
#include <random>
#include <ctime>
#include <time.h>
using namespace std;

//yy add
//yy add
void thread1(hipCtx_t ctx)
{
   hipCtx_t* pctx;
   int err;
   /*
   hipSetDevice(2);
   hipCtx_t tempcont;
   hipDevice_t dev;

   err = hipCtxGetDevice(&dev);
   if(err)
   {
       std::cout<<"Can't get device, err" << err<<std::endl;
   }
   err = hipCtxCreate(&tempcont,hipDeviceScheduleYield,dev);
   if(err)
   {
       std::cout<<"Can't create Context, err" << err << std::endl;
   }
   hipCtxGetCurrent(pctx);
   std::cout<<"new context"<<*pctx<<std::endl;
   */

   err=hipCtxPushCurrent(ctx);
   if(err){
   std::cout<<"Push Context ERR! "<<err<<std::endl;
   }
   //hipCtxGetCurrent(pctx);
   //std::cout<<"set context"<<*pctx<<std::endl;

   hipModule_t mod;
   hipModuleLoad(&mod, "/home/wuhao/HUST_Test/djx/json2kernel/resource/resnet18.ptx");
   sleep(1);

   hipModule_t mod1,mod2,mod3,mod4,mod5,mod6;
   hipModule_t mod7,mod8,mod9,mod10,mod11,mod12;
   hipModuleLoad(&mod1, "/home/wuhao/HUST_Test/yy/moduletest/temp1.ptx");
   hipModuleLoad(&mod2, "/home/wuhao/HUST_Test/yy/moduletest/temp2.ptx");
   hipModuleLoad(&mod3, "/home/wuhao/HUST_Test/yy/moduletest/temp3.ptx");
   hipModuleLoad(&mod4, "/home/wuhao/HUST_Test/yy/moduletest/temp4.ptx");
   hipModuleLoad(&mod5, "/home/wuhao/HUST_Test/yy/moduletest/temp5.ptx");
   hipModuleLoad(&mod6, "/home/wuhao/HUST_Test/yy/moduletest/temp6.ptx");
   hipModuleLoad(&mod7, "/home/wuhao/HUST_Test/yy/moduletest/temp7.ptx");
   hipModuleLoad(&mod8, "/home/wuhao/HUST_Test/yy/moduletest/temp8.ptx");
   hipModuleLoad(&mod9, "/home/wuhao/HUST_Test/yy/moduletest/temp9.ptx");
   hipModuleLoad(&mod10, "/home/wuhao/HUST_Test/yy/moduletest/temp10.ptx");
   hipModuleLoad(&mod11, "/home/wuhao/HUST_Test/yy/moduletest/temp11.ptx");
   hipModuleLoad(&mod12, "/home/wuhao/HUST_Test/yy/moduletest/temp12.ptx");
}
// add fininshed


int main()
{
    hipInit(0);
    hipSetDevice(2);
    //clock for collection

    //yy change
    hipCtx_t cont1;
    hipDevice_t dev;
    int err;
    err = hipCtxGetDevice(&dev);
    if(err)
    {
        std::cout<<"Can't get device, err" << err<<std::endl;
        return 0;
    }
    err = hipCtxCreate(&cont1,hipDeviceScheduleYield,dev);
    if(err)
    {
        std::cout<<"Can't create Context, err" << err << std::endl;
        return 0;
    }

    thread first=thread(thread1,cont1);
    //thread second=thread(thread1,cont1);
    first.join();
    //second.join();

    return 0;
}
