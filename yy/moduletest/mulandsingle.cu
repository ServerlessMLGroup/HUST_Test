#include "hip/hip_runtime.h"
#include <iostream>
#include <thread>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "unistd.h"
#include <thread>
#include <mutex>
#include <random>
#include <ctime>
#include <time.h>
using namespace std;

//yy add
//yy add
void thread1(hipCtx_t ctx)
{
    cpu_set_t mask;
    CPU_ZERO(&mask);
    CPU_SET(16, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0)
    {
            perror("pthread_setaffinity_np");
    }
   int err;
   hipCtx_t* pctx;

   err=hipCtxPushCurrent(ctx);
   if(err){
   std::cout<<"Push Context ERR! "<<err<<std::endl;
   }

   hipCtxGetCurrent(pctx);
   std::cout<<"set context"<<*pctx<<std::endl;

   size_t now;
   size_t total;
   hipMemGetInfo(&now,&total);
   std::cout<<"Size now"<<now<<std::endl;
   hipModule_t mod1,mod2,mod3,mod4,mod5,mod6;
   hipModuleLoad(&mod1, "/home/wuhao/HUST_Test/yy/moduletest/temp1.ptx");
   hipModuleLoad(&mod2, "/home/wuhao/HUST_Test/yy/moduletest/temp2.ptx");
   hipModuleLoad(&mod3, "/home/wuhao/HUST_Test/yy/moduletest/temp3.ptx");
   hipModuleLoad(&mod4, "/home/wuhao/HUST_Test/yy/moduletest/temp4.ptx");
   hipModuleLoad(&mod5, "/home/wuhao/HUST_Test/yy/moduletest/temp7.ptx");
   hipModuleLoad(&mod6, "/home/wuhao/HUST_Test/yy/moduletest/temp6.ptx");
   hipMemGetInfo(&now,&total);
   std::cout<<"Size now"<<now<<std::endl;
}

void thread2(hipCtx_t ctx)
{
   int err;
   hipCtx_t* pctx;

   err=hipCtxPushCurrent(ctx);
   if(err){
   std::cout<<"Push Context ERR! "<<err<<std::endl;
   }
   hipCtxGetCurrent(pctx);
   std::cout<<"set context"<<*pctx<<std::endl;

   hipModule_t mod1,mod2,mod3,mod4,mod5,mod6;
   hipModuleLoad(&mod1, "/home/wuhao/HUST_Test/yy/moduletest/temp7.ptx");
   hipModuleLoad(&mod2, "/home/wuhao/HUST_Test/yy/moduletest/temp8.ptx");
   hipModuleLoad(&mod3, "/home/wuhao/HUST_Test/yy/moduletest/temp9.ptx");
   hipModuleLoad(&mod4, "/home/wuhao/HUST_Test/yy/moduletest/temp10.ptx");
   hipModuleLoad(&mod5, "/home/wuhao/HUST_Test/yy/moduletest/temp11.ptx");
   hipModuleLoad(&mod6, "/home/wuhao/HUST_Test/yy/moduletest/temp12.ptx");
}

void thread3(hipCtx_t ctx)
{
   int err;
   hipCtx_t* pctx;

   err=hipCtxPushCurrent(ctx);
   if(err){
   std::cout<<"Push Context ERR! "<<err<<std::endl;
   }
   hipCtxGetCurrent(pctx);
   std::cout<<"set context"<<*pctx<<std::endl;

   hipModule_t mod1,mod2,mod3,mod4,mod5,mod6;
   hipModuleLoad(&mod1, "/home/wuhao/HUST_Test/yy/moduletest/temp7.ptx");
   hipModuleLoad(&mod2, "/home/wuhao/HUST_Test/yy/moduletest/temp8.ptx");
   hipModuleLoad(&mod3, "/home/wuhao/HUST_Test/yy/moduletest/temp9.ptx");
   hipModuleLoad(&mod4, "/home/wuhao/HUST_Test/yy/moduletest/temp10.ptx");
   hipModuleLoad(&mod5, "/home/wuhao/HUST_Test/yy/moduletest/temp11.ptx");
   hipModuleLoad(&mod6, "/home/wuhao/HUST_Test/yy/moduletest/temp12.ptx");
}



int main()
{
    hipInit(0);
    hipSetDevice(2);
    //clock for collection

    //yy change
    hipCtx_t cont1;
    hipDevice_t dev;
    int err;
    int temp;
    err = hipCtxGetDevice(&dev);
    if(err)
    {
        std::cout<<"Can't get device, err" << err<<std::endl;
        return 0;
    }
    err = hipCtxCreate(&cont1,hipDeviceScheduleYield,dev);
    if(err)
    {
        std::cout<<"Can't create Context, err" << err << std::endl;
        return 0;
    }

    thread first=thread(thread1,cont1);
    thread second=thread(thread2,cont1);
    first.join();
    second.join();

    thread third=thread(thread3,cont1);
    third.join();

    return 0;
}
