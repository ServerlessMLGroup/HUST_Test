#include "hip/hip_runtime.h"
#include <iostream>
#include <thread>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "unistd.h"
#include <thread>
#include <mutex>
#include <random>
#include <ctime>
#include <time.h>
using namespace std;

//yy add
//yy add
void thread1(hipCtx_t ctx)
{
    int err;
    err=hipCtxPushCurrent(ctx);
    if(err){
    std::cout<<"Push Context ERR! "<<err<<std::endl;
    }

   hipModule_t mod1,mod2,mod3,mod4,mod5,mod6;
   hipModuleLoad(&mod1, "/home/wuhao/HUST_Test/djx/json2kernel/resource/resnet18.ptx");
   hipModuleLoad(&mod2, "/home/wuhao/HUST_Test/djx/json2kernel/resource/resnet18.ptx");
   hipModuleLoad(&mod3, "/home/wuhao/HUST_Test/djx/json2kernel/resource/resnet18.ptx");
   hipModuleLoad(&mod4, "/home/wuhao/HUST_Test/djx/json2kernel/resource/resnet18.ptx");
   hipModuleLoad(&mod5, "/home/wuhao/HUST_Test/djx/json2kernel/resource/resnet18.ptx");
   hipModuleLoad(&mod6, "/home/wuhao/HUST_Test/djx/json2kernel/resource/resnet18.ptx");
}
// add fininshed


int main()
{
    hipInit(0);
    hipSetDevice(2);
    //clock for collection

    //yy change
    hipCtx_t cont1;
    hipDevice_t dev;
    int err;
    err = hipCtxGetDevice(&dev);
    if(err)
    {
        std::cout<<"Can't get device, err" << err<<std::endl;
        return 0;
    }
    err = hipCtxCreate(&cont1,hipDeviceScheduleYield,dev);
    if(err)
    {
        std::cout<<"Can't create Context, err" << err << std::endl;
        return 0;
    }

    thread first=thread(thread1,cont1);
    thread second=thread(thread1,cont1);
    first.join();
    second.join();

    thread third=thread(thread1,cont1);
    third.join();

    thread fourth=thread(thread1,cont1);
    thread fifth=thread(thread1,cont1);
    fourth.join();
    fifth.join();

    return 0;
}
