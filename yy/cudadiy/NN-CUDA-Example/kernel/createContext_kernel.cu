#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <iostream>
void crContext(int dev) {
	
    hipInit(0);
    // Get handle for device 0
    hipDevice_t cuDevice;
    hipDeviceGet(&cuDevice, dev);

    // Create context
    hipCtx_t cuContext;
    hipCtxCreate(&cuContext, 0, cuDevice);
}
