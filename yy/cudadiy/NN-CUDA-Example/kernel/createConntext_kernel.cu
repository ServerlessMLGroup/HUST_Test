void crContext(int dev) {
     // Get handle for device 0
    hipDevice_t cuDevice;
    hipDeviceGet(&cuDevice, dev);

    // Create context
    hipCtx_t cuContext;
    hipCtxCreate(&cuContext, 0, cuDevice);
}