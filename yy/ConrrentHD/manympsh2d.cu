#include "hip/hip_runtime.h"
#include <iostream>
#include <thread>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "unistd.h"
#include <thread>
#include <mutex>
#include <random>
#include <ctime>
#include <time.h>
#include <stdio.h>
using namespace std;

void thread1(hipCtx_t ctx,float* d_a,float* h_a,size_t size,int i)
{
    //set CPU
    clock_t start,finish;
    double time=0.0;
    /*
    cpu_set_t mask;
    CPU_ZERO(&mask);
    CPU_SET(16, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0) {
            perror("pthread_setaffinity_np");
    }
    */

    cout<<"one thread starts: "<<endl;
    int err;
    err=hipCtxPushCurrent(ctx);

    if(err){
    cout<<"Push Context ERR! "<<err<<endl;
    }
    for(int i=0;i < 10;i++)
    {
    start=clock();
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    finish=clock();
    time += (double)(finish-start)/CLOCKS_PER_SEC;
    }
    cout <<i<<" Timeuse: "<<time<<" (s)"<<endl;
}



int main()
{
    /*
    cpu_set_t mask;
    CPU_ZERO(&mask);
    CPU_SET(16, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0) {
            perror("pthread_setaffinity_np");
    }
    */


    hipInit(0);
    hipSetDevice(2);
    //clock for collection


    //data size
    int N = 209715200/2;
    size_t size = N * sizeof(float);

    //Context and memory
    cout<<"Create n contexts and their memory"<<endl;
    int err;
    hipCtx_t cont1,cont2,cont3,cont4,cont5,cont6,cont7,cont8,cont9,cont10;
    hipDevice_t dev;
    err = hipCtxGetDevice(&dev);
    if(err)
    {
        cout<<"Can't get device, err" << err<<endl;
        return 0;
    }

    err = hipCtxCreate(&cont1,hipDeviceScheduleYield,dev);
    if(err)
    {
        cout<<"Can't create Context, err" << err << endl;
        return 0;
    }
    float* d_A;
    hipMalloc(&d_A, size);
    float* h_A;
    //hipHostMalloc(&h_A, size);
    h_A = (float*)malloc(size);

    err = hipCtxCreate(&cont2,hipDeviceScheduleYield,dev);
    if(err)
    {
        cout<<"Can't create Context, err" << err << endl;
        return 0;
    }
    float* d_B;
    hipMalloc(&d_B, size);
    float* h_B;
    //hipHostMalloc(&h_B, size);
    h_B = (float*)malloc(size);

    err = hipCtxCreate(&cont3,hipDeviceScheduleYield,dev);
    if(err)
    {
        cout<<"Can't create Context, err" << err << endl;
        return 0;
    }
    float* d_C;
    hipMalloc(&d_C, size);
    float* h_C;
    //hipHostMalloc(&h_C, size);
    h_C = (float*)malloc(size);

    err = hipCtxCreate(&cont4,hipDeviceScheduleYield,dev);
    if(err)
    {
        cout<<"Can't create Context, err" << err << endl;
        return 0;
    }
    float* d_D;
    hipMalloc(&d_D, size);
    float* h_D;
    //hipHostMalloc(&h_D, size);
    h_D = (float*)malloc(size);

    err = hipCtxCreate(&cont5,hipDeviceScheduleYield,dev);
    if(err)
    {
        cout<<"Can't create Context, err" << err << endl;
        return 0;
    }
    float* d_E;
    hipMalloc(&d_E, size);
    float* h_E;
    //hipHostMalloc(&h_E, size);
    h_E = (float*)malloc(size);

    ///*
    err = hipCtxCreate(&cont6,hipDeviceScheduleYield,dev);
    if(err)
    {
        cout<<"Can't create Context, err" << err << endl;
        return 0;
    }
    float* d_F;
    hipMalloc(&d_F, size);
    float* h_F;
    //hipHostMalloc(&h_A, size);
    h_F = (float*)malloc(size);

    err = hipCtxCreate(&cont7,hipDeviceScheduleYield,dev);
    if(err)
    {
        cout<<"Can't create Context, err" << err << endl;
        return 0;
    }
    float* d_G;
    hipMalloc(&d_G, size);
    float* h_G;
    //hipHostMalloc(&h_B, size);
    h_G = (float*)malloc(size);

    err = hipCtxCreate(&cont8,hipDeviceScheduleYield,dev);
    if(err)
    {
        cout<<"Can't create Context, err" << err << endl;
        return 0;
    }
    float* d_H;
    hipMalloc(&d_H, size);
    float* h_H;
    //hipHostMalloc(&h_C, size);
    h_H = (float*)malloc(size);

    err = hipCtxCreate(&cont9,hipDeviceScheduleYield,dev);
    if(err)
    {
        cout<<"Can't create Context, err" << err << endl;
        return 0;
    }
    float* d_I;
    hipMalloc(&d_I, size);
    float* h_I;
    //hipHostMalloc(&h_D, size);
    h_I = (float*)malloc(size);

    err = hipCtxCreate(&cont10,hipDeviceScheduleYield,dev);
    if(err)
    {
        cout<<"Can't create Context, err" << err << endl;
        return 0;
    }
    float* d_J;
    hipMalloc(&d_J, size);
    float* h_J;
    //hipHostMalloc(&h_E, size);
    h_J = (float*)malloc(size);

    uniform_real_distribution<float> u(0,10);
    default_random_engine e(time(NULL));
    for(int i=0;i < N; ++i){
    *(h_A + i) = u(e);
	*(h_B + i) = u(e);
	*(h_C + i) = u(e);
	*(h_D + i) = u(e);
	*(h_E + i) = u(e);
	*(h_F + i) = u(e);
	*(h_G + i) = u(e);
	*(h_H + i) = u(e);
	*(h_I + i) = u(e);
	*(h_J + i) = u(e);
    }


    thread th1=thread(thread1,cont1,d_A,h_A,size,1);
    thread th2=thread(thread1,cont2,d_B,h_B,size,2);
    thread th3=thread(thread1,cont3,d_C,h_C,size,3);
    thread th4=thread(thread1,cont4,d_D,h_D,size,4);
    thread th5=thread(thread1,cont5,d_E,h_E,size,5);
    thread th6=thread(thread1,cont6,d_F,h_F,size,6);
    thread th7=thread(thread1,cont7,d_H,h_H,size,7);
    thread th8=thread(thread1,cont8,d_I,h_I,size,8);
    thread th9=thread(thread1,cont9,d_J,h_J,size,9);
    thread th10=thread(thread1,cont10,d_K,h_K,size,10);

    th1.join();
    th2.join();
    th3.join();
    th4.join();
    th5.join();
    th6.join();
    th7.join();
    th8.join();
    th9.join();
    th10.join();

    //Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_D);
    hipFree(d_E);
    hipFree(d_F);
    hipFree(d_G);
    hipFree(d_H);
    hipFree(d_I);
    hipFree(d_J);

    return 0;
}
