#include "hip/hip_runtime.h"
#include <iostream>
#include <thread>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "unistd.h"
#include <thread>
#include <mutex>
#include <random>
#include <ctime>
#include <time.h>
#include <stdio.h>
using namespace std;

void thread(hipCtx_t ctx,float* d_a,float* h_a,size_t size)
{
    //set CPU

    cpu_set_t mask;
    CPU_ZERO(&mask);
    CPU_SET(16, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0) {
            perror("pthread_setaffinity_np");
    }
    cout<<"one thread starts: "<<endl;
    int err;
    err=hipCtxPushCurrent(ctx);
    if(err){
    cout<<"Push Context ERR! "<<err<<endl;
    }
    for(int i=0;i < 10;i++)
    {
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    }
}

}

int main()
{
    hipInit(0);
    hipSetDevice(2);
    //clock for collection

    //data size
    int N = 209715200/20;
    size_t size = N * sizeof(float);

    //Context and memory
    cout<<"Create n contexts and their memory"<<endl;
    int err;
    hipCtx_t cont1,cont2,cont3,cont4,cont5,cont6,cont7,cont8,cont9,cont10;
    hipDevice_t dev;
    err = hipCtxGetDevice(&dev);
    if(err)
    {
        cout<<"Can't get device, err" << err<<endl;
        return 0;
    }

    err = hipCtxCreate(&cont1,hipDeviceScheduleYield,dev);
    if(err)
    {
        cout<<"Can't create Context, err" << err << endl;
        return 0;
    }
    float* d_A;
    hipMalloc(&d_A, size);
    float* h_A;
    hipHostMalloc(&h_A, size);

    err = hipCtxCreate(&cont2,hipDeviceScheduleYield,dev);
    if(err)
    {
        cout<<"Can't create Context, err" << err << endl;
        return 0;
    }
    float* d_B;
    hipMalloc(&d_B, size);
    float* h_B;
    hipHostMalloc(&h_B, size);

    err = hipCtxCreate(&cont3,hipDeviceScheduleYield,dev);
    if(err)
    {
        cout<<"Can't create Context, err" << err << endl;
        return 0;
    }
    float* d_C;
    hipMalloc(&d_C, size);
    float* h_C;
    hipHostMalloc(&h_C, size);

    err = hipCtxCreate(&cont4,hipDeviceScheduleYield,dev);
    if(err)
    {
        cout<<"Can't create Context, err" << err << endl;
        return 0;
    }
    float* d_D;
    hipMalloc(&d_D, size);
    float* h_D;
    hipHostMalloc(&h_D, size);

    err = hipCtxCreate(&cont5,hipDeviceScheduleYield,dev);
    if(err)
    {
        cout<<"Can't create Context, err" << err << endl;
        return 0;
    }
    float* d_E;
    hipMalloc(&d_E, size);
    float* h_E;
    hipHostMalloc(&h_E, size);

    uniform_real_distribution<float> u(0,10);
    default_random_engine e(time(NULL));
    for(int i=0;i < N; ++i){
    *(h_A + i) = u(e);
	*(h_B + i) = u(e);
	*(h_C + i) = u(e);
	*(h_D + i) = u(e);
	*(h_E + i) = u(e);
    }


    thread th1=thread(thread,cont1,d_A,h_A,size);
    thread th2=thread(thread,cont2,d_B,h_B,size);
    thread th3=thread(thread,cont3,d_C,h_C,size);
    thread th4=thread(thread,cont4,d_D,h_D,size);
    thread th5=thread(thread,cont5,d_E,h_E,size);

    th1.join();
    th2.join();
    th3.join();
    th4.join();
    th5.join();

    //Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_D);
    hipFree(d_E);

    return 0;
}
