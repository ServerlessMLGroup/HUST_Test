#include "hip/hip_runtime.h"
#include <iostream>
#include <thread>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "unistd.h"
#include <thread>
#include <mutex>
#include <random>
#include <ctime>
#include <time.h>
using namespace std;

//Mutex
mutex mtx1_1;
mutex mtx1_2;
mutex mtx2_1;
//clock_t
clock_t start1,finish1;
clock_t start1_2,finish1_2;
clock_t start2,finish2;
double singletime = 0.0;
double cotime1=0.0;
double cotime2=0.0;

void CUDART_CB thread1_1callback(void *data) {
    mtx1_1.lock();
    start1=clock();
}

void CUDART_CB thread1_2callback(void *data) {
    finish1=clock();
    singletime += (double)(finish1 - start1)/CLOCKS_PER_SEC;
    cout<<"This time single data transfer: "<<((double)(finish1-start1)/CLOCKS_PER_SEC)<<"(s)"<<endl;
}

void CUDART_CB thread1_3callback(void *data) {
    mtx2_1.unlock();
    mtx1_2.lock();
    start1_2=clock();
}

void CUDART_CB thread1_4callback(void *data) {
    finish1=clock();
    cotime1 += (double)(finish1_2-start1_2)/CLOCKS_PER_SEC;
    cout<<"This time cocurrent data transfer 1111: "<<((double)(finish1-start1)/CLOCKS_PER_SEC)<<"(s)"<<endl;
    mtx1_1.unlock();
}
void CUDART_CB thread1_5callback(void *data) {
    cout<<"single time: "<<singletime<<" s"<<endl;
    cout<<"cocurrent time1111: "<<cotime1<<" s"<<endl;
}

void CUDART_CB thread2_1callback(void *data) {
    mtx2_1.lock();
    start2=clock();
}

void CUDART_CB thread2_2callback(void *data) {
    finish2=clock();
    cotime2 += (double)(finish2-start2)/CLOCKS_PER_SEC;
    cout<<"This time cocurrent data transfer 2222: "<<((double)(finish2-start2)/CLOCKS_PER_SEC)<<"(s)"<<endl;
    mtx1_2.unlock();
}

void CUDART_CB thread2_3callback(void *data) {
    cout<<"cocurrent time2222: "<<cotime2<<" s"<<endl;
}

void thread1(hipStream_t stream,float* d_a,float* d_b,float* h_a,float* h_b,size_t size)
{
    hipHostFn_t fn1 = thread1_1callback;
    hipHostFn_t fn2 = thread1_2callback;
    hipHostFn_t fn3 = thread1_3callback;
    hipHostFn_t fn4 = thread1_4callback;
    hipHostFn_t fn5 = thread1_5callback;
    for(int i=0;i < 10;i++)
    {
    hipLaunchHostFunc(stream, fn1, 0);
    hipMemcpyAsync(d_a, h_a,size, hipMemcpyHostToDevice, stream);
    hipLaunchHostFunc(stream, fn2, 0);
    hipLaunchHostFunc(stream, fn3, 0);
    hipMemcpyAsync(d_b, h_b,size, hipMemcpyHostToDevice, stream);
    hipLaunchHostFunc(stream, fn4, 0);
    }
    //Should i add some code to exit the thread here?
    hipLaunchHostFunc(stream, fn5, 0);
}

void thread2(hipStream_t stream,float* d_c,float* h_c,size_t size)
{
    hipHostFn_t fn1 = thread2_1callback;
    hipHostFn_t fn2 = thread2_2callback;
    hipHostFn_t fn3 = thread2_3callback;
    for(int i=0;i < 10;i++)
    {
    hipLaunchHostFunc(stream, fn1, 0);
    hipMemcpyAsync(d_c, h_c,size, hipMemcpyHostToDevice, stream);
    hipLaunchHostFunc(stream, fn2, 0);
    }
    hipLaunchHostFunc(stream, fn3, 0);
}

int main()
{
    hipInit(0);
    hipSetDevice(2);
    //clock for collection

    //data size
    int N = 10485760;
    size_t size = N * sizeof(float);
    int err;

    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);


    cout<<"Allocate Host Memory"<<endl;
    // Allocate input vectors h_A and h_B in host memory
    float* h_A,h_B,h_C;
    hipHostMalloc(&h_A, size);
    hipHostMalloc(&h_B, size);
    hipHostMalloc(&h_C, size);

    uniform_real_distribution<float> u(0,10);
    default_random_engine e(time(NULL));
    for(int i=0;i < N; ++i){
    *(h_A + i) = u(e);
	*(h_B + i) = u(e);
	*(h_C + i) = u(e);
    }

    //Create Stream
    hipStream_t* firststream,secondstream;
    hipStreamCreate(firststream);
    hipStreamCreate(secondstream);

    //prepare
    mtx2_1.lock();
    thread first=thread(thread1,firststream,d_A,d_B,h_A,h_B,size);
    thread second=thread(thread2,secondstream,d_C,h_C,size);
    second.join();
    first.join();
    //Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
