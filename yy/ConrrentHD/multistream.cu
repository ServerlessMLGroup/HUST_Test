#include "hip/hip_runtime.h"
#include <iostream>
#include <thread>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "unistd.h"
#include <thread>
#include <mutex>
#include <random>
#include <ctime>
#include <time.h>
using namespace std;

//Mutex
mutex mtx1-1,mtx1-2,mtx2-1;
//clock_t
clock_t start1,finish1;
clock_t start2,finish2;
double singletime = 0.0;
double cotime1=0.0;
double cotime2=0.0;

void CUDART_CB thread1-1callback(void *data) {
    mtx1-1.lock();
    start1=clock();
}

void CUDART_CB thread1-2callback(void *data) {
    finish1=clock();
    singletime += (double)(finish1-start1)/CLOCKS_PER_SEC;
    cout<<"This time single data transfer: "<<((double)(finish1-start1)/CLOCKS_PER_SEC)<<"(s)"<<endl;
}

void CUDART_CB thread1-3callback(void *data) {
    mtx2-1.unlock();
    mtx1-2.lock();
    start1=clock();
}

void CUDART_CB thread1-4callback(void *data) {
    finish1=clock();
    cotime1 += (double)(finish1-start1)/CLOCKS_PER_SEC;
    cout<<"This time cocurrent data transfer 1111: "<<((double)(finish1-start1)/CLOCKS_PER_SEC)<<"(s)"<<endl;
    mtx1-1.unlock();
}
void CUDART_CB thread1-5callback(void *data) {
    cout<<"single time: "<<singletime<<" s"<<endl;
    cout<<"cocurrent time1111: "<<cotime1<<" s"<<endl;
}

void CUDART_CB thread2-1callback(void *data) {
    mtx2-1.lock();
    start2=clock();
}

void CUDART_CB thread2-2callback(void *data) {
    finish2=clock();
    cotime2 += (double)(finish1-start1)/CLOCKS_PER_SEC;
    cout<<"This time cocurrent data transfer 2222: "<<((double)(finish2-start2)/CLOCKS_PER_SEC)<<"(s)"<<endl;
    mtx1-2.unlock();
}

void CUDART_CB thread2-3callback(void *data) {
    cout<<"cocurrent time2222: "<<cotime2<<" s"<<endl;
}

void thread1(hipStream_t stream,float* d_a,float* d_b,float* h_a,float* h_b,size_t size)
{
    hipHostFn_t fn1 = thread1-1callback;
    hipHostFn_t fn2 = thread1-2callback;
    hipHostFn_t fn3 = thread1-3callback;
    hipHostFn_t fn4 = thread1-4callback;
    hipHostFn_t fn5 = thread1-5callback;
    for(int i=0;i < 10;i++)
    {
    hipLaunchHostFunc(stream, fn1, null);
    hipMemcpyAsync(d_a, h_a,size, hipMemcpyHostToDevice, stream);
    hipLaunchHostFunc(stream, fn2, null);
    hipLaunchHostFunc(stream, fn3, null);
    hipMemcpyAsync(d_b, h_b,size, hipMemcpyHostToDevice, stream);
    hipLaunchHostFunc(stream, fn4, null);
    }
    //Should i add some code to exit the thread here?
    hipLaunchHostFunc(stream, fn5, null);
}

void thread2(hipStream_t stream,float* d_c,float* h_c,size_t size)
{
    hipHostFn_t fn1 = thread2-1callback;
    hipHostFn_t fn2 = thread2-2callback;
    hipHostFn_t fn3 = thread2-3callback;
    for(int i=0;i < 10;i++)
    {
    hipLaunchHostFunc(stream, fn1, null);
    hipMemcpyAsync(d_c, h_c,size, hipMemcpyHostToDevice, stream);
    hipLaunchHostFunc(stream, fn2, null);
    }
    hipLaunchHostFunc(stream, fn3, null);
}

int main()
{
    hipInit(0);
    hipSetDevice(2);
    //clock for collection

    //data size
    int N = 10485760;
    size_t size = N * sizeof(float);
    int err;

    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);


    cout<<"Allocate Host Memory"<<endl;
    // Allocate input vectors h_A and h_B in host memory
    float* h_A,h_B,h_C;
    hipHostMalloc(&h_A, size);
    hipHostMalloc(&h_B, size);
    hipHostMalloc(&h_C, size);

    uniform_real_distribution<float> u(0,10);
    default_random_engine e(time(NULL));
    for(int i=0;i < N; ++i){
    *(h_A + i) = u(e);
	*(h_B + i) = u(e);
	*(h_C + i) = u(e);
    }

    //Create Stream
    hipStream_t* firststream,secondstream;
    hipStreamCreate(&firststream);
    hipStreamCreate(&secondstream);

    //prepare
    mtx2-1.lock();
    thread first=thread(thread1,firststream,d_A,d_B,h_A,h_B,size);
    thread second=thread(thread2,secondstream,d_C,h_C,size);
    second.join();
    first.join();
    //Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
