#include <iostream>
#include <pthread.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "unistd.h"
#include <thread>
#include <mutex>
#include <random>
#include <ctime>
#include <time.h>
#include <stdio.h>
#include<hip/hip_runtime.h>
using namespace std;
#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)

mutex workend2;
mutex workend1;
//diy thread

//void *thread1(void *dummy,void* d_A,void *h_A)
//
void thread1(hipCtx_t ctx,float* d_a,float* h_a,size_t size,int i)
{
    //set CPU
    /*
    cpu_set_t mask;
    CPU_ZERO(&mask);
    CPU_SET(16, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0)
    {
            perror("pthread_setaffinity_np");
    }
    */
    //set GPU
    //hipSetDevice(1);


    //yy change:huan yi ge wenjian hai yao gai makefile,wojiu yong zhe ge le
    //wo hui zai wo gaide mei yige difang jia shang zhushi yy
    //yy preparation

    hipEvent_t  start, stop;
    float time;
    hipEventCreateWithFlags(&start,0);
    hipEventCreateWithFlags(&stop,0);

    cout<<"one thread starts: "<<endl;
    int err;
    err=hipCtxPushCurrent(ctx);
    if(err){
    cout<<"Push Context ERR! "<<err<<endl;
    }

    hipStream_t tempstream;

    hipError_t cudaStatus;
    cudaStatus = hipStreamCreate(&tempstream);
    fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));

    /*
    int *flag = (int *)dummy;
    int *d_a = (int *)d_A;
    int *h_a = (int *)h_A;
    */

    if(i==1)
    {
    workend1.unlock();
    workend2.lock();
    }
    else
    {
    workend2.unlock();
    workend1.lock();
    }

    for(int i=1;i < 10;i++)
    {
    //hipEventRecord(start,0);
    hipMemcpyAsync(d_a, h_a,size, hipMemcpyHostToDevice, tempstream);
    //hipEventRecord(stop,0);
    //hipEventSynchronize(stop);
    //hipEventElapsedTime(&time, start, stop);
	//std::cout<< i <<" time: "<<1000*time<<" us"<<std::endl;
    }


    hipStreamSynchronize(tempstream);
}

/*
pthread_t ntid1;
pthread_t ntid2;
*/

int main()
{
    //preparation
    workend1.lock();
    workend2.lock();

    hipInit(0);
    hipSetDevice(1);
    /*
    cpu_set_t mask;
    CPU_ZERO(&mask);
    CPU_SET(16, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0) {
            perror("pthread_setaffinity_np");
    }
    */

    //Context
    cout<<"Create context"<<endl;
    int err;
    hipCtx_t cont1;
    hipDevice_t dev;
    err = hipCtxGetDevice(&dev);
    if(err)
    {
        cout<<"Can't get device, err" << err<<endl;
        return 0;
    }
    err = hipCtxCreate(&cont1,hipDeviceScheduleYield,dev);
    if(err)
    {
        cout<<"Can't create Context, err" << err << endl;
        return 0;
    }

    //40M data
    int N = 4*52428800/20;
    size_t size = N * sizeof(float);

    //allocate device variable(data)
    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);


    // Allocate input vectors h_A and h_B in host memory
    float* h_A;
    float* h_B;
    float* h_C;
    hipHostMalloc(&h_A, size);
    hipHostMalloc(&h_B, size);
    hipHostMalloc(&h_C, size);

    uniform_real_distribution<float> u(0,10);
    default_random_engine e(time(NULL));
    for(int i=0;i < N; ++i){
    /*
    *(h_A + i) = u(e);
	*(h_B + i) = u(e);
	*(h_C + i) = u(e);
    */
    *(h_A + i) = 1;
	*(h_B + i) = 1;
	*(h_C + i) = 1;
    }


    hipEvent_t  start, stop;
    float time;
    hipEventCreateWithFlags(&start,0);
    hipEventCreateWithFlags(&stop,0);

    for(int i=1;i < 10;i++)
    {
    hipEventRecord(start,0);
    hipMemcpyAsync(d_A, h_A,size, hipMemcpyHostToDevice, 0);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
	std::cout<< i <<" time: "<<1000*time<<" us"<<std::endl;
    }

    /*
    pthread_create(&ntid1, NULL, thread1, flag1,d_A,h_A);
    pthread_create(&ntid2, NULL, thread1, flag2,d_B,h_B);
    pthread_join(ntid1, NULL);
    pthread_join(ntid2, NULL);
    */

    //thread second=thread(thread1,cont1,d_B,h_B,size,1);
    //thread first=thread(thread1,cont1,d_A,h_A,size,2);
    //second.join();
    //first.join();

    //change,check whether the cudamemcpy works
    for(int i=0;i < N; ++i){
    /*
    *(h_A + i) = u(e);
	*(h_B + i) = u(e);
	*(h_C + i) = u(e);
    */
    *(h_A + i) = 0;
	*(h_B + i) = 0;
	*(h_C + i) = 0;
    }
    hipMemcpy(h_A, d_A,size, hipMemcpyDeviceToHost);
    for(int i=0;i < 10; ++i){
    //cout<<"now  data"<<*(h_A + i)<<endl;
    }


    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
.
    return 0;
}
