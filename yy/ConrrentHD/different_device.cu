#include "hip/hip_runtime.h"
#include <iostream>
#include <thread>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "unistd.h"
#include <thread>
#include <mutex>
#include <random>
#include <ctime>
#include <time.h>
using namespace std;

//Mutex
mutex mtx1,mtx2;

void thread1(hipCtx_t ctx,float* d_a,float* h_a,size_t size)
{
    clock_t start,finish;
    double singletime=0.0;
    double cotime=0.0;
    int err;
    err=hipCtxPushCurrent(ctx);
    if(err){
    cout<<"Push Context ERR! "<<err<<endl;
    }
    for(int i=0;i < 10;i++)
    {
    mtx2.unlock();
    mtx1.lock();
    start=clock();
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    finish=clock();
    cotime += (double)(finish-start)/CLOCKS_PER_SEC;
    }

    cout<<"device 3 time: "<<cotime<<" s"<<endl;
}

void thread2(hipCtx_t ctx,float* d_b,float* h_b,size_t size)
{
    clock_t start,finish;
    double singletime=0.0;
    int err;
    err=hipCtxPushCurrent(ctx);
    if(err){
    cout<<"Push Context ERR! "<<err<<endl;
    }
    for(int i=0;i < 10;i++)
    {
    mtx2.lock();
    start=clock();
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    finish=clock();
    singletime += (double)(finish-start)/CLOCKS_PER_SEC;
    mtx1.unlock();
    }
    cout<<"device 2 time: "<<singletime<<" s"<<endl;
}

int main()
{
    hipInit(0);
    hipSetDevice(3);
    //clock for collection

    //data size
    int N = 10485760;
    size_t size = N * sizeof(float);


    //device3
    cout<<"Create two context and their memory"<<endl;
    int err;
    hipCtx_t cont1,cont2;
    hipDevice_t dev;
    cout<<"Device 3: d_A"<<endl;
    err = hipCtxGetDevice(&dev);
    if(err)
    {
        cout<<"Can't get device, err" << err<<endl;
        return 0;
    }
    err = hipCtxCreate(&cont1,hipDeviceScheduleYield,dev);
    if(err)
    {
        cout<<"Can't create Context, err" << err << endl;
        return 0;
    }
    float* d_A;
    hipMalloc(&d_A, size);

    //device2
    hipSetDevice(2);
    err = hipCtxGetDevice(&dev);
    if(err)
    {
        cout<<"Can't get device, err" << err<<endl;
        return 0;
    }
    err = hipCtxCreate(&cont2,hipDeviceScheduleYield,dev);
    if(err)
    {
        cout<<"Can't create Context, err" << err<<endl;
        return 0;
    }
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);


    cout<<"Allocate Host Memory"<<endl;
    // Allocate input vectors h_A and h_B in host memory
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    uniform_real_distribution<float> u(0,10);
    default_random_engine e(time(NULL));
    for(int i=0;i < N; ++i){
    *(h_A + i) = u(e);
	*(h_B + i) = u(e);
	*(h_C + i) = u(e);
    }

    //prepare
    mtx2.lock();
    thread first=thread(thread1,cont1,d_A,h_A,size);
    thread second=thread(thread2,cont2,d_C,h_C,size);
    second.join();
    first.join();
    //Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
