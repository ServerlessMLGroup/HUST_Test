#include <iostream>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include <thread>
#include <random>
#include <ctime>
#include <time.h>

using namespace std;

enum class Unit{
    Byte, KB, MB, GB, TB, PB, EB
};


double convert(double size, Unit unit)
{
    double result = size;
    switch (unit)
    {
    case Unit::EB:
        result /= 1024;     // flow through
    case Unit::PB:
        result /= 1024;     // flow through
    case Unit::TB:
        result /= 1024;     // flow through
    case Unit::GB:
        result /= 1024;     // flow through
    case Unit::MB:
        result /= 1024;     // flow through
    case Unit::KB:
        result /= 1024;     // flow through
    case Unit::Byte:
        result /= 1;
    default:
        break;
    }
    return result;
}

void getMem() {
    size_t free, total;
    hipMemGetInfo(&free, &total);
    printf("Free mem = %.4f MB, Total = %.4f MB \n", convert(free, Unit::MB), convert(total, Unit::MB));
}

void getMembycu() {
    size_t free, total;
    int err = hipMemGetInfo(&free, &total);
    if (err) {
        cout<<"getMembycu error:"<<err<<endl;
    }
    else {
        printf("Free mem = %.4f MB, Total = %.4f MB \n", convert(free, Unit::MB), convert(total, Unit::MB)); 
    }
}


__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i]; 
}


int main()
{
    hipSetDevice(3);
    getMem();
    

    clock_t start,finish;
    double Times;
    
    cout<<"initialize variable at GPU"<<endl;
    int N = 10485760;
    size_t size = N * sizeof(float);

    cout<<"Allocate Host Memory"<<endl;
    // Allocate input vectors h_A and h_B in host memory
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    //Initialize input vectors
    uniform_real_distribution<float> u(0,10);
    default_random_engine e(time(NULL));
    for(int i=0;i < N; ++i){
        *(h_A + i) = u(e);
	*(h_B + i) = u(e);
	*(h_C + i) = u(e);
    }

    cout<<"Allocate Device Memory"<<endl;
    // Allocate vectors in device memory
    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);
    getMem();
  
    cout<<"MemCpy D2H"<<endl;
    start = clock();
    for(int i=0;i<1000;i++){
    hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
    }
    finish = clock();
    
    Times=(double)(finish-start)/CLOCKS_PER_SEC;
    cout<<"start time: "<<start<<endl;
    cout<<"finish time: "<<finish<<endl;
    cout<<"CLOCKS_PER_SEC: "<<CLOCKS_PER_SEC<<endl;
    cout<<"Time use(s): "<<Times<<endl;

    //Free Device Memory 
    hipFree(d_A);
    getMem();
    hipFree(d_B);
    getMem();
    hipFree(d_C);

    return 0;

}
