#include "hip/hip_runtime.h"
#include <iostream>
#include <thread>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "unistd.h"
#include <thread>
#include <mutex>
#include <random>
#include <ctime>
#include <time.h>
#include <stdio.h>
using namespace std;

//Mutex
mutex mtx1;
mutex mtx2;

void thread1(hipCtx_t ctx,float* d_a,float* d_b,float* h_a,float* h_b,size_t size)
{
    //set CPU

    cpu_set_t mask;
    CPU_ZERO(&mask);
    CPU_SET(16, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0) {
            perror("pthread_setaffinity_np");
    }

    /*
    float* h_A;
    float* h_B;
    hipHostMalloc(&h_A, size);
    hipHostMalloc(&h_B, size);
    */

    cout<<"game start "<<endl;

    clock_t start,finish;
    double singletime=0.0;
    double cotime=0.0;
    int err;
    err=hipCtxPushCurrent(ctx);
    if(err){
    cout<<"Push Context ERR! "<<err<<endl;
    }
    for(int i=0;i < 10;i++)
    {
    mtx1.lock();
    start=clock();
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    finish=clock();
    singletime += (double)(finish-start)/CLOCKS_PER_SEC;
    cout<<"This time single data transfer: "<<((double)(finish-start)/CLOCKS_PER_SEC)<<"(s)"<<endl;
    cout<<"1-1 timeline: "<<(double)(start)/CLOCKS_PER_SEC<<" to "<<(double)(finish)/CLOCKS_PER_SEC<<endl;
    mtx2.unlock();

    start=clock();
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    finish=clock();
    cotime += (double)(finish-start)/CLOCKS_PER_SEC;
    cout<<"This time concurrent data 111 transfer: "<<((double)(finish-start)/CLOCKS_PER_SEC)<<"(s)"<<endl;
    cout<<"1-2 timeline: "<<(double)(start)/CLOCKS_PER_SEC<<" to "<<(double)(finish)/CLOCKS_PER_SEC<<endl;
    }

    cout<<"single time: "<<singletime<<" s"<<endl;
    cout<<"cocurrent time1: "<<cotime<<" s"<<endl;
    /*
    while(1){
    sleep(1);
    cout<<"I'm alive"<<endl;
    }
    */

}

void thread2(hipCtx_t ctx,float* d_c,float* h_c,size_t size)
{

    cpu_set_t mask;
    CPU_ZERO(&mask);

    CPU_SET(16, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0) {
            perror("pthread_setaffinity_np");
    }

    /*
    float* h_C;
    hipHostMalloc(&h_C, size);
    */

    clock_t start,finish;
    double singletime=0.0;
    int err;
    err=hipCtxPushCurrent(ctx);
    if(err){
    cout<<"Push Context ERR! "<<err<<endl;
    }
    for(int i=0;i < 10;i++)
    {
    mtx2.lock();
    start=clock();
    hipMemcpy(d_c, h_c, size, hipMemcpyHostToDevice);
    finish=clock();
    singletime += (double)(finish-start)/CLOCKS_PER_SEC;
    cout<<"This time concurrent 222 data transfer: "<<((double)(finish-start)/CLOCKS_PER_SEC)<<"(s)"<<endl;
    cout<<"2-1 timeline: "<<(double)(start)/CLOCKS_PER_SEC<<" to "<<(double)(finish)/CLOCKS_PER_SEC<<endl;
    mtx1.unlock();
    }
    cout<<"cocurrent time2: "<<singletime<<" s"<<endl;
    cout<<"game end"<<endl;
   /*
    while(1){
    sleep(1);
    cout<<"I'm alive"<<endl;
    }
    */
}

int main()
{
    hipInit(0);
    hipSetDevice(1);
    //clock for collection

    //data size
    int N = 209715200;
    size_t size = N * sizeof(float);

    /*
    cpu_set_t mask;
    CPU_ZERO(&mask);
    CPU_SET(15, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0) {
            perror("pthread_setaffinity_np");
    }
    */


    //Context and memory
    cout<<"Create two context and their memory"<<endl;
    int err;
    hipCtx_t cont1,cont2;
    hipDevice_t dev;
    err = hipCtxGetDevice(&dev);
    if(err)
    {
        cout<<"Can't get device, err" << err<<endl;
        return 0;
    }

    err = hipCtxCreate(&cont1,hipDeviceScheduleYield,dev);
    if(err)
    {
        cout<<"Can't create Context, err" << err << endl;
        return 0;
    }
    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);

    //hipSetDevice(1);
    err = hipCtxCreate(&cont2,hipDeviceScheduleYield,dev);
    if(err)
    {
        cout<<"Can't create Context, err" << err<<endl;
        return 0;
    }
    float* d_C;
    hipMalloc(&d_C, size);


    cout<<"Allocate Host Memory"<<endl;
    // Allocate input vectors h_A and h_B in host memory

    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    /*
    float* h_A;
    float* h_B;
    float* h_C;
    hipHostMalloc(&h_A, size);
    hipHostMalloc(&h_B, size);
    hipHostMalloc(&h_C, size);
    */

    uniform_real_distribution<float> u(0,10);
    default_random_engine e(time(NULL));
    for(int i=0;i < N; ++i){
    *(h_A + i) = u(e);
	*(h_B + i) = u(e);
	*(h_C + i) = u(e);
    }

    //prepare
    mtx2.lock();
    thread first=thread(thread1,cont1,d_A,d_B,h_A,h_B,size);

    thread second=thread(thread2,cont2,d_C,h_C,size);
    second.join();
    first.join();
    //Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
