#include <iostream>
#include <thread>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "unistd.h"
#include <thread>
#include <mutex>
#include <random>
#include <ctime>
#include <time.h>
using namespace std;

int main()
{
    hipInit(0);
    hipSetDevice(1);
    //clock for collection

    //cpu_set_t mask;
    /*
    CPU_ZERO(&mask);
    CPU_SET(15, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0) {
            perror("pthread_setaffinity_np");
    }
    */

    //data size, 209715200 > 800 M
    int N = 209715200/20;
    size_t size = N * sizeof(float);

    //Alloc Device memory
    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);

    // Allocate input vectors
    float* h_A;
    float* h_B;
    float* h_C;
    hipHostMalloc(&h_A, size, hipHostMallocDefault);
    hipHostMalloc(&h_B, size, hipHostMallocDefault);
    hipHostMalloc(&h_C, size, hipHostMallocDefault);

    uniform_real_distribution<float> u(0,10);
    default_random_engine e(time(NULL));
    for(int i=0;i < N; ++i){
    *(h_A + i) = u(e);
	*(h_B + i) = u(e);
	*(h_C + i) = u(e);
    }

    //Create Stream
    hipStream_t firststream;
    hipStream_t secondstream;
    hipStreamCreate(&firststream);
    hipStreamCreate(&secondstream);

    //test
    /*
    cudaMemcpyAsync(d_C, h_C, size, cudaMemcpyHostToDevice, secondstream);
    cout<<"what?"<<endl;
    /*
    cudaMemcpyAsync(d_B, h_B, size, cudaMemcpyHostToDevice, firststream);
    //*/

    for(int i=0;i < 10;i++)
    {
    hipMemcpyAsync(d_C, h_C,size, hipMemcpyHostToDevice, secondstream);
    hipMemcpyAsync(d_B, h_B,size/2, hipMemcpyHostToDevice, firststream);
    }

    for(int i=0;i < 10;i++)
    {
    //cudaMemcpyAsync(d_A, h_A,size, cudaMemcpyHostToDevice, secondstream);
    }
    //Should i add some code to exit the thread here?

    //Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
