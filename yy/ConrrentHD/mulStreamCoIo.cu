#include <iostream>
#include <thread>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "unistd.h"
#include <thread>
#include <mutex>
#include <random>
#include <ctime>
#include <time.h>
using namespace std;

int main()
{
    hipInit(0);
    hipSetDevice(1);
    //clock for collection

    //cpu_set_t mask;
    /*
    CPU_ZERO(&mask);
    CPU_SET(15, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0) {
            perror("pthread_setaffinity_np");
    }
    */

    //data size, 209715200 > 800 M
    int N = 209715200/20;
    size_t size = N * sizeof(float);

    //Alloc Device memory
    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);
    float* d_D;
    hipMalloc(&d_D, size);

    // Allocate input vectors
    float* h_A;
    float* h_B;
    float* h_C;
    float* h_D;
    hipHostMalloc(&h_A, size, hipHostMallocDefault);
    hipHostMalloc(&h_B, size, hipHostMallocDefault);
    hipHostMalloc(&h_C, size, hipHostMallocDefault);
    hipHostMalloc(&h_D, size, hipHostMallocDefault);

    uniform_real_distribution<float> u(0,10);
    default_random_engine e(time(NULL));
    for(int i=0;i < N; ++i){
    *(h_A + i) = u(e);
	*(h_B + i) = u(e);
	*(h_C + i) = u(e);
	*(h_D + i) = u(e);
    }

    //Create Stream
    hipStream_t firststream;
    hipStream_t secondstream;
    hipStreamCreate(&firststream);
    hipStreamCreate(&secondstream);

    //test
    /*

    cout<<"what?"<<endl;
    */

    //cudaMemcpyAsync(d_D, h_D, size, cudaMemcpyHostToDevice, secondstream);
    hipMemcpyAsync(d_A, h_A, size/2, hipMemcpyHostToDevice, firststream);

    for(int i=0;i < 10;i++)
    {
    hipMemcpyAsync(d_C, h_C,size, hipMemcpyHostToDevice, secondstream);
    //cudaMemcpyAsync(d_B, h_B,size/2, cudaMemcpyHostToDevice, firststream);
    }

    for(int i=0;i < 10;i++)
    {
    hipMemcpyAsync(d_B, h_B,size/2, hipMemcpyHostToDevice, firststream);
    }
    //Should i add some code to exit the thread here?

    hipDeviceSynchronize();
    //Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
