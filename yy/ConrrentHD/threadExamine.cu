#include <iostream>
#include <thread>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "unistd.h"
#include <thread>
#include <mutex>
#include <random>
#include <ctime>
#include <time.h>
#include <stdio.h>

using namespace std;
#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)

//mutex workend2;
//mutex workend1;

//this is a normal kernel
__global__ void kernel(float n1, float n2, float n3, int stop) {
	for (int i = 0; i < stop; i++) {
		n1=sinf(n1);
		n2=n3/n2;
	}
}

//this is designed to timing based on the flag
//imagine it works in its own stream,"listening" to the flag and record the time
//it's a pity it didn't work well as the kernel launched in an original thread didn't worl
__global__ void kernel_timer(long long unsigned *times,int *flag) {
		unsigned long long mclk2;
		int i=0;
		while(i<11)
		{
		    while(flag[i] != 1) {
		        __nanosleep(5000); // 500us
		        //__syncthreads();
              }
		    if (threadIdx.x == 0){
		    asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk2));
		    times[i] = mclk2/ 1000000;
		    }
		    i++;
		}
}

//this kernel was designed to change the flag
__global__ void kernel_flager(int i,int *flag) {
		flag[i] = 1;
}


//synchronize funtion ,can be substituted by cudaDeviceSynchronize()
void CUDART_CB thread1_5callback(void *data) {
    //workend1.unlock();
}
void CUDART_CB thread2_3callback(void *data) {
    //workend2.unlock();
}

//diy thread
void thread1(hipStream_t stream,float* d_a,float* h_a,size_t size,long long unsigned *timeline,int number,int *flag)
{
    //set CPU
    /*
    cpu_set_t mask;
    CPU_ZERO(&mask);
    CPU_SET(16, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0)
    {
            perror("pthread_setaffinity_np");
    }
    */
    hipStream_t tempstream;
    hipStreamCreate(&tempstream);
    //compare whether the parameter stream has the same value as variable "firststream" outside
    cout << "In thread stream: "<<stream<<endl;

    //test whether the kernel worked ,it should work 67s,however,nothing happened
    kernel<<<1,32,0,tempstream>>>(1.0,2.0,3.0,1000000000);
    hipError_t cudaStatus;
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    //test wherther the parameter flag is effective,the line below worked well
    //flag[0] = 1;
    //compare whether the parameter flag has the same value as variable "flag" outside
    cout<<"In thread flag: "<<flag<<endl;

    //old code,designed for timing
    kernel_flager<<<1,1,0,tempstream>>>(0,flag);

    //data transfer loop
    //rotate for so many times,the total runtime didn't change
    //cout<<cudaMemcpy(d_a, h_a,size, cudaMemcpyHostToDevice);
    for(int i=1;i < 11;i++)
    {
    //cout<< cudaMemcpyAsync(d_a, h_a,size, cudaMemcpyHostToDevice, tempstream);
    //old code
    //kernel_flager<<<1,1,0,tempstream>>>(i,flag);
    }

}

int main()
{
    hipInit(0);
    hipSetDevice(1);

    /*
    cpu_set_t mask;
    CPU_ZERO(&mask);
    CPU_SET(16, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0) {
            perror("pthread_setaffinity_np");
    }
    */

    //40M data
    int N = 4*52428800/20;
    size_t size = N * sizeof(float);

    //allocate device variable(data)
    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);

    //create timeline and flag
    long long unsigned *timeline1;
	long long unsigned *timeline2;
    int *flag1;
    int *flag2;
    int *flag1h;
    int *flag2h;
    flag1h = (int*) malloc(11 * sizeof(int));
    flag2h = (int*) malloc(11 * sizeof(int));

    //for timeline and flag,create device variable
	size_t size2 = 11 * sizeof(long long unsigned);
	hipMalloc(&timeline1, size2);
    hipMalloc(&timeline2, size2);

    size_t size3 = 11*sizeof(int);
    hipMalloc(&flag1, size3);
    hipMalloc(&flag2, size3);

    //initialize the flags
    for(int i=0;i<11;i++)
    {
    flag1h[i]=0;
    flag2h[i]=0;
    }
    hipMemcpy(flag1, flag1h, sizeof(int) * 11, hipMemcpyHostToDevice);
    hipMemcpy(flag2, flag2h, sizeof(int) * 11, hipMemcpyHostToDevice);
    cout<<"Allocate Host Memory"<<endl;
    // Allocate input vectors h_A and h_B in host memory
    float* h_A;
    float* h_B;
    float* h_C;
    hipHostMalloc(&h_A, size, hipHostMallocDefault);
    hipHostMalloc(&h_B, size, hipHostMallocDefault);
    hipHostMalloc(&h_C, size, hipHostMallocDefault);

    uniform_real_distribution<float> u(0,10);
    default_random_engine e(time(NULL));
    for(int i=0;i < N; ++i){
    /*
    *(h_A + i) = u(e);
	*(h_B + i) = u(e);
	*(h_C + i) = u(e);
    */
    *(h_A + i) = 1;
	*(h_B + i) = 1;
	*(h_C + i) = 1;
    }

    //Create Stream,flagonestream and flag two stream was used to launch kernel_timer
    hipStream_t firststream;
    hipStream_t secondstream;
    //cudaStream_t flagonestream;
    //cudaStream_t flagtwostream;
    hipStreamCreate(&firststream);
    hipStreamCreate(&secondstream);
    //cudaStreamCreate(&flagonestream);
    //cudaStreamCreate(&flagtwostream);

    //kernel_timer<<<1,1,0,flagonestream>>>(timeline1,flag1);
    //kernel_timer<<<1,1,0,flagtwostream>>>(timeline2,flag2);


    //test whether memcpy works here
    //cudaMemcpyAsync(d_A, h_A,size/2, cudaMemcpyHostToDevice, firststream);
    //cudaMemcpyAsync(d_B, h_B,size, cudaMemcpyHostToDevice, secondstream);

    //prepare
    //workend1.lock();
    //workend2.lock();

    //test whether kernel works here
    //kernel<<<1,32,0,firststream>>>(1.0,2.0,3.0,1000000);

    //cudahostfunc to synchronize
    hipHostFn_t fn5 = thread1_5callback;
    hipHostFn_t fn8 = thread2_3callback;
    //cudaLaunchHostFunc(flagonestream, fn5, 0);
    //cudaLaunchHostFunc(flagtwostream, fn8, 0);

    //compare the flag in and out the original thread
    cout<<"flag: "<<flag1<<endl;
    cout << "Out of the thread stream: "<<firststream<<endl;

    thread first=thread(thread1,firststream,d_A,d_A,size,timeline1,1,flag1);
    //thread second=thread(thread1,secondstream,d_B,d_B,size,timeline2,2,flag2);
    //second.join();
    first.join();


    hipLaunchHostFunc(firststream, fn5, 0);
    //cudaLaunchHostFunc(secondstream, fn8, 0);


    cout<<"reach here"<<endl;
    //workend1.lock();
    //workend2.lock();

    //change,check whether the cudamemcpy works
    for(int i=0;i < N; ++i){
    /*
    *(h_A + i) = u(e);
	*(h_B + i) = u(e);
	*(h_C + i) = u(e);
    */
    *(h_A + i) = 0;
	*(h_B + i) = 0;
	*(h_C + i) = 0;
    }
    hipMemcpy(h_A, d_A,size, hipMemcpyDeviceToHost);
    for(int i=0;i < 10; ++i){
    cout<<"now  data"<<*(h_A + i)<<endl;
    }


    //check the kernel
    hipMemcpy(flag1h, flag1, sizeof(int) * 11, hipMemcpyDeviceToHost);
    for(int k=0;k< 11;k++)
    {
    printf("flag1-%d %d \n",k, flag1h[k]);
    }

    long long unsigned* timelineh1;
    long long unsigned* timelineh2;
    timelineh1 =(long long unsigned*)malloc(size2);
    timelineh2 =(long long unsigned*)malloc(size2);

    //output the timeline
    /*
    cudaMemcpy(timelineh1, timeline1, size2, cudaMemcpyDeviceToHost);
    cudaMemcpy(timelineh2, timeline2, size2, cudaMemcpyDeviceToHost);

    for(int k=0;k< 11;k++)
    {
    printf("Timeline0-%d %llu (s)\n",k, timelineh1[k]);
    }
    for(int k=0;k< 11;k++)
    {
    printf("Timeline1-%d %llu (s)\n",k, timelineh2[k]);
    }
    */

    //Free memory

    hipFree(timeline1);
    hipFree(timeline2);
    hipFree(flag1);
    hipFree(flag2);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
