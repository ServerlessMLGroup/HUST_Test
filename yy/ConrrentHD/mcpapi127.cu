#include <iostream>
#include <thread>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "unistd.h"
#include <thread>
#include <mutex>
#include <random>
#include <ctime>
#include <time.h>
using namespace std;

int main()
{
    hipInit(0);
    hipSetDevice(2);

    //cpu_set_t mask;
    /*
    CPU_ZERO(&mask);
    CPU_SET(15, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0) {
            perror("pthread_setaffinity_np");
    }
    */


    //data size, 262144 > 1 M
    int N = 262144;
    size_t size = N * sizeof(float);
    int datasize = 20;
    float* devicedata[datasize];
    float* hostdata[datasize];

    for(int i=0;i<datasize;i++)
    {
    hipMalloc(&devicedata[i], size);
    }

    for(int i=0;i<datasize;i++)
    {
    hipHostMalloc(&hostdata[i], size, hipHostMallocDefault);
    }

    uniform_real_distribution<float> u(0,10);
    default_random_engine e(time(NULL));
    for(int i=0;i < datasize; ++i){
    //*hostdata[i] = u(e);
    }

    //Create Stream
    hipStream_t firststream;
    hipStreamCreate(&firststream);

    for(int i=0;i<datasize;i++)
    {
    hipMemcpyAsync(devicedata[i], hostdata[i], size, hipMemcpyHostToDevice, firststream);
    }

    hipDeviceSynchronize();
    //Free memory
    for(int i=0;i<datasize;i++)
    {
    hipFree(devicedata[i]);
    }

    return 0;
}
