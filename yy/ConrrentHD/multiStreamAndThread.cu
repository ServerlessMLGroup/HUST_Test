#include <iostream>
#include <thread>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "unistd.h"
#include <thread>
#include <mutex>
#include <random>
#include <ctime>
#include <time.h>
#include <stdio.h>

using namespace std;
#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)

//Mutex
mutex mtx1_1;
mutex mtx1_2;
mutex mtx2_1;
mutex workend1;
mutex workend2;
//mutex test;
//clock_t
clock_t start1,finish1;
clock_t start1_2,finish1_2;
clock_t start2,finish2;
double singletime = 0.0;
double cotime1=0.0;
double cotime2=0.0;


__global__ void kernel_timer(long long unsigned *times,int *flag) {
		unsigned long long mclk2;
		int i=0;
		while(i<11)
		{
		    while(flag[i] != 1) {
		      //  if (threadIdx.x == 0)
		        //{
		        __nanosleep(5000); // 500us
	             //}
              }
		    if (threadIdx.x == 0){
		    asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk2));
		    times[i] = mclk2/ 1000000;
		    }
		    i++;
		}
}


__global__ void kernel_flager(int i,int *flag) {
		flag[i] = 1;
}


void CUDART_CB thread1_1callback(void *data) {
    //mtx1_1.lock();
    start1=clock();
    //test.unlock();
}


void CUDART_CB thread1_2callback(void *data) {
    finish1=clock();
    singletime += (double)(finish1 - start1)/CLOCKS_PER_SEC;
    cout<<"This time single data transfer: "<<((double)(finish1-start1)/CLOCKS_PER_SEC)<<"(s)"<<endl;
    cout<<"1-1 timeline: "<<(double)(start1)/CLOCKS_PER_SEC<<" to "<<(double)(finish1)/CLOCKS_PER_SEC<<endl;
}

void CUDART_CB thread1_3callback(void *data) {
    //mtx2_1.unlock();
    //mtx1_2.lock();
    start1_2=clock();
}

void CUDART_CB thread1_4callback(void *data) {
    finish1_2=clock();
    cotime1 += (double)(finish1_2-start1_2)/CLOCKS_PER_SEC;
    cout<<"This time cocurrent data transfer 1111: "<<((double)(finish1_2-start1_2)/CLOCKS_PER_SEC)<<"(s)"<<endl;
    cout<<"1-2 timeline: "<<(double)(start1_2)/CLOCKS_PER_SEC<<" to "<<(double)(finish1_2)/CLOCKS_PER_SEC<<endl;
    //mtx1_1.unlock();
}
void CUDART_CB thread1_5callback(void *data) {
    //cout<<"single time: "<<singletime<<" s"<<endl;
    //cout<<"cocurrent time1111: "<<cotime1<<" s"<<endl;
    workend1.unlock();
}

void CUDART_CB thread2_1callback(void *data) {
    //mtx2_1.lock();
    start2=clock();
}

void CUDART_CB thread2_2callback(void *data) {
    finish2=clock();
    cotime2 += (double)(finish2-start2)/CLOCKS_PER_SEC;
    cout<<"This time cocurrent data transfer 2222: "<<((double)(finish2-start2)/CLOCKS_PER_SEC)<<"(s)"<<endl;
    cout<<"2-1 timeline: "<<(double)(start2)/CLOCKS_PER_SEC<<" to "<<(double)(finish2)/CLOCKS_PER_SEC<<endl;
    //mtx1_2.unlock();
}

void CUDART_CB thread2_3callback(void *data) {
    //cout<<"cocurrent time2222: "<<cotime2<<" s"<<endl;
    workend2.unlock();
}




void thread1(hipStream_t stream,float* d_a,float* h_a,size_t size,long long unsigned *timeline,int number,int *flag)
{
    //set CPU
    cpu_set_t mask;
    CPU_ZERO(&mask);
    CPU_SET(16, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0)
    {
            perror("pthread_setaffinity_np");
    }
    kernel_flager<<<1,1,0,stream>>>(0,flag);

    for(int i=0;i < 10;i++)
    {
    hipMemcpyAsync(d_a, h_a,size, hipMemcpyHostToDevice, stream);
    kernel_flager<<<1,1,0,stream>>>(i+1,flag);
    }

}

int main()
{
    hipInit(0);
    hipSetDevice(1);

    cpu_set_t mask;
    CPU_ZERO(&mask);
    CPU_SET(16, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0) {
            perror("pthread_setaffinity_np");
    }

    int N = 4*52428800/4;
    size_t size = N * sizeof(float);

    double testtime;
    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);

    //cudaSetDevice(0);
    float* d_C;
    hipMalloc(&d_C, size);

    //create timeline and flag
    long long unsigned *timeline1;
	long long unsigned *timeline2;
    int *flag1;
    int *flag2;
    int *flag1h;
    int *flag2h;
    flag1h = (int*) malloc(11 * sizeof(int));
    flag2h = (int*) malloc(11 * sizeof(int));


	size_t size2 = 111 * sizeof(long long unsigned);
	hipMalloc(&timeline1, size2);
    hipMalloc(&timeline2, size2);
    size_t size3 = 111*sizeof(int);
    hipMalloc(&flag1, size3);
    hipMalloc(&flag2, size3);

    for(int i=0;i<10;i++)
    {
    flag1h[i]=1;
    flag2h[i]=1;
    }
    hipMemcpy(flag1, flag1h, sizeof(int) * 11, hipMemcpyHostToDevice);
    hipMemcpy(flag2, flag2h, sizeof(int) * 11, hipMemcpyHostToDevice);

    cout<<"Allocate Host Memory"<<endl;
    // Allocate input vectors h_A and h_B in host memory
    float* h_A;
    float* h_B;
    float* h_C;
    hipHostMalloc(&h_A, size, hipHostMallocDefault);
    hipHostMalloc(&h_B, size, hipHostMallocDefault);
    hipHostMalloc(&h_C, size, hipHostMallocDefault);

    uniform_real_distribution<float> u(0,10);
    default_random_engine e(time(NULL));
    for(int i=0;i < N; ++i){
    *(h_A + i) = u(e);
	*(h_B + i) = u(e);
	*(h_C + i) = u(e);
    }

    //Create Stream
    hipStream_t firststream;
    hipStream_t secondstream;
    hipStream_t flagonestream;
    hipStream_t flagtwostream;
    hipStreamCreate(&firststream);
    hipStreamCreate(&secondstream);
    hipStreamCreate(&flagonestream);
    hipStreamCreate(&flagtwostream);
    kernel_timer<<<1,1,0,flagonestream>>>(timeline1,flag1);
    kernel_timer<<<1,1,0,flagtwostream>>>(timeline2,flag2);

    //cudaMemcpyAsync(d_A, h_A,size/2, cudaMemcpyHostToDevice, firststream);
    //cudaMemcpyAsync(d_B, h_B,size, cudaMemcpyHostToDevice, secondstream);
    //prepare

    //mtx2_1.lock();
    workend1.lock();
    workend2.lock();

    //divide the formal funtion here
    hipHostFn_t fn1 = thread1_1callback;
    hipHostFn_t fn2 = thread1_2callback;
    hipHostFn_t fn3 = thread1_3callback;
    hipHostFn_t fn4 = thread1_4callback;
    hipHostFn_t fn5 = thread1_5callback;
    hipHostFn_t fn6 = thread2_1callback;
    hipHostFn_t fn7 = thread2_2callback;
    hipHostFn_t fn8 = thread2_3callback;
    
    thread first=thread(thread1,firststream,d_A,d_A,size,timeline1,1,flag1);
    thread second=thread(thread1,secondstream,d_B,d_B,size,timeline2,2,flag2);
    second.join();
    first.join();
    

    hipLaunchHostFunc(flagonestream, fn5, 0);

    hipLaunchHostFunc(flagtwostream, fn8, 0);

    cout<<"reach here"<<endl;
    workend1.lock();
    workend2.lock();


    long long unsigned* timelineh1;
    long long unsigned* timelineh2;
    timelineh1 =(long long unsigned*)malloc(size2);

    timelineh2 =(long long unsigned*)malloc(size2);


    hipMemcpy(timelineh1, timeline1, size, hipMemcpyDeviceToHost);
    hipMemcpy(timelineh2, timeline2, size, hipMemcpyDeviceToHost);

    for(int k=0;k< 11;k++)
    {
    printf("Timeline0-%d %llu (s)\n",k, timelineh1[k]);
    }
    for(int k=0;k< 11;k++)
    {
    printf("Timeline0-%d %llu (s)\n",k, timelineh2[k]);
    }

    cout<<"It can't be like this"<<endl;
    //Free memory
    hipFree(timeline1);
    hipFree(timeline2);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
