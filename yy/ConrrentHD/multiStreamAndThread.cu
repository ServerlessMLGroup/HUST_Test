#include "hip/hip_runtime.h"
#include <iostream>
#include <thread>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "unistd.h"
#include <thread>
#include <mutex>
#include <random>
#include <ctime>
#include <time.h>
#include <stdio.h>
using namespace std;

__global__ void kernel_timer(long long unsigned *times,int j) {
		unsigned long long mclk2;
		if (threadIdx.x == 0){
		asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk2));
		times[j] = mclk2/ 1000000;
		}
	}
}



//Mutex
mutex mtx1_1;
mutex mtx1_2;
mutex mtx2_1;
mutex workend1;
mutex workend2;
//mutex test;
//clock_t
clock_t start1,finish1;
clock_t start1_2,finish1_2;
clock_t start2,finish2;
double singletime = 0.0;
double cotime1=0.0;
double cotime2=0.0;

void CUDART_CB thread1_1callback(void *data) {
    //mtx1_1.lock();
    start1=clock();
    //test.unlock();
}


void CUDART_CB thread1_2callback(void *data) {
    finish1=clock();
    singletime += (double)(finish1 - start1)/CLOCKS_PER_SEC;
    cout<<"This time single data transfer: "<<((double)(finish1-start1)/CLOCKS_PER_SEC)<<"(s)"<<endl;
    cout<<"1-1 timeline: "<<(double)(start1)/CLOCKS_PER_SEC<<" to "<<(double)(finish1)/CLOCKS_PER_SEC<<endl;
}

void CUDART_CB thread1_3callback(void *data) {
    //mtx2_1.unlock();
    //mtx1_2.lock();
    start1_2=clock();
}

void CUDART_CB thread1_4callback(void *data) {
    finish1_2=clock();
    cotime1 += (double)(finish1_2-start1_2)/CLOCKS_PER_SEC;
    cout<<"This time cocurrent data transfer 1111: "<<((double)(finish1_2-start1_2)/CLOCKS_PER_SEC)<<"(s)"<<endl;
    cout<<"1-2 timeline: "<<(double)(start1_2)/CLOCKS_PER_SEC<<" to "<<(double)(finish1_2)/CLOCKS_PER_SEC<<endl;
    //mtx1_1.unlock();
}
void CUDART_CB thread1_5callback(void *data) {
    cout<<"single time: "<<singletime<<" s"<<endl;
    cout<<"cocurrent time1111: "<<cotime1<<" s"<<endl;
    workend1.unlock();
}

void CUDART_CB thread2_1callback(void *data) {
    //mtx2_1.lock();
    start2=clock();
}

void CUDART_CB thread2_2callback(void *data) {
    finish2=clock();
    cotime2 += (double)(finish2-start2)/CLOCKS_PER_SEC;
    cout<<"This time cocurrent data transfer 2222: "<<((double)(finish2-start2)/CLOCKS_PER_SEC)<<"(s)"<<endl;
    cout<<"2-1 timeline: "<<(double)(start2)/CLOCKS_PER_SEC<<" to "<<(double)(finish2)/CLOCKS_PER_SEC<<endl;
    //mtx1_2.unlock();
}

void CUDART_CB thread2_3callback(void *data) {
    cout<<"cocurrent time2222: "<<cotime2<<" s"<<endl;
    workend2.unlock();
}

void thread1(hipStream_t stream,float* d_a,float* h_a,size_t size,long long unsigned *timeline,int number)
{
    //set CPU
    cpu_set_t mask;
    CPU_ZERO(&mask);
    CPU_SET(16, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0)
    {
            perror("pthread_setaffinity_np");
    }
    kernel_timer <<<1, 1, 0, stream>>>(timeline,0);
    for(int i=0;i < 10;i++)
    {
    hipMemcpyAsync(d_a, h_a,size, hipMemcpyHostToDevice, stream);
    kernel_timer <<<1, 1, 0, stream>>>(timeline,i+1);
    }
    for(int k=0;k < 11;k++)
    {
    cout<<"Timeline"<<number<<"-"<<k<<" :"<<timeline[k]<<"(s)"<<endl;
    }
}

void thread2(hipStream_t stream,float* d_a,float* h_a,size_t size)
{
    //set CPU
    cpu_set_t mask;
    CPU_ZERO(&mask);
    CPU_SET(16, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0)
    {
            perror("pthread_setaffinity_np");
    }


    for(int i=0;i < 10;i++)
    {
    hipMemcpyAsync(d_a, h_a,size, hipMemcpyHostToDevice, stream);
    }

}

int main()
{
    hipInit(0);
    hipSetDevice(1);

    cpu_set_t mask;
    CPU_ZERO(&mask);
    CPU_SET(16, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0) {
            perror("pthread_setaffinity_np");
    }

    int N = 4*52428800;
    size_t size = N * sizeof(float);

    double testtime;
    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);

    //hipSetDevice(0);
    float* d_C;
    hipMalloc(&d_C, size);
    long long unsigned *timeline1;
	long long unsigned *timeline2;
	hipMalloc(&timeline1, 11 * sizeof(long long unsigned));
    hipMalloc(&timeline2, 11 * sizeof(long long unsigned));

    cout<<"Allocate Host Memory"<<endl;
    // Allocate input vectors h_A and h_B in host memory
    float* h_A;
    float* h_B;
    float* h_C;
    hipHostMalloc(&h_A, size);
    hipHostMalloc(&h_B, size);
    hipHostMalloc(&h_C, size);

    uniform_real_distribution<float> u(0,10);
    default_random_engine e(time(NULL));
    for(int i=0;i < N; ++i){
    *(h_A + i) = u(e);
	*(h_B + i) = u(e);
	*(h_C + i) = u(e);
    }



    //Create Stream
    hipStream_t firststream;
    hipStream_t secondstream;
    hipStreamCreate(&firststream);
    hipStreamCreate(&secondstream);

    //hipMemcpyAsync(d_A, h_A,size/2, hipMemcpyHostToDevice, firststream);
    //hipMemcpyAsync(d_B, h_B,size, hipMemcpyHostToDevice, secondstream);
    //prepare

    mtx2_1.lock();
    workend1.lock();
    workend2.lock();

    //divide the formal funtion here
    hipHostFn_t fn1 = thread1_1callback;
    hipHostFn_t fn2 = thread1_2callback;
    hipHostFn_t fn3 = thread1_3callback;
    hipHostFn_t fn4 = thread1_4callback;
    hipHostFn_t fn5 = thread1_5callback;
    hipHostFn_t fn6 = thread2_1callback;
    hipHostFn_t fn7 = thread2_2callback;
    hipHostFn_t fn8 = thread2_3callback;


    
    thread first=thread(thread1,firststream,d_A,d_A,size,timeline1,1);
    thread second=thread(thread1,secondstream,d_B,d_B,size,timeline2,2);
    second.join();
    first.join();
    

    hipLaunchHostFunc(firststream, fn5, 0);

    hipLaunchHostFunc(secondstream, fn8, 0);

    workend1.lock();
    workend2.lock();


    cout<<"It can't be like this"<<endl;
    //Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
