#include <iostream>
#include <thread>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "unistd.h"
#include <thread>
#include <mutex>
#include <random>
#include <ctime>
#include <time.h>
#include <stdio.h>

using namespace std;
#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)

//Mutex
mutex mtx1_1;
mutex mtx1_2;
mutex mtx2_1;
mutex workend1;
mutex workend2;
//mutex test;
//clock_t
clock_t start1,finish1;
clock_t start1_2,finish1_2;
clock_t start2,finish2;
double singletime = 0.0;
double cotime1=0.0;
double cotime2=0.0;

__global__ void kernel(float n1, float n2, float n3, int stop) {

	for (int i = 0; i < stop; i++) {
		n1=sinf(n1);
		n2=n3/n2;
	}

}


__global__ void kernel_timer(long long unsigned *times,int *flag) {
		unsigned long long mclk2;
		int i=0;
		while(i<11)
		{
		    
		    while(flag[0] != 1) {
		        __nanosleep(5000); // 500us
		        //__syncthreads();
              }

		    if (threadIdx.x == 0){
		    asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk2));
		    times[i] = mclk2/ 1000000;
		    }
		    i++;
		}
}


__global__ void kernel_flager(int i,int *flag) {
		flag[i] = 1;
}

void CUDART_CB thread1_5callback(void *data) {
    workend1.unlock();
}

void CUDART_CB thread2_3callback(void *data) {
    workend2.unlock();
}

void thread1(hipStream_t stream,float* d_a,float* h_a,size_t size,long long unsigned *timeline,int number,int *flag)
{
    //set CPU
    cpu_set_t mask;
    CPU_ZERO(&mask);
    CPU_SET(16, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0)
    {
            perror("pthread_setaffinity_np");
    }
    //kernel<<<1,1,0,stream>>>(1.0,2.0,3.0,100);
    kernel_flager<<<1,1,0,stream>>>(0,flag);

    for(int i=1;i < 11;i++)
    {
    //kernel<<<1,1,0,stream>>>(1.0,2.0,3.0,100);
    hipMemcpyAsync(d_a, h_a,size, hipMemcpyHostToDevice, stream);
    kernel_flager<<<1,1,0,stream>>>(i,flag);
    //kernel<<<1,1,0,stream>>>(1.0,2.0,3.0,100);
    }

}

int main()
{
    hipInit(0);
    hipSetDevice(2);

    cpu_set_t mask;
    CPU_ZERO(&mask);
    CPU_SET(16, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0) {
            perror("pthread_setaffinity_np");
    }

    int N = 4*52428800/20;
    size_t size = N * sizeof(float);

    double testtime;
    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);

    float* d_C;
    hipMalloc(&d_C, size);

    //create timeline and flag
    long long unsigned *timeline1;
	long long unsigned *timeline2;
    int *flag1;
    int *flag2;
    int *flag1h;
    int *flag2h;
    flag1h = (int*) malloc(11 * sizeof(int));
    flag2h = (int*) malloc(11 * sizeof(int));

    //use 111 instaead to check
	size_t size2 = 111 * sizeof(long long unsigned);
	hipMalloc(&timeline1, size2);
    hipMalloc(&timeline2, size2);

    size_t size3 = 11*sizeof(int);
    hipMalloc(&flag1, size3);
    hipMalloc(&flag2, size3);

    for(int i=0;i<11;i++)
    {
    flag1h[i]=0;
    flag2h[i]=0;
    }

    hipMemcpy(flag1, flag1h, sizeof(int) * 11, hipMemcpyHostToDevice);
    hipMemcpy(flag2, flag2h, sizeof(int) * 11, hipMemcpyHostToDevice);

    cout<<"Allocate Host Memory"<<endl;
    // Allocate input vectors h_A and h_B in host memory
    float* h_A;
    float* h_B;
    float* h_C;
    hipHostMalloc(&h_A, size, hipHostMallocDefault);
    hipHostMalloc(&h_B, size, hipHostMallocDefault);
    hipHostMalloc(&h_C, size, hipHostMallocDefault);

    uniform_real_distribution<float> u(0,10);
    default_random_engine e(time(NULL));
    for(int i=0;i < N; ++i){
    *(h_A + i) = u(e);
	*(h_B + i) = u(e);
	*(h_C + i) = u(e);
    }

    //Create Stream
    hipStream_t firststream;
    hipStream_t secondstream;
    hipStream_t flagonestream;
    hipStream_t flagtwostream;
    hipStreamCreate(&firststream);
    hipStreamCreate(&secondstream);
    hipStreamCreate(&flagonestream);
    hipStreamCreate(&flagtwostream);

    kernel_timer<<<1,1,0,flagonestream>>>(timeline1,flag1);
    kernel_timer<<<1,1,0,flagtwostream>>>(timeline2,flag2);



    //cudaMemcpyAsync(d_A, h_A,size/2, cudaMemcpyHostToDevice, firststream);
    //cudaMemcpyAsync(d_B, h_B,size, cudaMemcpyHostToDevice, secondstream);
    //prepare

    //mtx2_1.lock();
    workend1.lock();
    workend2.lock();

    //divide the formal funtion here
    hipHostFn_t fn5 = thread1_5callback;
    hipHostFn_t fn8 = thread2_3callback;
    //cudaLaunchHostFunc(flagonestream, fn5, 0);
    //cudaLaunchHostFunc(flagtwostream, fn8, 0);

    thread first=thread(thread1,firststream,d_A,d_A,size,timeline1,1,flag1);
    thread second=thread(thread1,secondstream,d_B,d_B,size,timeline2,2,flag2);
    second.join();
    first.join();


    hipLaunchHostFunc(firststream, fn5, 0);
    hipLaunchHostFunc(secondstream, fn8, 0);


    cout<<"reach here"<<endl;
    workend1.lock();
    workend2.lock();


    long long unsigned* timelineh1;
    long long unsigned* timelineh2;
    timelineh1 =(long long unsigned*)malloc(size2);

    timelineh2 =(long long unsigned*)malloc(size2);


    hipMemcpy(timelineh1, timeline1, size2, hipMemcpyDeviceToHost);
    hipMemcpy(timelineh2, timeline2, size2, hipMemcpyDeviceToHost);

    for(int k=0;k< 11;k++)
    {
    printf("Timeline0-%d %llu (s)\n",k, timelineh1[k]);
    }
    for(int k=0;k< 11;k++)
    {
    printf("Timeline1-%d %llu (s)\n",k, timelineh2[k]);
    }

    cout<<"It can't be like this"<<endl;
    //Free memory

    hipFree(timeline1);
    hipFree(timeline2);
    hipFree(flag1);
    hipFree(flag2);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
