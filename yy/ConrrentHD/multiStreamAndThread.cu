#include <iostream>
#include <thread>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "unistd.h"
#include <thread>
#include <mutex>
#include <random>
#include <ctime>
#include <time.h>
#include <stdio.h>
using namespace std;

//Mutex
mutex mtx1_1;
mutex mtx1_2;
mutex mtx2_1;
mutex workend1;
mutex workend2;
//mutex test;
//clock_t
clock_t start1,finish1;
clock_t start1_2,finish1_2;
clock_t start2,finish2;
double singletime = 0.0;
double cotime1=0.0;
double cotime2=0.0;

void CUDART_CB thread1_1callback(void *data) {
    //mtx1_1.lock();
    start1=clock();
    //test.unlock();
}


void CUDART_CB thread1_2callback(void *data) {
    finish1=clock();
    singletime += (double)(finish1 - start1)/CLOCKS_PER_SEC;
    cout<<"This time single data transfer: "<<((double)(finish1-start1)/CLOCKS_PER_SEC)<<"(s)"<<endl;
    cout<<"1-1 timeline: "<<(double)(start1)/CLOCKS_PER_SEC<<" to "<<(double)(finish1)/CLOCKS_PER_SEC<<endl;
}

void CUDART_CB thread1_3callback(void *data) {
    //mtx2_1.unlock();
    //mtx1_2.lock();
    start1_2=clock();
}

void CUDART_CB thread1_4callback(void *data) {
    finish1_2=clock();
    cotime1 += (double)(finish1_2-start1_2)/CLOCKS_PER_SEC;
    cout<<"This time cocurrent data transfer 1111: "<<((double)(finish1_2-start1_2)/CLOCKS_PER_SEC)<<"(s)"<<endl;
    cout<<"1-2 timeline: "<<(double)(start1_2)/CLOCKS_PER_SEC<<" to "<<(double)(finish1_2)/CLOCKS_PER_SEC<<endl;
    //mtx1_1.unlock();
}
void CUDART_CB thread1_5callback(void *data) {
    cout<<"single time: "<<singletime<<" s"<<endl;
    cout<<"cocurrent time1111: "<<cotime1<<" s"<<endl;
    workend1.unlock();
}

void CUDART_CB thread2_1callback(void *data) {
    //mtx2_1.lock();
    start2=clock();
}

void CUDART_CB thread2_2callback(void *data) {
    finish2=clock();
    cotime2 += (double)(finish2-start2)/CLOCKS_PER_SEC;
    cout<<"This time cocurrent data transfer 2222: "<<((double)(finish2-start2)/CLOCKS_PER_SEC)<<"(s)"<<endl;
    cout<<"2-1 timeline: "<<(double)(start2)/CLOCKS_PER_SEC<<" to "<<(double)(finish2)/CLOCKS_PER_SEC<<endl;
    //mtx1_2.unlock();
}

void CUDART_CB thread2_3callback(void *data) {
    cout<<"cocurrent time2222: "<<cotime2<<" s"<<endl;
    workend2.unlock();
}

void thread1(hipStream_t stream,float* d_a,float* h_a,size_t size)
{
    //set CPU
    cpu_set_t mask;
    CPU_ZERO(&mask);
    CPU_SET(16, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0)
    {
            perror("pthread_setaffinity_np");
    }

    for(int i=0;i < 10;i++)
    {
    hipMemcpyAsync(d_a, h_a,size, hipMemcpyHostToDevice, stream);
    }

}

void thread2(hipStream_t stream,float* d_a,float* h_a,size_t size)
{
    //set CPU
    cpu_set_t mask;
    CPU_ZERO(&mask);
    CPU_SET(16, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0)
    {
            perror("pthread_setaffinity_np");
    }

    for(int i=0;i < 10;i++)
    {
    hipMemcpyAsync(d_a, h_a,size, hipMemcpyHostToDevice, stream);
    }

}

int main()
{
    hipInit(0);
    hipSetDevice(1);

    cpu_set_t mask;
    CPU_ZERO(&mask);
    CPU_SET(16, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0) {
            perror("pthread_setaffinity_np");
    }

    int N = 4*52428800;
    size_t size = N * sizeof(float);

    double testtime;
    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);

    //cudaSetDevice(0);
    float* d_C;
    hipMalloc(&d_C, size);


    cout<<"Allocate Host Memory"<<endl;
    // Allocate input vectors h_A and h_B in host memory
    float* h_A;
    float* h_B;
    float* h_C;
    hipHostMalloc(&h_A, size, hipHostMallocDefault);
    hipHostMalloc(&h_B, size, hipHostMallocDefault);
    hipHostMalloc(&h_C, size, hipHostMallocDefault);

    uniform_real_distribution<float> u(0,10);
    default_random_engine e(time(NULL));
    for(int i=0;i < N; ++i){
    *(h_A + i) = u(e);
	*(h_B + i) = u(e);
	*(h_C + i) = u(e);
    }

    //Create Stream
    hipStream_t firststream;
    hipStream_t secondstream;
    hipStreamCreate(&firststream);
    hipStreamCreate(&secondstream);

    hipMemcpyAsync(d_A, h_A,size/2, hipMemcpyHostToDevice, firststream);
    hipMemcpyAsync(d_B, h_B,size, hipMemcpyHostToDevice, secondstream);
    //prepare

    mtx2_1.lock();
    workend1.lock();
    workend2.lock();
    
    //divide the formal funtion here
    hipHostFn_t fn1 = thread1_1callback;
    hipHostFn_t fn2 = thread1_2callback;
    hipHostFn_t fn3 = thread1_3callback;
    hipHostFn_t fn4 = thread1_4callback;
    hipHostFn_t fn5 = thread1_5callback;
    hipHostFn_t fn6 = thread2_1callback;
    hipHostFn_t fn7 = thread2_2callback;
    hipHostFn_t fn8 = thread2_3callback;


    /*
    thread first=thread(thread1,firststream,d_A,d_A,size);
    thread second=thread(thread2,secondstream,d_B,d_B,size);
    second.join();
    first.join();
    */

    hipLaunchHostFunc(firststream, fn5, 0);

    hipLaunchHostFunc(secondstream, fn8, 0);

    workend1.lock();
    workend2.lock();


    cout<<"It can't be like this"<<endl;
    //Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
