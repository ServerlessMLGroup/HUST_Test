#include <iostream>
#include <thread>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "unistd.h"
#include <thread>
#include <mutex>
#include <random>
#include <ctime>
#include <time.h>
using namespace std;

//Mutex
mutex mtx1_1;
mutex mtx1_2;
mutex mtx2_1;
mutex workend1;
mutex workend2;
//mutex test;
//clock_t
clock_t start1,finish1;
clock_t start1_2,finish1_2;
clock_t start2,finish2;
double singletime = 0.0;
double cotime1=0.0;
double cotime2=0.0;

void CUDART_CB thread1_1callback(void *data) {
    //mtx1_1.lock();
    start1=clock();
    //test.unlock();
}


void CUDART_CB thread1_2callback(void *data) {
    finish1=clock();
    singletime += (double)(finish1 - start1)/CLOCKS_PER_SEC;
    cout<<"This time single data transfer: "<<((double)(finish1-start1)/CLOCKS_PER_SEC)<<"(s)"<<endl;
    cout<<"1-1 timeline: "<<(double)(start1)/CLOCKS_PER_SEC<<" to "<<(double)(finish1)/CLOCKS_PER_SEC<<endl;
}

void CUDART_CB thread1_3callback(void *data) {
    //mtx2_1.unlock();
    //mtx1_2.lock();
    start1_2=clock();
}

void CUDART_CB thread1_4callback(void *data) {
    finish1_2=clock();
    cotime1 += (double)(finish1_2-start1_2)/CLOCKS_PER_SEC;
    cout<<"This time cocurrent data transfer 1111: "<<((double)(finish1_2-start1_2)/CLOCKS_PER_SEC)<<"(s)"<<endl;
    cout<<"1-2 timeline: "<<(double)(start1_2)/CLOCKS_PER_SEC<<" to "<<(double)(finish1_2)/CLOCKS_PER_SEC<<endl;
    //mtx1_1.unlock();
}
void CUDART_CB thread1_5callback(void *data) {
    cout<<"single time: "<<singletime<<" s"<<endl;
    cout<<"cocurrent time1111: "<<cotime1<<" s"<<endl;
    workend1.unlock();
}

void CUDART_CB thread2_1callback(void *data) {
    //mtx2_1.lock();
    start2=clock();
}

void CUDART_CB thread2_2callback(void *data) {
    finish2=clock();
    cotime2 += (double)(finish2-start2)/CLOCKS_PER_SEC;
    cout<<"This time cocurrent data transfer 2222: "<<((double)(finish2-start2)/CLOCKS_PER_SEC)<<"(s)"<<endl;
    cout<<"2-1 timeline: "<<(double)(start2)/CLOCKS_PER_SEC<<" to "<<(double)(finish2)/CLOCKS_PER_SEC<<endl;
    //mtx1_2.unlock();
}

void CUDART_CB thread2_3callback(void *data) {
    cout<<"cocurrent time2222: "<<cotime2<<" s"<<endl;
    workend2.unlock();
}


int main()
{
    hipInit(0);
    hipSetDevice(1);
    //clock for collection

    //cpu_set_t mask;
    /*
    CPU_ZERO(&mask);
    CPU_SET(15, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0) {
            perror("pthread_setaffinity_np");
    }
    */

    //data size
    //int N = 4*52428800;
    int N = 52428800/4;
    size_t size = N * sizeof(float);

    //
    double testtime;

    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);

    //cudaSetDevice(0);
    float* d_C;
    hipMalloc(&d_C, size);

    //test.lock();
    cout<<"Allocate Host Memory"<<endl;
    // Allocate input vectors h_A and h_B in host memory
    float* h_A;
    float* h_B;
    float* h_C;
    hipHostMalloc(&h_A, size, hipHostMallocDefault);
    hipHostMalloc(&h_B, size, hipHostMallocDefault);
    hipHostMalloc(&h_C, size, hipHostMallocDefault);

    uniform_real_distribution<float> u(0,10);
    default_random_engine e(time(NULL));
    for(int i=0;i < N; ++i){
    *(h_A + i) = u(e);
	*(h_B + i) = u(e);
	*(h_C + i) = u(e);
    }

    //Create Stream
    hipStream_t firststream;
    hipStream_t secondstream;
    hipStreamCreate(&firststream);
    hipStreamCreate(&secondstream);

    //prepare
    mtx2_1.lock();
    workend1.lock();
    workend2.lock();

    //divide the formal funtion here
    hipHostFn_t fn1 = thread1_1callback;
    hipHostFn_t fn2 = thread1_2callback;
    hipHostFn_t fn3 = thread1_3callback;
    hipHostFn_t fn4 = thread1_4callback;
    hipHostFn_t fn5 = thread1_5callback;
    hipHostFn_t fn6 = thread2_1callback;
    hipHostFn_t fn7 = thread2_2callback;
    hipHostFn_t fn8 = thread2_3callback;


    for(int i=0;i < 10;i++)
    {
    //cudaLaunchHostFunc(secondstream, fn6, 0);
    //cudaMemcpyAsync(d_C, h_C,size, cudaMemcpyHostToDevice, secondstream);
    //cudaLaunchHostFunc(secondstream, fn7, 0);
    /*
    cudaLaunchHostFunc(firststream, fn3, 0);
    cudaMemcpyAsync(d_B, h_B,size, cudaMemcpyHostToDevice, firststream);
    cudaLaunchHostFunc(firststream, fn4, 0);
    */
    //cudaLaunchHostFunc(secondstream, fn3, 0);
    hipMemcpyAsync(d_A, h_A, size/2, hipMemcpyHostToDevice, secondstream);
    //cudaMemcpyAsync(d_B, h_B, size, cudaMemcpyHostToDevice, firststream);
    //cudaLaunchHostFunc(secondstream, fn4, 0);

    }

    for(int i=0;i < 10;i++)
    {
    //cudaLaunchHostFunc(firststream, fn1, 0);
    //test.lock();
    //cout<<"Pass the test"<<endl;
    hipMemcpyAsync(d_B, h_B,size, hipMemcpyHostToDevice, firststream);
    //cudaLaunchHostFunc(firststream, fn2, 0);
    //cudaLaunchHostFunc(firststream, fn3, 0);
    //cudaMemcpyAsync(d_B, h_B,size, cudaMemcpyHostToDevice, firststream);
    //cudaLaunchHostFunc(firststream, fn4, 0);
    }
    //Should i add some code to exit the thread here?

    hipLaunchHostFunc(firststream, fn5, 0);


    hipLaunchHostFunc(secondstream, fn8, 0);

    workend1.lock();
    workend2.lock();
    cout<<"It can't be like this"<<endl;
    //Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
