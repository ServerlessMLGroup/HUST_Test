#include <iostream>
#include <thread>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "unistd.h"
#include <thread>
#include <mutex>
#include <random>
#include <ctime>
#include <time.h>
using namespace std;

int main()
{
    hipInit(0);
    hipSetDevice(2);

    //cpu_set_t mask;
    /*
    CPU_ZERO(&mask);
    CPU_SET(15, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0) {
            perror("pthread_setaffinity_np");
    }
    */


    //data size, 262144 > 1 M
    int N = 262144;
    size_t size = N * sizeof(float);
    size_t mergesize;

    //size of array datasize
    int datasize = 70;
    mergesize = datasize*size;

    float* devicedata[datasize];
    float* hostdata[datasize];
    float* mergedevicedata;
    float* mergehostdata;

    for(int i=0;i<datasize;i++)
    {
    hipMalloc(&devicedata[i], size);
    }
    hipMalloc(&mergedevicedata, mergesize);


    for(int i=0;i<datasize;i++)
    {
    hipHostMalloc(&hostdata[i], size, hipHostMallocDefault);
    }
    hipHostMalloc(&mergehostdata, mergesize, hipHostMallocDefault);


    uniform_real_distribution<float> u(0,10);
    default_random_engine e(time(NULL));
    for(int i=0;i < datasize; ++i){
    //*hostdata[i] = u(e);
    }

    //Create Stream
    hipStream_t firststream;
    hipStreamCreate(&firststream);

    //warm
    hipMemcpyAsync(devicedata[0], hostdata[0], size, hipMemcpyHostToDevice, firststream);
    hipDeviceSynchronize();


    //memcpy, scatter
    for(int i=0;i<datasize;i++)
    {
    hipMemcpyAsync(devicedata[i], hostdata[i], size, hipMemcpyHostToDevice, firststream);
    }

    hipDeviceSynchronize();

    //memcpy, merge
    hipMemcpyAsync(mergedevicedata, mergehostdata, mergesize, hipMemcpyHostToDevice, firststream);
    hipDeviceSynchronize();


    //Free memory
    for(int i=0;i<datasize;i++)
    {
    hipFree(devicedata[i]);
    }
    hipFree(mergedevicedata);

    return 0;
}
