#include <iostream>
#include <thread>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "unistd.h"
#include <thread>
#include <mutex>
#include <random>
#include <ctime>
#include <time.h>
using namespace std;

int main()
{
    hipInit(0);
    hipSetDevice(2);

    //cpu_set_t mask;
    /*
    CPU_ZERO(&mask);
    CPU_SET(15, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0) {
            perror("pthread_setaffinity_np");
    }
    */


    //data size, 262144 > 1 M
    int N = 262144*8/20;
    size_t size = N * sizeof(float);

    //size of array datasize
    int datasize = 20;
    float* devicedata[datasize];
    float* hostdata[datasize];

    for(int i=0;i<datasize;i++)
    {
    hipMalloc(&devicedata[i], size);
    }

    for(int i=0;i<datasize;i++)
    {
    hipHostMalloc(&hostdata[i], size, hipHostMallocDefault);
    }

    uniform_real_distribution<float> u(0,10);
    default_random_engine e(time(NULL));
    for(int i=0;i < datasize; ++i){
    //*hostdata[i] = u(e);
    }

    //Create Stream
    hipStream_t firststream;
    hipStreamCreate(&firststream);

    //warm
    hipMemcpyAsync(devicedata[0], hostdata[0], size, hipMemcpyHostToDevice, firststream);

    //memcpy
    for(int i=0;i<datasize;i++)
    {
    hipMemcpyAsync(devicedata[i], hostdata[i], size, hipMemcpyHostToDevice, firststream);
    }

    hipDeviceSynchronize();

    int cputime=1000;
    int tempint=0;
    for(int i=0;i<datasize;i++)
    {
    for(int j=0;j++;j<cputime)
    {
    tempint++;
    }
    hipMemcpyAsync(devicedata[i], hostdata[i], size, hipMemcpyHostToDevice, firststream);
    }

    //Free memory
    for(int i=0;i<datasize;i++)
    {
    hipFree(devicedata[i]);
    }

    return 0;
}
