#include "hip/hip_runtime.h"
#include <iostream>
#include <thread>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "unistd.h"
#include <thread>
#include <mutex>
#include <random>
#include <ctime>
#include <time.h>
#include <stdio.h>
using namespace std;


int main()
{
    hipInit(0);
    hipSetDevice(1);

    //set CPU
    clock_t start,finish;
    double time=0.0;
    /*
    cout<<"set cpu"<<endl;
    cpu_set_t mask;
    CPU_ZERO(&mask);
    CPU_SET(16, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0) {
            perror("pthread_setaffinity_np");
    }
    */



    //data size, 262144 > 1 M
    int N = 262144;
    size_t size = N * sizeof(float);
    float* hostdata1;
    float* hostdata2;
    float* hostdata3;
    float* hostdata4;
    float* hostdata5;
    float* hostdata6;
    float* hostdata7;
    float* hostdata8;
    float* hostdata9;
    float* hostdata0;
    float* devicedata;


    hipMalloc(&devicedata,size)
    hipHostMalloc(&hostdata, size);

    //Create Stream
    hipStream_t firststream;
    hipStreamCreate(&firststream);

    size = N * sizeof(float);
    start=clock();
    hipHostMalloc(&hostdata1, size);
    finish=clock();
    time = (double)(finish-start)/CLOCKS_PER_SEC;
    cout <<"1"<<" Timeuse: "<<time<<" (s)"<<endl;

    size = N *10 *sizeof(float);
    start=clock();
    hipHostMalloc(&hostdata2, size);
    finish=clock();
    time = (double)(finish-start)/CLOCKS_PER_SEC;
    cout <<"2"<<" Timeuse: "<<time<<" (s)"<<endl;

    size = N *20* sizeof(float);
    start=clock();
    hipHostMalloc(&hostdata3, size);
    finish=clock();
    time = (double)(finish-start)/CLOCKS_PER_SEC;
    cout <<"3"<<" Timeuse: "<<time<<" (s)"<<endl;

    size = N *100* sizeof(float);
    start=clock();
    hipHostMalloc(&hostdata4, size);
    finish=clock();
    time = (double)(finish-start)/CLOCKS_PER_SEC;
    cout <<"4"<<" Timeuse: "<<time<<" (s)"<<endl;

    size = N * 500*sizeof(float);
    start=clock();
    hipHostMalloc(&hostdata5, size);
    finish=clock();
    time = (double)(finish-start)/CLOCKS_PER_SEC;
    cout <<"5"<<" Timeuse: "<<time<<" (s)"<<endl;


    //
    cout <<"Add computation between malloc host"<<endl;
    int temp=0;
    for(int i=;i<1000;i++)
    {
    temp++;
    }
    size = N * sizeof(float);
    start=clock();
    hipHostMalloc(&hostdata6, size);
    finish=clock();
    time = (double)(finish-start)/CLOCKS_PER_SEC;
    cout <<"6"<<" Timeuse: "<<time<<" (s)"<<endl;

    for(int i=;i<1000;i++)
    {
    temp++;
    }
    size = N *10 *sizeof(float);
    start=clock();
    hipHostMalloc(&hostdata7, size);
    finish=clock();
    time = (double)(finish-start)/CLOCKS_PER_SEC;
    cout <<"7"<<" Timeuse: "<<time<<" (s)"<<endl;

    for(int i=;i<1000;i++)
    {
    temp++;
    }
    size = N *20* sizeof(float);
    start=clock();
    hipHostMalloc(&hostdata8, size);
    finish=clock();
    time = (double)(finish-start)/CLOCKS_PER_SEC;
    cout <<"8"<<" Timeuse: "<<time<<" (s)"<<endl;

    for(int i=;i<1000;i++)
    {
    temp++;
    }
    size = N *100* sizeof(float);
    start=clock();
    hipHostMalloc(&hostdata9, size);
    finish=clock();
    time = (double)(finish-start)/CLOCKS_PER_SEC;
    cout <<"9"<<" Timeuse: "<<time<<" (s)"<<endl;

    for(int i=;i<1000;i++)
    {
    temp++;
    }
    size = N * 500*sizeof(float);
    start=clock();
    hipHostMalloc(&hostdata0, size);
    finish=clock();
    time = (double)(finish-start)/CLOCKS_PER_SEC;
    cout <<"10"<<" Timeuse: "<<time<<" (s)"<<endl;


    //Free memory
    hipFree(devicedata);

    return 0;
}







