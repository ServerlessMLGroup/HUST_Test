#include "hip/hip_runtime.h"
#include <iostream>
#include <pthread.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "unistd.h"
#include <thread>
#include <mutex>
#include <random>
#include <ctime>
#include <time.h>
#include <stdio.h>
#include<hip/hip_runtime.h>
using namespace std;
#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)

mutex workend2;
mutex workend1;
//diy thread

__global__ void kernel(float n1, float n2, float n3, int stop) {
	for (int i = 0; i < stop; i++) {
		n1=cosf(n1);
		n3=n2/n3;
	}
}


void thread1(hipCtx_t ctx,float* d_a,float* h_a,size_t size,int i)
{
    //set CPU
    /*
    cpu_set_t mask;
    CPU_ZERO(&mask);
    CPU_SET(16, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0)
    {
            perror("pthread_setaffinity_np");
    }
    */

    cout<<"number: "<<i<<"   one thread starts: "<<endl;
    int err;
    err=hipCtxPushCurrent(ctx);
    if(err){
    cout<<"Push Context ERR! "<<err<<endl;
    }

    hipStream_t tempstream;

    hipError_t cudaStatus;
    cudaStatus = hipStreamCreate(&tempstream);
    fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));

    for(int i=1;i < 10;i++)
    {
    //hipMemcpyAsync(d_a, h_a,size, hipMemcpyHostToDevice, tempstream);
    kernel<<<1,1,0,tempstream>>>(1.0,2.0,3.0,10000000);
    }

    hipStreamSynchronize(tempstream);
    workend1.unlock();
}

void thread2(hipCtx_t ctx,int i)
{
    //set CPU
    /*
    cpu_set_t mask;
    CPU_ZERO(&mask);
    CPU_SET(16, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0)
    {
            perror("pthread_setaffinity_np");
    }
    */

    clock_t start,finish;
    double time=0.0;

    cout<<"number: "<<i<<"   one thread starts: "<<endl;
    int err;
    err=hipCtxPushCurrent(ctx);
    if(err){
    cout<<"Push Context ERR! "<<err<<endl;
    }

    float* hostdata1;
    float* hostdata2;
    float* hostdata3;
    float* hostdata4;
    float* hostdata5;

    //allocate locked memory
    int N = 262144;
    size_t size;

    size = N * sizeof(float);
    start=clock();
    hipHostMalloc(&hostdata1, size);
    finish=clock();
    time = (double)(finish-start)/CLOCKS_PER_SEC;
    cout <<"1"<<" Timeuse: "<<time<<" (s)"<<endl;

    size = N *10 *sizeof(float);
    start=clock();
    hipHostMalloc(&hostdata2, size);
    finish=clock();
    time = (double)(finish-start)/CLOCKS_PER_SEC;
    cout <<"2"<<" Timeuse: "<<time<<" (s)"<<endl;

    size = N *20* sizeof(float);
    start=clock();
    hipHostMalloc(&hostdata3, size);
    finish=clock();
    time = (double)(finish-start)/CLOCKS_PER_SEC;
    cout <<"3"<<" Timeuse: "<<time<<" (s)"<<endl;

    size = N *100* sizeof(float);
    start=clock();
    hipHostMalloc(&hostdata4, size);
    finish=clock();
    time = (double)(finish-start)/CLOCKS_PER_SEC;
    cout <<"4"<<" Timeuse: "<<time<<" (s)"<<endl;

    size = N * 500*sizeof(float);
    start=clock();
    hipHostMalloc(&hostdata5, size);
    finish=clock();
    time = (double)(finish-start)/CLOCKS_PER_SEC;
    cout <<"5"<<" Timeuse: "<<time<<" (s)"<<endl;

}

int main()
{

    hipInit(0);
    hipSetDevice(1);

    //synchronize thread 1
    workend1.lock();

    //set cpu
    /*
    cpu_set_t mask;
    CPU_ZERO(&mask);
    CPU_SET(16, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0) {
            perror("pthread_setaffinity_np");
    }
    */

    //Context
    cout<<"Create context"<<endl;
    int err;
    hipCtx_t cont1;
    hipDevice_t dev;
    err = hipCtxGetDevice(&dev);
    if(err)
    {
        cout<<"Can't get device, err" << err<<endl;
        return 0;
    }
    err = hipCtxCreate(&cont1,hipDeviceScheduleYield,dev);
    if(err)
    {
        cout<<"Can't create Context, err" << err << endl;
        return 0;
    }

    //400M data for data transfer
    int N = 262144;
    size_t size = N*400*sizeof(float);

    //allocate device variable(data)
    float* d_A;
    hipMalloc(&d_A, size);

    // Allocate input vectors h_A in host memory
    float* h_A;
    hipHostMalloc(&h_A, size);

    // create thread
    thread first=thread(thread1,cont1,d_A,h_A,size,1);
    thread second=thread(thread2,cont1,2);
    first.join();
    second.join();

    workend1.lock();
    hipFree(d_A);

    return 0;
}
