#include "hip/hip_runtime.h"
//this .cu file is made to test sleep
#include <stdio.h>
#include <stdlib.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)
// nvcc -arch=native ex.cu -o ex_sleep

#define GPU_RETURN_STATUS(cmd) \
{ \
    hipError_t result = cmd; \
    if (result != hipSuccess) { \
        std::cout << #cmd " error, return code:" << result << " | " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); \
    } \
}

// These are the inline versions for all of the SDK helper functions
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    const char *errorStr = NULL;
    errorStr = hipGetErrorString(err);
    fprintf(stderr,
            "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, "
            "line %i.\n",
            err, errorStr, file, line);
    exit(EXIT_FAILURE);
  }
}

__device__ uint get_smid(void) {

    uint ret;

    asm("mov.u32 %0, %smid;" : "=r"(ret) );

    return ret;

}

extern "C" __global__ void fused_nn_contrib_conv2d_winograd_without_weight_transform_add_kernel1(float* __restrict__ placeholder, float* __restrict__ data_pack, float* __restrict__ bgemm, int* flag) {
    unsigned int ns = 5;
    while(atomicAdd(flag, 0) == 0) { // 40us版本
        __nanosleep(ns); // 1us
        if (ns < 1000) {
            ns *= 2;
        }
    }

    float bgemm_local[8];
    __shared__ float placeholder_shared[1024];
    __shared__ float data_pack_shared[256];
    for (int co_c_init = 0; co_c_init < 4; ++co_c_init) {
      for (int p_c_init = 0; p_c_init < 2; ++p_c_init) {
        bgemm_local[(((co_c_init * 2) + p_c_init))] = 0.000000e+00f;
      }
    }
    for (int ci_outer = 0; ci_outer < 32; ++ci_outer) {
      __syncthreads();
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer < 8; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer) {
        placeholder_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 128) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)))] = placeholder[(((((((((int)blockIdx.z) * 262144) + (ci_outer * 8192)) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 1024)) + ((((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) >> 6) * 512)) + (((int)blockIdx.y) * 64)) + (((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) & 63)))];
      }
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer1 < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer1) {
        data_pack_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer1 * 128) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)))] = data_pack[((((((((int)blockIdx.z) * 8192) + (ci_outer * 256)) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer1 * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)))];
      }
      __syncthreads();
      for (int ci_inner = 0; ci_inner < 16; ++ci_inner) {
        for (int co_c = 0; co_c < 4; ++co_c) {
          for (int p_c = 0; p_c < 2; ++p_c) {
            bgemm_local[(((co_c * 2) + p_c))] = (bgemm_local[(((co_c * 2) + p_c))] + (placeholder_shared[((((ci_inner * 64) + (((int)threadIdx.y) * 4)) + co_c))] * data_pack_shared[((((ci_inner * 16) + (((int)threadIdx.x) * 2)) + p_c))]));
          }
        }
      }
    }
    for (int co_inner_inner_inner = 0; co_inner_inner_inner < 4; ++co_inner_inner_inner) {
      for (int p_inner_inner_inner = 0; p_inner_inner_inner < 2; ++p_inner_inner_inner) {
        bgemm[(((((((((int)blockIdx.z) * 8192) + (((int)blockIdx.y) * 1024)) + (((int)threadIdx.y) * 64)) + (co_inner_inner_inner * 16)) + (((int)threadIdx.x) * 2)) + p_inner_inner_inner))] = bgemm_local[(((co_inner_inner_inner * 2) + p_inner_inner_inner))];
      }
    }

}

extern "C" __global__ void fused_nn_contrib_conv2d_winograd_without_weight_transform_add_kernel0_warm(float* __restrict__ placeholder, float* __restrict__ data_pack, int* flag, long long unsigned* times, long long unsigned* sm) {
    // unsigned long long mclk;
	// if (threadIdx.x == 0) {
	// 	asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk));
	// 	times[blockIdx.x] = mclk / 1000;
	// }
    // if (threadIdx.x == 0) {
	// 	sm[blockIdx.x] = get_smid();
	// }

    // clock_t t0 = clock64();
    // clock_t t1 = t0;
    // while ((t1 - t0)/(1530000 * 1000.0f / 1000000) < 20) t1 = clock64(); // 20us, 1530000为kilohertz
    float d[16];
    float data_pack_local[16];
    for (int eps = 0; eps < 4; ++eps) {
        for (int nu = 0; nu < 4; ++nu) {
            d[(((eps * 4) + nu))] = (((((1 <= ((((((int)threadIdx.x) & 15) >> 2) * 2) + eps)) && (((((((int)threadIdx.x) & 15) >> 2) * 2) + eps) < 8)) && (1 <= (((((int)threadIdx.x) & 3) * 2) + nu))) && ((((((int)threadIdx.x) & 3) * 2) + nu) < 8)) ? placeholder[((((((((((int)blockIdx.x) * 392) + ((((int)threadIdx.x) >> 4) * 49)) + (((((int)threadIdx.x) & 15) >> 2) * 14)) + (eps * 7)) + ((((int)threadIdx.x) & 3) * 2)) + nu) - 8))] : 0.000000e+00f);
        }
    }
    data_pack_local[(0)] = 0.000000e+00f;
    data_pack_local[(0)] = (data_pack_local[(0)] + d[(0)]);
    data_pack_local[(0)] = (data_pack_local[(0)] + (d[(2)] * -1.000000e+00f));
    data_pack_local[(0)] = (data_pack_local[(0)] + (d[(8)] * -1.000000e+00f));
    data_pack_local[(0)] = (data_pack_local[(0)] + ((d[(10)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(1)] = 0.000000e+00f;
    data_pack_local[(1)] = (data_pack_local[(1)] + (d[(1)] * -1.000000e+00f));
    data_pack_local[(1)] = (data_pack_local[(1)] + d[(2)]);
    data_pack_local[(1)] = (data_pack_local[(1)] + ((d[(9)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(1)] = (data_pack_local[(1)] + (d[(10)] * -1.000000e+00f));
    data_pack_local[(2)] = 0.000000e+00f;
    data_pack_local[(2)] = (data_pack_local[(2)] + d[(1)]);
    data_pack_local[(2)] = (data_pack_local[(2)] + d[(2)]);
    data_pack_local[(2)] = (data_pack_local[(2)] + (d[(9)] * -1.000000e+00f));
    data_pack_local[(2)] = (data_pack_local[(2)] + (d[(10)] * -1.000000e+00f));
    data_pack_local[(3)] = 0.000000e+00f;
    data_pack_local[(3)] = (data_pack_local[(3)] + (d[(1)] * -1.000000e+00f));
    data_pack_local[(3)] = (data_pack_local[(3)] + d[(3)]);
    data_pack_local[(3)] = (data_pack_local[(3)] + ((d[(9)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(3)] = (data_pack_local[(3)] + (d[(11)] * -1.000000e+00f));
    data_pack_local[(4)] = 0.000000e+00f;
    data_pack_local[(4)] = (data_pack_local[(4)] + (d[(4)] * -1.000000e+00f));
    data_pack_local[(4)] = (data_pack_local[(4)] + ((d[(6)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(4)] = (data_pack_local[(4)] + d[(8)]);
    data_pack_local[(4)] = (data_pack_local[(4)] + (d[(10)] * -1.000000e+00f));
    data_pack_local[(5)] = 0.000000e+00f;
    data_pack_local[(5)] = (data_pack_local[(5)] + ((d[(5)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(5)] = (data_pack_local[(5)] + (d[(6)] * -1.000000e+00f));
    data_pack_local[(5)] = (data_pack_local[(5)] + (d[(9)] * -1.000000e+00f));
    data_pack_local[(5)] = (data_pack_local[(5)] + d[(10)]);
    data_pack_local[(6)] = 0.000000e+00f;
    data_pack_local[(6)] = (data_pack_local[(6)] + (d[(5)] * -1.000000e+00f));
    data_pack_local[(6)] = (data_pack_local[(6)] + (d[(6)] * -1.000000e+00f));
    data_pack_local[(6)] = (data_pack_local[(6)] + d[(9)]);
    data_pack_local[(6)] = (data_pack_local[(6)] + d[(10)]);
    data_pack_local[(7)] = 0.000000e+00f;
    data_pack_local[(7)] = (data_pack_local[(7)] + ((d[(5)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(7)] = (data_pack_local[(7)] + (d[(7)] * -1.000000e+00f));
    data_pack_local[(7)] = (data_pack_local[(7)] + (d[(9)] * -1.000000e+00f));
    data_pack_local[(7)] = (data_pack_local[(7)] + d[(11)]);
    data_pack_local[(8)] = 0.000000e+00f;
    data_pack_local[(8)] = (data_pack_local[(8)] + d[(4)]);
    data_pack_local[(8)] = (data_pack_local[(8)] + (d[(6)] * -1.000000e+00f));
    data_pack_local[(8)] = (data_pack_local[(8)] + d[(8)]);
    data_pack_local[(8)] = (data_pack_local[(8)] + (d[(10)] * -1.000000e+00f));
    data_pack_local[(9)] = 0.000000e+00f;
    data_pack_local[(9)] = (data_pack_local[(9)] + (d[(5)] * -1.000000e+00f));
    data_pack_local[(9)] = (data_pack_local[(9)] + d[(6)]);
    data_pack_local[(9)] = (data_pack_local[(9)] + (d[(9)] * -1.000000e+00f));
    data_pack_local[(9)] = (data_pack_local[(9)] + d[(10)]);
    data_pack_local[(10)] = 0.000000e+00f;
    data_pack_local[(10)] = (data_pack_local[(10)] + d[(5)]);
    data_pack_local[(10)] = (data_pack_local[(10)] + d[(6)]);
    data_pack_local[(10)] = (data_pack_local[(10)] + d[(9)]);
    data_pack_local[(10)] = (data_pack_local[(10)] + d[(10)]);
    data_pack_local[(11)] = 0.000000e+00f;
    data_pack_local[(11)] = (data_pack_local[(11)] + (d[(5)] * -1.000000e+00f));
    data_pack_local[(11)] = (data_pack_local[(11)] + d[(7)]);
    data_pack_local[(11)] = (data_pack_local[(11)] + (d[(9)] * -1.000000e+00f));
    data_pack_local[(11)] = (data_pack_local[(11)] + d[(11)]);
    data_pack_local[(12)] = 0.000000e+00f;
    data_pack_local[(12)] = (data_pack_local[(12)] + (d[(4)] * -1.000000e+00f));
    data_pack_local[(12)] = (data_pack_local[(12)] + ((d[(6)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(12)] = (data_pack_local[(12)] + d[(12)]);
    data_pack_local[(12)] = (data_pack_local[(12)] + (d[(14)] * -1.000000e+00f));
    data_pack_local[(13)] = 0.000000e+00f;
    data_pack_local[(13)] = (data_pack_local[(13)] + ((d[(5)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(13)] = (data_pack_local[(13)] + (d[(6)] * -1.000000e+00f));
    data_pack_local[(13)] = (data_pack_local[(13)] + (d[(13)] * -1.000000e+00f));
    data_pack_local[(13)] = (data_pack_local[(13)] + d[(14)]);
    data_pack_local[(14)] = 0.000000e+00f;
    data_pack_local[(14)] = (data_pack_local[(14)] + (d[(5)] * -1.000000e+00f));
    data_pack_local[(14)] = (data_pack_local[(14)] + (d[(6)] * -1.000000e+00f));
    data_pack_local[(14)] = (data_pack_local[(14)] + d[(13)]);
    data_pack_local[(14)] = (data_pack_local[(14)] + d[(14)]);
    data_pack_local[(15)] = 0.000000e+00f;
    data_pack_local[(15)] = (data_pack_local[(15)] + ((d[(5)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(15)] = (data_pack_local[(15)] + (d[(7)] * -1.000000e+00f));
    data_pack_local[(15)] = (data_pack_local[(15)] + (d[(13)] * -1.000000e+00f));
    data_pack_local[(15)] = (data_pack_local[(15)] + d[(15)]);
    for (int eps1 = 0; eps1 < 4; ++eps1) {
        for (int nu1 = 0; nu1 < 4; ++nu1) {
        data_pack[(((((eps1 * 32768) + (nu1 * 8192)) + (((int)blockIdx.x) * 128)) + ((int)threadIdx.x)))] = data_pack_local[(((eps1 * 4) + nu1))];
        }
    }

    // __syncthreads(); //new
    // unsigned long long mclk2;
	// if (threadIdx.x == 0) {
	// 	asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk2));
	// 	times[blockIdx.x + 64] = mclk2 / 1000;
	// }
    atomicAdd(flag + threadIdx.x + blockIdx.x * 128, 1);
    // flag[threadIdx.x + blockIdx.x * 128] = 1;
    // flag[0] = 1;

}

extern "C" __global__ void fused_nn_contrib_conv2d_winograd_without_weight_transform_add_kernel0(float* __restrict__ placeholder, float* __restrict__ data_pack, int* flag, long long unsigned* times, long long unsigned* sm) {
    // unsigned long long mclk;
	// if (threadIdx.x == 0) {
	// 	asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk));
	// 	times[blockIdx.x] = mclk / 1000;
	// }
    // if (threadIdx.x == 0) {
	// 	sm[blockIdx.x] = get_smid();
	// }

    // clock_t t0 = clock64();
    // clock_t t1 = t0;
    // while ((t1 - t0)/(1530000 * 1000.0f / 1000000) < 20) t1 = clock64(); // 20us, 1530000为kilohertz
    float d[16];
    float data_pack_local[16];
    for (int eps = 0; eps < 4; ++eps) {
        for (int nu = 0; nu < 4; ++nu) {
            d[(((eps * 4) + nu))] = (((((1 <= ((((((int)threadIdx.x) & 15) >> 2) * 2) + eps)) && (((((((int)threadIdx.x) & 15) >> 2) * 2) + eps) < 8)) && (1 <= (((((int)threadIdx.x) & 3) * 2) + nu))) && ((((((int)threadIdx.x) & 3) * 2) + nu) < 8)) ? placeholder[((((((((((int)blockIdx.x) * 392) + ((((int)threadIdx.x) >> 4) * 49)) + (((((int)threadIdx.x) & 15) >> 2) * 14)) + (eps * 7)) + ((((int)threadIdx.x) & 3) * 2)) + nu) - 8))] : 0.000000e+00f);
        }
    }
    data_pack_local[(0)] = 0.000000e+00f;
    data_pack_local[(0)] = (data_pack_local[(0)] + d[(0)]);
    data_pack_local[(0)] = (data_pack_local[(0)] + (d[(2)] * -1.000000e+00f));
    data_pack_local[(0)] = (data_pack_local[(0)] + (d[(8)] * -1.000000e+00f));
    data_pack_local[(0)] = (data_pack_local[(0)] + ((d[(10)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(1)] = 0.000000e+00f;
    data_pack_local[(1)] = (data_pack_local[(1)] + (d[(1)] * -1.000000e+00f));
    data_pack_local[(1)] = (data_pack_local[(1)] + d[(2)]);
    data_pack_local[(1)] = (data_pack_local[(1)] + ((d[(9)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(1)] = (data_pack_local[(1)] + (d[(10)] * -1.000000e+00f));
    data_pack_local[(2)] = 0.000000e+00f;
    data_pack_local[(2)] = (data_pack_local[(2)] + d[(1)]);
    data_pack_local[(2)] = (data_pack_local[(2)] + d[(2)]);
    data_pack_local[(2)] = (data_pack_local[(2)] + (d[(9)] * -1.000000e+00f));
    data_pack_local[(2)] = (data_pack_local[(2)] + (d[(10)] * -1.000000e+00f));
    data_pack_local[(3)] = 0.000000e+00f;
    data_pack_local[(3)] = (data_pack_local[(3)] + (d[(1)] * -1.000000e+00f));
    data_pack_local[(3)] = (data_pack_local[(3)] + d[(3)]);
    data_pack_local[(3)] = (data_pack_local[(3)] + ((d[(9)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(3)] = (data_pack_local[(3)] + (d[(11)] * -1.000000e+00f));
    data_pack_local[(4)] = 0.000000e+00f;
    data_pack_local[(4)] = (data_pack_local[(4)] + (d[(4)] * -1.000000e+00f));
    data_pack_local[(4)] = (data_pack_local[(4)] + ((d[(6)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(4)] = (data_pack_local[(4)] + d[(8)]);
    data_pack_local[(4)] = (data_pack_local[(4)] + (d[(10)] * -1.000000e+00f));
    data_pack_local[(5)] = 0.000000e+00f;
    data_pack_local[(5)] = (data_pack_local[(5)] + ((d[(5)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(5)] = (data_pack_local[(5)] + (d[(6)] * -1.000000e+00f));
    data_pack_local[(5)] = (data_pack_local[(5)] + (d[(9)] * -1.000000e+00f));
    data_pack_local[(5)] = (data_pack_local[(5)] + d[(10)]);
    data_pack_local[(6)] = 0.000000e+00f;
    data_pack_local[(6)] = (data_pack_local[(6)] + (d[(5)] * -1.000000e+00f));
    data_pack_local[(6)] = (data_pack_local[(6)] + (d[(6)] * -1.000000e+00f));
    data_pack_local[(6)] = (data_pack_local[(6)] + d[(9)]);
    data_pack_local[(6)] = (data_pack_local[(6)] + d[(10)]);
    data_pack_local[(7)] = 0.000000e+00f;
    data_pack_local[(7)] = (data_pack_local[(7)] + ((d[(5)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(7)] = (data_pack_local[(7)] + (d[(7)] * -1.000000e+00f));
    data_pack_local[(7)] = (data_pack_local[(7)] + (d[(9)] * -1.000000e+00f));
    data_pack_local[(7)] = (data_pack_local[(7)] + d[(11)]);
    data_pack_local[(8)] = 0.000000e+00f;
    data_pack_local[(8)] = (data_pack_local[(8)] + d[(4)]);
    data_pack_local[(8)] = (data_pack_local[(8)] + (d[(6)] * -1.000000e+00f));
    data_pack_local[(8)] = (data_pack_local[(8)] + d[(8)]);
    data_pack_local[(8)] = (data_pack_local[(8)] + (d[(10)] * -1.000000e+00f));
    data_pack_local[(9)] = 0.000000e+00f;
    data_pack_local[(9)] = (data_pack_local[(9)] + (d[(5)] * -1.000000e+00f));
    data_pack_local[(9)] = (data_pack_local[(9)] + d[(6)]);
    data_pack_local[(9)] = (data_pack_local[(9)] + (d[(9)] * -1.000000e+00f));
    data_pack_local[(9)] = (data_pack_local[(9)] + d[(10)]);
    data_pack_local[(10)] = 0.000000e+00f;
    data_pack_local[(10)] = (data_pack_local[(10)] + d[(5)]);
    data_pack_local[(10)] = (data_pack_local[(10)] + d[(6)]);
    data_pack_local[(10)] = (data_pack_local[(10)] + d[(9)]);
    data_pack_local[(10)] = (data_pack_local[(10)] + d[(10)]);
    data_pack_local[(11)] = 0.000000e+00f;
    data_pack_local[(11)] = (data_pack_local[(11)] + (d[(5)] * -1.000000e+00f));
    data_pack_local[(11)] = (data_pack_local[(11)] + d[(7)]);
    data_pack_local[(11)] = (data_pack_local[(11)] + (d[(9)] * -1.000000e+00f));
    data_pack_local[(11)] = (data_pack_local[(11)] + d[(11)]);
    data_pack_local[(12)] = 0.000000e+00f;
    data_pack_local[(12)] = (data_pack_local[(12)] + (d[(4)] * -1.000000e+00f));
    data_pack_local[(12)] = (data_pack_local[(12)] + ((d[(6)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(12)] = (data_pack_local[(12)] + d[(12)]);
    data_pack_local[(12)] = (data_pack_local[(12)] + (d[(14)] * -1.000000e+00f));
    data_pack_local[(13)] = 0.000000e+00f;
    data_pack_local[(13)] = (data_pack_local[(13)] + ((d[(5)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(13)] = (data_pack_local[(13)] + (d[(6)] * -1.000000e+00f));
    data_pack_local[(13)] = (data_pack_local[(13)] + (d[(13)] * -1.000000e+00f));
    data_pack_local[(13)] = (data_pack_local[(13)] + d[(14)]);
    data_pack_local[(14)] = 0.000000e+00f;
    data_pack_local[(14)] = (data_pack_local[(14)] + (d[(5)] * -1.000000e+00f));
    data_pack_local[(14)] = (data_pack_local[(14)] + (d[(6)] * -1.000000e+00f));
    data_pack_local[(14)] = (data_pack_local[(14)] + d[(13)]);
    data_pack_local[(14)] = (data_pack_local[(14)] + d[(14)]);
    data_pack_local[(15)] = 0.000000e+00f;
    data_pack_local[(15)] = (data_pack_local[(15)] + ((d[(5)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(15)] = (data_pack_local[(15)] + (d[(7)] * -1.000000e+00f));
    data_pack_local[(15)] = (data_pack_local[(15)] + (d[(13)] * -1.000000e+00f));
    data_pack_local[(15)] = (data_pack_local[(15)] + d[(15)]);
    for (int eps1 = 0; eps1 < 4; ++eps1) {
        for (int nu1 = 0; nu1 < 4; ++nu1) {
        data_pack[(((((eps1 * 32768) + (nu1 * 8192)) + (((int)blockIdx.x) * 128)) + ((int)threadIdx.x)))] = data_pack_local[(((eps1 * 4) + nu1))];
        }
    }

    // __syncthreads(); //new
    // unsigned long long mclk2;
	// if (threadIdx.x == 0) {
	// 	asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk2));
	// 	times[blockIdx.x + 64] = mclk2 / 1000;
	// }
    // atomicAdd(flag + threadIdx.x + blockIdx.x * 128, 1);
    // flag[threadIdx.x + blockIdx.x * 128] = 1;
    // flag[0] = 1;
    atomicAdd(flag + 0, 1);

}

__global__ void kernel_sleep(float n1, float n2, float n3, int stop, int* flag) {
    // #if __CUDA_ARCH__ >= 700
    // while(flag[0] != 1) {
    //     __nanosleep(1000); // 1us
    // }
    // #else
    // printf(">>> __CUDA_ARCH__ !\n");
    // #endif
    for (int i = 0; i < stop; i++) {
        n1=sinf(n1);
        n2=n3/n2;
    }

}


void run_kernel() {
	int num_streams = 2;
	hipStream_t streams[num_streams];
	for (int i = 0; i < num_streams; i++) {
		hipStreamCreate(&streams[i]);
	}




    // allocate resource
	long long unsigned *h_sm_ids = new long long unsigned[64 * 2];
	long long unsigned *d_sm_ids;
	hipMalloc(&d_sm_ids, 64 * sizeof(long long unsigned) * 2);

	long long unsigned *h_sm_ids2 = new long long unsigned[128 * 2];
	long long unsigned *d_sm_ids2;
	hipMalloc(&d_sm_ids2, 128 * sizeof(long long unsigned) * 2);

    float *h_args_55 = new float[25088]; // 55
    float *d_args_55;
    hipMalloc(&d_args_55, sizeof(float) * 25088);

    size_t size56 = 4194304*sizeof(float);
    float *h_args_56;
    hipHostMalloc(&h_args_56, size56);
    float *d_args_56;
    hipMalloc(&d_args_56, size56);

    size_t size75 = 451585*sizeof(float);
    float *h_args_75;
    hipHostMalloc(&h_args_75, size75);
    float *d_args_75;
    hipMalloc(&d_args_75, size75);

    size_t size76 = 451585*sizeof(float);
    float *h_args_76;
    hipHostMalloc(&h_args_76, size76);
    float *d_args_76;
    hipMalloc(&d_args_76, size76);




    // allocate flag
    int *flag ;
    size_t size = 9000*sizeof(int);
    hipHostMalloc(&flag, size);
    int *g_flag;
    hipMalloc((void **)&g_flag, size);

    for (int i = 0; i < 9000; ++i) {
        flag[i] = 0;
    }
    hipMemcpyAsync(g_flag, flag,size, hipMemcpyHostToDevice, streams[0]);
    hipDeviceSynchronize();

    // allocate flag
    int *sleep_times;
    int *g_sleep_times;
    sleep_times = (int*) malloc(1 * sizeof(int));
    sleep_times[0] = 0;
    hipMalloc((void **)&g_sleep_times, sizeof(int) * 1);
    hipMemcpy(g_sleep_times, sleep_times, sizeof(int) * 1, hipMemcpyHostToDevice);

	// // allocate sleep_time
	// long long unsigned *h_sleep_time = new long long unsigned[b_blocks];
	// long long unsigned *d_sleep_time;
	// hipMalloc(&d_sleep_time, b_blocks * sizeof(long long unsigned));

	// allocate kernel_sleep sm
	long long unsigned *h_sleep_sm = new long long unsigned[128];
	long long unsigned *d_sleep_sm;
	hipMalloc(&d_sleep_sm, 128 * sizeof(long long unsigned));

    // allocate kernel_sleep sm
	long long unsigned *h_sm = new long long unsigned[64];
	long long unsigned *d_sm;
	hipMalloc(&d_sm, 64 * sizeof(long long unsigned));


    // allocate warm flag
    int *flag_warm;
    int *g_flag_warm;
    flag_warm = (int*) malloc(1 * sizeof(int));
    flag_warm[0] = 0;
    hipMalloc((void **)&g_flag_warm, sizeof(int) * 9000);
    hipMemcpy(g_flag_warm, flag_warm, sizeof(int) * 9000, hipMemcpyHostToDevice);

    //change host flag
    for (int i = 0; i < 2; ++i) {
        flag[i] = 1;
    }
    size = 1*sizeof(int);

    

    // cuda launch kernel
	dim3 D_b_a = dim3(64, 1, 1);
	dim3 D_t_a = dim3(128, 1, 1);
	dim3 D_b_b = dim3(1, 8, 16);
	dim3 D_t_b = dim3(8, 16, 1);
    // warm-up
    for (int i = 0; i < 100; ++i) {
        fused_nn_contrib_conv2d_winograd_without_weight_transform_add_kernel0_warm <<<D_b_a, D_t_a, 0, streams[0]>>>(d_args_55, d_args_76, g_flag_warm, d_sm_ids, d_sm);
    }
	hipDeviceSynchronize();


    // sleep until flag changes
	fused_nn_contrib_conv2d_winograd_without_weight_transform_add_kernel1 <<<D_b_b, D_t_b, 0, streams[1]>>>(d_args_56, d_args_76, d_args_75, g_flag);
    hipMemcpyAsync(d_args_56, h_args_56,size56, hipMemcpyHostToDevice, streams[0]);
    hipMemcpyAsync(d_args_76, h_args_76,size76, hipMemcpyHostToDevice, streams[0]);
    hipMemcpyAsync(d_args_75, h_args_75,size75, hipMemcpyHostToDevice, streams[0]);
    hipMemcpyAsync(g_flag, flag,size, hipMemcpyHostToDevice, streams[0]);
	hipDeviceSynchronize();

    hipMemcpy(h_sm_ids, d_sm_ids, 64 * sizeof(long long unsigned) * 2, hipMemcpyDeviceToHost);
	hipMemcpy(h_sm_ids2, d_sm_ids2, 128 * sizeof(long long unsigned) * 2, hipMemcpyDeviceToHost);


	hipMemcpy(sleep_times, g_sleep_times, 1 * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_sleep_sm, d_sleep_sm, 128 * sizeof(long long unsigned), hipMemcpyDeviceToHost);
    hipMemcpy(h_sm, d_sm, 64 * sizeof(long long unsigned), hipMemcpyDeviceToHost);

    hipMemcpy(flag, g_flag, sizeof(int) * 9000, hipMemcpyDeviceToHost);

	long long unsigned maxm = 0, minm = 1768959725180341, max1 = 0, max2=0, min2=1768959725180341;
	long long unsigned maxm_e = 0, minm_e = 1768959725180341;
    printf("---1---\n");
	for (int i = 0; i < 64; i++) {
        // printf("%llu-%llu\n", h_sm_ids[i], h_sm_ids[i + 64]);
        maxm = max(maxm, h_sm_ids[i]);
        minm = min(minm, h_sm_ids[i]);
		maxm_e = max(maxm_e, h_sm_ids[i + 64]);
        minm_e = min(minm_e, h_sm_ids[i + 64]);
	    max1 = max(max1, h_sm_ids[i + 64] - h_sm_ids[i]);
	}
    printf("START_TIMING:max-%llu, min-%llu(us)\n", maxm, minm);
	printf("END_TIMING__:max-%llu, min-%llu(us)\n", maxm_e, minm_e);
	printf("DURATION:单block最大执行时间%llu(us)\n", max1);

	maxm = 0; minm = 1768959725180341;
	maxm_e = 0; minm_e = 1768959725180341;
	printf("---2---\n");
	for (int i = 0; i < 128; i++) {
		// printf("blcok%d:%llu-%llu   %llu \n",i, h_sm_ids2[i], h_sm_ids2[i + a_blocks] , h_sm_ids2[i + b_blocks]-h_sm_ids2[i]);
        // printf("%llu-%llu\n", h_sm_ids2[i], h_sm_ids2[i + 128]);
        maxm = max(maxm, h_sm_ids2[i]);
        minm = min(minm, h_sm_ids2[i]);
		maxm_e = max(maxm_e, h_sm_ids2[i + 128]);
        minm_e = min(minm_e, h_sm_ids2[i + 128]);
	    max2 = max(max2, h_sm_ids2[i + 128]-h_sm_ids2[i]);
	    min2 = min(min2, h_sm_ids2[i + 128]-h_sm_ids2[i]);
	}
    printf("START_TIMING:max-%llu, min-%llu(us)\n", maxm, minm);
	printf("END_TIMING__:max-%llu, min-%llu(us)\n", maxm_e, minm_e);
	printf("DURATION:单block最大执行时间%llu(us)  单block最大执行时间与最小的时间差%llu(us)\n", max2, max2 - min2);

	// printf("---sleep_times---\n");
	// for (int i = 0; i < b_blocks; i++) {
	// 	printf("block-%d : %llu\n", i, h_sleep_time[i]);
	// }

    printf("---first_sm---\n");
	for (int i = 0; i < 64; ++i) {
		//printf("block-%d %llu\n", i, h_sm[i]);
	}

	printf("---second_sm---\n");
	for (int i = 0; i < 128; ++i) {
		//printf("block-%d %llu\n", i, h_sleep_sm[i]);
	}
    printf("kernel1 sleep time:%d\n", sleep_times[0]);
    int total = 0;
    for (int i = 0; i < 64 * 128; ++i) {
        total += flag[i];
    }
    printf("total flag:%d\n", total);

	// hipFree(d_sm_ids);
	// hipFree(d_sm_ids2);

}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        printf("args num error! argc:%d", argc);
        exit(1);
    }
    int gpu_no = atoi(argv[1]);
    checkCudaErrors(hipSetDevice(gpu_no));

    // hipDeviceProp_t  prop;
    // hipGetDeviceProperties(&prop, 0);
    // clock_t clock_rate = prop.clockRate;
    // printf("clock_rate:%d\n", clock_rate); // 1530000
	run_kernel();

	return 0;
}

