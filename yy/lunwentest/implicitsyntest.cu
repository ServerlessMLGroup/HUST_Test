#include "hip/hip_runtime.h"
#define GPU_RETURN_STATUS(cmd) \
{ \
    hipError_t result = cmd; \
    if (result != hipSuccess) { \
        std::cout << #cmd " error, return code:" << result << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); \
    } \
}

__device__ int get_task(int* task_pool) {
    int* mutex = task_pool + POOL_MUTEX_BASE, capacity_ = task_pool[POOL_CAPACITY_BASE], *head_ = task_pool + POOL_HEAD_BASE, *tail_ = task_pool + POOL_TAIL_BASE, *tasks_ = task_pool + POOL_ARRAY_BASE;
    unsigned int ns = 10;
    while (atomicCAS(mutex, 0, 1) != 0) {
      // if (ns < 100) {
      //   ns += 10;
      // }
      __nanosleep(ns);
    };
    if ((*head_) == (*tail_)) {
        atomicExch(mutex, 0);
        return -1;
    }
    int task_id = tasks_[(*tail_)];
    (*tail_) = ((*tail_) + 1) % capacity_;
    atomicExch(mutex, 0);
    return task_id;
}

extern "C" __global__ void fused_nn_contrib_conv2d_winograd_without_weight_transform_add_2_kernel2(float* __restrict__ bgemm, float* __restrict__ T_add, float* __restrict__ placeholder) {
  float inverse[4];
  inverse[(0)] = 0.000000e+00f;
  inverse[(0)] = (inverse[(0)] + bgemm[(((((int)blockIdx.x) * 128) + ((int)threadIdx.x)))]);
  inverse[(0)] = (inverse[(0)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 25088))]);
  inverse[(0)] = (inverse[(0)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 50176))]);
  inverse[(0)] = (inverse[(0)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 100352))]);
  inverse[(0)] = (inverse[(0)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 125440))]);
  inverse[(0)] = (inverse[(0)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 150528))]);
  inverse[(0)] = (inverse[(0)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 200704))]);
  inverse[(0)] = (inverse[(0)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 225792))]);
  inverse[(0)] = (inverse[(0)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 250880))]);
  inverse[(1)] = 0.000000e+00f;
  inverse[(1)] = (inverse[(1)] + (bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 25088))] * -1.000000e+00f));
  inverse[(1)] = (inverse[(1)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 50176))]);
  inverse[(1)] = (inverse[(1)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 75264))]);
  inverse[(1)] = (inverse[(1)] + (bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 125440))] * -1.000000e+00f));
  inverse[(1)] = (inverse[(1)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 150528))]);
  inverse[(1)] = (inverse[(1)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 175616))]);
  inverse[(1)] = (inverse[(1)] + (bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 225792))] * -1.000000e+00f));
  inverse[(1)] = (inverse[(1)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 250880))]);
  inverse[(1)] = (inverse[(1)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 275968))]);
  inverse[(2)] = 0.000000e+00f;
  inverse[(2)] = (inverse[(2)] + (bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 100352))] * -1.000000e+00f));
  inverse[(2)] = (inverse[(2)] + (bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 125440))] * -1.000000e+00f));
  inverse[(2)] = (inverse[(2)] + (bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 150528))] * -1.000000e+00f));
  inverse[(2)] = (inverse[(2)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 200704))]);
  inverse[(2)] = (inverse[(2)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 225792))]);
  inverse[(2)] = (inverse[(2)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 250880))]);
  inverse[(2)] = (inverse[(2)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 301056))]);
  inverse[(2)] = (inverse[(2)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 326144))]);
  inverse[(2)] = (inverse[(2)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 351232))]);
  inverse[(3)] = 0.000000e+00f;
  inverse[(3)] = (inverse[(3)] + ((bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 125440))] * -1.000000e+00f) * -1.000000e+00f));
  inverse[(3)] = (inverse[(3)] + (bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 150528))] * -1.000000e+00f));
  inverse[(3)] = (inverse[(3)] + (bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 175616))] * -1.000000e+00f));
  inverse[(3)] = (inverse[(3)] + (bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 225792))] * -1.000000e+00f));
  inverse[(3)] = (inverse[(3)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 250880))]);
  inverse[(3)] = (inverse[(3)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 275968))]);
  inverse[(3)] = (inverse[(3)] + (bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 326144))] * -1.000000e+00f));
  inverse[(3)] = (inverse[(3)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 351232))]);
  inverse[(3)] = (inverse[(3)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 376320))]);
  for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
    for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
      T_add[((((((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) / 14) * 56) + (ax2_inner * 28)) + ((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) % 14) * 2)) + ax3_inner))] = (inverse[(((ax2_inner * 2) + ax3_inner))] + placeholder[((((((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) / 14) * 56) + (ax2_inner * 28)) + ((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) % 14) * 2)) + ax3_inner))]);
    }
  }
}

int main(int argc, char **argv) {
    if (argc < 2) {
        printf("args num error! argc:%d", argc);
    }
    int gpu_no = atoi(argv[1]);
    log("preate unique_ptr");
    hipCtx_t ctx;
    hipDevice_t device;
    hipError_t result;
    // init CUDA driver API
    GPU_RETURN_STATUS(hipInit(0));
    GPU_RETURN_STATUS(hipDeviceGet(&device, gpu_no));
    GPU_RETURN_STATUS(hipCtxCreate(&ctx, 0, device));

    //yy add stream
    hipStream_t firststream;
    hipStreamCreateWithFlags(&firststream,0);
    hipStream_t secondstream;
    hipStreamCreateWithFlags(&secondstream,0);
    //add fininshed

    std::vector<hipDeviceptr_t*> kernel_arg;
    //flag
    size_t storage_size1 = 1*sizeof(float);
    hipDeviceptr_t device_ptr1;
    GPU_RETURN_STATUS(hipMalloc((hipDeviceptr_t*)&device_ptr1, storage_size1));
    kernel_arg.push_back(&device_ptr1);

    //int block normal
    size_t storage_size2 = 1*sizeof(int);
    hipDeviceptr_t device_ptr2;
    GPU_RETURN_STATUS(hipMalloc((hipDeviceptr_t*)&device_ptr2, storage_size2));
    kernel_arg.push_back(&device_ptr2);

    //75
    size_t storage_size3 = 1806336;
    hipDeviceptr_t device_ptr3;
    GPU_RETURN_STATUS(hipMalloc((hipDeviceptr_t*)&device_ptr3, storage_size3));
    kernel_arg.push_back(&device_ptr3);

    //29
    size_t storage_size4 = 100352*sizeof(float);
    hipDeviceptr_t device_ptr4;
    GPU_RETURN_STATUS(hipMalloc((hipDeviceptr_t*)&device_ptr4, storage_size4));
    kernel_arg.push_back(&device_ptr4);

    //28
    size_t storage_size5 = 128*sizeof(float);
    hipDeviceptr_t device_ptr5;
    GPU_RETURN_STATUS(hipMalloc((hipDeviceptr_t*)&device_ptr5, storage_size5));
    kernel_arg.push_back(&device_ptr5);

    hipModule_t mod;
    GPU_RETURN_STATUS(hipModuleLoad(&mod, "/home/wuhao/HUST_Test/djx/json2kernel/resource/resnet18.ptx"));
    printf("load cuda kernels!\n");



    hipStreamSynchronize(firststream);
    std::string& func_name = "fused_nn_contrib_conv2d_winograd_without_weight_transform_add_2_kernel2";
    hipFunction_t kernel;
    GPU_RETURN_STATUS(
        hipModuleGetFunction(&kernel, mod, kernel_info.name.c_str())
    );


    hipFunction_t func = kernels[func_name];
    uint32_t *launch_params = kernel_info.launch_params;
    GPU_RETURN_STATUS(hipModuleLaunchKernel(func,
    launch_params[0], launch_params[1], launch_params[2],
    launch_params[3], launch_params[4], launch_params[5],
    0, secondstream, (void **)raw_args[j].data(), 0 // raw_args是json中指示的storage的下标
));


    hipStreamSynchronize(firststream);

    return 0;
}

