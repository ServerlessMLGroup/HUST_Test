#include "hip/hip_runtime.h"

__device__ int get_task(int* task_pool) {
    int* mutex = task_pool + POOL_MUTEX_BASE, capacity_ = task_pool[POOL_CAPACITY_BASE], *head_ = task_pool + POOL_HEAD_BASE, *tail_ = task_pool + POOL_TAIL_BASE, *tasks_ = task_pool + POOL_ARRAY_BASE;
    unsigned int ns = 10;
    while (atomicCAS(mutex, 0, 1) != 0) {
      // if (ns < 100) {
      //   ns += 10;
      // }
      __nanosleep(ns);
    };
    if ((*head_) == (*tail_)) {
        atomicExch(mutex, 0);
        return -1;
    }
    int task_id = tasks_[(*tail_)];
    (*tail_) = ((*tail_) + 1) % capacity_;
    atomicExch(mutex, 0);
    return task_id;
}

extern "C" __global__ void fused_nn_contrib_conv2d_winograd_without_weight_transform_add_2_kernel2(float* __restrict__ bgemm, float* __restrict__ T_add, float* __restrict__ placeholder) {
  float inverse[4];
  inverse[(0)] = 0.000000e+00f;
  inverse[(0)] = (inverse[(0)] + bgemm[(((((int)blockIdx.x) * 128) + ((int)threadIdx.x)))]);
  inverse[(0)] = (inverse[(0)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 25088))]);
  inverse[(0)] = (inverse[(0)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 50176))]);
  inverse[(0)] = (inverse[(0)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 100352))]);
  inverse[(0)] = (inverse[(0)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 125440))]);
  inverse[(0)] = (inverse[(0)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 150528))]);
  inverse[(0)] = (inverse[(0)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 200704))]);
  inverse[(0)] = (inverse[(0)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 225792))]);
  inverse[(0)] = (inverse[(0)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 250880))]);
  inverse[(1)] = 0.000000e+00f;
  inverse[(1)] = (inverse[(1)] + (bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 25088))] * -1.000000e+00f));
  inverse[(1)] = (inverse[(1)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 50176))]);
  inverse[(1)] = (inverse[(1)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 75264))]);
  inverse[(1)] = (inverse[(1)] + (bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 125440))] * -1.000000e+00f));
  inverse[(1)] = (inverse[(1)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 150528))]);
  inverse[(1)] = (inverse[(1)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 175616))]);
  inverse[(1)] = (inverse[(1)] + (bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 225792))] * -1.000000e+00f));
  inverse[(1)] = (inverse[(1)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 250880))]);
  inverse[(1)] = (inverse[(1)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 275968))]);
  inverse[(2)] = 0.000000e+00f;
  inverse[(2)] = (inverse[(2)] + (bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 100352))] * -1.000000e+00f));
  inverse[(2)] = (inverse[(2)] + (bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 125440))] * -1.000000e+00f));
  inverse[(2)] = (inverse[(2)] + (bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 150528))] * -1.000000e+00f));
  inverse[(2)] = (inverse[(2)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 200704))]);
  inverse[(2)] = (inverse[(2)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 225792))]);
  inverse[(2)] = (inverse[(2)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 250880))]);
  inverse[(2)] = (inverse[(2)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 301056))]);
  inverse[(2)] = (inverse[(2)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 326144))]);
  inverse[(2)] = (inverse[(2)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 351232))]);
  inverse[(3)] = 0.000000e+00f;
  inverse[(3)] = (inverse[(3)] + ((bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 125440))] * -1.000000e+00f) * -1.000000e+00f));
  inverse[(3)] = (inverse[(3)] + (bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 150528))] * -1.000000e+00f));
  inverse[(3)] = (inverse[(3)] + (bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 175616))] * -1.000000e+00f));
  inverse[(3)] = (inverse[(3)] + (bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 225792))] * -1.000000e+00f));
  inverse[(3)] = (inverse[(3)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 250880))]);
  inverse[(3)] = (inverse[(3)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 275968))]);
  inverse[(3)] = (inverse[(3)] + (bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 326144))] * -1.000000e+00f));
  inverse[(3)] = (inverse[(3)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 351232))]);
  inverse[(3)] = (inverse[(3)] + bgemm[((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 376320))]);
  for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
    for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
      T_add[((((((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) / 14) * 56) + (ax2_inner * 28)) + ((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) % 14) * 2)) + ax3_inner))] = (inverse[(((ax2_inner * 2) + ax3_inner))] + placeholder[((((((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) / 14) * 56) + (ax2_inner * 28)) + ((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) % 14) * 2)) + ax3_inner))]);
    }
  }
}