#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <thread>
#include <random>
#include <ctime>
using namespace std;


enum class Unit{
    Byte, KB, MB, GB, TB, PB, EB
};


double convert(double size, Unit unit)
{
    double result = size;
    switch (unit)
    {
    case Unit::EB:
        result /= 1024;     // flow through
    case Unit::PB:
        result /= 1024;     // flow through
    case Unit::TB:
        result /= 1024;     // flow through
    case Unit::GB:
        result /= 1024;     // flow through
    case Unit::MB:
        result /= 1024;     // flow through
    case Unit::KB:
        result /= 1024;     // flow through
    case Unit::Byte:
        result /= 1;
    default:
        break;
    }
    return result;
}

void getMem() {
    size_t free, total;
    hipMemGetInfo(&free, &total);
    printf("Free mem = %.4f MB, Total = %.4f MB \n", convert(free, Unit::MB), convert(total, Unit::MB));
}

void getMembycu() {
    size_t free, total;
    int err = hipMemGetInfo(&free, &total);
    if (err) {
        cout<<"getMembycu error:"<<err<<endl;
    }
    else {
        printf("Free mem = %.4f MB, Total = %.4f MB \n", convert(free, Unit::MB), convert(total, Unit::MB));
    }
}


__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}


int main()
{
    export CUDA_MPS_PINNED_DEVICE_MEM_LIMIT="2=512MB";
    //putenv("CUDA_MPS_PINNED_DEVICE_MEM_LIMIT=1G");
    hipSetDevice(2);
    hipCtx_t pctx;
    hipDevice_t dev;

    int err = hipCtxGetDevice(&dev);
    if(err){
        cout<<"hipCtxGetDevice error:"<<err<<endl;
        return 0;
    }

    getMem();

    //1048576 -> 1M
    size_t storage_size = 1048576*2000;
    float* h_A;
    hipHostMalloc(&h_A, storage_size);

    //nan dao cu driver api bu xing?
    /*
    hipDeviceptr_t device_ptr;
    int i=hipMalloc((hipDeviceptr_t*)&device_ptr, storage_size);
    */
    float* device_ptr;
    hipMalloc(&device_ptr,storage_size);



    if(i)
    {
    cout<<"error: "<<i<<endl;
    }
    hipStream_t firststream;
    hipStreamCreateWithFlags(&firststream,0);
    i=hipMemcpyHtoDAsync((hipDeviceptr_t)device_ptr,h_A,storage_size,firststream);
    if(i)
    {
    cout<<"error: "<<i<<endl;
    }
    hipDeviceSynchronize();

    getMem();

    /*
    getMem();
    cout<<"after first new context:"<<endl;
    err = hipCtxCreate(&pctx, hipDeviceScheduleYield, dev);
    if(err) {
        cout<<"hipCtxCreate error:"<<err<<endl;
        return 0;
    }
    getMem();

    cout<<"after second new context:"<<endl;
    err = hipCtxCreate(&pctx, hipDeviceScheduleYield, dev);
    if(err) {
        cout<<"hipCtxCreate error:"<<err<<endl;
        return 0;
    }
    getMem();
    */

    return 0;

}
