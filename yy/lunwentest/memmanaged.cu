#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <thread>
#include <random>
#include <ctime>
using namespace std;


enum class Unit{
    Byte, KB, MB, GB, TB, PB, EB
};


double convert(double size, Unit unit)
{
    double result = size;
    switch (unit)
    {
    case Unit::EB:
        result /= 1024;     // flow through
    case Unit::PB:
        result /= 1024;     // flow through
    case Unit::TB:
        result /= 1024;     // flow through
    case Unit::GB:
        result /= 1024;     // flow through
    case Unit::MB:
        result /= 1024;     // flow through
    case Unit::KB:
        result /= 1024;     // flow through
    case Unit::Byte:
        result /= 1;
    default:
        break;
    }
    return result;
}

void getMem() {
    size_t free, total;
    int j=hipMemGetInfo(&free, &total);
    if(j)
    {
       cout<<"get mem error: "<<j<<endl;
    }
    printf("Free mem = %.4f MB, Total = %.4f MB \n", convert(free, Unit::MB), convert(total, Unit::MB));
}

void getMembycu() {
    size_t free, total;
    int err = hipMemGetInfo(&free, &total);
    if (err) {
        cout<<"getMembycu error:"<<err<<endl;
    }
    else {
        printf("Free mem = %.4f MB, Total = %.4f MB \n", convert(free, Unit::MB), convert(total, Unit::MB));
    }
}


__global__ void VecAdd(float* h_A, float N)
{
    int tx =threadIdx.x;
    int bx =blockIdx.x;
    int offset=100*bx+tx;
    for(int i=0;i<1000;i++)
    {
    h_A[offset*1000+i]=h_A[offset*1000+i]+N;
    }
}


int main()
{

    //putenv("CUDA_MPS_PINNED_DEVICE_MEM_LIMIT=1G");
    hipSetDevice(2);
    hipCtx_t pctx;
    hipDevice_t dev;

    int i=0;
    int err = hipCtxGetDevice(&dev);
    if(err){
        cout<<"hipCtxGetDevice error:"<<err<<endl;
        return 0;
    }
    hipCtxGetCurrent(&pctx);

    getMem();

    //1048576 -> 1M
    //size_t storage_size = 1048576*400;
    size_t storage_size = 1048576*40;
    float* h_A;

    hipSetDevice(1);
    i=hipMallocManaged(&h_A,storage_size);

    if(i)
    {
    cout<<"cuda malloc managed error: "<<i<<endl;
    }


    i=hipCtxPushCurrent(pctx);
    if(i)
    {
    cout<<"push context error: "<<i<<endl;
    }


    for(int k=0;k<1000000;k++)
    {
        h_A[k]=1.0;
    }


    /*
    hipHostMalloc(&h_A, storage_size);

    //nan dao cu driver api bu xing?

    hipDeviceptr_t device_ptr;
    i=hipMalloc((hipDeviceptr_t*)&device_ptr, storage_size);


    float* device_ptr;
    i=hipMalloc(&device_ptr,storage_size);

    if(i)
    {
    cout<<"cuda malloc error: "<<i<<endl;
    }
    hipStream_t firststream;
    hipStreamCreateWithFlags(&firststream,0);
    i=hipMemcpyHtoDAsync((hipDeviceptr_t)device_ptr,h_A,storage_size,firststream);
    if(i)
    {
    cout<<"memcpy error: "<<i<<endl;
    }
    hipDeviceSynchronize();
    */


    /*
    i=hipMemPrefetchAsync(h_A,storage_size,2);
    if(i)
    {
    cout<<"prefetch error: "<<i<<endl;
    }
    */

    VecAdd<<<100,100>>>(h_A,1.0);
    hipError_t errd = hipGetLastError();  // add
    if (errd) cout << "CUDA error: " << hipGetErrorString(err) << endl; // add

    getMem();

    /*
    getMem();
    cout<<"after first new context:"<<endl;
    err = hipCtxCreate(&pctx, hipDeviceScheduleYield, dev);
    if(err) {
        cout<<"hipCtxCreate error:"<<err<<endl;
        return 0;
    }
    getMem();

    cout<<"after second new context:"<<endl;
    err = hipCtxCreate(&pctx, hipDeviceScheduleYield, dev);
    if(err) {
        cout<<"hipCtxCreate error:"<<err<<endl;
        return 0;
    }
    getMem();
    */

    return 0;

}
