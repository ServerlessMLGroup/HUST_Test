#include <iostream>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include <thread>
#include <random>
#include <ctime>
using namespace std;


enum class Unit{
    Byte, KB, MB, GB, TB, PB, EB
};


double convert(double size, Unit unit)
{
    double result = size;
    switch (unit)
    {
    case Unit::EB:
        result /= 1024;     // flow through
    case Unit::PB:
        result /= 1024;     // flow through
    case Unit::TB:
        result /= 1024;     // flow through
    case Unit::GB:
        result /= 1024;     // flow through
    case Unit::MB:
        result /= 1024;     // flow through
    case Unit::KB:
        result /= 1024;     // flow through
    case Unit::Byte:
        result /= 1;
    default:
        break;
    }
    return result;
}

void getMem() {
    size_t free, total;
    int j=hipMemGetInfo(&free, &total);
    if(j)
    {
       cout<<"get mem error: "<<j<<endl;
    }
    printf("Free mem = %.4f MB, Total = %.4f MB \n", convert(free, Unit::MB), convert(total, Unit::MB));
}

void getMembycu() {
    size_t free, total;
    int err = hipMemGetInfo(&free, &total);
    if (err) {
        cout<<"getMembycu error:"<<err<<endl;
    }
    else {
        printf("Free mem = %.4f MB, Total = %.4f MB \n", convert(free, Unit::MB), convert(total, Unit::MB));
    }
}


__global__ void VecAdd(float* h_A, float N)
{
    int tx =threadIdx.x;
    int bx =blockIdx.x;
    int offset=100*bx+tx;
    for(int i=0;i<50000;i++)
    {
    h_A[offset*50000+i]=h_A[offset*50000+i]+N;
    }
}


int main()
{

    //putenv("CUDA_MPS_PINNED_DEVICE_MEM_LIMIT=1G");
    hipSetDevice(2);
    hipCtx_t pctx;
    hipDevice_t dev;

    int i=0;
    int err = hipCtxGetDevice(&dev);
    if(err){
        cout<<"hipCtxGetDevice error:"<<err<<endl;
        return 0;
    }

    getMem();

    //1048576 -> 1M
    //size_t storage_size = 1048576*2000;
    size_t storage_size = 2000000000;
    float* h_A;
    i=hipMallocManaged(&h_A,storage_size);
    if(i)
    {
    cout<<"cuda malloc managed error: "<<i<<endl;
    }
    for(int k=0;k<500000000;k++)
    {
        h_A[k]=1.0;
    }


    /*
    cudaMallocHost(&h_A, storage_size);

    //nan dao cu driver api bu xing?

    CUdeviceptr device_ptr;
    i=cuMemAlloc((CUdeviceptr*)&device_ptr, storage_size);


    float* device_ptr;
    i=cudaMalloc(&device_ptr,storage_size);

    if(i)
    {
    cout<<"cuda malloc error: "<<i<<endl;
    }
    CUstream firststream;
    cuStreamCreate(&firststream,0);
    i=cuMemcpyHtoDAsync((CUdeviceptr)device_ptr,h_A,storage_size,firststream);
    if(i)
    {
    cout<<"memcpy error: "<<i<<endl;
    }
    cudaDeviceSynchronize();
    */

    /*
    i=cudaMemPrefetchAsync(h_A,storage_size,2);
    if(i)
    {
    cout<<"prefetch error: "<<i<<endl;
    }
    */
    VecAdd<<<100,100>>>(h_A,1.0);
    getMem();

    /*
    getMem();
    cout<<"after first new context:"<<endl;
    err = cuCtxCreate(&pctx, CU_CTX_SCHED_YIELD, dev);
    if(err) {
        cout<<"cuCtxCreate error:"<<err<<endl;
        return 0;
    }
    getMem();

    cout<<"after second new context:"<<endl;
    err = cuCtxCreate(&pctx, CU_CTX_SCHED_YIELD, dev);
    if(err) {
        cout<<"cuCtxCreate error:"<<err<<endl;
        return 0;
    }
    getMem();
    */

    return 0;

}
