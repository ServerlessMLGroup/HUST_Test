#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <thread>
#include <random>
#include <ctime>
using namespace std;


enum class Unit{
    Byte, KB, MB, GB, TB, PB, EB
};


double convert(double size, Unit unit)
{
    double result = size;
    switch (unit)
    {
    case Unit::EB:
        result /= 1024;     // flow through
    case Unit::PB:
        result /= 1024;     // flow through
    case Unit::TB:
        result /= 1024;     // flow through
    case Unit::GB:
        result /= 1024;     // flow through
    case Unit::MB:
        result /= 1024;     // flow through
    case Unit::KB:
        result /= 1024;     // flow through
    case Unit::Byte:
        result /= 1;
    default:
        break;
    }
    return result;
}

void getMem() {
    size_t free, total;
    int j=hipMemGetInfo(&free, &total);
    if(j)
    {
       cout<<"get mem error: "<<j<<endl;
    }
    printf("Free mem = %.4f MB, Total = %.4f MB \n", convert(free, Unit::MB), convert(total, Unit::MB));
}

void getMembycu() {
    size_t free, total;
    int err = hipMemGetInfo(&free, &total);
    if (err) {
        cout<<"getMembycu error:"<<err<<endl;
    }
    else {
        printf("Free mem = %.4f MB, Total = %.4f MB \n", convert(free, Unit::MB), convert(total, Unit::MB));
    }
}


__global__ void VecAdd(float* h_A, float N)
{
    int tx =threadIdx.x;
    int bx =blockIdx.x;
    int offset=100*bx+tx;
    for(int i=0;i<1000;i++)
    {
    h_A[offset*1000+i]=h_A[offset*1000+i]+N;
    }
}


int main()
{

    //putenv("CUDA_MPS_PINNED_DEVICE_MEM_LIMIT=1G");
    hipSetDevice(3);
    hipCtx_t pctx;
    hipDevice_t dev;

    int i=0;
    int err = hipCtxGetDevice(&dev);
    if(err){
        cout<<"hipCtxGetDevice error:"<<err<<endl;
        return 0;
    }
    hipCtxGetCurrent(&pctx);

    getMem();

    //1048576 -> 1M
    //size_t storage_size = 1048576*400;
    size_t storage_size = 1000*1048576;

    storage_size *=25;

    cout<<"storage_size: "<<storage_size<<endl;
    cout<<"size of size_t: "<<sizeof(size_t)<<endl;
    cout<<"size of float: "<<sizeof(float)<<endl;
    float* h_A;
    float* h_B;
    float* h_C;

    //cuda malloc
    float* device_ptr;
    i=hipMalloc(&device_ptr,storage_size);
    if(i)
    {
    cout<<"cuda malloc error: "<<i<<endl;
    }

    //cuda malloc managed
    i=hipMallocManaged(&h_A,storage_size);
    if(i)
    {
    cout<<"cuda malloc h_A managed error: "<<i<<endl;
    }



    //use mamaged mem
    /*
    for(int k=0;k<1000000;k++)
    {
        h_A[k]=1.0;
    }
    */

    //prefetch h_A
    i=hipMemPrefetchAsync(h_A,storage_size,3);
    if(i)
    {
    cout<<"prefetch error: "<<i<<endl;
    }

    //cuda malloc after prefetch
    float* device_ptr1;
    storage_size = 1048576*1000;
    i=hipMalloc(&device_ptr1,storage_size);
    if(i)
    {
    cout<<"cuda malloc error: "<<i<<endl;
    }

    //VecAdd<<<100,100>>>(h_A,1.0);
    hipDeviceSynchronize();
    for(int k=999990;k<1000000;k++)
    {
       //cout<<"after add "<<h_A[k]<<endl;
    }

    hipError_t errd = hipGetLastError();  // add
    if (errd) cout << "CUDA error: " << hipGetErrorString(errd) << endl; // add

    getMem();
    return 0;

}
