#include <iostream>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include <thread>
#include <random>
#include <ctime>
using namespace std;

enum class Unit{
    Byte, KB, MB, GB, TB, PB, EB
};


double convert(double size, Unit unit)
{
    double result = size;
    switch (unit)
    {
    case Unit::EB:
        result /= 1024;     // flow through
    case Unit::PB:
        result /= 1024;     // flow through
    case Unit::TB:
        result /= 1024;     // flow through
    case Unit::GB:
        result /= 1024;     // flow through
    case Unit::MB:
        result /= 1024;     // flow through
    case Unit::KB:
        result /= 1024;     // flow through
    case Unit::Byte:
        result /= 1;
    default:
        break;
    }
    return result;
}

void getMem() {
    size_t free, total;
    int err=hipMemGetInfo(&free, &total);
    if(err){
       cout<<"hipMemGetInfo error:"<<err<<endl;
       return;
    }
    printf("Free mem = %.4f MB, Total = %.4f MB ", convert(free, Unit::MB), convert(total, Unit::MB));
}

int main() {
    hipSetDevice(1);
    getMem();
    return 0;
}
