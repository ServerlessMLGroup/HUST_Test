#include <iostream>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <thread>
#include <random>
#include <ctime>
#include<cstdlib>
#include<string>
#include<cstdio>
#include<cstring>
#include<algorithm>
#include<stdio.h>
#include<unistd.h>
#include<sys/types.h>

using namespace std;

const int N = 300;

void Command0(void){

    std::string command = "CUDA_MPS_ACTIVE_THREAD_PERCENTAGE=40";
    putenv("CUDA_MPS_ACTIVE_THREAD_PERCENTAGE=40");

    cout<<"Parent set sm 40%: "<<endl;
    int err=hipSetDevice(0);
    int result = 0;
    if(err){
       cout<<"hipSetDevice error:"<<err<<endl;
       return;
    }
    hipCtx_t pctx;
    hipDevice_t dev;
    err=hipCtxGetDevice(&dev);
    if(err){
       cout<<"hipGetDevice error:"<<err<<endl;
       return;
    }
    err = hipCtxCreate(&pctx,hipDeviceScheduleYield,dev);
    if(err){
       cout<<"hipGetDevice error:"<<err<<endl;
       return;
    }
    err = hipDeviceGetAttribute(&result,hipDeviceAttributeMultiprocessorCount,0);
    if(err){
       cout<<"hipDeviceGetAttribute error:"<<err<<endl;
       return;
    }
    cout<< "Parent : hipDeviceAttributeMultiprocessorCount is: "<<result<<endl;
}

void Command1(void){

    putenv("CUDA_MPS_ACTIVE_THREAD_PERCENTAGE=20");

    cout<<"set sm 20%: "<<endl;
    int err=hipSetDevice(0);
    int result = 0;
    if(err){
       cout<<"hipSetDevice error:"<<err<<endl;
       return;
    }
    hipCtx_t pctx;
    hipDevice_t dev;
    err=hipCtxGetDevice(&dev);
    if(err){
       cout<<"hipGetDevice error:"<<err<<endl;
       return;
    }
    err = hipCtxCreate(&pctx,hipDeviceScheduleYield,dev);
    if(err){
       cout<<"hipGetDevice error:"<<err<<endl;
       return;
    }
    err = hipDeviceGetAttribute(&result,hipDeviceAttributeMultiprocessorCount,0);
    if(err){
       cout<<"hipDeviceGetAttribute error:"<<err<<endl;
       return;
    }
    cout<< "Child: hipDeviceAttributeMultiprocessorCount is: "<<result<<endl;
}

enum class Unit{
    Byte, KB, MB, GB, TB, PB, EB
};


double convert(double size, Unit unit)
{
    double result = size;
    switch (unit)
    {
    case Unit::EB:
        result /= 1024;     // flow through
    case Unit::PB:
        result /= 1024;     // flow through
    case Unit::TB:
        result /= 1024;     // flow through
    case Unit::GB:
        result /= 1024;     // flow through
    case Unit::MB:
        result /= 1024;     // flow through
    case Unit::KB:
        result /= 1024;     // flow through
    case Unit::Byte:
        result /= 1;
    default:
        break;
    }
    return result;
}

void getMem() {
    size_t free, total;
    int err=hipMemGetInfo(&free, &total);
    if(err){
       cout<<"hipMemGetInfo error:"<<err<<endl;
       return;
    }
    printf("Free mem = %.4f MB, Total = %.4f MB \n", convert(free, Unit::MB), convert(total, Unit::MB));
}

int main(void) {
    putenv("CUDA_MPS_ACTIVE_THREAD_PERCENTAGE=40");

    cout<<"set sm 40%: "<<endl;
    int err=hipSetDevice(0);
    int result = 0;
    if(err){
       cout<<"hipSetDevice error:"<<err<<endl;
       return;
    }
    hipCtx_t pctx;
    hipDevice_t dev;
    err=hipCtxGetDevice(&dev);
    if(err){
       cout<<"hipGetDevice error:"<<err<<endl;
       return;
    }
    err = hipCtxCreate(&pctx,hipDeviceScheduleYield,dev);
    if(err){
       cout<<"hipGetDevice error:"<<err<<endl;
       return;
    }
    err = hipDeviceGetAttribute(&result,hipDeviceAttributeMultiprocessorCount,0);
    if(err){
       cout<<"First:hipDeviceGetAttribute error:"<<err<<endl;
       return;
    }
    cout<< ": hipDeviceAttributeMultiprocessorCount is: "<<result<<endl;

    cout<<"set sm 20%: "<<endl;
    putenv("CUDA_MPS_ACTIVE_THREAD_PERCENTAGE=20");
    err = hipCtxCreate(&pctx,hipDeviceScheduleYield,dev);
    if(err){
       cout<<"hipGetDevice error:"<<err<<endl;
       return;
    }
    err = hipDeviceGetAttribute(&result,hipDeviceAttributeMultiprocessorCount,0);
    if(err){
       cout<<"First:hipDeviceGetAttribute error:"<<err<<endl;
       return;
    }
    cout<< ": hipDeviceAttributeMultiprocessorCount is: "<<result<<endl;

    return 0;
}
