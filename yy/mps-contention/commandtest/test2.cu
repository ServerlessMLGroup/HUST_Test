#include <iostream>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <thread>
#include <random>
#include <ctime>
#include<cstdlib>
#include<string>
#include<cstdio>
#include<cstring>
#include<algorithm>
#include<stdio.h>
#include<unistd.h>
#include<sys/types.h>

using namespace std;

const int N = 300;

void Command0(void){

    std::string command = "CUDA_MPS_ACTIVE_THREAD_PERCENTAGE=40";
    putenv("CUDA_MPS_ACTIVE_THREAD_PERCENTAGE=40");

    cout<<"Parent set sm 40%: "<<endl;
    int err=hipSetDevice(0);
    int result = 0;
    if(err){
       cout<<"hipSetDevice error:"<<err<<endl;
       return;
    }
    hipCtx_t pctx;
    hipDevice_t dev;
    err=hipCtxGetDevice(&dev);
    if(err){
       cout<<"hipGetDevice error:"<<err<<endl;
       return;
    }
    err = hipCtxCreate(&pctx,hipDeviceScheduleYield,dev);
    if(err){
       cout<<"hipGetDevice error:"<<err<<endl;
       return;
    }
    err = hipDeviceGetAttribute(&result,hipDeviceAttributeMultiprocessorCount,0);
    if(err){
       cout<<"hipDeviceGetAttribute error:"<<err<<endl;
       return;
    }
    cout<< "Parent : hipDeviceAttributeMultiprocessorCount is: "<<result<<endl;
}

void Command1(void){

    putenv("CUDA_MPS_ACTIVE_THREAD_PERCENTAGE=20");

    cout<<"set sm 20%: "<<endl;
    int err=hipSetDevice(0);
    int result = 0;
    if(err){
       cout<<"hipSetDevice error:"<<err<<endl;
       return;
    }
    hipCtx_t pctx;
    hipDevice_t dev;
    err=hipCtxGetDevice(&dev);
    if(err){
       cout<<"hipGetDevice error:"<<err<<endl;
       return;
    }
    err = hipCtxCreate(&pctx,hipDeviceScheduleYield,dev);
    if(err){
       cout<<"hipGetDevice error:"<<err<<endl;
       return;
    }
    err = hipDeviceGetAttribute(&result,hipDeviceAttributeMultiprocessorCount,0);
    if(err){
       cout<<"hipDeviceGetAttribute error:"<<err<<endl;
       return;
    }
    cout<< "Child: hipDeviceAttributeMultiprocessorCount is: "<<result<<endl;
}

enum class Unit{
    Byte, KB, MB, GB, TB, PB, EB
};


double convert(double size, Unit unit)
{
    double result = size;
    switch (unit)
    {
    case Unit::EB:
        result /= 1024;     // flow through
    case Unit::PB:
        result /= 1024;     // flow through
    case Unit::TB:
        result /= 1024;     // flow through
    case Unit::GB:
        result /= 1024;     // flow through
    case Unit::MB:
        result /= 1024;     // flow through
    case Unit::KB:
        result /= 1024;     // flow through
    case Unit::Byte:
        result /= 1;
    default:
        break;
    }
    return result;
}

void getMem() {
    size_t free, total;
    int err=hipMemGetInfo(&free, &total);
    if(err){
       cout<<"hipMemGetInfo error:"<<err<<endl;
       return;
    }
    printf("Free mem = %.4f MB, Total = %.4f MB \n", convert(free, Unit::MB), convert(total, Unit::MB));
}

int main(void) {
    int num=3;
    pid_t pid = 0;
    pid = fork();           //创建一个子进程,fork()函数没有参数。
    printf("pid is %d\n",getpid());     //获取进程的pid
    if (0 < pid)        //父进程得到的pid大于0,这段代码是父进程中执行的
    {
        Command0();
        num++;
        printf("I am parent!,num is %d\n",num);
    }
    else if(0 == pid)   //子进程得到的返回值是0，这段代码在子进程中执行
    {
        Command1();
        num--;
        printf("I am son!,num is %d\n",num);
    }
   else                 //创建进程失败
   {
       //有两种情况会失败：
       //1.进程数目达到OS的最大值
       //2.进程创建时内存不够了。
       printf("fork error!\n");
       exit(-1);
   }
   return 0;
}
