#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <thread>
#include <random>
#include <ctime>
using namespace std;

enum class Unit{
    Byte, KB, MB, GB, TB, PB, EB
};


double convert(double size, Unit unit)
{
    double result = size;
    switch (unit)
    {
    case Unit::EB:
        result /= 1024;     // flow through
    case Unit::PB:
        result /= 1024;     // flow through
    case Unit::TB:
        result /= 1024;     // flow through
    case Unit::GB:
        result /= 1024;     // flow through
    case Unit::MB:
        result /= 1024;     // flow through
    case Unit::KB:
        result /= 1024;     // flow through
    case Unit::Byte:
        result /= 1;
    default:
        break;
    }
    return result;
}

void getMem() {
    size_t free, total;
    hipMemGetInfo(&free, &total);
    printf("Free mem = %.4f MB, Total = %.4f MB \n", convert(free, Unit::MB), convert(total, Unit::MB));
}

void getMembycu() {
    size_t free, total;
    int err = hipMemGetInfo(&free, &total);
    if (err) {
        cout<<"getMembycu error:"<<err<<endl;
    }
    else {
        printf("Free mem = %.4f MB, Total = %.4f MB \n", convert(free, Unit::MB), convert(total, Unit::MB)); 
    }
}


__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i]; 
}


int main()
{
    hipSetDevice(3);
    hipCtx_t pctx;
    hipDevice_t dev;
    getMem();
    cout<<"after new context:"<<endl;
    int err = hipCtxGetDevice(&dev);
    if(err){
        cout<<"hipCtxGetDevice error:"<<err<<endl;
        return 0;
    }
    err = hipCtxCreate(&pctx, hipDeviceScheduleYield, dev);
    if(err) {
        cout<<"hipCtxCreate error:"<<err<<endl;
        return 0;
    }

    getMem();
    cout<<"initialize variable at GPU"<<endl;
    int N = 10485760;
    size_t size = N * sizeof(float);

    // Allocate input vectors h_A and h_B in host memory
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // Initialize input vectors
    uniform_real_distribution<float> u(0, 10);
    default_random_engine e(time(NULL));

    for(int i = 0; i < N; ++i) {
        *(h_A + i) = u(e);
        *(h_B + i) = u(e);
        *(h_C + i) = u(e);
    }
    cout<<"initialize for three times * size"<<endl;
    // Allocate vectors in device memory
    float* d_A;
    hipMalloc(&d_A, size);
    getMem();
    float* d_B;
    hipMalloc(&d_B, size);
    getMem();
    float* d_C;
    hipMalloc(&d_C, size);
    getMem();
  
    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    
    cout<<"after VecAdd"<<endl;
    getMem();


    // create a new context and use variable d_A,test whther it's ok
    cout<<"after new context initialization:"<<endl;
    hipCtx_t pctxnew;
    err = hipCtxCreate(&pctxnew, hipDeviceScheduleYield, dev);
    getMem();
    int err2=0;
    if(err){
        cout<<"cuCtx new create error: "<<err<<endl;
    }
    err2 = hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
    if(err2){
        cout<<"Failed to use variable under another context though belong to th same thread:"<<err2<<endl;
    }
    cout<<"new context transfer data from d_B to h_B"<<endl;
    getMem();

    cout<<"new context vecadd"<<endl;
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    getMem();

    hipCtxPopCurrent(&pctxnew);
    cout<<"throw new context"<<endl;
    getMem();

    //destroy the new context
    

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    cout<<"Memcpy fron d_c to h_C"<<endl;
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    getMem();
    hipFree(d_A);
    getMem();
    hipFree(d_B);
    getMem();
    hipFree(d_C);

    getMem();
    return 0;

}
