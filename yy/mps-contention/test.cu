#include <iostream>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <thread>
#include <random>
#include <ctime>
#include<cstdlib>
#include<string>
#include<cstdio>
#include<cstring>
#include<algorithm>
using namespace std;

const int N = 300;

void Command0(void){
    char line[N];
    FILE *fp;
    string cmd = "echo set_active_thread_percentage 359 10 | nvidia-cuda-mps-control";
    引号内是你的linux指令
    // 系统调用
    const char *sysCommand = cmd.data();
    if ((fp = popen(sysCommand, "r")) == NULL) {
        cout << "error" << endl;
        return;
    }
    while (fgets(line, sizeof(line)-1, fp) != NULL){
        cout << line ;
    }
    pclose(fp);
}

void Command1(void){
    char line[N];
    FILE *fp;
    string cmd = "echo set_active_thread_percentage 359 20 | nvidia-cuda-mps-control";
    引号内是你的linux指令
    // 系统调用
    const char *sysCommand = cmd.data();
    if ((fp = popen(sysCommand, "r")) == NULL) {
        cout << "error" << endl;
        return;
    }
    while (fgets(line, sizeof(line)-1, fp) != NULL){
        cout << line ;
    }
    pclose(fp);
}

enum class Unit{
    Byte, KB, MB, GB, TB, PB, EB
};


double convert(double size, Unit unit)
{
    double result = size;
    switch (unit)
    {
    case Unit::EB:
        result /= 1024;     // flow through
    case Unit::PB:
        result /= 1024;     // flow through
    case Unit::TB:
        result /= 1024;     // flow through
    case Unit::GB:
        result /= 1024;     // flow through
    case Unit::MB:
        result /= 1024;     // flow through
    case Unit::KB:
        result /= 1024;     // flow through
    case Unit::Byte:
        result /= 1;
    default:
        break;
    }
    return result;
}

void getMem() {
    size_t free, total;
    int err=hipMemGetInfo(&free, &total);
    if(err){
       cout<<"hipMemGetInfo error:"<<err<<endl;
       return;
    }
    printf("Free mem = %.4f MB, Total = %.4f MB \n", convert(free, Unit::MB), convert(total, Unit::MB));
}

int main(void) {
    Command0();
    cout<<"set sm 10%: "<<endl;
    int err=hipSetDevice(0);
    int result = 0;
    if(err){
       cout<<"hipSetDevice error:"<<err<<endl;
       return;
    }
    hipCtx_t pctx;
    hipDevice_t dev;
    err=hipCtxGetDevice(&dev);
    if(err){
       cout<<"hipGetDevice error:"<<err<<endl;
       return;
    }
    err = hipCtxCreate(&pctx,hipDeviceScheduleYield,dev);
    if(err){
       cout<<"hipGetDevice error:"<<err<<endl;
       return;
    }
    err = hipDeviceGetAttribute(&result,hipDeviceAttributeMultiprocessorCount,0);
    if(err){
       cout<<"hipDeviceGetAttribute error:"<<err<<endl;
       return;
    }
    cout<< "hipDeviceAttributeMultiprocessorCount is: "<<result<<endl;

    Command1();
    cout<<"set sm 20%: "<<endl;
    hipCtx_t pctx2;
    err = hipCtxCreate(&pctx2,hipDeviceScheduleYield,dev);
    if(err){
       cout<<"hipGetDevice error:"<<err<<endl;
       return;
    }
    err = hipDeviceGetAttribute(&result,hipDeviceAttributeMultiprocessorCount,0);
    if(err){
       cout<<"hipDeviceGetAttribute error:"<<err<<endl;
       return;
    }
    cout<< "hipDeviceAttributeMultiprocessorCount is: "<<result<<endl;

}
