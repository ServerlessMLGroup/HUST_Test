#include <iostream>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <thread>
#include <random>
#include <ctime>
#include<cstdlib>
#include<string>
#include<cstdio>
#include<cstring>
#include<algorithm>
using namespace std;

const int N = 300;

void Command0(void){
    char line[N];
    FILE *fp;
    string cmd = "echo set_active_thread_percentage 849 40 | nvidia-cuda-mps-control";
    //引号内是你的linux指令
    // 系统调用
    const char *sysCommand = cmd.data();
    if ((fp = popen(sysCommand, "r")) == NULL) {
        cout << "error" << endl;
        return;
    }
    while (fgets(line, sizeof(line)-1, fp) != NULL){
        cout << line ;
    }
    pclose(fp);

    cout<<"Parent set sm 40%: "<<endl;
    int err=hipSetDevice(0);
    int result = 0;
    if(err){
       cout<<"hipSetDevice error:"<<err<<endl;
       return 0;
    }
    hipCtx_t pctx;
    hipDevice_t dev;
    err=hipCtxGetDevice(&dev);
    if(err){
       cout<<"hipGetDevice error:"<<err<<endl;
       return 0;
    }
    err = hipCtxCreate(&pctx,hipDeviceScheduleYield,dev);
    if(err){
       cout<<"hipGetDevice error:"<<err<<endl;
       return 0;
    }
    err = hipDeviceGetAttribute(&result,hipDeviceAttributeMultiprocessorCount,0);
    if(err){
       cout<<"hipDeviceGetAttribute error:"<<err<<endl;
       return 0;
    }
    cout<< "Parent : hipDeviceAttributeMultiprocessorCount is: "<<result<<endl;
}

void Command1(void){
    char line[N];
    FILE *fp;
    string cmd = "echo set_active_thread_percentage 849 20 | nvidia-cuda-mps-control";
    //引号内是你的linux指令
    // 系统调用
    const char *sysCommand = cmd.data();
    if ((fp = popen(sysCommand, "r")) == NULL) {
        cout << "error" << endl;
        return;
    }
    while (fgets(line, sizeof(line)-1, fp) != NULL){
        cout << line ;
    }
    pclose(fp);

     cout<<"set sm 20%: "<<endl;
    int err=hipSetDevice(0);
    int result = 0;
    if(err){
       cout<<"hipSetDevice error:"<<err<<endl;
       return 0;
    }
    hipCtx_t pctx;
    hipDevice_t dev;
    err=hipCtxGetDevice(&dev);
    if(err){
       cout<<"hipGetDevice error:"<<err<<endl;
       return 0;
    }
    err = hipCtxCreate(&pctx,hipDeviceScheduleYield,dev);
    if(err){
       cout<<"hipGetDevice error:"<<err<<endl;
       return 0;
    }
    err = hipDeviceGetAttribute(&result,hipDeviceAttributeMultiprocessorCount,0);
    if(err){
       cout<<"hipDeviceGetAttribute error:"<<err<<endl;
       return 0;
    }
    cout<< "Child: hipDeviceAttributeMultiprocessorCount is: "<<result<<endl;
}

enum class Unit{
    Byte, KB, MB, GB, TB, PB, EB
};


double convert(double size, Unit unit)
{
    double result = size;
    switch (unit)
    {
    case Unit::EB:
        result /= 1024;     // flow through
    case Unit::PB:
        result /= 1024;     // flow through
    case Unit::TB:
        result /= 1024;     // flow through
    case Unit::GB:
        result /= 1024;     // flow through
    case Unit::MB:
        result /= 1024;     // flow through
    case Unit::KB:
        result /= 1024;     // flow through
    case Unit::Byte:
        result /= 1;
    default:
        break;
    }
    return result;
}

void getMem() {
    size_t free, total;
    int err=hipMemGetInfo(&free, &total);
    if(err){
       cout<<"hipMemGetInfo error:"<<err<<endl;
       return;
    }
    printf("Free mem = %.4f MB, Total = %.4f MB \n", convert(free, Unit::MB), convert(total, Unit::MB));
}

int main(void) {
    int num=3
    pid_t pid = 0;
    pid = fork();           //创建一个子进程,fork()函数没有参数。
    printf("pid is %d\n",getpid());     //获取进程的pid
    if (0 < pid)        //父进程得到的pid大于0,这段代码是父进程中执行的
    {
        Command0()
        num++;
        printf("I am parent!,num is %d\n",num);
    }
    else if(0 == pid)   //子进程得到的返回值是0，这段代码在子进程中执行
    {
        Command1()
        num--;
        printf("I am son!,num is %d\n",num);
    }
   else                 //创建进程失败
   {
       //有两种情况会失败：
       //1.进程数目达到OS的最大值
       //2.进程创建时内存不够了。
       printf("fork error!\n");
       exit(-1);
   }

}
