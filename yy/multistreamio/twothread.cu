#include "hip/hip_runtime.h"
#include <iostream>
#include <pthread.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "unistd.h"
#include <thread>
#include <mutex>
#include <random>
#include <ctime>
#include <time.h>
#include <stdio.h>
#include<hip/hip_runtime.h>
using namespace std;
#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)

mutex workend2;
mutex workend1;
//diy thread

__global__ void kernel(float n1, float n2, float n3, int stop) {
	for (int i = 0; i < stop; i++) {
		n1=cosf(n1);
		n3=n2/n3;
	}
}


//void *thread1(void *dummy,void* d_A,void *h_A)
//
void thread1(hipCtx_t ctx,float* d_a,float* h_a,size_t size,int i)
{
    //set CPU
    /*
    cpu_set_t mask;
    CPU_ZERO(&mask);
    CPU_SET(16, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0)
    {
            perror("pthread_setaffinity_np");
    }
    */
    //set GPU
    //hipSetDevice(1);

    //yy change:huan yi ge wenjian hai yao gai makefile,wojiu yong zhe ge le
    //wo hui zai wo gaide mei yige difang jia shang zhushi yy
    //yy preparation

    cout<<"one thread starts: "<<endl;
    int err;
    err=hipCtxPushCurrent(ctx);
    if(err){
    cout<<"Push Context ERR! "<<err<<endl;
    }

    hipStream_t tempstream;
    hipError_t cudaStatus;
    cudaStatus = hipStreamCreate(&tempstream);
    fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));

    /*
    int *flag = (int *)dummy;
    int *d_a = (int *)d_A;
    int *h_a = (int *)h_A;
    */

    if(i==1)
    {
    workend2.unlock();
    workend1.lock();
    }
    else
    {
    workend2.lock();
    workend1.unlock();
    }

    for(int j=0;j<10;j++)
    {
    hipMemcpyAsync(d_a, h_a,size, hipMemcpyHostToDevice, tempstream);
    }
    hipStreamSynchronize(tempstream);
}

/*
pthread_t ntid1;
pthread_t ntid2;
*/

int main()
{
    //preparation
    workend1.lock();
    workend2.lock();

    hipInit(0);
    hipSetDevice(2);
    /*
    cpu_set_t mask;
    CPU_ZERO(&mask);
    CPU_SET(16, &mask); //指定该线程使用的CPU
    if (pthread_setaffinity_np(pthread_self(), sizeof(mask), &mask) < 0) {
            perror("pthread_setaffinity_np");
    }
    */

    //Context
    cout<<"Create context"<<endl;
    int err;
    hipCtx_t cont1;
    hipDevice_t dev;
    err = hipCtxGetDevice(&dev);
    if(err)
    {
        cout<<"Can't get device, err" << err<<endl;
        return 0;
    }
    err = hipCtxCreate(&cont1,hipDeviceScheduleYield,dev);
    if(err)
    {
        cout<<"Can't create Context, err" << err << endl;
        return 0;
    }

    //262144 1M
    int N = 262144/50;
    size_t size = N * sizeof(float);

    //allocate device variable(data)
    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);

    // Allocate input vectors h_A and h_B in host memory
    float* h_A;
    float* h_B;
    float* h_C;
    hipHostMalloc(&h_A, size);
    hipHostMalloc(&h_B, size);
    hipHostMalloc(&h_C, size);

    uniform_real_distribution<float> u(0,10);
    default_random_engine e(time(NULL));
    for(int i=0;i < N; ++i){
    /*
    *(h_A + i) = u(e);
	*(h_B + i) = u(e);
	*(h_C + i) = u(e);
    */
    *(h_A + i) = 1;
	*(h_B + i) = 1;
	*(h_C + i) = 1;
    }

    thread second=thread(thread1,cont1,d_B,h_B,size,1);
    thread first=thread(thread1,cont1,d_A,h_A,size,2);
    second.join();
    first.join();

    //change,check whether the cudamemcpy works
    for(int i=0;i < N; ++i){
    /*
    *(h_A + i) = u(e);
	*(h_B + i) = u(e);
	*(h_C + i) = u(e);
    */
    *(h_A + i) = 0;
	*(h_B + i) = 0;
	*(h_C + i) = 0;
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
