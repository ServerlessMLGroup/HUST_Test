
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void kernel(float n1, float n2, float n3, long long unsigned *times, int stop) {
	unsigned long long mclk;
	if (threadIdx.x == 0) {
		asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk));
	}

	for (int i = 0; i < stop; i++) {
		n1=sinf(n1);
		n2=n3/n2;
	}

	__syncthreads();

	if (threadIdx.x == 0) {
		unsigned long long mclk2;
		asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk2));
		times[blockIdx.x] = mclk2 - mclk;
	}

}

void run_kernel(int a_blocks, int b_blocks, int a_threads, int b_threads) {
	int num_streams = 2;
	hipStream_t streams[num_streams];
	for (int i = 0; i < num_streams; i++) {
		hipStreamCreate(&streams[i]);
	}

	long long unsigned *h_sm_ids = new long long unsigned[a_blocks];
	long long unsigned *d_sm_ids;
	hipMalloc(&d_sm_ids, a_blocks * sizeof(long long unsigned));

	long long unsigned *h_sm_ids2 = new long long unsigned[b_blocks];
	long long unsigned *d_sm_ids2;
	hipMalloc(&d_sm_ids2, b_blocks * sizeof(long long unsigned));

	dim3 Dba = dim3(a_threads);
	dim3 Dga = dim3(a_blocks,1,1);
	dim3 Dbb = dim3(b_threads);
	dim3 Dgb = dim3(b_blocks,1,1);
	kernel <<<Dga, Dba, 0, streams[0]>>>(15.6, 64.9, 134.7, d_sm_ids, 5000000);
	kernel <<<Dgb, Dbb, 0, streams[1]>>>(98.2, 3.6, 17.8, d_sm_ids2, 5000000);

	hipDeviceSynchronize();

	hipMemcpy(h_sm_ids, d_sm_ids, a_blocks * sizeof(long long unsigned), hipMemcpyDeviceToHost);
	hipMemcpy(h_sm_ids2, d_sm_ids2, b_blocks * sizeof(long long unsigned), hipMemcpyDeviceToHost);

	for (int i = 0; i < a_blocks; i++) {
		printf("%llu\n", h_sm_ids[i]);
	}

	for (int i = 0; i < b_blocks; i++) {
		printf("%llu\n", h_sm_ids2[i]);
	}

	hipFree(d_sm_ids);
	hipFree(d_sm_ids2);

}

int main() {
	run_kernel(67, 8, 512, 32);

	return 0;
}

