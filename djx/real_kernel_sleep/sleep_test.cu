#include <stdio.h>
#include <stdlib.h>
#include<hip/hip_runtime.h>


#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)
// nvcc -arch=native ex.cu -o ex_sleep

#define GPU_RETURN_STATUS(cmd) \
{ \
    hipError_t result = cmd; \
    if (result != hipSuccess) { \
        std::cout << #cmd " error, return code:" << result << " | " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); \
    } \
}

// These are the inline versions for all of the SDK helper functions
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    const char *errorStr = NULL;
    errorStr = hipGetErrorString(err);
    fprintf(stderr,
            "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, "
            "line %i.\n",
            err, errorStr, file, line);
    exit(EXIT_FAILURE);
  }
}


__global__ void kernel_sleep_warm() {
    #if __CUDA_ARCH__ >= 700
    for (int i = 0; i < 100; ++i) {
        __nanosleep(1000); // 1us
    }
    #else
    printf(">>> __CUDA_ARCH__ !\n");
    #endif
}

__global__ void kernel_sleep0() {
    #if __CUDA_ARCH__ >= 700
    for (int i = 0; i < 100; ++i) {
        __nanosleep(1000); // 1us
    }
    #else
    printf(">>> __CUDA_ARCH__ !\n");
    #endif
}

__global__ void kernel_sleep1() {
    #if __CUDA_ARCH__ >= 700
    for (int i = 0; i < 100; ++i) {
        __nanosleep(1000); // 1us
    }
    #else
    printf(">>> __CUDA_ARCH__ !\n");
    #endif
}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        printf("args num error! argc:%d", argc);
        exit(1);
    }
    int gpu_no = atoi(argv[1]);
    checkCudaErrors(hipSetDevice(gpu_no));

    int num_streams = 2;
	hipStream_t streams[num_streams];
	for (int i = 0; i < num_streams; i++) {
		hipStreamCreate(&streams[i]);
	}

    dim3 D_b_a = dim3(64, 1, 1);
	dim3 D_t_a = dim3(128, 1, 1);
	dim3 D_b_b = dim3(1, 8, 16);
	dim3 D_t_b = dim3(8, 16, 1);
    // warm-up
    for (int i = 0; i < 100; ++i) {
        kernel_sleep_warm <<<D_b_a, D_t_a, 0, streams[0]>>>();
    }
    hipDeviceSynchronize();
	kernel_sleep0<<<D_b_a, D_t_a, 0, streams[0]>>>();
    kernel_sleep1<<<D_b_a, D_t_a, 0, streams[1]>>>();
    hipDeviceSynchronize();
	return 0;
}