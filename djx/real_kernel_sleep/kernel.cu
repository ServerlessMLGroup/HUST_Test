#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)
// nvcc -arch=native ex.cu -o ex_sleep

#define GPU_RETURN_STATUS(cmd) \
{ \
    hipError_t result = cmd; \
    if (result != hipSuccess) { \
        std::cout << #cmd " error, return code:" << result << " | " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); \
    } \
}

// These are the inline versions for all of the SDK helper functions
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    const char *errorStr = NULL;
    errorStr = hipGetErrorString(err);
    fprintf(stderr,
            "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, "
            "line %i.\n",
            err, errorStr, file, line);
    exit(EXIT_FAILURE);
  }
}

__device__ uint get_smid(void) {

    uint ret;
  
    asm("mov.u32 %0, %smid;" : "=r"(ret) );
  
    return ret;
  
}

extern "C" __global__ void fused_nn_contrib_conv2d_winograd_without_weight_transform_add_kernel1(float* __restrict__ placeholder, float* __restrict__ data_pack, float* __restrict__ bgemm, long long unsigned* times) {
    float bgemm_local[8];
    __shared__ float placeholder_shared[1024];
    __shared__ float data_pack_shared[256];
    for (int co_c_init = 0; co_c_init < 4; ++co_c_init) {
      for (int p_c_init = 0; p_c_init < 2; ++p_c_init) {
        bgemm_local[(((co_c_init * 2) + p_c_init))] = 0.000000e+00f;
      }
    }
    for (int ci_outer = 0; ci_outer < 32; ++ci_outer) {
      __syncthreads();
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer < 8; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer) {
        placeholder_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 128) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)))] = placeholder[(((((((((int)blockIdx.z) * 262144) + (ci_outer * 8192)) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 1024)) + ((((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) >> 6) * 512)) + (((int)blockIdx.y) * 64)) + (((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) & 63)))];
      }
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer1 < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer1) {
        data_pack_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer1 * 128) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)))] = data_pack[((((((((int)blockIdx.z) * 8192) + (ci_outer * 256)) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer1 * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)))];
      }
      __syncthreads();
      for (int ci_inner = 0; ci_inner < 16; ++ci_inner) {
        for (int co_c = 0; co_c < 4; ++co_c) {
          for (int p_c = 0; p_c < 2; ++p_c) {
            bgemm_local[(((co_c * 2) + p_c))] = (bgemm_local[(((co_c * 2) + p_c))] + (placeholder_shared[((((ci_inner * 64) + (((int)threadIdx.y) * 4)) + co_c))] * data_pack_shared[((((ci_inner * 16) + (((int)threadIdx.x) * 2)) + p_c))]));
          }
        }
      }
    }
    for (int co_inner_inner_inner = 0; co_inner_inner_inner < 4; ++co_inner_inner_inner) {
      for (int p_inner_inner_inner = 0; p_inner_inner_inner < 2; ++p_inner_inner_inner) {
        bgemm[(((((((((int)blockIdx.z) * 8192) + (((int)blockIdx.y) * 1024)) + (((int)threadIdx.y) * 64)) + (co_inner_inner_inner * 16)) + (((int)threadIdx.x) * 2)) + p_inner_inner_inner))] = bgemm_local[(((co_inner_inner_inner * 2) + p_inner_inner_inner))];
      }
    }

}

extern "C" __global__ void fused_nn_contrib_conv2d_winograd_without_weight_transform_add_kernel0(float* __restrict__ placeholder, float* __restrict__ data_pack, long long unsigned* times) {
  float d[16];
    float data_pack_local[16];
    for (int eps = 0; eps < 4; ++eps) {
      for (int nu = 0; nu < 4; ++nu) {
          d[(((eps * 4) + nu))] = (((((1 <= ((((((int)threadIdx.x) & 15) >> 2) * 2) + eps)) && (((((((int)threadIdx.x) & 15) >> 2) * 2) + eps) < 8)) && (1 <= (((((int)threadIdx.x) & 3) * 2) + nu))) && ((((((int)threadIdx.x) & 3) * 2) + nu) < 8)) ? placeholder[((((((((((int)blockIdx.x) * 392) + ((((int)threadIdx.x) >> 4) * 49)) + (((((int)threadIdx.x) & 15) >> 2) * 14)) + (eps * 7)) + ((((int)threadIdx.x) & 3) * 2)) + nu) - 8))] : 0.000000e+00f);
      }
  }
  data_pack_local[(0)] = 0.000000e+00f;
  data_pack_local[(0)] = (data_pack_local[(0)] + d[(0)]);
  data_pack_local[(0)] = (data_pack_local[(0)] + (d[(2)] * -1.000000e+00f));
  data_pack_local[(0)] = (data_pack_local[(0)] + (d[(8)] * -1.000000e+00f));
  data_pack_local[(0)] = (data_pack_local[(0)] + ((d[(10)] * -1.000000e+00f) * -1.000000e+00f));
  data_pack_local[(1)] = 0.000000e+00f;
  data_pack_local[(1)] = (data_pack_local[(1)] + (d[(1)] * -1.000000e+00f));
  data_pack_local[(1)] = (data_pack_local[(1)] + d[(2)]);
  data_pack_local[(1)] = (data_pack_local[(1)] + ((d[(9)] * -1.000000e+00f) * -1.000000e+00f));
  data_pack_local[(1)] = (data_pack_local[(1)] + (d[(10)] * -1.000000e+00f));
  data_pack_local[(2)] = 0.000000e+00f;
  data_pack_local[(2)] = (data_pack_local[(2)] + d[(1)]);
  data_pack_local[(2)] = (data_pack_local[(2)] + d[(2)]);
  data_pack_local[(2)] = (data_pack_local[(2)] + (d[(9)] * -1.000000e+00f));
  data_pack_local[(2)] = (data_pack_local[(2)] + (d[(10)] * -1.000000e+00f));
  data_pack_local[(3)] = 0.000000e+00f;
  data_pack_local[(3)] = (data_pack_local[(3)] + (d[(1)] * -1.000000e+00f));
  data_pack_local[(3)] = (data_pack_local[(3)] + d[(3)]);
  data_pack_local[(3)] = (data_pack_local[(3)] + ((d[(9)] * -1.000000e+00f) * -1.000000e+00f));
  data_pack_local[(3)] = (data_pack_local[(3)] + (d[(11)] * -1.000000e+00f));
  data_pack_local[(4)] = 0.000000e+00f;
  data_pack_local[(4)] = (data_pack_local[(4)] + (d[(4)] * -1.000000e+00f));
  data_pack_local[(4)] = (data_pack_local[(4)] + ((d[(6)] * -1.000000e+00f) * -1.000000e+00f));
  data_pack_local[(4)] = (data_pack_local[(4)] + d[(8)]);
  data_pack_local[(4)] = (data_pack_local[(4)] + (d[(10)] * -1.000000e+00f));
  data_pack_local[(5)] = 0.000000e+00f;
  data_pack_local[(5)] = (data_pack_local[(5)] + ((d[(5)] * -1.000000e+00f) * -1.000000e+00f));
  data_pack_local[(5)] = (data_pack_local[(5)] + (d[(6)] * -1.000000e+00f));
  data_pack_local[(5)] = (data_pack_local[(5)] + (d[(9)] * -1.000000e+00f));
  data_pack_local[(5)] = (data_pack_local[(5)] + d[(10)]);
  data_pack_local[(6)] = 0.000000e+00f;
  data_pack_local[(6)] = (data_pack_local[(6)] + (d[(5)] * -1.000000e+00f));
  data_pack_local[(6)] = (data_pack_local[(6)] + (d[(6)] * -1.000000e+00f));
  data_pack_local[(6)] = (data_pack_local[(6)] + d[(9)]);
  data_pack_local[(6)] = (data_pack_local[(6)] + d[(10)]);
  data_pack_local[(7)] = 0.000000e+00f;
  data_pack_local[(7)] = (data_pack_local[(7)] + ((d[(5)] * -1.000000e+00f) * -1.000000e+00f));
  data_pack_local[(7)] = (data_pack_local[(7)] + (d[(7)] * -1.000000e+00f));
  data_pack_local[(7)] = (data_pack_local[(7)] + (d[(9)] * -1.000000e+00f));
  data_pack_local[(7)] = (data_pack_local[(7)] + d[(11)]);
  data_pack_local[(8)] = 0.000000e+00f;
  data_pack_local[(8)] = (data_pack_local[(8)] + d[(4)]);
  data_pack_local[(8)] = (data_pack_local[(8)] + (d[(6)] * -1.000000e+00f));
  data_pack_local[(8)] = (data_pack_local[(8)] + d[(8)]);
  data_pack_local[(8)] = (data_pack_local[(8)] + (d[(10)] * -1.000000e+00f));
  data_pack_local[(9)] = 0.000000e+00f;
  data_pack_local[(9)] = (data_pack_local[(9)] + (d[(5)] * -1.000000e+00f));
  data_pack_local[(9)] = (data_pack_local[(9)] + d[(6)]);
  data_pack_local[(9)] = (data_pack_local[(9)] + (d[(9)] * -1.000000e+00f));
  data_pack_local[(9)] = (data_pack_local[(9)] + d[(10)]);
  data_pack_local[(10)] = 0.000000e+00f;
  data_pack_local[(10)] = (data_pack_local[(10)] + d[(5)]);
  data_pack_local[(10)] = (data_pack_local[(10)] + d[(6)]);
  data_pack_local[(10)] = (data_pack_local[(10)] + d[(9)]);
  data_pack_local[(10)] = (data_pack_local[(10)] + d[(10)]);
  data_pack_local[(11)] = 0.000000e+00f;
  data_pack_local[(11)] = (data_pack_local[(11)] + (d[(5)] * -1.000000e+00f));
  data_pack_local[(11)] = (data_pack_local[(11)] + d[(7)]);
  data_pack_local[(11)] = (data_pack_local[(11)] + (d[(9)] * -1.000000e+00f));
  data_pack_local[(11)] = (data_pack_local[(11)] + d[(11)]);
  data_pack_local[(12)] = 0.000000e+00f;
  data_pack_local[(12)] = (data_pack_local[(12)] + (d[(4)] * -1.000000e+00f));
  data_pack_local[(12)] = (data_pack_local[(12)] + ((d[(6)] * -1.000000e+00f) * -1.000000e+00f));
  data_pack_local[(12)] = (data_pack_local[(12)] + d[(12)]);
  data_pack_local[(12)] = (data_pack_local[(12)] + (d[(14)] * -1.000000e+00f));
  data_pack_local[(13)] = 0.000000e+00f;
  data_pack_local[(13)] = (data_pack_local[(13)] + ((d[(5)] * -1.000000e+00f) * -1.000000e+00f));
  data_pack_local[(13)] = (data_pack_local[(13)] + (d[(6)] * -1.000000e+00f));
  data_pack_local[(13)] = (data_pack_local[(13)] + (d[(13)] * -1.000000e+00f));
  data_pack_local[(13)] = (data_pack_local[(13)] + d[(14)]);
  data_pack_local[(14)] = 0.000000e+00f;
  data_pack_local[(14)] = (data_pack_local[(14)] + (d[(5)] * -1.000000e+00f));
  data_pack_local[(14)] = (data_pack_local[(14)] + (d[(6)] * -1.000000e+00f));
  data_pack_local[(14)] = (data_pack_local[(14)] + d[(13)]);
  data_pack_local[(14)] = (data_pack_local[(14)] + d[(14)]);
  data_pack_local[(15)] = 0.000000e+00f;
  data_pack_local[(15)] = (data_pack_local[(15)] + ((d[(5)] * -1.000000e+00f) * -1.000000e+00f));
  data_pack_local[(15)] = (data_pack_local[(15)] + (d[(7)] * -1.000000e+00f));
  data_pack_local[(15)] = (data_pack_local[(15)] + (d[(13)] * -1.000000e+00f));
  data_pack_local[(15)] = (data_pack_local[(15)] + d[(15)]);
  for (int eps1 = 0; eps1 < 4; ++eps1) {
      for (int nu1 = 0; nu1 < 4; ++nu1) {
      data_pack[(((((eps1 * 32768) + (nu1 * 8192)) + (((int)blockIdx.x) * 128)) + ((int)threadIdx.x)))] = data_pack_local[(((eps1 * 4) + nu1))];
      }
  }
}


void run_kernel() {
	int num_streams = 2;
	hipStream_t streams[num_streams];
	for (int i = 0; i < num_streams; i++) {
		hipStreamCreate(&streams[i]);
	}
	

    long long unsigned *h_sm_ids = new long long unsigned[64 * 2];
    long long unsigned *d_sm_ids;
    hipMalloc(&d_sm_ids, 64 * sizeof(long long unsigned) * 2);
    
    long long unsigned *h_sm_ids2 = new long long unsigned[128 * 2];
    long long unsigned *d_sm_ids2;
    hipMalloc(&d_sm_ids2, 128 * sizeof(long long unsigned) * 2);

    float *h_args_55 = new float[25088]; // 55
    float *d_args_55;
    hipMalloc(&d_args_55, sizeof(float) * 25088);

    float *h_args_56 = new float[4194304]; // 56
    float *d_args_56;
    hipMalloc(&d_args_56, sizeof(float) * 4194304);

    float *h_args_75 = new float[1806336 / 4 + 1]; // 55
    float *d_args_75;
    hipMalloc(&d_args_75, sizeof(float) * 1806336 / 4 + 4);

    float *h_args_76 = new float[1806336 / 4 + 1]; // 55
    float *d_args_76;
    hipMalloc(&d_args_76, sizeof(float) * 1806336 / 4 + 4);


    // allocate flag
    int *flag;
    int *g_flag;
    flag = (int*) malloc(1 * sizeof(int));
    flag[0] = 0;
    hipMalloc((void **)&g_flag, sizeof(int) * 1);
    hipMemcpy(g_flag, flag, sizeof(int) * 1, hipMemcpyHostToDevice);


    // allocate warm flag
    int *flag_warm;
    int *g_flag_warm;
    flag_warm = (int*) malloc(1 * sizeof(int));
    flag_warm[0] = 0;
    hipMalloc((void **)&g_flag_warm, sizeof(int) * 1);
    hipMemcpy(g_flag_warm, flag_warm, sizeof(int) * 1, hipMemcpyHostToDevice);

    // cuda launch kernel
    dim3 D_b_a = dim3(64, 1, 1);
    dim3 D_t_a = dim3(128, 1, 1);
    dim3 D_b_b = dim3(1, 8, 16);
    dim3 D_t_b = dim3(8, 16, 1);
    // warm-up
    for (int i = 0; i < 100; ++i) {
        fused_nn_contrib_conv2d_winograd_without_weight_transform_add_kernel0 <<<D_b_a, D_t_a, 0, streams[0]>>>(d_args_55, d_args_76, d_sm_ids);
    }
    hipDeviceSynchronize();
    // test kernel
    fused_nn_contrib_conv2d_winograd_without_weight_transform_add_kernel0 <<<D_b_a, D_t_a, 0, streams[0]>>>(d_args_55, d_args_76, d_sm_ids);
    hipDeviceSynchronize();
    // sleep until kernel finish
    fused_nn_contrib_conv2d_winograd_without_weight_transform_add_kernel1 <<<D_b_b, D_t_b, 0, streams[1]>>>(d_args_56, d_args_76, d_args_75, d_sm_ids2);
    
	
    hipDeviceSynchronize();
    
    hipMemcpy(h_sm_ids, d_sm_ids, 64 * sizeof(long long unsigned) * 2, hipMemcpyDeviceToHost);
    hipMemcpy(h_sm_ids2, d_sm_ids2, 128 * sizeof(long long unsigned) * 2, hipMemcpyDeviceToHost);
    

    // hipMemcpy(h_sleep_time, d_sleep_time, b_blocks * sizeof(long long unsigned), hipMemcpyDeviceToHost);
    // hipMemcpy(h_sleep_sm, d_sleep_sm, b_blocks * sizeof(long long unsigned), hipMemcpyDeviceToHost);

    long long unsigned maxm = 0, minm = 1768959725180341, max1 = 0, max2=0, min2=1768959725180341;
    long long unsigned maxm_e = 0, minm_e = 1768959725180341;
    printf("---1---\n");
    for (int i = 0; i < 64; i++) {
          // printf("%llu-%llu\n", h_sm_ids[i], h_sm_ids[i + 64]);
          maxm = max(maxm, h_sm_ids[i]);
          minm = min(minm, h_sm_ids[i]);
          maxm_e = max(maxm_e, h_sm_ids[i + 64]);
          minm_e = min(minm_e, h_sm_ids[i + 64]);
          max1 = max(max1, h_sm_ids[i + 64] - h_sm_ids[i]);
    }
    printf("START_TIMING:max-%llu, min-%llu(us)\n", maxm, minm);
    printf("END_TIMING__:max-%llu, min-%llu(us)\n", maxm_e, minm_e);
    printf("DURATION:单block最大执行时间%llu(us)\n", max1);
          
    maxm = 0; minm = 1768959725180341;
    maxm_e = 0; minm_e = 1768959725180341;
    printf("---2---\n");
    for (int i = 0; i < 128; i++) {
      // printf("blcok%d:%llu-%llu   %llu \n",i, h_sm_ids2[i], h_sm_ids2[i + a_blocks] , h_sm_ids2[i + b_blocks]-h_sm_ids2[i]);
          // printf("%llu-%llu\n", h_sm_ids2[i], h_sm_ids2[i + 128]);
          maxm = max(maxm, h_sm_ids2[i]);
          minm = min(minm, h_sm_ids2[i]);
          maxm_e = max(maxm_e, h_sm_ids2[i + 128]);
          minm_e = min(minm_e, h_sm_ids2[i + 128]);
          max2 = max(max2, h_sm_ids2[i + 128]-h_sm_ids2[i]);
          min2 = min(min2, h_sm_ids2[i + 128]-h_sm_ids2[i]);
    }
    printf("START_TIMING:max-%llu, min-%llu(us)\n", maxm, minm);
    printf("END_TIMING__:max-%llu, min-%llu(us)\n", maxm_e, minm_e);
    printf("DURATION:单block最大执行时间%llu(us)  单block最大执行时间与最小的时间差%llu(us)\n", max2, max2 - min2);

}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        printf("args num error! argc:%d", argc);
        exit(1);
    }
    int gpu_no = atoi(argv[1]);
    checkCudaErrors(hipSetDevice(gpu_no));
	run_kernel();

	return 0;
}

