#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <mutex>
#include <queue>
#define BLOCK_NUM 200
#define THREAD_NUM 128
#define SM_NUM 80
#define TASK_NUM 200
#define RESULT_NUM SM_NUM
#define FLAG_LENGTH 65535
#define FLAG_BLOCK_BASE 0
#define FLAG_SM_BASE (FLAG_BLOCK_BASE + BLOCK_NUM)
#define FLAG_RESULT_BASE (FLAG_SM_BASE + SM_NUM)
#define FLAG_TIME_BASE (FLAG_RESULT_BASE + RESULT_NUM)
#define POOL_CAPACITY 300
#define POOL_LENGTH POOL_CAPACITY + 4
#define POOL_CAPACITY_BASE 0
#define POOL_HEAD_BASE 1
#define POOL_TAIL_BASE 2
#define POOL_MUTEX_BASE 3
#define POOL_ARRAY_BASE 4
// nvcc -arch=native task_pool.cu tool.cu -o pool

__device__ void push_task(int* task_pool, int id) {
    int* mutex = task_pool + POOL_MUTEX_BASE, capacity_ = task_pool[POOL_CAPACITY_BASE], *head_ = task_pool + POOL_HEAD_BASE, *tail_ = task_pool + POOL_TAIL_BASE, *tasks_ = task_pool + POOL_ARRAY_BASE;
    // printf("into push %d\n", id);
    while (atomicCAS(mutex, 0, 1) != 0);
    // printf("into push %d\n", id);
    tasks_[*head_] = id;
    *head_ = (*head_ + 1) % capacity_;
    // printf("push_task %d, head = %d, tail = %d, capacity = %d\n", id, *head_, *tail_, capacity_);
    atomicExch(mutex, 0);
}

__device__ int get_task(int* task_pool) {
    int* mutex = task_pool + POOL_MUTEX_BASE, capacity_ = task_pool[POOL_CAPACITY_BASE], *head_ = task_pool + POOL_HEAD_BASE, *tail_ = task_pool + POOL_TAIL_BASE, *tasks_ = task_pool + POOL_ARRAY_BASE;
    unsigned int ns = 10;
    while (atomicCAS(mutex, 0, 1) != 0) {
      // if (ns < 100) {
      //   ns += 10;
      // }
      __nanosleep(ns);
    };
    if ((*head_) == (*tail_)) {
        atomicExch(mutex, 0);
        return -1;  
    }
    int task_id = tasks_[(*tail_)];
    (*tail_) = ((*tail_) + 1) % capacity_;
    atomicExch(mutex, 0);
    return task_id;
}

extern void initDevice(int argc, char *argv[]);

__device__ uint get_smid(void) {
  uint ret;
  asm("mov.u32 %0, %smid;" : "=r"(ret) );
  return ret;
}

__device__ dim3 Dim_block = dim3(BLOCK_NUM, 1, 1);
__device__ dim3 Dim_thread = dim3(THREAD_NUM, 1, 1);


__global__ void workload() {
    int n1 = 15.6, n2 = 64.9, n3 = 134.7;
    for (int i = 0; i < 50000; i++) {
        n1=sinf(n1);
        n2=n3/n2;
    }
    __syncthreads();
}

__global__ void ElasticKernel(int *flag, int* task_pool) {
    int* sm_flag = flag + FLAG_SM_BASE, *block_flag = flag + FLAG_BLOCK_BASE, *result_flag = flag + FLAG_RESULT_BASE, *times = flag + FLAG_TIME_BASE;
    unsigned int ns = 5;
    int smid = get_smid();
    if (threadIdx.x == 0 && atomicAdd(sm_flag + smid, 1) == 0) atomicAdd(block_flag + blockIdx.x, 1);
    __syncthreads();

    if (atomicAdd(block_flag + blockIdx.x, 0) == 0) return ;
    __syncthreads();
    __shared__ int BlockSyn[128 + 5];
    BlockSyn[threadIdx.x] = 0;

    // if (threadIdx.x == 0 )printf("%d\n", smid); 已验证仍然均匀分布

    if (threadIdx.x == 0) {
      int id = 0, task_num = 0, i = 0;
      unsigned long long mclk[2];
      asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk[(i++) % 2]));
      while((id = get_task(task_pool)) != -1) {
        asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk[(i++) % 2]));
        times[smid * TASK_NUM + task_num] = (mclk[(i + 1) % 2] - mclk[i % 2]) / 1000;
        ++task_num;
        atomicAdd(result_flag + smid, 1);

        // workload 目前与blockIdx无关
        int n1 = 15.6, n2 = 64.9, n3 = 134.7;
        for (int ii = 0; ii < 500; ii++) {
            n1=sinf(n1);
            n2=n3/n2;
        }
      }
      times[smid * TASK_NUM + task_num] = -1;
      for (int i = 0; i < THREAD_NUM; ++i) {
        atomicAdd(BlockSyn + i, 1);
      }
    }
    else {
      while (atomicAdd(BlockSyn + threadIdx.x, 0) == 0) {
        __nanosleep(ns);
        if (ns < 1000) {
            ns *= 2;
        }
      }
    }

}

__device__ void init_pool(int *flags, int *task_pool) {
  int* mutex = task_pool + POOL_MUTEX_BASE, *capacity_ = task_pool + POOL_CAPACITY_BASE, *head_ = task_pool + POOL_HEAD_BASE, *tail_ = task_pool + POOL_TAIL_BASE, *tasks_ = task_pool + POOL_ARRAY_BASE;
  atomicExch(mutex, 0);
  atomicExch(capacity_, POOL_CAPACITY);
  atomicExch(head_, 0);
  atomicExch(tail_, 0);
  for (int i = 0; i < TASK_NUM; ++i) {
    push_task(task_pool, i);
    //atomicAdd(flags + FLAG_RESULT_BASE + RESULT_NUM, 1);
  }
}

__global__ void LaunchKernel(int *flags, int *task_pool) { //smid = 0
    init_pool(flags, task_pool);
    //printf("push tasks finish!\n");
    ElasticKernel<<<Dim_block, Dim_thread>>>(flags, task_pool); // 如果指定stream则会导致无效
    // cudaStreamSynchronize(stream);
}


int main(int argc, char *argv[]) {
    // init device
    initDevice(argc, argv);

    // allocate stream
    int num_streams = 2;
    hipStream_t streams[num_streams];
    for (int i = 0; i < num_streams; i++) {
      hipStreamCreate(&streams[i]);
    }
    // allocate flag
    int *flag = new int[FLAG_LENGTH];
    int *g_flag;
    for (int i = 0; i < FLAG_LENGTH; ++i) {
        flag[i] = 0;
    }
    hipMalloc((void **)&g_flag, sizeof(int) * FLAG_LENGTH);
    hipMemcpy(g_flag, flag, sizeof(int) * FLAG_LENGTH, hipMemcpyHostToDevice);
    
    int *task_pool = new int[POOL_LENGTH];
    int *g_task_pool;
    for (int i = 0; i < POOL_LENGTH; ++i) {
        task_pool[i] = 0;
    }
    hipMalloc((void **)&g_task_pool, sizeof(int) * POOL_LENGTH);
    hipMemcpy(g_task_pool, task_pool, sizeof(int) * POOL_LENGTH, hipMemcpyHostToDevice);
    // cuda launch kernel
    // warm-up
    for (int i = 0; i < 50; ++i) {
        workload <<<Dim_block, Dim_thread, 0, streams[0]>>> ();
        // LaunchKernel <<<1, 1, 0, streams[0]>>> (g_flag, g_task_pool);
    }
    hipDeviceSynchronize();
    // flush
    hipMemcpy(g_flag, flag, sizeof(int) * FLAG_LENGTH, hipMemcpyHostToDevice);
    hipMemcpy(g_task_pool, task_pool, sizeof(int) * POOL_LENGTH, hipMemcpyHostToDevice);

    LaunchKernel <<<1, 1, 0, streams[0]>>> (g_flag, g_task_pool);
    hipDeviceSynchronize();

    hipMemcpy(flag, g_flag, sizeof(int) * FLAG_LENGTH, hipMemcpyDeviceToHost);

    int total = 0;
    for (int i = FLAG_RESULT_BASE; i < FLAG_RESULT_BASE + RESULT_NUM; ++i) {
      printf("sm %d get %d task\n", (i - FLAG_RESULT_BASE), flag[i]);
      total += flag[i];
    }
    int i = FLAG_TIME_BASE;
    for (; ; ) {
      int sm_id = (i - FLAG_TIME_BASE) / TASK_NUM;
      if (sm_id == 80) break;
      printf("sm %d task duration:", sm_id);
      while (flag[i] != -1) {
        printf("%d ", flag[i]);
        ++i;
      }
      printf("\n");
      i = (sm_id + 1) * TASK_NUM + FLAG_TIME_BASE;
    }
    printf("total task get:%d\n", total);
    //printf("total task push:%d\n", flag[FLAG_RESULT_BASE + RESULT_NUM]);
}