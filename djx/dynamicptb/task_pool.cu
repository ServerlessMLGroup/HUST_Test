#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <mutex>
#include <queue>
#define BLOCK_NUM 200
#define THREAD_NUM 128
#define SM_NUM 80
#define TASK_NUM 200
#define RESULT_NUM SM_NUM
#define FLAG_LENGTH 65535
#define FLAG_BLOCK_BASE 0
#define FLAG_SM_BASE FLAG_BLOCK_BASE + BLOCK_NUM
#define FLAG_RESULT_BASE FLAG_SM_BASE + SM_NUM
// nvcc -arch=native task_pool.cu tool.cu -o pool

template <typename TASK>
class CudaTaskPool {
 public:
   CudaTaskPool(size_t capacity) : capacity_(capacity) {
    int state = 0;
    hipMalloc((void**)&mutex, sizeof(int));
    hipMemcpy(mutex, &state, sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void**)&tasks_, capacity_ * sizeof(TASK));
  }

   ~CudaTaskPool() {
    hipFree(tasks_);
    hipFree(mutex);
  }

  __device__ void push(int id) {
    while (atomicCAS(mutex, 0, 1) != 0);
    tasks_[head_].block_id = id;
    head_ = (head_ + 1) % capacity_;
    atomicExch(mutex, 0);
  }

  __device__ int get() {
    while (atomicCAS(mutex, 0, 1) != 0);
    if (head_ == tail_) {
      return -1;  
    }
    TASK task = tasks_[tail_];
    tail_ = (tail_ + 1) % capacity_;
    atomicExch(mutex, 0);
    return task.block_id;
  }

  __device__ size_t size() const {
    return (head_ + capacity_ - tail_) % capacity_;
  }

 private:
  TASK* tasks_;
  size_t capacity_;
  size_t head_ = 0;
  size_t tail_ = 0;
  int *mutex;
};

extern void initDevice(int argc, char *argv[]);

__device__ uint get_smid(void) {
  uint ret;
  asm("mov.u32 %0, %smid;" : "=r"(ret) );
  return ret;
}


struct Task{
    int block_id;
    __host__ __device__ Task(int data) : block_id(data) {}
};

__global__ void workload() {
  int n1 = 15.6, n2 = 64.9, n3 = 134.7;
    for (int i = 0; i < 500000; i++) {
        n1=sinf(n1);
        n2=n3/n2;
    }
    __syncthreads();
}

__device__ void ElasticKernel(int *flag, CudaTaskPool<Task>& task_pool) {
    int* sm_flag = flag + FLAG_SM_BASE, *block_flag = flag + FLAG_BLOCK_BASE, *result_flag = flag + FLAG_RESULT_BASE;
    unsigned int ns = 5;
    int smid = get_smid();
    if (threadIdx.x == 0 && atomicAdd(sm_flag + smid, 1) == 0) atomicAdd(block_flag + blockIdx.x, 1);
    __syncthreads();

    if (atomicAdd(block_flag + blockIdx.x, 0) == 0) return ;
    __syncthreads();

    __shared__ int BlockSyn[128 + 5];
    BlockSyn[threadIdx.x] = 0;

    if (threadIdx.x == 0) {
      int id;
      while(id = task_pool.get()) {
        if (id == -1) break;
        // printf("block %d get task %d\n", blockIdx.x, id);
        atomicAdd(result_flag + smid, 1);
      }
      for (int i = 0; i < THREAD_NUM; ++i) {
        atomicAdd(BlockSyn + i, 1);
      }
    }
    else {
      while (atomicAdd(BlockSyn + threadIdx.x, 0) == 0) {
        __nanosleep(ns);
        if (ns < 1000) {
            ns *= 2;
        }
      }
    }

}

__global__ void LaunchKernel(int *flags, CudaTaskPool<Task>& task_pool) {
  for (int i = 0; i < TASK_NUM; ++i) {
    task_pool.push(i);
    atomicAdd(flags + FLAG_RESULT_BASE + RESULT_NUM + 1, 1);
  }
  ElasticKernel(flags, task_pool);
}


int main(int argc, char *argv[]) {
    // init device
    initDevice(argc, argv);

    // allocate stream
    int num_streams = 2;
    hipStream_t streams[num_streams];
    for (int i = 0; i < num_streams; i++) {
      hipStreamCreate(&streams[i]);
    }
    // allocate flag
    int *flag = new int[FLAG_LENGTH];
    int *g_flag;
    for (int i = 0; i < FLAG_LENGTH; ++i) {
        flag[i] = 0;
    }
    hipMalloc((void **)&g_flag, sizeof(int) * FLAG_LENGTH);
    hipMemcpy(g_flag, flag, sizeof(int) * FLAG_LENGTH, hipMemcpyHostToDevice);

    // allocate pool
    CudaTaskPool<Task>* task_pool;
    hipMallocManaged(&task_pool, sizeof(CudaTaskPool<Task>));

    // 在 CPU 端初始化 task_pool
    new(task_pool) CudaTaskPool<Task>(TASK_NUM);


    // CudaTaskPool<Task> *d_task_pool;
    // cudaMalloc(&d_task_pool, sizeof(CudaTaskPool<Task>));
    // cudaMemcpy(d_task_pool, &task_pool, sizeof(CudaTaskPool<Task>), cudaMemcpyHostToDevice);

    // cuda launch kernel
    dim3 Dim_block = dim3(BLOCK_NUM, 1, 1);
    dim3 Dim_thread = dim3(THREAD_NUM, 1, 1);
    // warm-up
    for (int i = 0; i < 100; ++i) {
        workload <<<Dim_block, Dim_thread, 0, streams[0]>>> ();
    }
    hipDeviceSynchronize();
    hipMemcpy(g_flag, flag, sizeof(int) * FLAG_LENGTH, hipMemcpyHostToDevice);

    hipMemcpy(flag, g_flag, sizeof(int) * FLAG_LENGTH, hipMemcpyDeviceToHost);

    int total = 0;
    for (int i = FLAG_RESULT_BASE; i < FLAG_RESULT_BASE + RESULT_NUM; ++i) {
      printf("sm %d get %d task\n", i - FLAG_RESULT_BASE, flag[i]);
      total += flag[i];
    }
    printf("total task:%d\n", total);
    printf("total push:%d\n", flag[FLAG_RESULT_BASE + RESULT_NUM]);

    LaunchKernel <<<Dim_block, Dim_thread, 0, streams[0]>>> (g_flag, *task_pool);
    hipDeviceSynchronize();


}