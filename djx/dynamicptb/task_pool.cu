#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <mutex>
#include <queue>
#define BLOCK_NUM 200
#define THREAD_NUM 128
#define SM_NUM 80
#define TASK_NUM 200
#define RESULT_NUM BLOCK_NUM * TASK_NUM
#define FLAG_LENGTH 65535
#define FLAG_BLOCK_BASE 0
#define FLAG_SM_BASE FLAG_BLOCK_BASE + BLOCK_NUM
#define FLAG_RESULT_BASE FLAG_SM_BASE + SM_NUM
// nvcc -arch=native task_pool.cu tool.cu -o pool

template <typename TASK>
class CudaTaskPool {
 public:
   CudaTaskPool(size_t capacity) : capacity_(capacity) {
    int state = 0;
    hipMalloc((int*)&&mutex, sizeof(int));
    hipMemcpy(mutex, &state, sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void**)&tasks_, capacity_ * sizeof(TASK));
  }

   ~CudaTaskPool() {
    hipFree(tasks_);
    hipFree(&mutex);
  }

  __device__ void push(TASK task) {
    while (atomicCAS(mutex, 0, 1) != 0);
    tasks_[head_] = std::move(task);
    head_ = (head_ + 1) % capacity_;
    atomicExch(mutex, 0);
  }

  __device__ TASK* get() {
    while (atomicCAS(mutex, 0, 1) != 0);
    if (head_ == tail_) {
      return nullptr;
    }
    TASK task = std::move(tasks_[tail_]);
    tail_ = (tail_ + 1) % capacity_;
    atomicExch(mutex, 0);
    return &task;
  }

  __device__ size_t size() const {
    return (head_ + capacity_ - tail_) % capacity_;
  }

 private:
  TASK* tasks_;
  size_t capacity_;
  size_t head_ = 0;
  size_t tail_ = 0;
  int *mutex;
};

extern void initDevice(void);
extern __device__ uint get_smid(void);

struct Task{
    int block_id;
    Task(int data) : block_id(data) {}
};

__global__ void workload() {
  int n1 = 15.6, n2 = 64.9, n3 = 134.7;
    for (int i = 0; i < 500000; i++) {
        n1=sinf(n1);
        n2=n3/n2;
    }
    __syncthreads();
}

__device__ void ElasticKernel(int *flag, CudaTaskPool<Task>& task_pool) {
    int* sm_flag = flag + FLAG_SM_BASE, *block_flag = flag + FLAG_BLOCK_BASE;
    unsigned int ns = 5;
    int smid = get_smid();
    if (threadIdx.x == 0 && atomicAdd(sm_flag + smid, 1) == 0) atomicAdd(block_flag + blockIdx.x, 1);
    __syncthreads();

    if (atomicAdd(block_flag + blockIdx.x, 0) == 0) return ;
    __syncthreads();

    __shared__ int BlockSyn[128 + 5];
    BlockSyn[threadIdx.x] = 0;

    if (threadIdx.x == 0) {
      Task *task;
      while((task = task_pool.get()) != nullptr) {
        printf("block %d get task %d\n", blockIdx.x, task->block_id);
      }
      for (int i = 0; i < THREAD_NUM; ++i) {
        atomicAdd(BlockSyn + i, 1);
      }
    }
    else {
      while (atomicAdd(BlockSyn + threadIdx.x, 0) == 0) {
        __nanosleep(ns);
        if (ns < 1000) {
            ns *= 2;
        }
      }
    }

}

__global__ void LaunchKernel(int *flags, CudaTaskPool<Task>& task_pool) {
  ElasticKernel(flags, task_pool);
}


int main() {
    // init device
    initDevice();

    // allocate stream
    int num_streams = 2;
    hipStream_t streams[num_streams];
    for (int i = 0; i < num_streams; i++) {
      hipStreamCreate(&streams[i]);
    }
    // allocate flag
    int *flag = new int[FLAG_LENGTH];
    int *g_flag;
    for (int i = 0; i < FLAG_LENGTH; ++i) {
        flag[i] = 0;
    }
    hipMalloc((void **)&g_flag, sizeof(int) * FLAG_LENGTH);
    hipMemcpy(g_flag, flag, sizeof(int) * FLAG_LENGTH, hipMemcpyHostToDevice);

    // allocate pool
    CudaTaskPool<Task> task_pool(1024);
    // 创建任务队列
    std::vector<Task*> tasks;
    for (int i = 0; i < TASK_NUM; ++i) {
      tasks.push_back(new Task(i));
    }

    // 将任务加入任务池
    for (Task *task : tasks) {
      task_pool.push(std::move(*task));
    }
    CudaTaskPool<Task> *d_task_pool;
    hipMalloc(&d_task_pool, sizeof(CudaTaskPool<Task>));
    hipMemcpy(d_task_pool, &task_pool, sizeof(CudaTaskPool<Task>), hipMemcpyHostToDevice);

    // cuda launch kernel
    dim3 Dim_block = dim3(BLOCK_NUM, 1, 1);
    dim3 Dim_thread = dim3(THREAD_NUM, 1, 1);
    // warm-up
    for (int i = 0; i < 100; ++i) {
        workload <<<Dim_block, Dim_thread, 0, streams[0]>>> ();
    }
    hipDeviceSynchronize();
    hipMemcpy(g_flag, flag, sizeof(int) * FLAG_LENGTH, hipMemcpyHostToDevice);

    

    LaunchKernel <<<Dim_block, Dim_thread, 0, streams[0]>>> (g_flag, *d_task_pool);

    // 释放任务
    for (Task* task : tasks) {
      delete task;
    }

}