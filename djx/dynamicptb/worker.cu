#include "hip/hip_runtime.h"
__global__ int pull_taskID(task_queue){
    int top_block_id = task_queue.front(); // 原子操作
    task_queue.pop(); // 原子操作
    return top_block_id;
}


__global__ void Elastic_Kernel((origin_block_num, origin_thread_num, mem, streamid), &kernel_task_queue) {
    smID = get_current_smID();
    kill_redundant_worker();
    while(!empty_task()){
        if(!is_current_sm_idle(smID)) {
            sleep(10us);
            continue;
        }
        block_id = pull_taskID(kernel_task_queue);
        ... run as block_id:
    }
}

__global__ void Launch_kernel()