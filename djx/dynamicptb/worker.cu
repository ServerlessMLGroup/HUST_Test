#include "hip/hip_runtime.h"
typedef struct TASK {
    block_id,
    ...
}TASK;

__device__ int pull_taskID(&task_pool, smID){
    TASK task_info = task_pool.GetTASK(); // concurrent
    return task_info;
}

__device__ void Elastic_Kernel((streamid, sm_num, original_block_num), &kernel_task_pool) {
    smID = get_current_smID();
    kill_redundant_worker(); // 适应sm
    while(!empty_task(kernel_task_pool)){
        if(!is_current_sm_idle(smID)) {
            nanosleep();
            continue;
        }
        task_info = pull_taskID(kernel_task_pool, smID);
        ... run as task_info.block_id:
    }
}

__global__ void Launch_Kernel(args..., flags...) {
    init_flags(flags...);
    kernel_task_pool = init_task_pool();
    Elastic_Kernel((...), &kernel_task_pool);
}
