#include <stdio.h>
#include <stdlib.h>
#include<hip/hip_runtime.h>

#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)
// nvcc -arch=native ex.cu -o ex_sleep

#define GPU_RETURN_STATUS(cmd) \
{ \
    hipError_t result = cmd; \
    if (result != hipSuccess) { \
        std::cout << #cmd " error, return code:" << result << " | " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); \
    } \
}

// These are the inline versions for all of the SDK helper functions
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    const char *errorStr = NULL;
    errorStr = hipGetErrorString(err);
    fprintf(stderr,
            "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, "
            "line %i.\n",
            err, errorStr, file, line);
    exit(EXIT_FAILURE);
  }
}

__device__ uint get_smid(void) {

    uint ret;
  
    asm("mov.u32 %0, %smid;" : "=r"(ret) );
  
    return ret;
  
}

extern "C" __global__ void fused_nn_contrib_conv2d_winograd_without_weight_transform_add_kernel0_warm(float* __restrict__ placeholder, float* __restrict__ data_pack, int* flag, long long unsigned* times, long long unsigned* sm) {
    float d[16];
    float data_pack_local[16];
    for (int eps = 0; eps < 4; ++eps) {
        for (int nu = 0; nu < 4; ++nu) {
            d[(((eps * 4) + nu))] = (((((1 <= ((((((int)threadIdx.x) & 15) >> 2) * 2) + eps)) && (((((((int)threadIdx.x) & 15) >> 2) * 2) + eps) < 8)) && (1 <= (((((int)threadIdx.x) & 3) * 2) + nu))) && ((((((int)threadIdx.x) & 3) * 2) + nu) < 8)) ? placeholder[((((((((((int)blockIdx.x) * 392) + ((((int)threadIdx.x) >> 4) * 49)) + (((((int)threadIdx.x) & 15) >> 2) * 14)) + (eps * 7)) + ((((int)threadIdx.x) & 3) * 2)) + nu) - 8))] : 0.000000e+00f);
        }
    }
    data_pack_local[(0)] = 0.000000e+00f;
    data_pack_local[(0)] = (data_pack_local[(0)] + d[(0)]);
    data_pack_local[(0)] = (data_pack_local[(0)] + (d[(2)] * -1.000000e+00f));
    data_pack_local[(0)] = (data_pack_local[(0)] + (d[(8)] * -1.000000e+00f));
    data_pack_local[(0)] = (data_pack_local[(0)] + ((d[(10)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(1)] = 0.000000e+00f;
    data_pack_local[(1)] = (data_pack_local[(1)] + (d[(1)] * -1.000000e+00f));
    data_pack_local[(1)] = (data_pack_local[(1)] + d[(2)]);
    data_pack_local[(1)] = (data_pack_local[(1)] + ((d[(9)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(1)] = (data_pack_local[(1)] + (d[(10)] * -1.000000e+00f));
    data_pack_local[(2)] = 0.000000e+00f;
    data_pack_local[(2)] = (data_pack_local[(2)] + d[(1)]);
    data_pack_local[(2)] = (data_pack_local[(2)] + d[(2)]);
    data_pack_local[(2)] = (data_pack_local[(2)] + (d[(9)] * -1.000000e+00f));
    data_pack_local[(2)] = (data_pack_local[(2)] + (d[(10)] * -1.000000e+00f));
    data_pack_local[(3)] = 0.000000e+00f;
    data_pack_local[(3)] = (data_pack_local[(3)] + (d[(1)] * -1.000000e+00f));
    data_pack_local[(3)] = (data_pack_local[(3)] + d[(3)]);
    data_pack_local[(3)] = (data_pack_local[(3)] + ((d[(9)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(3)] = (data_pack_local[(3)] + (d[(11)] * -1.000000e+00f));
    data_pack_local[(4)] = 0.000000e+00f;
    data_pack_local[(4)] = (data_pack_local[(4)] + (d[(4)] * -1.000000e+00f));
    data_pack_local[(4)] = (data_pack_local[(4)] + ((d[(6)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(4)] = (data_pack_local[(4)] + d[(8)]);
    data_pack_local[(4)] = (data_pack_local[(4)] + (d[(10)] * -1.000000e+00f));
    data_pack_local[(5)] = 0.000000e+00f;
    data_pack_local[(5)] = (data_pack_local[(5)] + ((d[(5)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(5)] = (data_pack_local[(5)] + (d[(6)] * -1.000000e+00f));
    data_pack_local[(5)] = (data_pack_local[(5)] + (d[(9)] * -1.000000e+00f));
    data_pack_local[(5)] = (data_pack_local[(5)] + d[(10)]);
    data_pack_local[(6)] = 0.000000e+00f;
    data_pack_local[(6)] = (data_pack_local[(6)] + (d[(5)] * -1.000000e+00f));
    data_pack_local[(6)] = (data_pack_local[(6)] + (d[(6)] * -1.000000e+00f));
    data_pack_local[(6)] = (data_pack_local[(6)] + d[(9)]);
    data_pack_local[(6)] = (data_pack_local[(6)] + d[(10)]);
    data_pack_local[(7)] = 0.000000e+00f;
    data_pack_local[(7)] = (data_pack_local[(7)] + ((d[(5)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(7)] = (data_pack_local[(7)] + (d[(7)] * -1.000000e+00f));
    data_pack_local[(7)] = (data_pack_local[(7)] + (d[(9)] * -1.000000e+00f));
    data_pack_local[(7)] = (data_pack_local[(7)] + d[(11)]);
    data_pack_local[(8)] = 0.000000e+00f;
    data_pack_local[(8)] = (data_pack_local[(8)] + d[(4)]);
    data_pack_local[(8)] = (data_pack_local[(8)] + (d[(6)] * -1.000000e+00f));
    data_pack_local[(8)] = (data_pack_local[(8)] + d[(8)]);
    data_pack_local[(8)] = (data_pack_local[(8)] + (d[(10)] * -1.000000e+00f));
    data_pack_local[(9)] = 0.000000e+00f;
    data_pack_local[(9)] = (data_pack_local[(9)] + (d[(5)] * -1.000000e+00f));
    data_pack_local[(9)] = (data_pack_local[(9)] + d[(6)]);
    data_pack_local[(9)] = (data_pack_local[(9)] + (d[(9)] * -1.000000e+00f));
    data_pack_local[(9)] = (data_pack_local[(9)] + d[(10)]);
    data_pack_local[(10)] = 0.000000e+00f;
    data_pack_local[(10)] = (data_pack_local[(10)] + d[(5)]);
    data_pack_local[(10)] = (data_pack_local[(10)] + d[(6)]);
    data_pack_local[(10)] = (data_pack_local[(10)] + d[(9)]);
    data_pack_local[(10)] = (data_pack_local[(10)] + d[(10)]);
    data_pack_local[(11)] = 0.000000e+00f;
    data_pack_local[(11)] = (data_pack_local[(11)] + (d[(5)] * -1.000000e+00f));
    data_pack_local[(11)] = (data_pack_local[(11)] + d[(7)]);
    data_pack_local[(11)] = (data_pack_local[(11)] + (d[(9)] * -1.000000e+00f));
    data_pack_local[(11)] = (data_pack_local[(11)] + d[(11)]);
    data_pack_local[(12)] = 0.000000e+00f;
    data_pack_local[(12)] = (data_pack_local[(12)] + (d[(4)] * -1.000000e+00f));
    data_pack_local[(12)] = (data_pack_local[(12)] + ((d[(6)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(12)] = (data_pack_local[(12)] + d[(12)]);
    data_pack_local[(12)] = (data_pack_local[(12)] + (d[(14)] * -1.000000e+00f));
    data_pack_local[(13)] = 0.000000e+00f;
    data_pack_local[(13)] = (data_pack_local[(13)] + ((d[(5)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(13)] = (data_pack_local[(13)] + (d[(6)] * -1.000000e+00f));
    data_pack_local[(13)] = (data_pack_local[(13)] + (d[(13)] * -1.000000e+00f));
    data_pack_local[(13)] = (data_pack_local[(13)] + d[(14)]);
    data_pack_local[(14)] = 0.000000e+00f;
    data_pack_local[(14)] = (data_pack_local[(14)] + (d[(5)] * -1.000000e+00f));
    data_pack_local[(14)] = (data_pack_local[(14)] + (d[(6)] * -1.000000e+00f));
    data_pack_local[(14)] = (data_pack_local[(14)] + d[(13)]);
    data_pack_local[(14)] = (data_pack_local[(14)] + d[(14)]);
    data_pack_local[(15)] = 0.000000e+00f;
    data_pack_local[(15)] = (data_pack_local[(15)] + ((d[(5)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(15)] = (data_pack_local[(15)] + (d[(7)] * -1.000000e+00f));
    data_pack_local[(15)] = (data_pack_local[(15)] + (d[(13)] * -1.000000e+00f));
    data_pack_local[(15)] = (data_pack_local[(15)] + d[(15)]);
    for (int eps1 = 0; eps1 < 4; ++eps1) {
        for (int nu1 = 0; nu1 < 4; ++nu1) {
        data_pack[(((((eps1 * 32768) + (nu1 * 8192)) + (((int)blockIdx.x) * 128)) + ((int)threadIdx.x)))] = data_pack_local[(((eps1 * 4) + nu1))];
        }
    }
    
}

// sm_flag指示i号sm是保留的原先几号block
extern "C" __global__ void fused_nn_contrib_conv2d_winograd_without_weight_transform_add_kernel0(float* __restrict__ placeholder, float* __restrict__ data_pack, int* sm_flag, long long unsigned* worker_num, int* block_flag, long long unsigned* time) {
    unsigned int ns = 5;
    int smid = get_smid();
    if (threadIdx.x == 0 && atomicAdd(sm_flag + smid, 1) == 0) atomicAdd(block_flag + blockIdx.x, 1);
    __syncthreads();
    // while(atomicAdd(flag, 0) == 0) { // 40us版本
    //     __nanosleep(ns); 
    //     if (ns < 1000) {
    //         ns *= 2;
    //     }
    // }

    if (atomicAdd(block_flag + blockIdx.x, 0) == 0) return ;
    __syncthreads();
    
    // unsigned long long mclk;
    // if (threadIdx.x == 1) {
    //     asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk));
    //     time[get_smid()] = mclk / 1000;
    // }
    // while ((t1 - t0)/(1530000 * 1000.0f / 1000000) < 20) t1 = clock64(); // 20us, 1530000为kilohertz

    // if (threadIdx.x == 0) printf("%d %d\n", smid, blockIdx.x);
    // for (int i = 0; i < 100; ++i) { // 模拟多次主动感知
    ns = 5;
    __shared__ int BlockSyn[128 + 5];
    BlockSyn[threadIdx.x] = 0;
    if (threadIdx.x == 0) {
        while (atomicAdd(worker_num + smid, 0) == 0) {
            //if (threadIdx.x == 0) time[smid] += 1; 
            __nanosleep(10);
            // if (ns < 1000) {
            //     ns *= 2;
            // }
        }
        for(int i = 0; i < 128 + 5; ++i) {
            atomicAdd(BlockSyn + i, 1);
        }
    }
    else {
        while (atomicAdd(BlockSyn + threadIdx.x, 0) == 0) {
            //if (threadIdx.x == 0) time[smid] += 1; 
            __nanosleep(10);
            // if (ns < 1000) {
            //     ns *= 2;
            // }
        }
    }
    // if (threadIdx.x == 0) {
    //     asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk2));
    //     time[get_smid() + 80] = mclk2 / 1000;
    // }
    // if (get_smid() != smid - 1) printf("error in %d-%d\n", smid - 1, get_smid());
    unsigned long long mclk;
    if (threadIdx.x == 1) {
        asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk));
        time[get_smid()] = mclk / 1000;
    }
    float d[16];
    float data_pack_local[16];
    for (int eps = 0; eps < 4; ++eps) {
        for (int nu = 0; nu < 4; ++nu) {
            d[(((eps * 4) + nu))] = (((((1 <= ((((((int)threadIdx.x) & 15) >> 2) * 2) + eps)) && (((((((int)threadIdx.x) & 15) >> 2) * 2) + eps) < 8)) && (1 <= (((((int)threadIdx.x) & 3) * 2) + nu))) && ((((((int)threadIdx.x) & 3) * 2) + nu) < 8)) ? placeholder[((((((((((int)blockIdx.x) * 392) + ((((int)threadIdx.x) >> 4) * 49)) + (((((int)threadIdx.x) & 15) >> 2) * 14)) + (eps * 7)) + ((((int)threadIdx.x) & 3) * 2)) + nu) - 8))] : 0.000000e+00f);
        }
    }
    data_pack_local[(0)] = 0.000000e+00f;
    data_pack_local[(0)] = (data_pack_local[(0)] + d[(0)]);
    data_pack_local[(0)] = (data_pack_local[(0)] + (d[(2)] * -1.000000e+00f));
    data_pack_local[(0)] = (data_pack_local[(0)] + (d[(8)] * -1.000000e+00f));
    data_pack_local[(0)] = (data_pack_local[(0)] + ((d[(10)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(1)] = 0.000000e+00f;
    data_pack_local[(1)] = (data_pack_local[(1)] + (d[(1)] * -1.000000e+00f));
    data_pack_local[(1)] = (data_pack_local[(1)] + d[(2)]);
    data_pack_local[(1)] = (data_pack_local[(1)] + ((d[(9)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(1)] = (data_pack_local[(1)] + (d[(10)] * -1.000000e+00f));
    data_pack_local[(2)] = 0.000000e+00f;
    data_pack_local[(2)] = (data_pack_local[(2)] + d[(1)]);
    data_pack_local[(2)] = (data_pack_local[(2)] + d[(2)]);
    data_pack_local[(2)] = (data_pack_local[(2)] + (d[(9)] * -1.000000e+00f));
    data_pack_local[(2)] = (data_pack_local[(2)] + (d[(10)] * -1.000000e+00f));
    data_pack_local[(3)] = 0.000000e+00f;
    data_pack_local[(3)] = (data_pack_local[(3)] + (d[(1)] * -1.000000e+00f));
    data_pack_local[(3)] = (data_pack_local[(3)] + d[(3)]);
    data_pack_local[(3)] = (data_pack_local[(3)] + ((d[(9)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(3)] = (data_pack_local[(3)] + (d[(11)] * -1.000000e+00f));
    data_pack_local[(4)] = 0.000000e+00f;
    data_pack_local[(4)] = (data_pack_local[(4)] + (d[(4)] * -1.000000e+00f));
    data_pack_local[(4)] = (data_pack_local[(4)] + ((d[(6)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(4)] = (data_pack_local[(4)] + d[(8)]);
    data_pack_local[(4)] = (data_pack_local[(4)] + (d[(10)] * -1.000000e+00f));
    data_pack_local[(5)] = 0.000000e+00f;
    data_pack_local[(5)] = (data_pack_local[(5)] + ((d[(5)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(5)] = (data_pack_local[(5)] + (d[(6)] * -1.000000e+00f));
    data_pack_local[(5)] = (data_pack_local[(5)] + (d[(9)] * -1.000000e+00f));
    data_pack_local[(5)] = (data_pack_local[(5)] + d[(10)]);
    data_pack_local[(6)] = 0.000000e+00f;
    data_pack_local[(6)] = (data_pack_local[(6)] + (d[(5)] * -1.000000e+00f));
    data_pack_local[(6)] = (data_pack_local[(6)] + (d[(6)] * -1.000000e+00f));
    data_pack_local[(6)] = (data_pack_local[(6)] + d[(9)]);
    data_pack_local[(6)] = (data_pack_local[(6)] + d[(10)]);
    data_pack_local[(7)] = 0.000000e+00f;
    data_pack_local[(7)] = (data_pack_local[(7)] + ((d[(5)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(7)] = (data_pack_local[(7)] + (d[(7)] * -1.000000e+00f));
    data_pack_local[(7)] = (data_pack_local[(7)] + (d[(9)] * -1.000000e+00f));
    data_pack_local[(7)] = (data_pack_local[(7)] + d[(11)]);
    data_pack_local[(8)] = 0.000000e+00f;
    data_pack_local[(8)] = (data_pack_local[(8)] + d[(4)]);
    data_pack_local[(8)] = (data_pack_local[(8)] + (d[(6)] * -1.000000e+00f));
    data_pack_local[(8)] = (data_pack_local[(8)] + d[(8)]);
    data_pack_local[(8)] = (data_pack_local[(8)] + (d[(10)] * -1.000000e+00f));
    data_pack_local[(9)] = 0.000000e+00f;
    data_pack_local[(9)] = (data_pack_local[(9)] + (d[(5)] * -1.000000e+00f));
    data_pack_local[(9)] = (data_pack_local[(9)] + d[(6)]);
    data_pack_local[(9)] = (data_pack_local[(9)] + (d[(9)] * -1.000000e+00f));
    data_pack_local[(9)] = (data_pack_local[(9)] + d[(10)]);
    data_pack_local[(10)] = 0.000000e+00f;
    data_pack_local[(10)] = (data_pack_local[(10)] + d[(5)]);
    data_pack_local[(10)] = (data_pack_local[(10)] + d[(6)]);
    data_pack_local[(10)] = (data_pack_local[(10)] + d[(9)]);
    data_pack_local[(10)] = (data_pack_local[(10)] + d[(10)]);
    data_pack_local[(11)] = 0.000000e+00f;
    data_pack_local[(11)] = (data_pack_local[(11)] + (d[(5)] * -1.000000e+00f));
    data_pack_local[(11)] = (data_pack_local[(11)] + d[(7)]);
    data_pack_local[(11)] = (data_pack_local[(11)] + (d[(9)] * -1.000000e+00f));
    data_pack_local[(11)] = (data_pack_local[(11)] + d[(11)]);
    data_pack_local[(12)] = 0.000000e+00f;
    data_pack_local[(12)] = (data_pack_local[(12)] + (d[(4)] * -1.000000e+00f));
    data_pack_local[(12)] = (data_pack_local[(12)] + ((d[(6)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(12)] = (data_pack_local[(12)] + d[(12)]);
    data_pack_local[(12)] = (data_pack_local[(12)] + (d[(14)] * -1.000000e+00f));
    data_pack_local[(13)] = 0.000000e+00f;
    data_pack_local[(13)] = (data_pack_local[(13)] + ((d[(5)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(13)] = (data_pack_local[(13)] + (d[(6)] * -1.000000e+00f));
    data_pack_local[(13)] = (data_pack_local[(13)] + (d[(13)] * -1.000000e+00f));
    data_pack_local[(13)] = (data_pack_local[(13)] + d[(14)]);
    data_pack_local[(14)] = 0.000000e+00f;
    data_pack_local[(14)] = (data_pack_local[(14)] + (d[(5)] * -1.000000e+00f));
    data_pack_local[(14)] = (data_pack_local[(14)] + (d[(6)] * -1.000000e+00f));
    data_pack_local[(14)] = (data_pack_local[(14)] + d[(13)]);
    data_pack_local[(14)] = (data_pack_local[(14)] + d[(14)]);
    data_pack_local[(15)] = 0.000000e+00f;
    data_pack_local[(15)] = (data_pack_local[(15)] + ((d[(5)] * -1.000000e+00f) * -1.000000e+00f));
    data_pack_local[(15)] = (data_pack_local[(15)] + (d[(7)] * -1.000000e+00f));
    data_pack_local[(15)] = (data_pack_local[(15)] + (d[(13)] * -1.000000e+00f));
    data_pack_local[(15)] = (data_pack_local[(15)] + d[(15)]);
    for (int eps1 = 0; eps1 < 4; ++eps1) {
        for (int nu1 = 0; nu1 < 4; ++nu1) {
        data_pack[(((((eps1 * 32768) + (nu1 * 8192)) + (((int)blockIdx.x) * 128)) + ((int)threadIdx.x)))] = data_pack_local[(((eps1 * 4) + nu1))];
        }
    }

    unsigned long long mclk2; 
    if (threadIdx.x == 1) {
        if (smid == 0) {
            for (int i = 0; i < 80; ++i) {
                atomicExch(worker_num + i, 1);
            }
        }
        asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk2));
        time[get_smid() + 80] = (mclk2) / 1000;
    }
    
}


void run_kernel() {
	int num_streams = 2;
	hipStream_t streams[num_streams];
	for (int i = 0; i < num_streams; i++) {
		hipStreamCreate(&streams[i]);
	}
	
    // allocate kernel_sleep sm
	long long unsigned *h_sm = new long long unsigned[64];
	long long unsigned *d_sm;
	hipMalloc(&d_sm, 64 * sizeof(long long unsigned));

    long long unsigned *h_sm_ids = new long long unsigned[64 * 2];
	long long unsigned *d_sm_ids;
	hipMalloc(&d_sm_ids, 64 * sizeof(long long unsigned) * 2);

    // // allocate resource
    float *h_args_55 = new float[25088]; // 55
    float *d_args_55;
    hipMalloc(&d_args_55, sizeof(float) * 25088);

    float *h_args_56 = new float[4194304]; // 56
    float *d_args_56;
    hipMalloc(&d_args_56, sizeof(float) * 4194304);

    float *h_args_75 = new float[1806336 / 4 + 1]; // 55
    float *d_args_75;
    hipMalloc(&d_args_75, sizeof(float) * 1806336 / 4 + 4);

    float *h_args_76 = new float[1806336 / 4 + 1]; // 55
    float *d_args_76;
    hipMalloc(&d_args_76, sizeof(float) * 1806336 / 4 + 4);


    // allocate flag
    int *sm_flag = new int[85];
    int *g_sm_flag;
    for (int i = 0; i < 85; ++i) {
        sm_flag[i] = 0;
    }
    hipMalloc((void **)&g_sm_flag, sizeof(int) * 85);
    hipMemcpy(g_sm_flag, sm_flag, sizeof(int) * 85, hipMemcpyHostToDevice);

    int *block_flag = new int[300];
    int *g_block_flag;
    for (int i = 0; i < 300; ++i) {
        block_flag[i] = 0;
    }
    hipMalloc((void **)&g_block_flag, sizeof(int) * 300);
    hipMemcpy(g_block_flag, block_flag, sizeof(int) * 300, hipMemcpyHostToDevice);

    // allocate kernel_sleep sm
	long long unsigned *worker_num = new long long unsigned[80];
    for (int i = 0; i < 40; ++i) {
        worker_num[i] = 1;
        worker_num[80 - i - 1] = 0;
    }
	long long unsigned *d_worker_num;
	hipMalloc(&d_worker_num, 80 * sizeof(long long unsigned));
    hipMemcpy(d_worker_num, worker_num, sizeof(long long unsigned) * 80, hipMemcpyHostToDevice);

    long long unsigned *time = new long long unsigned[200];
	long long unsigned *d_time;
	hipMalloc(&d_time, 200 * sizeof(long long unsigned));


    // allocate warm flag
    int *flag_warm;
    int *g_flag_warm;
    flag_warm = (int*) malloc(1 * sizeof(int));
    flag_warm[0] = 0;
    hipMalloc((void **)&g_flag_warm, sizeof(int) * 9000);
    hipMemcpy(g_flag_warm, flag_warm, sizeof(int) * 9000, hipMemcpyHostToDevice);

    // cuda launch kernel
	dim3 D_b_a = dim3(200, 1, 1);
	dim3 D_t_a = dim3(128, 1, 1);
    // warm-up
    for (int i = 0; i < 100; ++i) {
        fused_nn_contrib_conv2d_winograd_without_weight_transform_add_kernel0_warm <<<D_b_a, D_t_a, 0, streams[0]>>>(d_args_55, d_args_76, g_flag_warm, d_sm_ids, d_sm);
    }
	hipDeviceSynchronize();
    // test kernel
	fused_nn_contrib_conv2d_winograd_without_weight_transform_add_kernel0 <<<D_b_a, D_t_a, 0, streams[0]>>>(d_args_55, d_args_76, g_sm_flag, d_worker_num, g_block_flag, d_time);
    // sleep until kernel finish
	// fused_nn_contrib_conv2d_winograd_without_weight_transform_add_kernel1 <<<D_b_b, D_t_b, 0, streams[1]>>>(d_args_56, d_args_76, d_args_75, g_flag, d_sm_ids2, d_sleep_sm, g_sleep_times);
	// kernel_sleep <<<D_b_b, D_t_b, 0, streams[1]>>>(15.6, 64.9, 134.7, 1000, g_flag);
	hipDeviceSynchronize();

    // cudaMemcpy(h_sm_ids, d_sm_ids, 64 * sizeof(long long unsigned) * 2, cudaMemcpyDeviceToHost);
	hipMemcpy(time, d_time, 200 * sizeof(long long unsigned), hipMemcpyDeviceToHost);
    for (int i = 0; i < 80; ++i) {
        printf("sm-%d---start_time:%llu end_time:%llu\n", i, time[i] , time[i + 80]);
    }
	

}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        printf("args num error! argc:%d", argc);
        exit(1);
    }
    int gpu_no = atoi(argv[1]);
    checkCudaErrors(hipSetDevice(gpu_no));

    // cudaDeviceProp  prop;
    // cudaGetDeviceProperties(&prop, 0); 
    // clock_t clock_rate = prop.clockRate;
    // printf("clock_rate:%d\n", clock_rate); // 1530000
	run_kernel();

	return 0;
}

