#include <hip/hip_runtime.h>
#include <iostream>

class Array {
public:
  Array(size_t size) : size_(size) {
    hipMalloc(&data_, size_ * sizeof(int));
  }

  ~Array() {
    hipFree(data_);
  }

  int* data() { return data_; }
  size_t size() const { return size_; }

private:
  int* data_;
  size_t size_;
};

__global__ void kernel(Array array) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < array.size()) {
    array.data()[idx] = idx;
  }
}

int main() {
  const size_t size = 1024;
  Array array(size);

  kernel<<<1, size>>>(array);
  hipDeviceSynchronize();

  int* data = new int[size];
  hipMemcpy(data, array.data(), size * sizeof(int), hipMemcpyDeviceToHost);

  for (size_t i = 0; i < size; ++i) {
    std::cout << data[i] << std::endl;
  }

  delete[] data;
  return 0;
}
