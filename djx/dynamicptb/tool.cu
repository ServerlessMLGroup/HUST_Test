#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>
#include <stdio.h>

#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)

#define GPU_RETURN_STATUS(cmd) \
{ \
    hipError_t result = cmd; \
    if (result != hipSuccess) { \
        std::cout << #cmd " error, return code:" << result << " | " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); \
    } \
}

// These are the inline versions for all of the SDK helper functions
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    const char *errorStr = NULL;
    errorStr = hipGetErrorString(err);
    fprintf(stderr,
            "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, "
            "line %i.\n",
            err, errorStr, file, line);
    exit(EXIT_FAILURE);
  }
}

void initDevice() {
    if (argc < 2) {
      printf("args num error! argc:%d", argc);
      exit(1);
    }
    int gpu_no = atoi(argv[1]);
    checkCudaErrors(hipSetDevice(gpu_no));
}

__device__ uint get_smid(void) {
    uint ret;
    asm("mov.u32 %0, %smid;" : "=r"(ret) );
    return ret;
}
