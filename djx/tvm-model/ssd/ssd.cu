
#include <hip/hip_runtime.h>

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 700)
#define __shfl_sync(mask, var, lane, width) \
        __shfl((var), (lane), (width))

#define __shfl_down_sync(mask, var, offset, width) \
        __shfl_down((var), (offset), (width))

#define __shfl_up_sync(mask, var, offset, width) \
        __shfl_up((var), (offset), (width))
#endif

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_4_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[4];
  __shared__ float pad_temp_shared[784];
  __shared__ float placeholder_shared[128];
  #pragma unroll
  for (int ff_init = 0; ff_init < 2; ++ff_init) {
    compute[(ff_init)] = 0.000000e+00f;
    compute[((ff_init + 2))] = 0.000000e+00f;
  }
  for (int rc_outer = 0; rc_outer < 64; ++rc_outer) {
    __syncthreads();
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      pad_temp_shared[(((((((int)threadIdx.z) * 196) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = placeholder[(((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 14)) / 7) * 196)) + (((int)blockIdx.y) * 98)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 14)) % 7) * 14)) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 14)))];
    }
    if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) >> 3)) < 16) {
      if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 5)) + ((int)threadIdx.x)) < 128) {
        if (((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) < 32) {
          if (((int)threadIdx.x) < 5) {
            placeholder_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 5)) + ((int)threadIdx.x)))] = placeholder1[((((((((int)blockIdx.z) * 8192) + (((int)threadIdx.z) * 2048)) + ((((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) >> 3) * 512)) + (rc_outer * 8)) + (((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) & 7)))];
          }
        }
      }
    }
    __syncthreads();
    #pragma unroll
    for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
      #pragma unroll
      for (int ff = 0; ff < 2; ++ff) {
        compute[(ff)] = (compute[(ff)] + (pad_temp_shared[((((rc_inner * 98) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)))] * placeholder_shared[((((((int)threadIdx.z) * 16) + (ff * 8)) + rc_inner))]));
        compute[((ff + 2))] = (compute[((ff + 2))] + (pad_temp_shared[((((rc_inner * 98) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)))] * placeholder_shared[(((((((int)threadIdx.z) * 16) + (ff * 8)) + rc_inner) + 64))]));
      }
    }
  }
  #pragma unroll
  for (int ax1_inner_inner_inner = 0; ax1_inner_inner_inner < 2; ++ax1_inner_inner_inner) {
    T_relu[(((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 392)) + (ax1_inner_inner_inner * 196)) + (((int)blockIdx.y) * 98)) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)))] = max((compute[(ax1_inner_inner_inner)] + placeholder2[((((((int)blockIdx.z) * 16) + (((int)threadIdx.z) * 2)) + ax1_inner_inner_inner))]), 0.000000e+00f);
    T_relu[((((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 392)) + (ax1_inner_inner_inner * 196)) + (((int)blockIdx.y) * 98)) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 1568))] = max((compute[((ax1_inner_inner_inner + 2))] + placeholder2[(((((((int)blockIdx.z) * 16) + (((int)threadIdx.z) * 2)) + ax1_inner_inner_inner) + 8))]), 0.000000e+00f);
  }
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_18_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[16];
  __shared__ float pad_temp_shared[297];
  __shared__ float placeholder_shared[288];
  compute[(0)] = 0.000000e+00f;
  compute[(4)] = 0.000000e+00f;
  compute[(8)] = 0.000000e+00f;
  compute[(12)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  compute[(5)] = 0.000000e+00f;
  compute[(9)] = 0.000000e+00f;
  compute[(13)] = 0.000000e+00f;
  compute[(2)] = 0.000000e+00f;
  compute[(6)] = 0.000000e+00f;
  compute[(10)] = 0.000000e+00f;
  compute[(14)] = 0.000000e+00f;
  compute[(3)] = 0.000000e+00f;
  compute[(7)] = 0.000000e+00f;
  compute[(11)] = 0.000000e+00f;
  compute[(15)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 3; ++rc_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      if ((((((int)threadIdx.z) * 38) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 297) {
        if (((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 38) {
          pad_temp_shared[((((((int)threadIdx.z) * 38) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((1 <= ((((int)blockIdx.y) * 8) + ((((((int)threadIdx.z) * 38) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 33))) && (1 <= ((((int)blockIdx.x) * 32) + ((((((int)threadIdx.z) * 38) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 33)))) ? placeholder[(((((((rc_outer * 50176) + (((int)blockIdx.y) * 1792)) + (((((((int)threadIdx.z) * 38) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 33) * 224)) + (((int)blockIdx.x) * 32)) + ((((((int)threadIdx.z) * 38) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 33)) - 225))] : 0.000000e+00f);
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
      if (((((int)threadIdx.z) * 4) + (((int)threadIdx.x) / 3)) < 32) {
        if (((((int)threadIdx.z) * 12) + ((int)threadIdx.x)) < 96) {
          if ((((((int)threadIdx.z) * 36) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 288) {
            if (((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 36) {
              placeholder_shared[((((((int)threadIdx.z) * 36) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = placeholder1[((((((((int)threadIdx.z) * 108) + ((((int)threadIdx.x) / 3) * 27)) + (rc_outer * 9)) + ((((int)threadIdx.x) % 3) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))];
            }
          }
        }
      }
    }
    __syncthreads();
    for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
      for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
        compute[(0)] = (compute[(0)] + (pad_temp_shared[((((ry_inner * 33) + (((int)threadIdx.x) * 2)) + rx_inner))] * placeholder_shared[((((((int)threadIdx.z) * 9) + (ry_inner * 3)) + rx_inner))]));
        compute[(4)] = (compute[(4)] + (pad_temp_shared[((((ry_inner * 33) + (((int)threadIdx.x) * 2)) + rx_inner))] * placeholder_shared[(((((((int)threadIdx.z) * 9) + (ry_inner * 3)) + rx_inner) + 72))]));
        compute[(8)] = (compute[(8)] + (pad_temp_shared[((((ry_inner * 33) + (((int)threadIdx.x) * 2)) + rx_inner))] * placeholder_shared[(((((((int)threadIdx.z) * 9) + (ry_inner * 3)) + rx_inner) + 144))]));
        compute[(12)] = (compute[(12)] + (pad_temp_shared[((((ry_inner * 33) + (((int)threadIdx.x) * 2)) + rx_inner))] * placeholder_shared[(((((((int)threadIdx.z) * 9) + (ry_inner * 3)) + rx_inner) + 216))]));
        compute[(1)] = (compute[(1)] + (pad_temp_shared[(((((ry_inner * 33) + (((int)threadIdx.x) * 2)) + rx_inner) + 66))] * placeholder_shared[((((((int)threadIdx.z) * 9) + (ry_inner * 3)) + rx_inner))]));
        compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((ry_inner * 33) + (((int)threadIdx.x) * 2)) + rx_inner) + 66))] * placeholder_shared[(((((((int)threadIdx.z) * 9) + (ry_inner * 3)) + rx_inner) + 72))]));
        compute[(9)] = (compute[(9)] + (pad_temp_shared[(((((ry_inner * 33) + (((int)threadIdx.x) * 2)) + rx_inner) + 66))] * placeholder_shared[(((((((int)threadIdx.z) * 9) + (ry_inner * 3)) + rx_inner) + 144))]));
        compute[(13)] = (compute[(13)] + (pad_temp_shared[(((((ry_inner * 33) + (((int)threadIdx.x) * 2)) + rx_inner) + 66))] * placeholder_shared[(((((((int)threadIdx.z) * 9) + (ry_inner * 3)) + rx_inner) + 216))]));
        compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((ry_inner * 33) + (((int)threadIdx.x) * 2)) + rx_inner) + 132))] * placeholder_shared[((((((int)threadIdx.z) * 9) + (ry_inner * 3)) + rx_inner))]));
        compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((ry_inner * 33) + (((int)threadIdx.x) * 2)) + rx_inner) + 132))] * placeholder_shared[(((((((int)threadIdx.z) * 9) + (ry_inner * 3)) + rx_inner) + 72))]));
        compute[(10)] = (compute[(10)] + (pad_temp_shared[(((((ry_inner * 33) + (((int)threadIdx.x) * 2)) + rx_inner) + 132))] * placeholder_shared[(((((((int)threadIdx.z) * 9) + (ry_inner * 3)) + rx_inner) + 144))]));
        compute[(14)] = (compute[(14)] + (pad_temp_shared[(((((ry_inner * 33) + (((int)threadIdx.x) * 2)) + rx_inner) + 132))] * placeholder_shared[(((((((int)threadIdx.z) * 9) + (ry_inner * 3)) + rx_inner) + 216))]));
        compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((ry_inner * 33) + (((int)threadIdx.x) * 2)) + rx_inner) + 198))] * placeholder_shared[((((((int)threadIdx.z) * 9) + (ry_inner * 3)) + rx_inner))]));
        compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((ry_inner * 33) + (((int)threadIdx.x) * 2)) + rx_inner) + 198))] * placeholder_shared[(((((((int)threadIdx.z) * 9) + (ry_inner * 3)) + rx_inner) + 72))]));
        compute[(11)] = (compute[(11)] + (pad_temp_shared[(((((ry_inner * 33) + (((int)threadIdx.x) * 2)) + rx_inner) + 198))] * placeholder_shared[(((((((int)threadIdx.z) * 9) + (ry_inner * 3)) + rx_inner) + 144))]));
        compute[(15)] = (compute[(15)] + (pad_temp_shared[(((((ry_inner * 33) + (((int)threadIdx.x) * 2)) + rx_inner) + 198))] * placeholder_shared[(((((((int)threadIdx.z) * 9) + (ry_inner * 3)) + rx_inner) + 216))]));
      }
    }
  }
  T_relu[(((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)))] = max((compute[(0)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
  T_relu[((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 100352))] = max((compute[(4)] + placeholder2[((((int)threadIdx.z) + 8))]), 0.000000e+00f);
  T_relu[((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 200704))] = max((compute[(8)] + placeholder2[((((int)threadIdx.z) + 16))]), 0.000000e+00f);
  T_relu[((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 301056))] = max((compute[(12)] + placeholder2[((((int)threadIdx.z) + 24))]), 0.000000e+00f);
  T_relu[((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 112))] = max((compute[(1)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
  T_relu[((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 100464))] = max((compute[(5)] + placeholder2[((((int)threadIdx.z) + 8))]), 0.000000e+00f);
  T_relu[((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 200816))] = max((compute[(9)] + placeholder2[((((int)threadIdx.z) + 16))]), 0.000000e+00f);
  T_relu[((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 301168))] = max((compute[(13)] + placeholder2[((((int)threadIdx.z) + 24))]), 0.000000e+00f);
  T_relu[((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 224))] = max((compute[(2)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
  T_relu[((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 100576))] = max((compute[(6)] + placeholder2[((((int)threadIdx.z) + 8))]), 0.000000e+00f);
  T_relu[((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 200928))] = max((compute[(10)] + placeholder2[((((int)threadIdx.z) + 16))]), 0.000000e+00f);
  T_relu[((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 301280))] = max((compute[(14)] + placeholder2[((((int)threadIdx.z) + 24))]), 0.000000e+00f);
  T_relu[((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 336))] = max((compute[(3)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
  T_relu[((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 100688))] = max((compute[(7)] + placeholder2[((((int)threadIdx.z) + 8))]), 0.000000e+00f);
  T_relu[((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 201040))] = max((compute[(11)] + placeholder2[((((int)threadIdx.z) + 16))]), 0.000000e+00f);
  T_relu[((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 301392))] = max((compute[(15)] + placeholder2[((((int)threadIdx.z) + 24))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_16_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[16];
  __shared__ float pad_temp_shared[1792];
  __shared__ float placeholder_shared[1024];
  for (int yy_init = 0; yy_init < 2; ++yy_init) {
    compute[(yy_init)] = 0.000000e+00f;
    compute[((yy_init + 4))] = 0.000000e+00f;
    compute[((yy_init + 8))] = 0.000000e+00f;
    compute[((yy_init + 12))] = 0.000000e+00f;
    compute[((yy_init + 2))] = 0.000000e+00f;
    compute[((yy_init + 6))] = 0.000000e+00f;
    compute[((yy_init + 10))] = 0.000000e+00f;
    compute[((yy_init + 14))] = 0.000000e+00f;
  }
  for (int rc_outer = 0; rc_outer < 2; ++rc_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      pad_temp_shared[((((((int)threadIdx.z) * 112) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = placeholder[(((((((rc_outer * 200704) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 224)) + ((((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 56) * 112)) + (((int)blockIdx.x) * 56)) + (((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 56)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
      if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) >> 4)) < 64) {
        if ((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 1024) {
          if (((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 64) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = placeholder1[(((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) >> 4) * 32)) + (rc_outer * 16)) + (((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) & 15)))];
          }
        }
      }
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
      for (int yy = 0; yy < 2; ++yy) {
        compute[(yy)] = (compute[(yy)] + (pad_temp_shared[((((rc_inner * 112) + (yy * 56)) + ((int)threadIdx.x)))] * placeholder_shared[(((((int)threadIdx.z) * 16) + rc_inner))]));
        compute[((yy + 4))] = (compute[((yy + 4))] + (pad_temp_shared[((((rc_inner * 112) + (yy * 56)) + ((int)threadIdx.x)))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 256))]));
        compute[((yy + 8))] = (compute[((yy + 8))] + (pad_temp_shared[((((rc_inner * 112) + (yy * 56)) + ((int)threadIdx.x)))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 512))]));
        compute[((yy + 12))] = (compute[((yy + 12))] + (pad_temp_shared[((((rc_inner * 112) + (yy * 56)) + ((int)threadIdx.x)))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 768))]));
        compute[((yy + 2))] = (compute[((yy + 2))] + (pad_temp_shared[(((((rc_inner * 112) + (yy * 56)) + ((int)threadIdx.x)) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 16) + rc_inner))]));
        compute[((yy + 6))] = (compute[((yy + 6))] + (pad_temp_shared[(((((rc_inner * 112) + (yy * 56)) + ((int)threadIdx.x)) + 28))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 256))]));
        compute[((yy + 10))] = (compute[((yy + 10))] + (pad_temp_shared[(((((rc_inner * 112) + (yy * 56)) + ((int)threadIdx.x)) + 28))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 512))]));
        compute[((yy + 14))] = (compute[((yy + 14))] + (pad_temp_shared[(((((rc_inner * 112) + (yy * 56)) + ((int)threadIdx.x)) + 28))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 768))]));
      }
    }
  }
  for (int ax2_inner_inner_inner = 0; ax2_inner_inner_inner < 2; ++ax2_inner_inner_inner) {
    T_relu[((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 224)) + (ax2_inner_inner_inner * 112)) + (((int)blockIdx.x) * 56)) + ((int)threadIdx.x)))] = max((compute[(ax2_inner_inner_inner)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
    T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 224)) + (ax2_inner_inner_inner * 112)) + (((int)blockIdx.x) * 56)) + ((int)threadIdx.x)) + 200704))] = max((compute[((ax2_inner_inner_inner + 4))] + placeholder2[((((int)threadIdx.z) + 16))]), 0.000000e+00f);
    T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 224)) + (ax2_inner_inner_inner * 112)) + (((int)blockIdx.x) * 56)) + ((int)threadIdx.x)) + 401408))] = max((compute[((ax2_inner_inner_inner + 8))] + placeholder2[((((int)threadIdx.z) + 32))]), 0.000000e+00f);
    T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 224)) + (ax2_inner_inner_inner * 112)) + (((int)blockIdx.x) * 56)) + ((int)threadIdx.x)) + 602112))] = max((compute[((ax2_inner_inner_inner + 12))] + placeholder2[((((int)threadIdx.z) + 48))]), 0.000000e+00f);
    T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 224)) + (ax2_inner_inner_inner * 112)) + (((int)blockIdx.x) * 56)) + ((int)threadIdx.x)) + 28))] = max((compute[((ax2_inner_inner_inner + 2))] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
    T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 224)) + (ax2_inner_inner_inner * 112)) + (((int)blockIdx.x) * 56)) + ((int)threadIdx.x)) + 200732))] = max((compute[((ax2_inner_inner_inner + 6))] + placeholder2[((((int)threadIdx.z) + 16))]), 0.000000e+00f);
    T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 224)) + (ax2_inner_inner_inner * 112)) + (((int)blockIdx.x) * 56)) + ((int)threadIdx.x)) + 401436))] = max((compute[((ax2_inner_inner_inner + 10))] + placeholder2[((((int)threadIdx.z) + 32))]), 0.000000e+00f);
    T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 224)) + (ax2_inner_inner_inner * 112)) + (((int)blockIdx.x) * 56)) + ((int)threadIdx.x)) + 602140))] = max((compute[((ax2_inner_inner_inner + 14))] + placeholder2[((((int)threadIdx.z) + 48))]), 0.000000e+00f);
  }
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_9_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  __shared__ float PaddedInput_shared[900];
  __shared__ float placeholder_shared[9];
  float PaddedInput_shared_local[24];
  float placeholder_shared_local[9];
  float DepthwiseConv2d[4];
  PaddedInput_shared[(((((int)threadIdx.y) * 28) + ((int)threadIdx.x)))] = ((((30 <= ((((int)threadIdx.y) * 28) + ((int)threadIdx.x))) && (1 <= (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) % 30))) && ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) % 30) < 29)) ? placeholder[(((((((int)blockIdx.z) * 784) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) / 30) * 28)) + (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) % 30)) - 29))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 196))] = (((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 16) % 30)) && (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 16) % 30) < 29)) ? placeholder[(((((((int)blockIdx.z) * 784) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 196) / 30) * 28)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 16) % 30)) - 29))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 392))] = (((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 2) % 30)) && (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 2) % 30) < 29)) ? placeholder[(((((((int)blockIdx.z) * 784) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 392) / 30) * 28)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 2) % 30)) - 29))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 588))] = (((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 18) % 30)) && (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 18) % 30) < 29)) ? placeholder[(((((((int)blockIdx.z) * 784) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 588) / 30) * 28)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 18) % 30)) - 29))] : 0.000000e+00f);
  if (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) < 116) {
    if (((int)threadIdx.y) < 5) {
      PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 784))] = ((((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) < 86) && (1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 4) % 30))) && (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 4) % 30) < 29)) ? placeholder[(((((((int)blockIdx.z) * 784) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 784) / 30) * 28)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 4) % 30)) - 29))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) < 9) {
    if (((int)threadIdx.y) < 1) {
      placeholder_shared[(((((int)threadIdx.y) * 28) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.y) * 28) + (((int)blockIdx.z) * 9)) + ((int)threadIdx.x)))];
    }
  }
  __syncthreads();
  PaddedInput_shared_local[(0)] = PaddedInput_shared[(((((int)threadIdx.y) * 60) + ((int)threadIdx.x)))];
  PaddedInput_shared_local[(12)] = PaddedInput_shared[((((((int)threadIdx.y) * 60) + ((int)threadIdx.x)) + 420))];
  PaddedInput_shared_local[(1)] = PaddedInput_shared[((((((int)threadIdx.y) * 60) + ((int)threadIdx.x)) + 1))];
  PaddedInput_shared_local[(13)] = PaddedInput_shared[((((((int)threadIdx.y) * 60) + ((int)threadIdx.x)) + 421))];
  PaddedInput_shared_local[(2)] = PaddedInput_shared[((((((int)threadIdx.y) * 60) + ((int)threadIdx.x)) + 2))];
  PaddedInput_shared_local[(14)] = PaddedInput_shared[((((((int)threadIdx.y) * 60) + ((int)threadIdx.x)) + 422))];
  PaddedInput_shared_local[(3)] = PaddedInput_shared[((((((int)threadIdx.y) * 60) + ((int)threadIdx.x)) + 30))];
  PaddedInput_shared_local[(15)] = PaddedInput_shared[((((((int)threadIdx.y) * 60) + ((int)threadIdx.x)) + 450))];
  PaddedInput_shared_local[(4)] = PaddedInput_shared[((((((int)threadIdx.y) * 60) + ((int)threadIdx.x)) + 31))];
  PaddedInput_shared_local[(16)] = PaddedInput_shared[((((((int)threadIdx.y) * 60) + ((int)threadIdx.x)) + 451))];
  PaddedInput_shared_local[(5)] = PaddedInput_shared[((((((int)threadIdx.y) * 60) + ((int)threadIdx.x)) + 32))];
  PaddedInput_shared_local[(17)] = PaddedInput_shared[((((((int)threadIdx.y) * 60) + ((int)threadIdx.x)) + 452))];
  PaddedInput_shared_local[(6)] = PaddedInput_shared[((((((int)threadIdx.y) * 60) + ((int)threadIdx.x)) + 60))];
  PaddedInput_shared_local[(18)] = PaddedInput_shared[((((((int)threadIdx.y) * 60) + ((int)threadIdx.x)) + 480))];
  PaddedInput_shared_local[(7)] = PaddedInput_shared[((((((int)threadIdx.y) * 60) + ((int)threadIdx.x)) + 61))];
  PaddedInput_shared_local[(19)] = PaddedInput_shared[((((((int)threadIdx.y) * 60) + ((int)threadIdx.x)) + 481))];
  PaddedInput_shared_local[(8)] = PaddedInput_shared[((((((int)threadIdx.y) * 60) + ((int)threadIdx.x)) + 62))];
  PaddedInput_shared_local[(20)] = PaddedInput_shared[((((((int)threadIdx.y) * 60) + ((int)threadIdx.x)) + 482))];
  PaddedInput_shared_local[(9)] = PaddedInput_shared[((((((int)threadIdx.y) * 60) + ((int)threadIdx.x)) + 90))];
  PaddedInput_shared_local[(21)] = PaddedInput_shared[((((((int)threadIdx.y) * 60) + ((int)threadIdx.x)) + 510))];
  PaddedInput_shared_local[(10)] = PaddedInput_shared[((((((int)threadIdx.y) * 60) + ((int)threadIdx.x)) + 91))];
  PaddedInput_shared_local[(22)] = PaddedInput_shared[((((((int)threadIdx.y) * 60) + ((int)threadIdx.x)) + 511))];
  PaddedInput_shared_local[(11)] = PaddedInput_shared[((((((int)threadIdx.y) * 60) + ((int)threadIdx.x)) + 92))];
  PaddedInput_shared_local[(23)] = PaddedInput_shared[((((((int)threadIdx.y) * 60) + ((int)threadIdx.x)) + 512))];
  placeholder_shared_local[(0)] = placeholder_shared[(0)];
  placeholder_shared_local[(1)] = placeholder_shared[(1)];
  placeholder_shared_local[(2)] = placeholder_shared[(2)];
  placeholder_shared_local[(3)] = placeholder_shared[(3)];
  placeholder_shared_local[(4)] = placeholder_shared[(4)];
  placeholder_shared_local[(5)] = placeholder_shared[(5)];
  placeholder_shared_local[(6)] = placeholder_shared[(6)];
  placeholder_shared_local[(7)] = placeholder_shared[(7)];
  placeholder_shared_local[(8)] = placeholder_shared[(8)];
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  DepthwiseConv2d[(2)] = 0.000000e+00f;
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(0)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(12)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(1)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(13)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(2)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(14)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(3)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(15)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(4)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(16)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(5)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(17)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(6)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(18)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(7)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(19)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(8)] * placeholder_shared_local[(8)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(20)] * placeholder_shared_local[(8)]));
  DepthwiseConv2d[(1)] = 0.000000e+00f;
  DepthwiseConv2d[(3)] = 0.000000e+00f;
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(3)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(15)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(4)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(16)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(5)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(17)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(6)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(18)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(7)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(19)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(8)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(20)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(9)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(21)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(10)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(22)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(11)] * placeholder_shared_local[(8)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(23)] * placeholder_shared_local[(8)]));
  T_relu[((((((int)blockIdx.z) * 784) + (((int)threadIdx.y) * 56)) + ((int)threadIdx.x)))] = max((DepthwiseConv2d[(0)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[(((((((int)blockIdx.z) * 784) + (((int)threadIdx.y) * 56)) + ((int)threadIdx.x)) + 392))] = max((DepthwiseConv2d[(2)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[(((((((int)blockIdx.z) * 784) + (((int)threadIdx.y) * 56)) + ((int)threadIdx.x)) + 28))] = max((DepthwiseConv2d[(1)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[(((((((int)blockIdx.z) * 784) + (((int)threadIdx.y) * 56)) + ((int)threadIdx.x)) + 420))] = max((DepthwiseConv2d[(3)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[2];
  __shared__ float pad_temp_shared[784];
  __shared__ float placeholder_shared[256];
  compute[(0)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 64; ++rc_outer) {
    __syncthreads();
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      pad_temp_shared[(((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = placeholder[(((((rc_outer * 784) + (((int)threadIdx.z) * 98)) + (((int)threadIdx.y) * 14)) + ((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)))];
    }
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) >> 4)) < 16) {
      if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 5)) + ((int)threadIdx.x)) < 256) {
        if (((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) < 32) {
          if (((int)threadIdx.x) < 5) {
            placeholder_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 5)) + ((int)threadIdx.x)))] = placeholder1[((((((((int)blockIdx.z) * 16384) + (((int)threadIdx.z) * 2048)) + ((((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) >> 4) * 1024)) + (rc_outer * 16)) + (((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) & 15)))];
          }
        }
      }
    }
    __syncthreads();
    #pragma unroll
    for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((rc_inner * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] * placeholder_shared[(((((int)threadIdx.z) * 16) + rc_inner))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((rc_inner * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 128))]));
    }
  }
  T_relu[(((((((int)blockIdx.z) * 784) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = max((compute[(0)] + placeholder2[(((((int)blockIdx.z) * 16) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 784) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 392))] = max((compute[(1)] + placeholder2[((((((int)blockIdx.z) * 16) + ((int)threadIdx.z)) + 8))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_11_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  __shared__ float PaddedInput_shared[3249];
  __shared__ float placeholder_shared[9];
  float PaddedInput_shared_local[15];
  float placeholder_shared_local[9];
  float DepthwiseConv2d[2];
  PaddedInput_shared[(((((int)threadIdx.y) * 28) + ((int)threadIdx.x)))] = (((57 <= ((((int)threadIdx.y) * 28) + ((int)threadIdx.x))) && (1 <= (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) % 57))) ? placeholder[(((((((int)blockIdx.z) * 3136) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) / 57) * 56)) + (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) % 57)) - 57))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 392))] = ((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 50) % 57)) ? placeholder[(((((((int)blockIdx.z) * 3136) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 392) / 57) * 56)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 50) % 57)) - 57))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 784))] = ((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 43) % 57)) ? placeholder[(((((((int)blockIdx.z) * 3136) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 784) / 57) * 56)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 43) % 57)) - 57))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1176))] = ((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 36) % 57)) ? placeholder[(((((((int)blockIdx.z) * 3136) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1176) / 57) * 56)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 36) % 57)) - 57))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1568))] = ((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 29) % 57)) ? placeholder[(((((((int)blockIdx.z) * 3136) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1568) / 57) * 56)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 29) % 57)) - 57))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1960))] = ((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 22) % 57)) ? placeholder[(((((((int)blockIdx.z) * 3136) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1960) / 57) * 56)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 22) % 57)) - 57))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 2352))] = ((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 15) % 57)) ? placeholder[(((((((int)blockIdx.z) * 3136) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 2352) / 57) * 56)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 15) % 57)) - 57))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 2744))] = ((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 8) % 57)) ? placeholder[(((((((int)blockIdx.z) * 3136) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 2744) / 57) * 56)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 8) % 57)) - 57))] : 0.000000e+00f);
  if (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) < 113) {
    if (((int)threadIdx.y) < 5) {
      PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 3136))] = ((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1) % 57)) ? placeholder[(((((((int)blockIdx.z) * 3136) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 3136) / 57) * 56)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1) % 57)) - 57))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) < 9) {
    if (((int)threadIdx.y) < 1) {
      placeholder_shared[(((((int)threadIdx.y) * 28) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.y) * 28) + (((int)blockIdx.z) * 9)) + ((int)threadIdx.x)))];
    }
  }
  __syncthreads();
  PaddedInput_shared_local[(0)] = PaddedInput_shared[(((((int)threadIdx.y) * 228) + (((int)threadIdx.x) * 2)))];
  PaddedInput_shared_local[(1)] = PaddedInput_shared[((((((int)threadIdx.y) * 228) + (((int)threadIdx.x) * 2)) + 1))];
  PaddedInput_shared_local[(2)] = PaddedInput_shared[((((((int)threadIdx.y) * 228) + (((int)threadIdx.x) * 2)) + 2))];
  PaddedInput_shared_local[(3)] = PaddedInput_shared[((((((int)threadIdx.y) * 228) + (((int)threadIdx.x) * 2)) + 57))];
  PaddedInput_shared_local[(4)] = PaddedInput_shared[((((((int)threadIdx.y) * 228) + (((int)threadIdx.x) * 2)) + 58))];
  PaddedInput_shared_local[(5)] = PaddedInput_shared[((((((int)threadIdx.y) * 228) + (((int)threadIdx.x) * 2)) + 59))];
  PaddedInput_shared_local[(6)] = PaddedInput_shared[((((((int)threadIdx.y) * 228) + (((int)threadIdx.x) * 2)) + 114))];
  PaddedInput_shared_local[(7)] = PaddedInput_shared[((((((int)threadIdx.y) * 228) + (((int)threadIdx.x) * 2)) + 115))];
  PaddedInput_shared_local[(8)] = PaddedInput_shared[((((((int)threadIdx.y) * 228) + (((int)threadIdx.x) * 2)) + 116))];
  PaddedInput_shared_local[(9)] = PaddedInput_shared[((((((int)threadIdx.y) * 228) + (((int)threadIdx.x) * 2)) + 171))];
  PaddedInput_shared_local[(10)] = PaddedInput_shared[((((((int)threadIdx.y) * 228) + (((int)threadIdx.x) * 2)) + 172))];
  PaddedInput_shared_local[(11)] = PaddedInput_shared[((((((int)threadIdx.y) * 228) + (((int)threadIdx.x) * 2)) + 173))];
  PaddedInput_shared_local[(12)] = PaddedInput_shared[((((((int)threadIdx.y) * 228) + (((int)threadIdx.x) * 2)) + 228))];
  PaddedInput_shared_local[(13)] = PaddedInput_shared[((((((int)threadIdx.y) * 228) + (((int)threadIdx.x) * 2)) + 229))];
  PaddedInput_shared_local[(14)] = PaddedInput_shared[((((((int)threadIdx.y) * 228) + (((int)threadIdx.x) * 2)) + 230))];
  placeholder_shared_local[(0)] = placeholder_shared[(0)];
  placeholder_shared_local[(1)] = placeholder_shared[(1)];
  placeholder_shared_local[(2)] = placeholder_shared[(2)];
  placeholder_shared_local[(3)] = placeholder_shared[(3)];
  placeholder_shared_local[(4)] = placeholder_shared[(4)];
  placeholder_shared_local[(5)] = placeholder_shared[(5)];
  placeholder_shared_local[(6)] = placeholder_shared[(6)];
  placeholder_shared_local[(7)] = placeholder_shared[(7)];
  placeholder_shared_local[(8)] = placeholder_shared[(8)];
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(0)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(1)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(2)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(3)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(4)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(5)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(6)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(7)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(8)] * placeholder_shared_local[(8)]));
  DepthwiseConv2d[(1)] = 0.000000e+00f;
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(6)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(7)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(8)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(9)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(10)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(11)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(12)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(13)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(14)] * placeholder_shared_local[(8)]));
  T_relu[((((((int)blockIdx.z) * 784) + (((int)threadIdx.y) * 56)) + ((int)threadIdx.x)))] = max((DepthwiseConv2d[(0)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[(((((((int)blockIdx.z) * 784) + (((int)threadIdx.y) * 56)) + ((int)threadIdx.x)) + 28))] = max((DepthwiseConv2d[(1)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_3_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  __shared__ float PaddedInput_shared[225];
  __shared__ float placeholder_shared[9];
  float PaddedInput_shared_local[9];
  float placeholder_shared_local[9];
  float DepthwiseConv2d[1];
  PaddedInput_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))] = (((15 <= ((((int)threadIdx.y) * 7) + ((int)threadIdx.x))) && (1 <= (((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) % 15))) ? placeholder[(((((((int)blockIdx.z) * 196) + ((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) / 15) * 14)) + (((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) % 15)) - 15))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 49))] = ((1 <= ((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 4) % 15)) ? placeholder[(((((((int)blockIdx.z) * 196) + (((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 49) / 15) * 14)) + ((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 4) % 15)) - 15))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 98))] = ((1 <= ((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 8) % 15)) ? placeholder[(((((((int)blockIdx.z) * 196) + (((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 98) / 15) * 14)) + ((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 8) % 15)) - 15))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 147))] = ((1 <= ((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 12) % 15)) ? placeholder[(((((((int)blockIdx.z) * 196) + (((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 147) / 15) * 14)) + ((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 12) % 15)) - 15))] : 0.000000e+00f);
  if (((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) < 29) {
    if (((int)threadIdx.y) < 5) {
      PaddedInput_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 196))] = ((1 <= ((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1) % 15)) ? placeholder[(((((((int)blockIdx.z) * 196) + (((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 196) / 15) * 14)) + ((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1) % 15)) - 15))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) < 9) {
    if (((int)threadIdx.y) < 2) {
      placeholder_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))] = placeholder1[((((((int)blockIdx.z) * 9) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))];
    }
  }
  __syncthreads();
  PaddedInput_shared_local[(0)] = PaddedInput_shared[(((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 2)))];
  PaddedInput_shared_local[(1)] = PaddedInput_shared[((((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 2)) + 1))];
  PaddedInput_shared_local[(2)] = PaddedInput_shared[((((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 2)) + 2))];
  PaddedInput_shared_local[(3)] = PaddedInput_shared[((((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 2)) + 15))];
  PaddedInput_shared_local[(4)] = PaddedInput_shared[((((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 2)) + 16))];
  PaddedInput_shared_local[(5)] = PaddedInput_shared[((((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 2)) + 17))];
  PaddedInput_shared_local[(6)] = PaddedInput_shared[((((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 2)) + 30))];
  PaddedInput_shared_local[(7)] = PaddedInput_shared[((((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 2)) + 31))];
  PaddedInput_shared_local[(8)] = PaddedInput_shared[((((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 2)) + 32))];
  placeholder_shared_local[(0)] = placeholder_shared[(0)];
  placeholder_shared_local[(1)] = placeholder_shared[(1)];
  placeholder_shared_local[(2)] = placeholder_shared[(2)];
  placeholder_shared_local[(3)] = placeholder_shared[(3)];
  placeholder_shared_local[(4)] = placeholder_shared[(4)];
  placeholder_shared_local[(5)] = placeholder_shared[(5)];
  placeholder_shared_local[(6)] = placeholder_shared[(6)];
  placeholder_shared_local[(7)] = placeholder_shared[(7)];
  placeholder_shared_local[(8)] = placeholder_shared[(8)];
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(0)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(1)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(2)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(3)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(4)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(5)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(6)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(7)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(8)] * placeholder_shared_local[(8)]));
  T_relu[((((((int)blockIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = max((DepthwiseConv2d[(0)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_7_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  __shared__ float PaddedInput_shared[841];
  __shared__ float placeholder_shared[9];
  float PaddedInput_shared_local[9];
  float placeholder_shared_local[9];
  float DepthwiseConv2d[1];
  PaddedInput_shared[(((((int)threadIdx.y) * 14) + ((int)threadIdx.x)))] = (((29 <= ((((int)threadIdx.y) * 14) + ((int)threadIdx.x))) && (1 <= (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) % 29))) ? placeholder[(((((((int)blockIdx.z) * 784) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) / 29) * 28)) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) % 29)) - 29))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 196))] = ((1 <= ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 22) % 29)) ? placeholder[(((((((int)blockIdx.z) * 784) + (((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 196) / 29) * 28)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 22) % 29)) - 29))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 392))] = ((1 <= ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 15) % 29)) ? placeholder[(((((((int)blockIdx.z) * 784) + (((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 392) / 29) * 28)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 15) % 29)) - 29))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 588))] = ((1 <= ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 8) % 29)) ? placeholder[(((((((int)blockIdx.z) * 784) + (((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 588) / 29) * 28)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 8) % 29)) - 29))] : 0.000000e+00f);
  if (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) < 57) {
    if (((int)threadIdx.y) < 5) {
      PaddedInput_shared[((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 784))] = ((1 <= ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 1) % 29)) ? placeholder[(((((((int)blockIdx.z) * 784) + (((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 784) / 29) * 28)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 1) % 29)) - 29))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) < 9) {
    if (((int)threadIdx.y) < 1) {
      placeholder_shared[(((((int)threadIdx.y) * 14) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.y) * 14) + (((int)blockIdx.z) * 9)) + ((int)threadIdx.x)))];
    }
  }
  __syncthreads();
  PaddedInput_shared_local[(0)] = PaddedInput_shared[(((((int)threadIdx.y) * 58) + (((int)threadIdx.x) * 2)))];
  PaddedInput_shared_local[(1)] = PaddedInput_shared[((((((int)threadIdx.y) * 58) + (((int)threadIdx.x) * 2)) + 1))];
  PaddedInput_shared_local[(2)] = PaddedInput_shared[((((((int)threadIdx.y) * 58) + (((int)threadIdx.x) * 2)) + 2))];
  PaddedInput_shared_local[(3)] = PaddedInput_shared[((((((int)threadIdx.y) * 58) + (((int)threadIdx.x) * 2)) + 29))];
  PaddedInput_shared_local[(4)] = PaddedInput_shared[((((((int)threadIdx.y) * 58) + (((int)threadIdx.x) * 2)) + 30))];
  PaddedInput_shared_local[(5)] = PaddedInput_shared[((((((int)threadIdx.y) * 58) + (((int)threadIdx.x) * 2)) + 31))];
  PaddedInput_shared_local[(6)] = PaddedInput_shared[((((((int)threadIdx.y) * 58) + (((int)threadIdx.x) * 2)) + 58))];
  PaddedInput_shared_local[(7)] = PaddedInput_shared[((((((int)threadIdx.y) * 58) + (((int)threadIdx.x) * 2)) + 59))];
  PaddedInput_shared_local[(8)] = PaddedInput_shared[((((((int)threadIdx.y) * 58) + (((int)threadIdx.x) * 2)) + 60))];
  placeholder_shared_local[(0)] = placeholder_shared[(0)];
  placeholder_shared_local[(1)] = placeholder_shared[(1)];
  placeholder_shared_local[(2)] = placeholder_shared[(2)];
  placeholder_shared_local[(3)] = placeholder_shared[(3)];
  placeholder_shared_local[(4)] = placeholder_shared[(4)];
  placeholder_shared_local[(5)] = placeholder_shared[(5)];
  placeholder_shared_local[(6)] = placeholder_shared[(6)];
  placeholder_shared_local[(7)] = placeholder_shared[(7)];
  placeholder_shared_local[(8)] = placeholder_shared[(8)];
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(0)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(1)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(2)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(3)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(4)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(5)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(6)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(7)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(8)] * placeholder_shared_local[(8)]));
  T_relu[((((((int)blockIdx.z) * 196) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)))] = max((DepthwiseConv2d[(0)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_6_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[4];
  __shared__ float pad_temp_shared[784];
  __shared__ float placeholder_shared[128];
  compute[(0)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  compute[(2)] = 0.000000e+00f;
  compute[(3)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 32; ++rc_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      pad_temp_shared[(((((((int)threadIdx.z) * 196) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = placeholder[(((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 14)) / 7) * 196)) + (((int)blockIdx.y) * 98)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 14)) % 7) * 14)) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 14)))];
    }
    if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) >> 3)) < 16) {
      if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 5)) + ((int)threadIdx.x)) < 128) {
        if (((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) < 32) {
          if (((int)threadIdx.x) < 5) {
            placeholder_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 5)) + ((int)threadIdx.x)))] = placeholder1[((((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 1024)) + ((((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) >> 3) * 256)) + (rc_outer * 8)) + (((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) & 7)))];
          }
        }
      }
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((rc_inner * 98) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)))] * placeholder_shared[(((((int)threadIdx.z) * 8) + rc_inner))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((rc_inner * 98) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)))] * placeholder_shared[((((((int)threadIdx.z) * 8) + rc_inner) + 32))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((rc_inner * 98) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)))] * placeholder_shared[((((((int)threadIdx.z) * 8) + rc_inner) + 64))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((rc_inner * 98) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)))] * placeholder_shared[((((((int)threadIdx.z) * 8) + rc_inner) + 96))]));
    }
  }
  T_relu[((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 98)) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)))] = max((compute[(0)] + placeholder2[(((((int)blockIdx.z) * 16) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[(((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 98)) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 784))] = max((compute[(1)] + placeholder2[((((((int)blockIdx.z) * 16) + ((int)threadIdx.z)) + 4))]), 0.000000e+00f);
  T_relu[(((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 98)) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 1568))] = max((compute[(2)] + placeholder2[((((((int)blockIdx.z) * 16) + ((int)threadIdx.z)) + 8))]), 0.000000e+00f);
  T_relu[(((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 98)) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 2352))] = max((compute[(3)] + placeholder2[((((((int)blockIdx.z) * 16) + ((int)threadIdx.z)) + 12))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_12_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[32];
  __shared__ float pad_temp_shared[896];
  __shared__ float placeholder_shared[512];
  for (int ff_init = 0; ff_init < 2; ++ff_init) {
    compute[(ff_init)] = 0.000000e+00f;
    compute[((ff_init + 8))] = 0.000000e+00f;
    compute[((ff_init + 16))] = 0.000000e+00f;
    compute[((ff_init + 24))] = 0.000000e+00f;
    compute[((ff_init + 4))] = 0.000000e+00f;
    compute[((ff_init + 12))] = 0.000000e+00f;
    compute[((ff_init + 20))] = 0.000000e+00f;
    compute[((ff_init + 28))] = 0.000000e+00f;
    compute[((ff_init + 2))] = 0.000000e+00f;
    compute[((ff_init + 10))] = 0.000000e+00f;
    compute[((ff_init + 18))] = 0.000000e+00f;
    compute[((ff_init + 26))] = 0.000000e+00f;
    compute[((ff_init + 6))] = 0.000000e+00f;
    compute[((ff_init + 14))] = 0.000000e+00f;
    compute[((ff_init + 22))] = 0.000000e+00f;
    compute[((ff_init + 30))] = 0.000000e+00f;
  }
  for (int rc_outer = 0; rc_outer < 16; ++rc_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      pad_temp_shared[((((((int)threadIdx.z) * 112) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = placeholder[((((((rc_outer * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) >> 3)) < 64) {
        if ((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 512) {
          if (((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 64) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = placeholder1[((((((((int)blockIdx.z) * 8192) + (((int)threadIdx.z) * 1024)) + ((((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) >> 3) * 128)) + (rc_outer * 8)) + (((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) & 7)))];
          }
        }
      }
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
      for (int ff = 0; ff < 2; ++ff) {
        compute[(ff)] = (compute[(ff)] + (pad_temp_shared[(((rc_inner * 112) + ((int)threadIdx.x)))] * placeholder_shared[((((((int)threadIdx.z) * 16) + (ff * 8)) + rc_inner))]));
        compute[((ff + 8))] = (compute[((ff + 8))] + (pad_temp_shared[(((rc_inner * 112) + ((int)threadIdx.x)))] * placeholder_shared[(((((((int)threadIdx.z) * 16) + (ff * 8)) + rc_inner) + 128))]));
        compute[((ff + 16))] = (compute[((ff + 16))] + (pad_temp_shared[(((rc_inner * 112) + ((int)threadIdx.x)))] * placeholder_shared[(((((((int)threadIdx.z) * 16) + (ff * 8)) + rc_inner) + 256))]));
        compute[((ff + 24))] = (compute[((ff + 24))] + (pad_temp_shared[(((rc_inner * 112) + ((int)threadIdx.x)))] * placeholder_shared[(((((((int)threadIdx.z) * 16) + (ff * 8)) + rc_inner) + 384))]));
        compute[((ff + 4))] = (compute[((ff + 4))] + (pad_temp_shared[((((rc_inner * 112) + ((int)threadIdx.x)) + 56))] * placeholder_shared[((((((int)threadIdx.z) * 16) + (ff * 8)) + rc_inner))]));
        compute[((ff + 12))] = (compute[((ff + 12))] + (pad_temp_shared[((((rc_inner * 112) + ((int)threadIdx.x)) + 56))] * placeholder_shared[(((((((int)threadIdx.z) * 16) + (ff * 8)) + rc_inner) + 128))]));
        compute[((ff + 20))] = (compute[((ff + 20))] + (pad_temp_shared[((((rc_inner * 112) + ((int)threadIdx.x)) + 56))] * placeholder_shared[(((((((int)threadIdx.z) * 16) + (ff * 8)) + rc_inner) + 256))]));
        compute[((ff + 28))] = (compute[((ff + 28))] + (pad_temp_shared[((((rc_inner * 112) + ((int)threadIdx.x)) + 56))] * placeholder_shared[(((((((int)threadIdx.z) * 16) + (ff * 8)) + rc_inner) + 384))]));
        compute[((ff + 2))] = (compute[((ff + 2))] + (pad_temp_shared[((((rc_inner * 112) + ((int)threadIdx.x)) + 28))] * placeholder_shared[((((((int)threadIdx.z) * 16) + (ff * 8)) + rc_inner))]));
        compute[((ff + 10))] = (compute[((ff + 10))] + (pad_temp_shared[((((rc_inner * 112) + ((int)threadIdx.x)) + 28))] * placeholder_shared[(((((((int)threadIdx.z) * 16) + (ff * 8)) + rc_inner) + 128))]));
        compute[((ff + 18))] = (compute[((ff + 18))] + (pad_temp_shared[((((rc_inner * 112) + ((int)threadIdx.x)) + 28))] * placeholder_shared[(((((((int)threadIdx.z) * 16) + (ff * 8)) + rc_inner) + 256))]));
        compute[((ff + 26))] = (compute[((ff + 26))] + (pad_temp_shared[((((rc_inner * 112) + ((int)threadIdx.x)) + 28))] * placeholder_shared[(((((((int)threadIdx.z) * 16) + (ff * 8)) + rc_inner) + 384))]));
        compute[((ff + 6))] = (compute[((ff + 6))] + (pad_temp_shared[((((rc_inner * 112) + ((int)threadIdx.x)) + 84))] * placeholder_shared[((((((int)threadIdx.z) * 16) + (ff * 8)) + rc_inner))]));
        compute[((ff + 14))] = (compute[((ff + 14))] + (pad_temp_shared[((((rc_inner * 112) + ((int)threadIdx.x)) + 84))] * placeholder_shared[(((((((int)threadIdx.z) * 16) + (ff * 8)) + rc_inner) + 128))]));
        compute[((ff + 22))] = (compute[((ff + 22))] + (pad_temp_shared[((((rc_inner * 112) + ((int)threadIdx.x)) + 84))] * placeholder_shared[(((((((int)threadIdx.z) * 16) + (ff * 8)) + rc_inner) + 256))]));
        compute[((ff + 30))] = (compute[((ff + 30))] + (pad_temp_shared[((((rc_inner * 112) + ((int)threadIdx.x)) + 84))] * placeholder_shared[(((((((int)threadIdx.z) * 16) + (ff * 8)) + rc_inner) + 384))]));
      }
    }
  }
  for (int ax1_inner_inner_inner = 0; ax1_inner_inner_inner < 2; ++ax1_inner_inner_inner) {
    T_relu[((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 6272)) + (ax1_inner_inner_inner * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)))] = max((compute[(ax1_inner_inner_inner)] + placeholder2[((((((int)blockIdx.z) * 64) + (((int)threadIdx.z) * 2)) + ax1_inner_inner_inner))]), 0.000000e+00f);
    T_relu[(((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 6272)) + (ax1_inner_inner_inner * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 50176))] = max((compute[((ax1_inner_inner_inner + 8))] + placeholder2[(((((((int)blockIdx.z) * 64) + (((int)threadIdx.z) * 2)) + ax1_inner_inner_inner) + 16))]), 0.000000e+00f);
    T_relu[(((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 6272)) + (ax1_inner_inner_inner * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 100352))] = max((compute[((ax1_inner_inner_inner + 16))] + placeholder2[(((((((int)blockIdx.z) * 64) + (((int)threadIdx.z) * 2)) + ax1_inner_inner_inner) + 32))]), 0.000000e+00f);
    T_relu[(((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 6272)) + (ax1_inner_inner_inner * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 150528))] = max((compute[((ax1_inner_inner_inner + 24))] + placeholder2[(((((((int)blockIdx.z) * 64) + (((int)threadIdx.z) * 2)) + ax1_inner_inner_inner) + 48))]), 0.000000e+00f);
    T_relu[(((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 6272)) + (ax1_inner_inner_inner * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 56))] = max((compute[((ax1_inner_inner_inner + 4))] + placeholder2[((((((int)blockIdx.z) * 64) + (((int)threadIdx.z) * 2)) + ax1_inner_inner_inner))]), 0.000000e+00f);
    T_relu[(((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 6272)) + (ax1_inner_inner_inner * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 50232))] = max((compute[((ax1_inner_inner_inner + 12))] + placeholder2[(((((((int)blockIdx.z) * 64) + (((int)threadIdx.z) * 2)) + ax1_inner_inner_inner) + 16))]), 0.000000e+00f);
    T_relu[(((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 6272)) + (ax1_inner_inner_inner * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 100408))] = max((compute[((ax1_inner_inner_inner + 20))] + placeholder2[(((((((int)blockIdx.z) * 64) + (((int)threadIdx.z) * 2)) + ax1_inner_inner_inner) + 32))]), 0.000000e+00f);
    T_relu[(((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 6272)) + (ax1_inner_inner_inner * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 150584))] = max((compute[((ax1_inner_inner_inner + 28))] + placeholder2[(((((((int)blockIdx.z) * 64) + (((int)threadIdx.z) * 2)) + ax1_inner_inner_inner) + 48))]), 0.000000e+00f);
    T_relu[(((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 6272)) + (ax1_inner_inner_inner * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 28))] = max((compute[((ax1_inner_inner_inner + 2))] + placeholder2[((((((int)blockIdx.z) * 64) + (((int)threadIdx.z) * 2)) + ax1_inner_inner_inner))]), 0.000000e+00f);
    T_relu[(((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 6272)) + (ax1_inner_inner_inner * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 50204))] = max((compute[((ax1_inner_inner_inner + 10))] + placeholder2[(((((((int)blockIdx.z) * 64) + (((int)threadIdx.z) * 2)) + ax1_inner_inner_inner) + 16))]), 0.000000e+00f);
    T_relu[(((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 6272)) + (ax1_inner_inner_inner * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 100380))] = max((compute[((ax1_inner_inner_inner + 18))] + placeholder2[(((((((int)blockIdx.z) * 64) + (((int)threadIdx.z) * 2)) + ax1_inner_inner_inner) + 32))]), 0.000000e+00f);
    T_relu[(((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 6272)) + (ax1_inner_inner_inner * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 150556))] = max((compute[((ax1_inner_inner_inner + 26))] + placeholder2[(((((((int)blockIdx.z) * 64) + (((int)threadIdx.z) * 2)) + ax1_inner_inner_inner) + 48))]), 0.000000e+00f);
    T_relu[(((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 6272)) + (ax1_inner_inner_inner * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 84))] = max((compute[((ax1_inner_inner_inner + 6))] + placeholder2[((((((int)blockIdx.z) * 64) + (((int)threadIdx.z) * 2)) + ax1_inner_inner_inner))]), 0.000000e+00f);
    T_relu[(((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 6272)) + (ax1_inner_inner_inner * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 50260))] = max((compute[((ax1_inner_inner_inner + 14))] + placeholder2[(((((((int)blockIdx.z) * 64) + (((int)threadIdx.z) * 2)) + ax1_inner_inner_inner) + 16))]), 0.000000e+00f);
    T_relu[(((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 6272)) + (ax1_inner_inner_inner * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 100436))] = max((compute[((ax1_inner_inner_inner + 22))] + placeholder2[(((((((int)blockIdx.z) * 64) + (((int)threadIdx.z) * 2)) + ax1_inner_inner_inner) + 32))]), 0.000000e+00f);
    T_relu[(((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 6272)) + (ax1_inner_inner_inner * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 150612))] = max((compute[((ax1_inner_inner_inner + 30))] + placeholder2[(((((((int)blockIdx.z) * 64) + (((int)threadIdx.z) * 2)) + ax1_inner_inner_inner) + 48))]), 0.000000e+00f);
  }
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_17_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  __shared__ float PaddedInput_shared[2052];
  __shared__ float placeholder_shared[9];
  float PaddedInput_shared_local[30];
  float placeholder_shared_local[9];
  float DepthwiseConv2d[8];
  PaddedInput_shared[(((((int)threadIdx.y) * 112) + ((int)threadIdx.x)))] = ((((1 <= ((((int)blockIdx.y) * 16) + (((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) / 114))) && (1 <= (((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) % 114))) && ((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) % 114) < 113)) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1792)) + ((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) / 114) * 112)) + (((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) % 114)) - 113))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 224))] = (((1 <= ((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 110) % 114)) && (((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 110) % 114) < 113)) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1792)) + (((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 224) / 114) * 112)) + ((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 110) % 114)) - 113))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 448))] = (((1 <= ((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 106) % 114)) && (((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 106) % 114) < 113)) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1792)) + (((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 448) / 114) * 112)) + ((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 106) % 114)) - 113))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 672))] = (((1 <= ((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 102) % 114)) && (((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 102) % 114) < 113)) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1792)) + (((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 672) / 114) * 112)) + ((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 102) % 114)) - 113))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 896))] = (((1 <= ((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 98) % 114)) && (((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 98) % 114) < 113)) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1792)) + (((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 896) / 114) * 112)) + ((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 98) % 114)) - 113))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 1120))] = (((1 <= ((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 94) % 114)) && (((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 94) % 114) < 113)) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1792)) + (((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 1120) / 114) * 112)) + ((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 94) % 114)) - 113))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 1344))] = (((1 <= ((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 90) % 114)) && (((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 90) % 114) < 113)) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1792)) + (((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 1344) / 114) * 112)) + ((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 90) % 114)) - 113))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 1568))] = (((1 <= ((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 86) % 114)) && (((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 86) % 114) < 113)) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1792)) + (((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 1568) / 114) * 112)) + ((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 86) % 114)) - 113))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 1792))] = ((((((((int)blockIdx.y) * 16) + ((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 1792) / 114)) < 113) && (1 <= ((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 82) % 114))) && (((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 82) % 114) < 113)) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1792)) + (((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 1792) / 114) * 112)) + ((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 82) % 114)) - 113))] : 0.000000e+00f);
  if (((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) < 36) {
    if (((int)threadIdx.y) < 1) {
      PaddedInput_shared[((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 2016))] = (((((((int)blockIdx.y) * 16) + ((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 2016) / 114)) < 113) && (((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) < 35)) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1792)) + (((((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 2016) / 114) * 112)) + (((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) + 78)) - 113))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.y) * 112) + ((int)threadIdx.x)) < 9) {
    if (((int)threadIdx.y) < 1) {
      placeholder_shared[(((((int)threadIdx.y) * 112) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.y) * 112) + (((int)blockIdx.z) * 9)) + ((int)threadIdx.x)))];
    }
  }
  __syncthreads();
  PaddedInput_shared_local[(0)] = PaddedInput_shared[(((((int)threadIdx.y) * 912) + ((int)threadIdx.x)))];
  PaddedInput_shared_local[(1)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 1))];
  PaddedInput_shared_local[(2)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 2))];
  PaddedInput_shared_local[(3)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 114))];
  PaddedInput_shared_local[(4)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 115))];
  PaddedInput_shared_local[(5)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 116))];
  PaddedInput_shared_local[(6)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 228))];
  PaddedInput_shared_local[(7)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 229))];
  PaddedInput_shared_local[(8)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 230))];
  PaddedInput_shared_local[(9)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 342))];
  PaddedInput_shared_local[(10)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 343))];
  PaddedInput_shared_local[(11)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 344))];
  PaddedInput_shared_local[(12)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 456))];
  PaddedInput_shared_local[(13)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 457))];
  PaddedInput_shared_local[(14)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 458))];
  PaddedInput_shared_local[(15)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 570))];
  PaddedInput_shared_local[(16)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 571))];
  PaddedInput_shared_local[(17)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 572))];
  PaddedInput_shared_local[(18)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 684))];
  PaddedInput_shared_local[(19)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 685))];
  PaddedInput_shared_local[(20)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 686))];
  PaddedInput_shared_local[(21)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 798))];
  PaddedInput_shared_local[(22)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 799))];
  PaddedInput_shared_local[(23)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 800))];
  PaddedInput_shared_local[(24)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 912))];
  PaddedInput_shared_local[(25)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 913))];
  PaddedInput_shared_local[(26)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 914))];
  PaddedInput_shared_local[(27)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 1026))];
  PaddedInput_shared_local[(28)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 1027))];
  PaddedInput_shared_local[(29)] = PaddedInput_shared[((((((int)threadIdx.y) * 912) + ((int)threadIdx.x)) + 1028))];
  placeholder_shared_local[(0)] = placeholder_shared[(0)];
  placeholder_shared_local[(1)] = placeholder_shared[(1)];
  placeholder_shared_local[(2)] = placeholder_shared[(2)];
  placeholder_shared_local[(3)] = placeholder_shared[(3)];
  placeholder_shared_local[(4)] = placeholder_shared[(4)];
  placeholder_shared_local[(5)] = placeholder_shared[(5)];
  placeholder_shared_local[(6)] = placeholder_shared[(6)];
  placeholder_shared_local[(7)] = placeholder_shared[(7)];
  placeholder_shared_local[(8)] = placeholder_shared[(8)];
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(0)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(1)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(2)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(3)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(4)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(5)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(6)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(7)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(8)] * placeholder_shared_local[(8)]));
  DepthwiseConv2d[(1)] = 0.000000e+00f;
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(3)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(4)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(5)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(6)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(7)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(8)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(9)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(10)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(11)] * placeholder_shared_local[(8)]));
  DepthwiseConv2d[(2)] = 0.000000e+00f;
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(6)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(7)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(8)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(9)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(10)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(11)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(12)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(13)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(14)] * placeholder_shared_local[(8)]));
  DepthwiseConv2d[(3)] = 0.000000e+00f;
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(9)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(10)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(11)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(12)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(13)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(14)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(15)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(16)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(17)] * placeholder_shared_local[(8)]));
  DepthwiseConv2d[(4)] = 0.000000e+00f;
  DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared_local[(12)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared_local[(13)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared_local[(14)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared_local[(15)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared_local[(16)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared_local[(17)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared_local[(18)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared_local[(19)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared_local[(20)] * placeholder_shared_local[(8)]));
  DepthwiseConv2d[(5)] = 0.000000e+00f;
  DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared_local[(15)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared_local[(16)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared_local[(17)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared_local[(18)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared_local[(19)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared_local[(20)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared_local[(21)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared_local[(22)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared_local[(23)] * placeholder_shared_local[(8)]));
  DepthwiseConv2d[(6)] = 0.000000e+00f;
  DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared_local[(18)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared_local[(19)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared_local[(20)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared_local[(21)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared_local[(22)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared_local[(23)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared_local[(24)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared_local[(25)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared_local[(26)] * placeholder_shared_local[(8)]));
  DepthwiseConv2d[(7)] = 0.000000e+00f;
  DepthwiseConv2d[(7)] = (DepthwiseConv2d[(7)] + (PaddedInput_shared_local[(21)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(7)] = (DepthwiseConv2d[(7)] + (PaddedInput_shared_local[(22)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(7)] = (DepthwiseConv2d[(7)] + (PaddedInput_shared_local[(23)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(7)] = (DepthwiseConv2d[(7)] + (PaddedInput_shared_local[(24)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(7)] = (DepthwiseConv2d[(7)] + (PaddedInput_shared_local[(25)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(7)] = (DepthwiseConv2d[(7)] + (PaddedInput_shared_local[(26)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(7)] = (DepthwiseConv2d[(7)] + (PaddedInput_shared_local[(27)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(7)] = (DepthwiseConv2d[(7)] + (PaddedInput_shared_local[(28)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(7)] = (DepthwiseConv2d[(7)] + (PaddedInput_shared_local[(29)] * placeholder_shared_local[(8)]));
  T_relu[(((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 896)) + ((int)threadIdx.x)))] = max((DepthwiseConv2d[(0)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 896)) + ((int)threadIdx.x)) + 112))] = max((DepthwiseConv2d[(1)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 896)) + ((int)threadIdx.x)) + 224))] = max((DepthwiseConv2d[(2)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 896)) + ((int)threadIdx.x)) + 336))] = max((DepthwiseConv2d[(3)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 896)) + ((int)threadIdx.x)) + 448))] = max((DepthwiseConv2d[(4)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 896)) + ((int)threadIdx.x)) + 560))] = max((DepthwiseConv2d[(5)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 896)) + ((int)threadIdx.x)) + 672))] = max((DepthwiseConv2d[(6)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 896)) + ((int)threadIdx.x)) + 784))] = max((DepthwiseConv2d[(7)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_15_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  __shared__ float PaddedInput_shared[1017];
  __shared__ float placeholder_shared[9];
  float PaddedInput_shared_local[18];
  float placeholder_shared_local[9];
  float DepthwiseConv2d[2];
  PaddedInput_shared[(((((int)threadIdx.y) * 56) + ((int)threadIdx.x)))] = (((1 <= ((int)blockIdx.y)) && (1 <= ((((int)threadIdx.y) * 56) + ((int)threadIdx.x)))) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 896)) + (((int)threadIdx.y) * 56)) + ((int)threadIdx.x)) - 113))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 112))] = (((1 <= ((((int)blockIdx.y) * 8) + ((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 112) / 113))) && (1 <= ((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 112) % 113))) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 896)) + (((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 112) / 113) * 112)) + ((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 112) % 113)) - 113))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 224))] = ((1 <= ((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 111) % 113)) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 896)) + (((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 224) / 113) * 112)) + ((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 111) % 113)) - 113))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 336))] = ((1 <= ((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 110) % 113)) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 896)) + (((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 336) / 113) * 112)) + ((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 110) % 113)) - 113))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 448))] = ((1 <= ((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 109) % 113)) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 896)) + (((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 448) / 113) * 112)) + ((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 109) % 113)) - 113))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 560))] = ((1 <= ((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 108) % 113)) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 896)) + (((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 560) / 113) * 112)) + ((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 108) % 113)) - 113))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 672))] = ((1 <= ((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 107) % 113)) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 896)) + (((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 672) / 113) * 112)) + ((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 107) % 113)) - 113))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 784))] = ((1 <= ((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 106) % 113)) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 896)) + (((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 784) / 113) * 112)) + ((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 106) % 113)) - 113))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 896))] = ((1 <= ((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 105) % 113)) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 896)) + (((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 896) / 113) * 112)) + ((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 105) % 113)) - 113))] : 0.000000e+00f);
  if (((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) < 9) {
    if (((int)threadIdx.y) < 1) {
      PaddedInput_shared[((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 1008))] = placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 896)) + (((((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 1008) / 113) * 112)) + (((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) + 104)) - 113))];
    }
  }
  if (((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) < 9) {
    if (((int)threadIdx.y) < 1) {
      placeholder_shared[(((((int)threadIdx.y) * 56) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.y) * 56) + (((int)blockIdx.z) * 9)) + ((int)threadIdx.x)))];
    }
  }
  __syncthreads();
  PaddedInput_shared_local[(0)] = PaddedInput_shared[(((((int)threadIdx.y) * 226) + (((int)threadIdx.x) * 2)))];
  PaddedInput_shared_local[(9)] = PaddedInput_shared[((((((int)threadIdx.y) * 226) + (((int)threadIdx.x) * 2)) + 452))];
  PaddedInput_shared_local[(1)] = PaddedInput_shared[((((((int)threadIdx.y) * 226) + (((int)threadIdx.x) * 2)) + 1))];
  PaddedInput_shared_local[(10)] = PaddedInput_shared[((((((int)threadIdx.y) * 226) + (((int)threadIdx.x) * 2)) + 453))];
  PaddedInput_shared_local[(2)] = PaddedInput_shared[((((((int)threadIdx.y) * 226) + (((int)threadIdx.x) * 2)) + 2))];
  PaddedInput_shared_local[(11)] = PaddedInput_shared[((((((int)threadIdx.y) * 226) + (((int)threadIdx.x) * 2)) + 454))];
  PaddedInput_shared_local[(3)] = PaddedInput_shared[((((((int)threadIdx.y) * 226) + (((int)threadIdx.x) * 2)) + 113))];
  PaddedInput_shared_local[(12)] = PaddedInput_shared[((((((int)threadIdx.y) * 226) + (((int)threadIdx.x) * 2)) + 565))];
  PaddedInput_shared_local[(4)] = PaddedInput_shared[((((((int)threadIdx.y) * 226) + (((int)threadIdx.x) * 2)) + 114))];
  PaddedInput_shared_local[(13)] = PaddedInput_shared[((((((int)threadIdx.y) * 226) + (((int)threadIdx.x) * 2)) + 566))];
  PaddedInput_shared_local[(5)] = PaddedInput_shared[((((((int)threadIdx.y) * 226) + (((int)threadIdx.x) * 2)) + 115))];
  PaddedInput_shared_local[(14)] = PaddedInput_shared[((((((int)threadIdx.y) * 226) + (((int)threadIdx.x) * 2)) + 567))];
  PaddedInput_shared_local[(6)] = PaddedInput_shared[((((((int)threadIdx.y) * 226) + (((int)threadIdx.x) * 2)) + 226))];
  PaddedInput_shared_local[(15)] = PaddedInput_shared[((((((int)threadIdx.y) * 226) + (((int)threadIdx.x) * 2)) + 678))];
  PaddedInput_shared_local[(7)] = PaddedInput_shared[((((((int)threadIdx.y) * 226) + (((int)threadIdx.x) * 2)) + 227))];
  PaddedInput_shared_local[(16)] = PaddedInput_shared[((((((int)threadIdx.y) * 226) + (((int)threadIdx.x) * 2)) + 679))];
  PaddedInput_shared_local[(8)] = PaddedInput_shared[((((((int)threadIdx.y) * 226) + (((int)threadIdx.x) * 2)) + 228))];
  PaddedInput_shared_local[(17)] = PaddedInput_shared[((((((int)threadIdx.y) * 226) + (((int)threadIdx.x) * 2)) + 680))];
  placeholder_shared_local[(0)] = placeholder_shared[(0)];
  placeholder_shared_local[(1)] = placeholder_shared[(1)];
  placeholder_shared_local[(2)] = placeholder_shared[(2)];
  placeholder_shared_local[(3)] = placeholder_shared[(3)];
  placeholder_shared_local[(4)] = placeholder_shared[(4)];
  placeholder_shared_local[(5)] = placeholder_shared[(5)];
  placeholder_shared_local[(6)] = placeholder_shared[(6)];
  placeholder_shared_local[(7)] = placeholder_shared[(7)];
  placeholder_shared_local[(8)] = placeholder_shared[(8)];
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  DepthwiseConv2d[(1)] = 0.000000e+00f;
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(0)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(9)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(1)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(10)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(2)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(11)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(3)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(12)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(4)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(13)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(5)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(14)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(6)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(15)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(7)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(16)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(8)] * placeholder_shared_local[(8)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(17)] * placeholder_shared_local[(8)]));
  T_relu[(((((((int)blockIdx.z) * 3136) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + ((int)threadIdx.x)))] = max((DepthwiseConv2d[(0)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 3136) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + ((int)threadIdx.x)) + 112))] = max((DepthwiseConv2d[(1)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_batch_flatten_kernel0(float* __restrict__ tensor, float* __restrict__ placeholder) {
  tensor[(((int)threadIdx.x))] = placeholder[(((int)threadIdx.x))];
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_14_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[16];
  __shared__ float pad_temp_shared[1792];
  __shared__ float placeholder_shared[1024];
  compute[(0)] = 0.000000e+00f;
  compute[(4)] = 0.000000e+00f;
  compute[(8)] = 0.000000e+00f;
  compute[(12)] = 0.000000e+00f;
  compute[(2)] = 0.000000e+00f;
  compute[(6)] = 0.000000e+00f;
  compute[(10)] = 0.000000e+00f;
  compute[(14)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  compute[(5)] = 0.000000e+00f;
  compute[(9)] = 0.000000e+00f;
  compute[(13)] = 0.000000e+00f;
  compute[(3)] = 0.000000e+00f;
  compute[(7)] = 0.000000e+00f;
  compute[(11)] = 0.000000e+00f;
  compute[(15)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 4; ++rc_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      pad_temp_shared[((((((int)threadIdx.z) * 112) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = placeholder[((((((rc_outer * 50176) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
      if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) >> 4)) < 64) {
        if ((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 1024) {
          if (((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 64) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = placeholder1[((((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + ((((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) >> 4) * 64)) + (rc_outer * 16)) + (((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) & 15)))];
          }
        }
      }
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((rc_inner * 112) + ((int)threadIdx.x)))] * placeholder_shared[(((((int)threadIdx.z) * 16) + rc_inner))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((rc_inner * 112) + ((int)threadIdx.x)))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 256))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[(((rc_inner * 112) + ((int)threadIdx.x)))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 512))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[(((rc_inner * 112) + ((int)threadIdx.x)))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 768))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((rc_inner * 112) + ((int)threadIdx.x)) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 16) + rc_inner))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((rc_inner * 112) + ((int)threadIdx.x)) + 56))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 256))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((rc_inner * 112) + ((int)threadIdx.x)) + 56))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 512))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((rc_inner * 112) + ((int)threadIdx.x)) + 56))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 768))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((rc_inner * 112) + ((int)threadIdx.x)) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 16) + rc_inner))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((rc_inner * 112) + ((int)threadIdx.x)) + 28))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 256))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((rc_inner * 112) + ((int)threadIdx.x)) + 28))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 512))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((rc_inner * 112) + ((int)threadIdx.x)) + 28))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 768))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((rc_inner * 112) + ((int)threadIdx.x)) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 16) + rc_inner))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((rc_inner * 112) + ((int)threadIdx.x)) + 84))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 256))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((rc_inner * 112) + ((int)threadIdx.x)) + 84))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 512))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((rc_inner * 112) + ((int)threadIdx.x)) + 84))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 768))]));
    }
  }
  T_relu[(((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)))] = max((compute[(0)] + placeholder2[(((((int)blockIdx.z) * 64) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 50176))] = max((compute[(4)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 16))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 100352))] = max((compute[(8)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 32))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 150528))] = max((compute[(12)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 48))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 56))] = max((compute[(2)] + placeholder2[(((((int)blockIdx.z) * 64) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 50232))] = max((compute[(6)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 16))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 100408))] = max((compute[(10)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 32))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 150584))] = max((compute[(14)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 48))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 28))] = max((compute[(1)] + placeholder2[(((((int)blockIdx.z) * 64) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 50204))] = max((compute[(5)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 16))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 100380))] = max((compute[(9)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 32))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 150556))] = max((compute[(13)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 48))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 84))] = max((compute[(3)] + placeholder2[(((((int)blockIdx.z) * 64) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 50260))] = max((compute[(7)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 16))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 100436))] = max((compute[(11)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 32))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + ((int)threadIdx.x)) + 150612))] = max((compute[(15)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 48))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_1_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  __shared__ float PaddedInput_shared[324];
  __shared__ float placeholder_shared[36];
  float PaddedInput_shared_local[9];
  float placeholder_shared_local[9];
  float DepthwiseConv2d[1];
  PaddedInput_shared[((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = (((((9 <= ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 81)) && (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 81) < 72)) && (1 <= ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 9))) && (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 9) < 8)) ? placeholder[((((((((int)blockIdx.z) * 196) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) / 81) * 49)) + ((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 81) / 9) * 7)) + ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 9)) - 8))] : 0.000000e+00f);
  if ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 128) {
    if (((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) < 19) {
      if (((int)threadIdx.z) < 3) {
        PaddedInput_shared[(((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 196))] = (((((9 <= (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 34) % 81)) && ((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 34) % 81) < 72)) && (1 <= (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 7) % 9))) && ((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 7) % 9) < 8)) ? placeholder[((((((((int)blockIdx.z) * 196) + ((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 196) / 81) * 49)) + (((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 34) % 81) / 9) * 7)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 7) % 9)) - 8))] : 0.000000e+00f);
      }
    }
  }
  if ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 36) {
    if (((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) < 6) {
      if (((int)threadIdx.z) < 1) {
        placeholder_shared[((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = placeholder1[(((((((int)threadIdx.z) * 49) + (((int)blockIdx.z) * 36)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))];
      }
    }
  }
  __syncthreads();
  PaddedInput_shared_local[(0)] = PaddedInput_shared[((((((int)threadIdx.z) * 81) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)))];
  PaddedInput_shared_local[(1)] = PaddedInput_shared[(((((((int)threadIdx.z) * 81) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 1))];
  PaddedInput_shared_local[(2)] = PaddedInput_shared[(((((((int)threadIdx.z) * 81) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 2))];
  PaddedInput_shared_local[(3)] = PaddedInput_shared[(((((((int)threadIdx.z) * 81) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 9))];
  PaddedInput_shared_local[(4)] = PaddedInput_shared[(((((((int)threadIdx.z) * 81) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 10))];
  PaddedInput_shared_local[(5)] = PaddedInput_shared[(((((((int)threadIdx.z) * 81) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 11))];
  PaddedInput_shared_local[(6)] = PaddedInput_shared[(((((((int)threadIdx.z) * 81) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 18))];
  PaddedInput_shared_local[(7)] = PaddedInput_shared[(((((((int)threadIdx.z) * 81) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 19))];
  PaddedInput_shared_local[(8)] = PaddedInput_shared[(((((((int)threadIdx.z) * 81) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 20))];
  placeholder_shared_local[(0)] = placeholder_shared[((((int)threadIdx.z) * 9))];
  placeholder_shared_local[(1)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 1))];
  placeholder_shared_local[(2)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 2))];
  placeholder_shared_local[(3)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 3))];
  placeholder_shared_local[(4)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 4))];
  placeholder_shared_local[(5)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 5))];
  placeholder_shared_local[(6)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 6))];
  placeholder_shared_local[(7)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 7))];
  placeholder_shared_local[(8)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 8))];
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(0)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(1)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(2)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(3)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(4)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(5)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(6)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(7)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(8)] * placeholder_shared_local[(8)]));
  T_relu[(((((((int)blockIdx.z) * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = max((DepthwiseConv2d[(0)] + placeholder2[(((((int)blockIdx.z) * 4) + ((int)threadIdx.z)))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_10_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[8];
  __shared__ float pad_temp_shared[896];
  __shared__ float placeholder_shared[1024];
  compute[(0)] = 0.000000e+00f;
  compute[(2)] = 0.000000e+00f;
  compute[(4)] = 0.000000e+00f;
  compute[(6)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  compute[(3)] = 0.000000e+00f;
  compute[(5)] = 0.000000e+00f;
  compute[(7)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      pad_temp_shared[((((((int)threadIdx.z) * 56) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = placeholder[((((((rc_outer * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
      if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) >> 4)) < 64) {
        if ((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 1024) {
          if (((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 64) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = placeholder1[((((((((int)blockIdx.z) * 8192) + (((int)threadIdx.z) * 512)) + ((((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) >> 4) * 128)) + (rc_outer * 16)) + (((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) & 15)))];
          }
        }
      }
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((rc_inner * 56) + ((int)threadIdx.x)))] * placeholder_shared[(((((int)threadIdx.z) * 16) + rc_inner))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((rc_inner * 56) + ((int)threadIdx.x)))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 256))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((rc_inner * 56) + ((int)threadIdx.x)))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 512))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((rc_inner * 56) + ((int)threadIdx.x)))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 768))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((rc_inner * 56) + ((int)threadIdx.x)) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 16) + rc_inner))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((rc_inner * 56) + ((int)threadIdx.x)) + 28))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 256))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((rc_inner * 56) + ((int)threadIdx.x)) + 28))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 512))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((rc_inner * 56) + ((int)threadIdx.x)) + 28))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 768))]));
    }
  }
  T_relu[(((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + ((int)threadIdx.x)))] = max((compute[(0)] + placeholder2[(((((int)blockIdx.z) * 64) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + ((int)threadIdx.x)) + 12544))] = max((compute[(2)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 16))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + ((int)threadIdx.x)) + 25088))] = max((compute[(4)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 32))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + ((int)threadIdx.x)) + 37632))] = max((compute[(6)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 48))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + ((int)threadIdx.x)) + 28))] = max((compute[(1)] + placeholder2[(((((int)blockIdx.z) * 64) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + ((int)threadIdx.x)) + 12572))] = max((compute[(3)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 16))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + ((int)threadIdx.x)) + 25116))] = max((compute[(5)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 32))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + ((int)threadIdx.x)) + 37660))] = max((compute[(7)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 48))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_softmax_kernel0(float* __restrict__ placeholder, float* __restrict__ T_softmax_norm) {
  float normal_reduce_temp0[1];
  float red_buf0[1];
  float T_softmax_exp[32];
  float normal_reduce_temp01[1];
  float red_buf01[1];
  normal_reduce_temp0[(0)] = -3.402823e+38f;
  for (int k_inner = 0; k_inner < 32; ++k_inner) {
    if (((((int)threadIdx.x) * 32) + k_inner) < 1000) {
      normal_reduce_temp0[(0)] = max(normal_reduce_temp0[(0)], placeholder[(((((int)threadIdx.x) * 32) + k_inner))]);
    }
  }
  unsigned int mask[1];
  float t0[1];
  red_buf0[(0)] = normal_reduce_temp0[(0)];
  mask[(0)] = __activemask();
  t0[(0)] = __shfl_down_sync(mask[(0)], red_buf0[(0)], 16, 32);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  t0[(0)] = __shfl_down_sync(mask[(0)], red_buf0[(0)], 8, 32);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  t0[(0)] = __shfl_down_sync(mask[(0)], red_buf0[(0)], 4, 32);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  t0[(0)] = __shfl_down_sync(mask[(0)], red_buf0[(0)], 2, 32);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  t0[(0)] = __shfl_down_sync(mask[(0)], red_buf0[(0)], 1, 32);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  red_buf0[(0)] = __shfl_sync(mask[(0)], red_buf0[(0)], 0, 32);
  for (int i1_inner_outer = 0; i1_inner_outer < 8; ++i1_inner_outer) {
    for (int i1_inner_inner_s = 0; i1_inner_inner_s < 4; ++i1_inner_inner_s) {
      if ((((((int)threadIdx.x) * 32) + (i1_inner_outer * 4)) + i1_inner_inner_s) < 1000) {
        T_softmax_exp[(((i1_inner_outer * 4) + i1_inner_inner_s))] = __expf((placeholder[((((((int)threadIdx.x) * 32) + (i1_inner_outer * 4)) + i1_inner_inner_s))] - red_buf0[(0)]));
      }
    }
  }
  normal_reduce_temp01[(0)] = 0.000000e+00f;
  for (int k_inner1 = 0; k_inner1 < 32; ++k_inner1) {
    if (((((int)threadIdx.x) * 32) + k_inner1) < 1000) {
      normal_reduce_temp01[(0)] = (normal_reduce_temp01[(0)] + __shfl_sync(__activemask(), T_softmax_exp[(k_inner1)], ((int)threadIdx.x), 32));
    }
  }
  unsigned int mask1[1];
  float t01[1];
  red_buf01[(0)] = normal_reduce_temp01[(0)];
  mask1[(0)] = __activemask();
  t01[(0)] = __shfl_down_sync(mask1[(0)], red_buf01[(0)], 16, 32);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  t01[(0)] = __shfl_down_sync(mask1[(0)], red_buf01[(0)], 8, 32);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  t01[(0)] = __shfl_down_sync(mask1[(0)], red_buf01[(0)], 4, 32);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  t01[(0)] = __shfl_down_sync(mask1[(0)], red_buf01[(0)], 2, 32);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  t01[(0)] = __shfl_down_sync(mask1[(0)], red_buf01[(0)], 1, 32);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  red_buf01[(0)] = __shfl_sync(mask1[(0)], red_buf01[(0)], 0, 32);
  for (int i1_inner_outer1 = 0; i1_inner_outer1 < 8; ++i1_inner_outer1) {
    for (int i1_inner_inner_s1 = 0; i1_inner_inner_s1 < 4; ++i1_inner_inner_s1) {
      if ((((((int)threadIdx.x) * 32) + (i1_inner_outer1 * 4)) + i1_inner_inner_s1) < 1000) {
        T_softmax_norm[((((((int)threadIdx.x) * 32) + (i1_inner_outer1 * 4)) + i1_inner_inner_s1))] = (__shfl_sync(__activemask(), T_softmax_exp[(((i1_inner_outer1 * 4) + i1_inner_inner_s1))], ((int)threadIdx.x), 32) / red_buf01[(0)]);
      }
    }
  }
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_2_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[4];
  __shared__ float pad_temp_shared[392];
  __shared__ float placeholder_shared[128];
  for (int ff_init = 0; ff_init < 2; ++ff_init) {
    compute[(ff_init)] = 0.000000e+00f;
    compute[((ff_init + 2))] = 0.000000e+00f;
  }
  for (int rc_outer = 0; rc_outer < 64; ++rc_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      pad_temp_shared[(((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = placeholder[(((((rc_outer * 392) + (((int)threadIdx.z) * 98)) + (((int)threadIdx.y) * 14)) + ((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)))];
    }
    if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) >> 3)) < 16) {
      if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 5)) + ((int)threadIdx.x)) < 128) {
        if (((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) < 32) {
          if (((int)threadIdx.x) < 5) {
            placeholder_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 5)) + ((int)threadIdx.x)))] = placeholder1[((((((((int)blockIdx.z) * 8192) + (((int)threadIdx.z) * 2048)) + ((((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) >> 3) * 512)) + (rc_outer * 8)) + (((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) & 7)))];
          }
        }
      }
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
      for (int ff = 0; ff < 2; ++ff) {
        compute[(ff)] = (compute[(ff)] + (pad_temp_shared[((((rc_inner * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] * placeholder_shared[((((((int)threadIdx.z) * 16) + (ff * 8)) + rc_inner))]));
        compute[((ff + 2))] = (compute[((ff + 2))] + (pad_temp_shared[((((rc_inner * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] * placeholder_shared[(((((((int)threadIdx.z) * 16) + (ff * 8)) + rc_inner) + 64))]));
      }
    }
  }
  for (int ax1_inner_inner_inner = 0; ax1_inner_inner_inner < 2; ++ax1_inner_inner_inner) {
    T_relu[((((((((int)blockIdx.z) * 784) + (((int)threadIdx.z) * 98)) + (ax1_inner_inner_inner * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = max((compute[(ax1_inner_inner_inner)] + placeholder2[((((((int)blockIdx.z) * 16) + (((int)threadIdx.z) * 2)) + ax1_inner_inner_inner))]), 0.000000e+00f);
    T_relu[(((((((((int)blockIdx.z) * 784) + (((int)threadIdx.z) * 98)) + (ax1_inner_inner_inner * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 392))] = max((compute[((ax1_inner_inner_inner + 2))] + placeholder2[(((((((int)blockIdx.z) * 16) + (((int)threadIdx.z) * 2)) + ax1_inner_inner_inner) + 8))]), 0.000000e+00f);
  }
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_13_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  __shared__ float PaddedInput_shared[580];
  __shared__ float placeholder_shared[9];
  float PaddedInput_shared_local[36];
  float placeholder_shared_local[9];
  float DepthwiseConv2d[8];
  PaddedInput_shared[(((int)threadIdx.x))] = (((1 <= ((int)blockIdx.y)) && (1 <= ((int)threadIdx.x))) ? placeholder[(((((((int)blockIdx.z) * 3136) + (((int)blockIdx.y) * 448)) + ((int)threadIdx.x)) - 57))] : 0.000000e+00f);
  PaddedInput_shared[((((int)threadIdx.x) + 56))] = ((((1 <= ((((int)blockIdx.y) * 8) + ((((int)threadIdx.x) + 56) / 58))) && (1 <= ((((int)threadIdx.x) + 56) % 58))) && (((((int)threadIdx.x) + 56) % 58) < 57)) ? placeholder[((((((((int)blockIdx.z) * 3136) + (((int)blockIdx.y) * 448)) + (((((int)threadIdx.x) + 56) / 58) * 56)) + ((((int)threadIdx.x) + 56) % 58)) - 57))] : 0.000000e+00f);
  PaddedInput_shared[((((int)threadIdx.x) + 112))] = (((1 <= ((((int)threadIdx.x) + 54) % 58)) && (((((int)threadIdx.x) + 54) % 58) < 57)) ? placeholder[((((((((int)blockIdx.z) * 3136) + (((int)blockIdx.y) * 448)) + (((((int)threadIdx.x) + 112) / 58) * 56)) + ((((int)threadIdx.x) + 54) % 58)) - 57))] : 0.000000e+00f);
  PaddedInput_shared[((((int)threadIdx.x) + 168))] = (((1 <= ((((int)threadIdx.x) + 52) % 58)) && (((((int)threadIdx.x) + 52) % 58) < 57)) ? placeholder[((((((((int)blockIdx.z) * 3136) + (((int)blockIdx.y) * 448)) + (((((int)threadIdx.x) + 168) / 58) * 56)) + ((((int)threadIdx.x) + 52) % 58)) - 57))] : 0.000000e+00f);
  PaddedInput_shared[((((int)threadIdx.x) + 224))] = (((1 <= ((((int)threadIdx.x) + 50) % 58)) && (((((int)threadIdx.x) + 50) % 58) < 57)) ? placeholder[((((((((int)blockIdx.z) * 3136) + (((int)blockIdx.y) * 448)) + (((((int)threadIdx.x) + 224) / 58) * 56)) + ((((int)threadIdx.x) + 50) % 58)) - 57))] : 0.000000e+00f);
  PaddedInput_shared[((((int)threadIdx.x) + 280))] = (((1 <= ((((int)threadIdx.x) + 48) % 58)) && (((((int)threadIdx.x) + 48) % 58) < 57)) ? placeholder[((((((((int)blockIdx.z) * 3136) + (((int)blockIdx.y) * 448)) + (((((int)threadIdx.x) + 280) / 58) * 56)) + ((((int)threadIdx.x) + 48) % 58)) - 57))] : 0.000000e+00f);
  PaddedInput_shared[((((int)threadIdx.x) + 336))] = (((1 <= ((((int)threadIdx.x) + 46) % 58)) && (((((int)threadIdx.x) + 46) % 58) < 57)) ? placeholder[((((((((int)blockIdx.z) * 3136) + (((int)blockIdx.y) * 448)) + (((((int)threadIdx.x) + 336) / 58) * 56)) + ((((int)threadIdx.x) + 46) % 58)) - 57))] : 0.000000e+00f);
  PaddedInput_shared[((((int)threadIdx.x) + 392))] = (((1 <= ((((int)threadIdx.x) + 44) % 58)) && (((((int)threadIdx.x) + 44) % 58) < 57)) ? placeholder[((((((((int)blockIdx.z) * 3136) + (((int)blockIdx.y) * 448)) + (((((int)threadIdx.x) + 392) / 58) * 56)) + ((((int)threadIdx.x) + 44) % 58)) - 57))] : 0.000000e+00f);
  PaddedInput_shared[((((int)threadIdx.x) + 448))] = (((1 <= ((((int)threadIdx.x) + 42) % 58)) && (((((int)threadIdx.x) + 42) % 58) < 57)) ? placeholder[((((((((int)blockIdx.z) * 3136) + (((int)blockIdx.y) * 448)) + (((((int)threadIdx.x) + 448) / 58) * 56)) + ((((int)threadIdx.x) + 42) % 58)) - 57))] : 0.000000e+00f);
  PaddedInput_shared[((((int)threadIdx.x) + 504))] = ((((((((int)blockIdx.y) * 8) + ((((int)threadIdx.x) + 504) / 58)) < 57) && (1 <= ((((int)threadIdx.x) + 40) % 58))) && (((((int)threadIdx.x) + 40) % 58) < 57)) ? placeholder[((((((((int)blockIdx.z) * 3136) + (((int)blockIdx.y) * 448)) + (((((int)threadIdx.x) + 504) / 58) * 56)) + ((((int)threadIdx.x) + 40) % 58)) - 57))] : 0.000000e+00f);
  if (((int)threadIdx.x) < 20) {
    PaddedInput_shared[((((int)threadIdx.x) + 560))] = (((((((int)blockIdx.y) * 8) + ((((int)threadIdx.x) + 560) / 58)) < 57) && (((int)threadIdx.x) < 19)) ? placeholder[((((((((int)blockIdx.z) * 3136) + (((int)blockIdx.y) * 448)) + (((((int)threadIdx.x) + 560) / 58) * 56)) + (((int)threadIdx.x) + 38)) - 57))] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 9) {
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((int)blockIdx.z) * 9) + ((int)threadIdx.x)))];
  }
  __syncthreads();
  PaddedInput_shared_local[(0)] = PaddedInput_shared[(((int)threadIdx.x))];
  PaddedInput_shared_local[(18)] = PaddedInput_shared[((((int)threadIdx.x) + 232))];
  PaddedInput_shared_local[(1)] = PaddedInput_shared[((((int)threadIdx.x) + 1))];
  PaddedInput_shared_local[(19)] = PaddedInput_shared[((((int)threadIdx.x) + 233))];
  PaddedInput_shared_local[(2)] = PaddedInput_shared[((((int)threadIdx.x) + 2))];
  PaddedInput_shared_local[(20)] = PaddedInput_shared[((((int)threadIdx.x) + 234))];
  PaddedInput_shared_local[(3)] = PaddedInput_shared[((((int)threadIdx.x) + 58))];
  PaddedInput_shared_local[(21)] = PaddedInput_shared[((((int)threadIdx.x) + 290))];
  PaddedInput_shared_local[(4)] = PaddedInput_shared[((((int)threadIdx.x) + 59))];
  PaddedInput_shared_local[(22)] = PaddedInput_shared[((((int)threadIdx.x) + 291))];
  PaddedInput_shared_local[(5)] = PaddedInput_shared[((((int)threadIdx.x) + 60))];
  PaddedInput_shared_local[(23)] = PaddedInput_shared[((((int)threadIdx.x) + 292))];
  PaddedInput_shared_local[(6)] = PaddedInput_shared[((((int)threadIdx.x) + 116))];
  PaddedInput_shared_local[(24)] = PaddedInput_shared[((((int)threadIdx.x) + 348))];
  PaddedInput_shared_local[(7)] = PaddedInput_shared[((((int)threadIdx.x) + 117))];
  PaddedInput_shared_local[(25)] = PaddedInput_shared[((((int)threadIdx.x) + 349))];
  PaddedInput_shared_local[(8)] = PaddedInput_shared[((((int)threadIdx.x) + 118))];
  PaddedInput_shared_local[(26)] = PaddedInput_shared[((((int)threadIdx.x) + 350))];
  PaddedInput_shared_local[(9)] = PaddedInput_shared[((((int)threadIdx.x) + 174))];
  PaddedInput_shared_local[(27)] = PaddedInput_shared[((((int)threadIdx.x) + 406))];
  PaddedInput_shared_local[(10)] = PaddedInput_shared[((((int)threadIdx.x) + 175))];
  PaddedInput_shared_local[(28)] = PaddedInput_shared[((((int)threadIdx.x) + 407))];
  PaddedInput_shared_local[(11)] = PaddedInput_shared[((((int)threadIdx.x) + 176))];
  PaddedInput_shared_local[(29)] = PaddedInput_shared[((((int)threadIdx.x) + 408))];
  PaddedInput_shared_local[(12)] = PaddedInput_shared[((((int)threadIdx.x) + 232))];
  PaddedInput_shared_local[(30)] = PaddedInput_shared[((((int)threadIdx.x) + 464))];
  PaddedInput_shared_local[(13)] = PaddedInput_shared[((((int)threadIdx.x) + 233))];
  PaddedInput_shared_local[(31)] = PaddedInput_shared[((((int)threadIdx.x) + 465))];
  PaddedInput_shared_local[(14)] = PaddedInput_shared[((((int)threadIdx.x) + 234))];
  PaddedInput_shared_local[(32)] = PaddedInput_shared[((((int)threadIdx.x) + 466))];
  PaddedInput_shared_local[(15)] = PaddedInput_shared[((((int)threadIdx.x) + 290))];
  PaddedInput_shared_local[(33)] = PaddedInput_shared[((((int)threadIdx.x) + 522))];
  PaddedInput_shared_local[(16)] = PaddedInput_shared[((((int)threadIdx.x) + 291))];
  PaddedInput_shared_local[(34)] = PaddedInput_shared[((((int)threadIdx.x) + 523))];
  PaddedInput_shared_local[(17)] = PaddedInput_shared[((((int)threadIdx.x) + 292))];
  PaddedInput_shared_local[(35)] = PaddedInput_shared[((((int)threadIdx.x) + 524))];
  placeholder_shared_local[(0)] = placeholder_shared[(0)];
  placeholder_shared_local[(1)] = placeholder_shared[(1)];
  placeholder_shared_local[(2)] = placeholder_shared[(2)];
  placeholder_shared_local[(3)] = placeholder_shared[(3)];
  placeholder_shared_local[(4)] = placeholder_shared[(4)];
  placeholder_shared_local[(5)] = placeholder_shared[(5)];
  placeholder_shared_local[(6)] = placeholder_shared[(6)];
  placeholder_shared_local[(7)] = placeholder_shared[(7)];
  placeholder_shared_local[(8)] = placeholder_shared[(8)];
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  DepthwiseConv2d[(4)] = 0.000000e+00f;
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(0)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared_local[(18)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(1)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared_local[(19)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(2)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared_local[(20)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(3)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared_local[(21)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(4)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared_local[(22)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(5)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared_local[(23)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(6)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared_local[(24)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(7)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared_local[(25)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(8)] * placeholder_shared_local[(8)]));
  DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared_local[(26)] * placeholder_shared_local[(8)]));
  DepthwiseConv2d[(1)] = 0.000000e+00f;
  DepthwiseConv2d[(5)] = 0.000000e+00f;
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(3)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared_local[(21)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(4)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared_local[(22)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(5)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared_local[(23)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(6)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared_local[(24)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(7)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared_local[(25)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(8)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared_local[(26)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(9)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared_local[(27)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(10)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared_local[(28)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(11)] * placeholder_shared_local[(8)]));
  DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared_local[(29)] * placeholder_shared_local[(8)]));
  DepthwiseConv2d[(2)] = 0.000000e+00f;
  DepthwiseConv2d[(6)] = 0.000000e+00f;
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(6)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared_local[(24)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(7)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared_local[(25)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(8)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared_local[(26)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(9)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared_local[(27)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(10)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared_local[(28)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(11)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared_local[(29)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(12)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared_local[(30)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(13)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared_local[(31)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(14)] * placeholder_shared_local[(8)]));
  DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared_local[(32)] * placeholder_shared_local[(8)]));
  DepthwiseConv2d[(3)] = 0.000000e+00f;
  DepthwiseConv2d[(7)] = 0.000000e+00f;
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(9)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(7)] = (DepthwiseConv2d[(7)] + (PaddedInput_shared_local[(27)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(10)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(7)] = (DepthwiseConv2d[(7)] + (PaddedInput_shared_local[(28)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(11)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(7)] = (DepthwiseConv2d[(7)] + (PaddedInput_shared_local[(29)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(12)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(7)] = (DepthwiseConv2d[(7)] + (PaddedInput_shared_local[(30)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(13)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(7)] = (DepthwiseConv2d[(7)] + (PaddedInput_shared_local[(31)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(14)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(7)] = (DepthwiseConv2d[(7)] + (PaddedInput_shared_local[(32)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(15)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(7)] = (DepthwiseConv2d[(7)] + (PaddedInput_shared_local[(33)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(16)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(7)] = (DepthwiseConv2d[(7)] + (PaddedInput_shared_local[(34)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(17)] * placeholder_shared_local[(8)]));
  DepthwiseConv2d[(7)] = (DepthwiseConv2d[(7)] + (PaddedInput_shared_local[(35)] * placeholder_shared_local[(8)]));
  T_relu[((((((int)blockIdx.z) * 3136) + (((int)blockIdx.y) * 448)) + ((int)threadIdx.x)))] = max((DepthwiseConv2d[(0)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[(((((((int)blockIdx.z) * 3136) + (((int)blockIdx.y) * 448)) + ((int)threadIdx.x)) + 224))] = max((DepthwiseConv2d[(4)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[(((((((int)blockIdx.z) * 3136) + (((int)blockIdx.y) * 448)) + ((int)threadIdx.x)) + 56))] = max((DepthwiseConv2d[(1)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[(((((((int)blockIdx.z) * 3136) + (((int)blockIdx.y) * 448)) + ((int)threadIdx.x)) + 280))] = max((DepthwiseConv2d[(5)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[(((((((int)blockIdx.z) * 3136) + (((int)blockIdx.y) * 448)) + ((int)threadIdx.x)) + 112))] = max((DepthwiseConv2d[(2)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[(((((((int)blockIdx.z) * 3136) + (((int)blockIdx.y) * 448)) + ((int)threadIdx.x)) + 336))] = max((DepthwiseConv2d[(6)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[(((((((int)blockIdx.z) * 3136) + (((int)blockIdx.y) * 448)) + ((int)threadIdx.x)) + 168))] = max((DepthwiseConv2d[(3)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[(((((((int)blockIdx.z) * 3136) + (((int)blockIdx.y) * 448)) + ((int)threadIdx.x)) + 392))] = max((DepthwiseConv2d[(7)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_5_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  __shared__ float PaddedInput_shared[256];
  __shared__ float placeholder_shared[9];
  float PaddedInput_shared_local[63];
  float placeholder_shared_local[9];
  float DepthwiseConv2d[7];
  PaddedInput_shared[(((((int)threadIdx.y) * 14) + ((int)threadIdx.x)))] = ((((16 <= ((((int)threadIdx.y) * 14) + ((int)threadIdx.x))) && (1 <= (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15))) && ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15) < 15)) ? placeholder[(((((((int)blockIdx.z) * 196) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 4) * 14)) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15)) - 15))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 28))] = (((1 <= ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 12) & 15)) && (((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 12) & 15) < 15)) ? placeholder[(((((((int)blockIdx.z) * 196) + (((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 28) >> 4) * 14)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 12) & 15)) - 15))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 56))] = (((1 <= ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 8) & 15)) && (((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 8) & 15) < 15)) ? placeholder[(((((((int)blockIdx.z) * 196) + (((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 56) >> 4) * 14)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 8) & 15)) - 15))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 84))] = (((1 <= ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 4) & 15)) && (((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 4) & 15) < 15)) ? placeholder[(((((((int)blockIdx.z) * 196) + (((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 84) >> 4) * 14)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 4) & 15)) - 15))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 112))] = (((1 <= (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15)) && ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15) < 15)) ? placeholder[(((((((int)blockIdx.z) * 196) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 4) * 14)) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15)) + 83))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 140))] = (((1 <= ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 12) & 15)) && (((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 12) & 15) < 15)) ? placeholder[(((((((int)blockIdx.z) * 196) + (((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 140) >> 4) * 14)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 12) & 15)) - 15))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 168))] = (((1 <= ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 8) & 15)) && (((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 8) & 15) < 15)) ? placeholder[(((((((int)blockIdx.z) * 196) + (((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 168) >> 4) * 14)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 8) & 15)) - 15))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 196))] = (((1 <= ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 4) & 15)) && (((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 4) & 15) < 15)) ? placeholder[(((((((int)blockIdx.z) * 196) + (((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 196) >> 4) * 14)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 4) & 15)) - 15))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 224))] = ((((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) < 16) && (1 <= (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15))) && ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15) < 15)) ? placeholder[(((((((int)blockIdx.z) * 196) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 181))] : 0.000000e+00f);
  if (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) < 4) {
    if (((int)threadIdx.y) < 1) {
      PaddedInput_shared[((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) + 252))] = 0.000000e+00f;
    }
  }
  if (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) < 9) {
    if (((int)threadIdx.y) < 1) {
      placeholder_shared[(((((int)threadIdx.y) * 14) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.y) * 14) + (((int)blockIdx.z) * 9)) + ((int)threadIdx.x)))];
    }
  }
  __syncthreads();
  PaddedInput_shared_local[(0)] = PaddedInput_shared[(((((int)threadIdx.y) * 16) + ((int)threadIdx.x)))];
  PaddedInput_shared_local[(9)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 32))];
  PaddedInput_shared_local[(18)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 64))];
  PaddedInput_shared_local[(27)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 96))];
  PaddedInput_shared_local[(36)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 128))];
  PaddedInput_shared_local[(45)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 160))];
  PaddedInput_shared_local[(54)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 192))];
  PaddedInput_shared_local[(1)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 1))];
  PaddedInput_shared_local[(10)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 33))];
  PaddedInput_shared_local[(19)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 65))];
  PaddedInput_shared_local[(28)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 97))];
  PaddedInput_shared_local[(37)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 129))];
  PaddedInput_shared_local[(46)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 161))];
  PaddedInput_shared_local[(55)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 193))];
  PaddedInput_shared_local[(2)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 2))];
  PaddedInput_shared_local[(11)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 34))];
  PaddedInput_shared_local[(20)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 66))];
  PaddedInput_shared_local[(29)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 98))];
  PaddedInput_shared_local[(38)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 130))];
  PaddedInput_shared_local[(47)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 162))];
  PaddedInput_shared_local[(56)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 194))];
  PaddedInput_shared_local[(3)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 16))];
  PaddedInput_shared_local[(12)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 48))];
  PaddedInput_shared_local[(21)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 80))];
  PaddedInput_shared_local[(30)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 112))];
  PaddedInput_shared_local[(39)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 144))];
  PaddedInput_shared_local[(48)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 176))];
  PaddedInput_shared_local[(57)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 208))];
  PaddedInput_shared_local[(4)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 17))];
  PaddedInput_shared_local[(13)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 49))];
  PaddedInput_shared_local[(22)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 81))];
  PaddedInput_shared_local[(31)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 113))];
  PaddedInput_shared_local[(40)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 145))];
  PaddedInput_shared_local[(49)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 177))];
  PaddedInput_shared_local[(58)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 209))];
  PaddedInput_shared_local[(5)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 18))];
  PaddedInput_shared_local[(14)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 50))];
  PaddedInput_shared_local[(23)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 82))];
  PaddedInput_shared_local[(32)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 114))];
  PaddedInput_shared_local[(41)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 146))];
  PaddedInput_shared_local[(50)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 178))];
  PaddedInput_shared_local[(59)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 210))];
  PaddedInput_shared_local[(6)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 32))];
  PaddedInput_shared_local[(15)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 64))];
  PaddedInput_shared_local[(24)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 96))];
  PaddedInput_shared_local[(33)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 128))];
  PaddedInput_shared_local[(42)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 160))];
  PaddedInput_shared_local[(51)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 192))];
  PaddedInput_shared_local[(60)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 224))];
  PaddedInput_shared_local[(7)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 33))];
  PaddedInput_shared_local[(16)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 65))];
  PaddedInput_shared_local[(25)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 97))];
  PaddedInput_shared_local[(34)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 129))];
  PaddedInput_shared_local[(43)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 161))];
  PaddedInput_shared_local[(52)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 193))];
  PaddedInput_shared_local[(61)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 225))];
  PaddedInput_shared_local[(8)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 34))];
  PaddedInput_shared_local[(17)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 66))];
  PaddedInput_shared_local[(26)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 98))];
  PaddedInput_shared_local[(35)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 130))];
  PaddedInput_shared_local[(44)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 162))];
  PaddedInput_shared_local[(53)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 194))];
  PaddedInput_shared_local[(62)] = PaddedInput_shared[((((((int)threadIdx.y) * 16) + ((int)threadIdx.x)) + 226))];
  placeholder_shared_local[(0)] = placeholder_shared[(0)];
  placeholder_shared_local[(1)] = placeholder_shared[(1)];
  placeholder_shared_local[(2)] = placeholder_shared[(2)];
  placeholder_shared_local[(3)] = placeholder_shared[(3)];
  placeholder_shared_local[(4)] = placeholder_shared[(4)];
  placeholder_shared_local[(5)] = placeholder_shared[(5)];
  placeholder_shared_local[(6)] = placeholder_shared[(6)];
  placeholder_shared_local[(7)] = placeholder_shared[(7)];
  placeholder_shared_local[(8)] = placeholder_shared[(8)];
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  DepthwiseConv2d[(1)] = 0.000000e+00f;
  DepthwiseConv2d[(2)] = 0.000000e+00f;
  DepthwiseConv2d[(3)] = 0.000000e+00f;
  DepthwiseConv2d[(4)] = 0.000000e+00f;
  DepthwiseConv2d[(5)] = 0.000000e+00f;
  DepthwiseConv2d[(6)] = 0.000000e+00f;
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(0)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(9)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(18)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(27)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared_local[(36)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared_local[(45)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared_local[(54)] * placeholder_shared_local[(0)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(1)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(10)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(19)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(28)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared_local[(37)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared_local[(46)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared_local[(55)] * placeholder_shared_local[(1)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(2)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(11)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(20)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(29)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared_local[(38)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared_local[(47)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared_local[(56)] * placeholder_shared_local[(2)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(3)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(12)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(21)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(30)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared_local[(39)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared_local[(48)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared_local[(57)] * placeholder_shared_local[(3)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(4)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(13)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(22)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(31)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared_local[(40)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared_local[(49)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared_local[(58)] * placeholder_shared_local[(4)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(5)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(14)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(23)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(32)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared_local[(41)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared_local[(50)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared_local[(59)] * placeholder_shared_local[(5)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(6)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(15)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(24)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(33)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared_local[(42)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared_local[(51)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared_local[(60)] * placeholder_shared_local[(6)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(7)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(16)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(25)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(34)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared_local[(43)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared_local[(52)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared_local[(61)] * placeholder_shared_local[(7)]));
  DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared_local[(8)] * placeholder_shared_local[(8)]));
  DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared_local[(17)] * placeholder_shared_local[(8)]));
  DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared_local[(26)] * placeholder_shared_local[(8)]));
  DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared_local[(35)] * placeholder_shared_local[(8)]));
  DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared_local[(44)] * placeholder_shared_local[(8)]));
  DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared_local[(53)] * placeholder_shared_local[(8)]));
  DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared_local[(62)] * placeholder_shared_local[(8)]));
  T_relu[((((((int)blockIdx.z) * 196) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)))] = max((DepthwiseConv2d[(0)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[(((((((int)blockIdx.z) * 196) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 28))] = max((DepthwiseConv2d[(1)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[(((((((int)blockIdx.z) * 196) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 56))] = max((DepthwiseConv2d[(2)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[(((((((int)blockIdx.z) * 196) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 84))] = max((DepthwiseConv2d[(3)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[(((((((int)blockIdx.z) * 196) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 112))] = max((DepthwiseConv2d[(4)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[(((((((int)blockIdx.z) * 196) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 140))] = max((DepthwiseConv2d[(5)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[(((((((int)blockIdx.z) * 196) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 168))] = max((DepthwiseConv2d[(6)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_global_avg_pool2d_kernel0(float* __restrict__ placeholder, float* __restrict__ tensor) {
  float tensor1[1];
  tensor1[(0)] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 7; ++rv0) {
    for (int rv1 = 0; rv1 < 7; ++rv1) {
      if (((int)threadIdx.y) < 1) {
        tensor1[(0)] = (tensor1[(0)] + placeholder[((((((((int)threadIdx.y) * 50176) + (((int)blockIdx.x) * 392)) + (((int)threadIdx.x) * 49)) + (rv0 * 7)) + rv1))]);
      }
    }
  }
  if (((int)threadIdx.y) < 1) {
    tensor[((((((int)threadIdx.y) * 1024) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)))] = (tensor1[(0)] * 2.040816e-02f);
  }
}

extern "C" __global__ void fused_nn_dense_add_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float T_dense_rf[1];
  __shared__ float red_buf0[64];
  __shared__ float T_dense[1];
  T_dense_rf[(0)] = 0.000000e+00f;
  for (int k_outer = 0; k_outer < 16; ++k_outer) {
    T_dense_rf[(0)] = (T_dense_rf[(0)] + (placeholder[(((k_outer * 64) + ((int)threadIdx.x)))] * placeholder1[((((((int)blockIdx.x) * 1024) + (k_outer * 64)) + ((int)threadIdx.x)))]));
  }
  __syncthreads();
  ((volatile float*)red_buf0)[(((int)threadIdx.x))] = T_dense_rf[(0)];
  __syncthreads();
  if (((int)threadIdx.x) < 32) {
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 32))]);
  }
  __syncthreads();
  if (((int)threadIdx.x) < 16) {
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 16))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 8))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 4))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 2))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 1))]);
  }
  __syncthreads();
  if (((int)threadIdx.x) == 0) {
    T_dense[(0)] = ((volatile float*)red_buf0)[(0)];
  }
  if (((int)threadIdx.x) == 0) {
    T_add[(((int)blockIdx.x))] = (T_dense[(0)] + placeholder2[(((int)blockIdx.x))]);
  }
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_8_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[8];
  __shared__ float pad_temp_shared[896];
  __shared__ float placeholder_shared[1024];
  compute[(0)] = 0.000000e+00f;
  compute[(2)] = 0.000000e+00f;
  compute[(4)] = 0.000000e+00f;
  compute[(6)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  compute[(3)] = 0.000000e+00f;
  compute[(5)] = 0.000000e+00f;
  compute[(7)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 16; ++rc_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      pad_temp_shared[((((((int)threadIdx.z) * 56) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = placeholder[((((((rc_outer * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
      if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) >> 4)) < 64) {
        if ((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 1024) {
          if (((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 64) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = placeholder1[((((((((int)blockIdx.z) * 16384) + (((int)threadIdx.z) * 1024)) + ((((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) >> 4) * 256)) + (rc_outer * 16)) + (((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) & 15)))];
          }
        }
      }
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((rc_inner * 56) + ((int)threadIdx.x)))] * placeholder_shared[(((((int)threadIdx.z) * 16) + rc_inner))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((rc_inner * 56) + ((int)threadIdx.x)))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 256))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((rc_inner * 56) + ((int)threadIdx.x)))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 512))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((rc_inner * 56) + ((int)threadIdx.x)))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 768))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((rc_inner * 56) + ((int)threadIdx.x)) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 16) + rc_inner))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((rc_inner * 56) + ((int)threadIdx.x)) + 28))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 256))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((rc_inner * 56) + ((int)threadIdx.x)) + 28))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 512))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((rc_inner * 56) + ((int)threadIdx.x)) + 28))] * placeholder_shared[((((((int)threadIdx.z) * 16) + rc_inner) + 768))]));
    }
  }
  T_relu[(((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + ((int)threadIdx.x)))] = max((compute[(0)] + placeholder2[(((((int)blockIdx.z) * 64) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + ((int)threadIdx.x)) + 12544))] = max((compute[(2)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 16))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + ((int)threadIdx.x)) + 25088))] = max((compute[(4)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 32))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + ((int)threadIdx.x)) + 37632))] = max((compute[(6)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 48))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + ((int)threadIdx.x)) + 28))] = max((compute[(1)] + placeholder2[(((((int)blockIdx.z) * 64) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + ((int)threadIdx.x)) + 12572))] = max((compute[(3)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 16))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + ((int)threadIdx.x)) + 25116))] = max((compute[(5)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 32))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + ((int)threadIdx.x)) + 37660))] = max((compute[(7)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 48))]), 0.000000e+00f);
}

