#include <stdio.h>
#include <stdlib.h>
#include<hip/hip_runtime.h>

#include <iostream>
#include <sys/time.h>
#include <math.h>


#define LAUNCH_THREADX 7
#define LAUNCH_THREADY 1
#define LAUNCH_THREADZ 4

#define LAUNCH_BLOCKX 1
#define ORI_BLOCKX 1
#define LAUNCH_BLOCKY 1
#define ORI_BLOCKY 1
#define LAUNCH_BLOCKZ 512 * 5 *4 // 5是额外部分，满足多层覆盖
#define ORI_BLOCKZ 512

#define SM_NUM 32
#define WORKER_NUM_PERSM 16

#define BLOCK_NUM LAUNCH_BLOCKZ * LAUNCH_BLOCKY * LAUNCH_BLOCKX
#define FLAG_LENGTH 65535
#define FLAG_BLOCK_BASE 0
#define FLAG_SM_BASE (FLAG_BLOCK_BASE + 1)
#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)
// nvcc -arch=native main.cu -o main

#define GPU_RETURN_STATUS(cmd) \
{ \
    hipError_t result = cmd; \
    if (result != hipSuccess) { \
        std::cout << #cmd " error, return code:" << result << " | " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); \
    } \
}

// These are the inline versions for all of the SDK helper functions
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    const char *errorStr = NULL;
    errorStr = hipGetErrorString(err);
    fprintf(stderr,
            "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, "
            "line %i.\n",
            err, errorStr, file, line);
    exit(EXIT_FAILURE);
  }
}

__device__ uint get_smid(void) {

    uint ret;

    asm("mov.u32 %0, %smid;" : "=r"(ret) );

    return ret;

}

// #if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 700)
// #define __shfl_sync(mask, var, lane, width) \
//         __shfl((var), (lane), (width))

// #define __shfl_down_sync(mask, var, offset, width) \
//         __shfl_down((var), (offset), (width))

// #define __shfl_up_sync(mask, var, offset, width) \
//         __shfl_up((var), (offset), (width))
// #endif

extern "C" __global__ void fused_nn_conv2d_add_multiply_add_nn_relu_kernel0(int *worker,int number,int *flag, float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3, float* __restrict__ placeholder4) {
    int* sm_flag = flag;
    __shared__ int basicoffset;
    int offset;
    int smid;

    //judge whether to continue work,which work to fetch
    if(threadIdx.x+threadIdx.y+threadIdx.z == 0)
    {
       basicoffset=-1;
       smid = get_smid();

       int blocknumber=atomicAdd(sm_flag + smid, 1);
       atomicAdd(worker + smid, 1);

       /*
       if(smid>63)
       {
       for(int sleeptime=0;sleeptime<400;sleeptime++)
       {
        __nanosleep(10);
       }
       }
       */

       //judge whther sm id is right
       if((smid < number*SM_NUM)&&(smid >= (number-1)*SM_NUM))
       {
            //judge whether worker is enough
            //get the basic offset for the block
            if(blocknumber< WORKER_NUM_PERSM)
            {

                basicoffset = WORKER_NUM_PERSM*(smid-(number-1)*SM_NUM) + blocknumber;
                //printf("smid %d\n", smid);
            }
       }
       else
       {
       for(int sleeptime=0;sleeptime<50;sleeptime++)
       {
        __nanosleep(10);
       }
       return;

       }

    }
    __syncthreads();
    if (basicoffset < 0)
    {


    return ;
    }

    //every thread has its own offset
    offset = basicoffset;
    // if ((threadIdx.x + threadIdx.y + threadIdx.z) == 0 && (number == 1)) {
    //     printf("smid %d\n", smid);
    // }

    while(offset < (ORI_BLOCKX * ORI_BLOCKY * ORI_BLOCKZ)) {
        int vx = (offset)/(ORI_BLOCKY * ORI_BLOCKZ);
        int vy = (offset - (vx * ORI_BLOCKY * ORI_BLOCKZ)) / ORI_BLOCKZ;
        int vz = offset - (vx * ORI_BLOCKY * ORI_BLOCKZ) - vy * ORI_BLOCKZ;
        offset += SM_NUM * WORKER_NUM_PERSM;
        // begin original
        float compute[56];
        __shared__ float pad_temp_shared[196];
        __shared__ float placeholder_shared[128];
        compute[(0)] = 0.000000e+00f;
        compute[(28)] = 0.000000e+00f;
        compute[(4)] = 0.000000e+00f;
        compute[(32)] = 0.000000e+00f;
        compute[(8)] = 0.000000e+00f;
        compute[(36)] = 0.000000e+00f;
        compute[(12)] = 0.000000e+00f;
        compute[(40)] = 0.000000e+00f;
        compute[(16)] = 0.000000e+00f;
        compute[(44)] = 0.000000e+00f;
        compute[(20)] = 0.000000e+00f;
        compute[(48)] = 0.000000e+00f;
        compute[(24)] = 0.000000e+00f;
        compute[(52)] = 0.000000e+00f;
        compute[(1)] = 0.000000e+00f;
        compute[(29)] = 0.000000e+00f;
        compute[(5)] = 0.000000e+00f;
        compute[(33)] = 0.000000e+00f;
        compute[(9)] = 0.000000e+00f;
        compute[(37)] = 0.000000e+00f;
        compute[(13)] = 0.000000e+00f;
        compute[(41)] = 0.000000e+00f;
        compute[(17)] = 0.000000e+00f;
        compute[(45)] = 0.000000e+00f;
        compute[(21)] = 0.000000e+00f;
        compute[(49)] = 0.000000e+00f;
        compute[(25)] = 0.000000e+00f;
        compute[(53)] = 0.000000e+00f;
        compute[(2)] = 0.000000e+00f;
        compute[(30)] = 0.000000e+00f;
        compute[(6)] = 0.000000e+00f;
        compute[(34)] = 0.000000e+00f;
        compute[(10)] = 0.000000e+00f;
        compute[(38)] = 0.000000e+00f;
        compute[(14)] = 0.000000e+00f;
        compute[(42)] = 0.000000e+00f;
        compute[(18)] = 0.000000e+00f;
        compute[(46)] = 0.000000e+00f;
        compute[(22)] = 0.000000e+00f;
        compute[(50)] = 0.000000e+00f;
        compute[(26)] = 0.000000e+00f;
        compute[(54)] = 0.000000e+00f;
        compute[(3)] = 0.000000e+00f;
        compute[(31)] = 0.000000e+00f;
        compute[(7)] = 0.000000e+00f;
        compute[(35)] = 0.000000e+00f;
        compute[(11)] = 0.000000e+00f;
        compute[(39)] = 0.000000e+00f;
        compute[(15)] = 0.000000e+00f;
        compute[(43)] = 0.000000e+00f;
        compute[(19)] = 0.000000e+00f;
        compute[(47)] = 0.000000e+00f;
        compute[(23)] = 0.000000e+00f;
        compute[(51)] = 0.000000e+00f;
        compute[(27)] = 0.000000e+00f;
        compute[(55)] = 0.000000e+00f;
        for (int rc_outer = 0; rc_outer < 128; ++rc_outer) {
        for (int ry_outer = 0; ry_outer < 3; ++ry_outer) {
            for (int rx_outer = 0; rx_outer < 3; ++rx_outer) {
            __syncthreads();
            pad_temp_shared[(((((int)threadIdx.z) * 49) + (((int)threadIdx.x) * 7)))] = ((((1 <= (((int)threadIdx.x) + ry_outer)) && ((((int)threadIdx.x) + ry_outer) < 8)) && (1 <= rx_outer)) ? placeholder[(((((((((((int)vz) >> 4) * 25088) + (rc_outer * 196)) + (((int)threadIdx.z) * 49)) + (ry_outer * 7)) + (((int)threadIdx.x) * 7)) + rx_outer) - 8))] : 0.000000e+00f);
            pad_temp_shared[((((((int)threadIdx.z) * 49) + (((int)threadIdx.x) * 7)) + 1))] = (((1 <= (((int)threadIdx.x) + ry_outer)) && ((((int)threadIdx.x) + ry_outer) < 8)) ? placeholder[(((((((((((int)vz) >> 4) * 25088) + (rc_outer * 196)) + (((int)threadIdx.z) * 49)) + (ry_outer * 7)) + (((int)threadIdx.x) * 7)) + rx_outer) - 7))] : 0.000000e+00f);
            pad_temp_shared[((((((int)threadIdx.z) * 49) + (((int)threadIdx.x) * 7)) + 2))] = (((1 <= (((int)threadIdx.x) + ry_outer)) && ((((int)threadIdx.x) + ry_outer) < 8)) ? placeholder[(((((((((((int)vz) >> 4) * 25088) + (rc_outer * 196)) + (((int)threadIdx.z) * 49)) + (ry_outer * 7)) + (((int)threadIdx.x) * 7)) + rx_outer) - 6))] : 0.000000e+00f);
            pad_temp_shared[((((((int)threadIdx.z) * 49) + (((int)threadIdx.x) * 7)) + 3))] = (((1 <= (((int)threadIdx.x) + ry_outer)) && ((((int)threadIdx.x) + ry_outer) < 8)) ? placeholder[(((((((((((int)vz) >> 4) * 25088) + (rc_outer * 196)) + (((int)threadIdx.z) * 49)) + (ry_outer * 7)) + (((int)threadIdx.x) * 7)) + rx_outer) - 5))] : 0.000000e+00f);
            pad_temp_shared[((((((int)threadIdx.z) * 49) + (((int)threadIdx.x) * 7)) + 4))] = (((1 <= (((int)threadIdx.x) + ry_outer)) && ((((int)threadIdx.x) + ry_outer) < 8)) ? placeholder[(((((((((((int)vz) >> 4) * 25088) + (rc_outer * 196)) + (((int)threadIdx.z) * 49)) + (ry_outer * 7)) + (((int)threadIdx.x) * 7)) + rx_outer) - 4))] : 0.000000e+00f);
            pad_temp_shared[((((((int)threadIdx.z) * 49) + (((int)threadIdx.x) * 7)) + 5))] = (((1 <= (((int)threadIdx.x) + ry_outer)) && ((((int)threadIdx.x) + ry_outer) < 8)) ? placeholder[(((((((((((int)vz) >> 4) * 25088) + (rc_outer * 196)) + (((int)threadIdx.z) * 49)) + (ry_outer * 7)) + (((int)threadIdx.x) * 7)) + rx_outer) - 3))] : 0.000000e+00f);
            pad_temp_shared[((((((int)threadIdx.z) * 49) + (((int)threadIdx.x) * 7)) + 6))] = ((((1 <= (((int)threadIdx.x) + ry_outer)) && ((((int)threadIdx.x) + ry_outer) < 8)) && (rx_outer < 2)) ? placeholder[(((((((((((int)vz) >> 4) * 25088) + (rc_outer * 196)) + (((int)threadIdx.z) * 49)) + (ry_outer * 7)) + (((int)threadIdx.x) * 7)) + rx_outer) - 2))] : 0.000000e+00f);
            placeholder_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.x) * 5)))] = placeholder1[(((((((((((int)vz) & 15) * 147456) + (((int)threadIdx.z) * 36864)) + (((((int)threadIdx.x) * 5) >> 2) * 4608)) + (rc_outer * 36)) + (((((int)threadIdx.x) * 5) & 3) * 9)) + (ry_outer * 3)) + rx_outer))];
            placeholder_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.x) * 5)) + 1))] = placeholder1[(((((((((((int)vz) & 15) * 147456) + (((int)threadIdx.z) * 36864)) + ((((((int)threadIdx.x) * 5) + 1) >> 2) * 4608)) + (rc_outer * 36)) + ((((((int)threadIdx.x) * 5) + 1) & 3) * 9)) + (ry_outer * 3)) + rx_outer))];
            if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 5) + 2) >> 2)) < 32) {
                if (((((int)threadIdx.z) * 32) + (((int)threadIdx.x) * 5)) < 126) {
                if (((int)threadIdx.x) < 6) {
                    placeholder_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.x) * 5)) + 2))] = placeholder1[(((((((((((int)vz) & 15) * 147456) + (((int)threadIdx.z) * 36864)) + ((((((int)threadIdx.x) * 5) + 2) >> 2) * 4608)) + (rc_outer * 36)) + ((((((int)threadIdx.x) * 5) + 2) & 3) * 9)) + (ry_outer * 3)) + rx_outer))];
                }
                }
            }
            if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 5) + 3) >> 2)) < 32) {
                if (((((int)threadIdx.z) * 32) + (((int)threadIdx.x) * 5)) < 125) {
                if (((int)threadIdx.x) < 6) {
                    placeholder_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.x) * 5)) + 3))] = placeholder1[(((((((((((int)vz) & 15) * 147456) + (((int)threadIdx.z) * 36864)) + ((((((int)threadIdx.x) * 5) + 3) >> 2) * 4608)) + (rc_outer * 36)) + ((((((int)threadIdx.x) * 5) + 3) & 3) * 9)) + (ry_outer * 3)) + rx_outer))];
                }
                }
            }
            if (((((int)threadIdx.z) * 8) + ((((int)threadIdx.x) * 5) >> 2)) < 31) {
                if (((((int)threadIdx.z) * 32) + (((int)threadIdx.x) * 5)) < 124) {
                if (((int)threadIdx.x) < 6) {
                    placeholder_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.x) * 5)) + 4))] = placeholder1[((((((((((((int)vz) & 15) * 147456) + (((int)threadIdx.z) * 36864)) + (((((int)threadIdx.x) * 5) >> 2) * 4608)) + (rc_outer * 36)) + (((((int)threadIdx.x) * 5) & 3) * 9)) + (ry_outer * 3)) + rx_outer) + 4608))];
                }
                }
            }
            __syncthreads();
            compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 16))]));
            compute[(28)] = (compute[(28)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 64))]));
            compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 7))] * placeholder_shared[((((int)threadIdx.z) * 16))]));
            compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 7))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 64))]));
            compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 16))]));
            compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 64))]));
            compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 21))] * placeholder_shared[((((int)threadIdx.z) * 16))]));
            compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 21))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 64))]));
            compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[((((int)threadIdx.z) * 16))]));
            compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 64))]));
            compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 35))] * placeholder_shared[((((int)threadIdx.z) * 16))]));
            compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 35))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 64))]));
            compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[((((int)threadIdx.z) * 16))]));
            compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 64))]));
            compute[(1)] = (compute[(1)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 4))]));
            compute[(29)] = (compute[(29)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 68))]));
            compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 7))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 4))]));
            compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 7))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 68))]));
            compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 4))]));
            compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 68))]));
            compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 21))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 4))]));
            compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 21))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 68))]));
            compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 4))]));
            compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 68))]));
            compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 35))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 4))]));
            compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 35))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 68))]));
            compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 4))]));
            compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 68))]));
            compute[(2)] = (compute[(2)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 8))]));
            compute[(30)] = (compute[(30)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 72))]));
            compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 7))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 8))]));
            compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 7))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 72))]));
            compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 8))]));
            compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 72))]));
            compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 21))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 8))]));
            compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 21))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 72))]));
            compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 8))]));
            compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 72))]));
            compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 35))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 8))]));
            compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 35))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 72))]));
            compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 8))]));
            compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 72))]));
            compute[(3)] = (compute[(3)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 12))]));
            compute[(31)] = (compute[(31)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 76))]));
            compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 7))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 12))]));
            compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 7))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 76))]));
            compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 12))]));
            compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 76))]));
            compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 21))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 12))]));
            compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 21))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 76))]));
            compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 12))]));
            compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 76))]));
            compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 35))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 12))]));
            compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 35))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 76))]));
            compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 12))]));
            compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 76))]));
            compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 49))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 1))]));
            compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 49))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 65))]));
            compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 1))]));
            compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 65))]));
            compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 63))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 1))]));
            compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 63))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 65))]));
            compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 1))]));
            compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 65))]));
            compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 77))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 1))]));
            compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 77))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 65))]));
            compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 1))]));
            compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 65))]));
            compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 91))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 1))]));
            compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 91))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 65))]));
            compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 49))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 5))]));
            compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 49))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 69))]));
            compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 5))]));
            compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 69))]));
            compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 63))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 5))]));
            compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 63))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 69))]));
            compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 5))]));
            compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 69))]));
            compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 77))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 5))]));
            compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 77))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 69))]));
            compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 5))]));
            compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 69))]));
            compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 91))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 5))]));
            compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 91))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 69))]));
            compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 49))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 9))]));
            compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 49))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 73))]));
            compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 9))]));
            compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 73))]));
            compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 63))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 9))]));
            compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 63))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 73))]));
            compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 9))]));
            compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 73))]));
            compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 77))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 9))]));
            compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 77))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 73))]));
            compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 9))]));
            compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 73))]));
            compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 91))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 9))]));
            compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 91))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 73))]));
            compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 49))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 13))]));
            compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 49))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 77))]));
            compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 13))]));
            compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 77))]));
            compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 63))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 13))]));
            compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 63))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 77))]));
            compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 13))]));
            compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 77))]));
            compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 77))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 13))]));
            compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 77))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 77))]));
            compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 13))]));
            compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 77))]));
            compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 91))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 13))]));
            compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 91))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 77))]));
            compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 2))]));
            compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 66))]));
            compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 105))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 2))]));
            compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 105))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 66))]));
            compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 2))]));
            compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 66))]));
            compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 119))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 2))]));
            compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 119))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 66))]));
            compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 2))]));
            compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 66))]));
            compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 133))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 2))]));
            compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 133))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 66))]));
            compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 2))]));
            compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 66))]));
            compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 6))]));
            compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 70))]));
            compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 105))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 6))]));
            compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 105))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 70))]));
            compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 6))]));
            compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 70))]));
            compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 119))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 6))]));
            compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 119))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 70))]));
            compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 6))]));
            compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 70))]));
            compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 133))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 6))]));
            compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 133))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 70))]));
            compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 6))]));
            compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 70))]));
            compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 10))]));
            compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 74))]));
            compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 105))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 10))]));
            compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 105))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 74))]));
            compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 10))]));
            compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 74))]));
            compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 119))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 10))]));
            compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 119))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 74))]));
            compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 10))]));
            compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 74))]));
            compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 133))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 10))]));
            compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 133))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 74))]));
            compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 10))]));
            compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 74))]));
            compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 14))]));
            compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 78))]));
            compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 105))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 14))]));
            compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 105))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 78))]));
            compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 14))]));
            compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 78))]));
            compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 119))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 14))]));
            compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 119))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 78))]));
            compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 14))]));
            compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 78))]));
            compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 133))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 14))]));
            compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 133))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 78))]));
            compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 14))]));
            compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 78))]));
            compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 147))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 3))]));
            compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 147))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 67))]));
            compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 3))]));
            compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 67))]));
            compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 161))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 3))]));
            compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 161))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 67))]));
            compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 3))]));
            compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 67))]));
            compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 175))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 3))]));
            compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 175))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 67))]));
            compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 3))]));
            compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 67))]));
            compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 189))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 3))]));
            compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 189))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 67))]));
            compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 147))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 7))]));
            compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 147))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 71))]));
            compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 7))]));
            compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 71))]));
            compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 161))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 7))]));
            compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 161))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 71))]));
            compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 7))]));
            compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 71))]));
            compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 175))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 7))]));
            compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 175))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 71))]));
            compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 7))]));
            compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 71))]));
            compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 189))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 7))]));
            compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 189))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 71))]));
            compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 147))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 11))]));
            compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 147))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 75))]));
            compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 11))]));
            compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 75))]));
            compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 161))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 11))]));
            compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 161))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 75))]));
            compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 11))]));
            compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 75))]));
            compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 175))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 11))]));
            compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 175))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 75))]));
            compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 11))]));
            compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 75))]));
            compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 189))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 11))]));
            compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 189))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 75))]));
            compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 147))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 15))]));
            compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 147))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 79))]));
            compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 15))]));
            compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 79))]));
            compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 161))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 15))]));
            compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 161))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 79))]));
            compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 15))]));
            compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 79))]));
            compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 175))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 15))]));
            compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 175))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 79))]));
            compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 15))]));
            compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 79))]));
            compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 189))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 15))]));
            compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 189))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 79))]));
            }
        }
        }
        T_relu[((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)))] = max((((compute[(0)] + placeholder2[((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)))]) * placeholder3[((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)))]) + placeholder4[((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 784))] = max((((compute[(28)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 784))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 16))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 16))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 7))] = max((((compute[(4)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 7))]) * placeholder3[((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)))]) + placeholder4[((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 791))] = max((((compute[(32)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 791))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 16))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 16))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 14))] = max((((compute[(8)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 14))]) * placeholder3[((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)))]) + placeholder4[((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 798))] = max((((compute[(36)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 798))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 16))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 16))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 21))] = max((((compute[(12)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 21))]) * placeholder3[((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)))]) + placeholder4[((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 805))] = max((((compute[(40)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 805))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 16))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 16))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 28))] = max((((compute[(16)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 28))]) * placeholder3[((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)))]) + placeholder4[((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 812))] = max((((compute[(44)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 812))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 16))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 16))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 35))] = max((((compute[(20)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 35))]) * placeholder3[((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)))]) + placeholder4[((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 819))] = max((((compute[(48)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 819))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 16))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 16))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 42))] = max((((compute[(24)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 42))]) * placeholder3[((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)))]) + placeholder4[((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 826))] = max((((compute[(52)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 826))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 16))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 16))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 49))] = max((((compute[(1)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 49))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 1))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 1))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 833))] = max((((compute[(29)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 833))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 17))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 17))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 56))] = max((((compute[(5)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 56))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 1))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 1))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 840))] = max((((compute[(33)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 840))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 17))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 17))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 63))] = max((((compute[(9)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 63))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 1))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 1))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 847))] = max((((compute[(37)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 847))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 17))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 17))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 70))] = max((((compute[(13)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 70))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 1))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 1))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 854))] = max((((compute[(41)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 854))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 17))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 17))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 77))] = max((((compute[(17)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 77))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 1))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 1))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 861))] = max((((compute[(45)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 861))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 17))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 17))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 84))] = max((((compute[(21)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 84))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 1))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 1))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 868))] = max((((compute[(49)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 868))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 17))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 17))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 91))] = max((((compute[(25)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 91))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 1))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 1))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 875))] = max((((compute[(53)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 875))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 17))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 17))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 98))] = max((((compute[(2)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 98))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 2))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 2))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 882))] = max((((compute[(30)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 882))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 18))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 18))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 105))] = max((((compute[(6)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 105))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 2))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 2))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 889))] = max((((compute[(34)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 889))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 18))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 18))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 112))] = max((((compute[(10)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 112))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 2))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 2))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 896))] = max((((compute[(38)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 896))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 18))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 18))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 119))] = max((((compute[(14)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 119))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 2))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 2))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 903))] = max((((compute[(42)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 903))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 18))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 18))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 126))] = max((((compute[(18)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 126))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 2))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 2))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 910))] = max((((compute[(46)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 910))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 18))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 18))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 133))] = max((((compute[(22)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 133))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 2))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 2))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 917))] = max((((compute[(50)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 917))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 18))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 18))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 140))] = max((((compute[(26)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 140))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 2))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 2))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 924))] = max((((compute[(54)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 924))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 18))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 18))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 147))] = max((((compute[(3)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 147))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 3))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 3))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 931))] = max((((compute[(31)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 931))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 19))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 19))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 154))] = max((((compute[(7)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 154))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 3))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 3))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 938))] = max((((compute[(35)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 938))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 19))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 19))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 161))] = max((((compute[(11)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 161))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 3))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 3))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 945))] = max((((compute[(39)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 945))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 19))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 19))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 168))] = max((((compute[(15)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 168))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 3))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 3))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 952))] = max((((compute[(43)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 952))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 19))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 19))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 175))] = max((((compute[(19)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 175))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 3))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 3))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 959))] = max((((compute[(47)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 959))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 19))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 19))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 182))] = max((((compute[(23)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 182))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 3))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 3))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 966))] = max((((compute[(51)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 966))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 19))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 19))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 189))] = max((((compute[(27)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 189))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 3))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 3))]), 0.000000e+00f);
        T_relu[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 973))] = max((((compute[(55)] + placeholder2[(((((((int)vz) * 1568) + (((int)threadIdx.z) * 196)) + ((int)threadIdx.x)) + 973))]) * placeholder3[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 19))]) + placeholder4[(((((((int)vz) & 15) * 32) + (((int)threadIdx.z) * 4)) + 19))]), 0.000000e+00f);

        __syncthreads();
    }
}

int main(int argc, char *argv[]) {
    // init device
    if (argc < 2) {
        printf("args num error! argc:%d", argc);
        exit(1);
    }
    int gpu_no = atoi(argv[1]);
    checkCudaErrors(hipSetDevice(gpu_no));

    // allocate stream
    int num_streams = 2;
    hipStream_t streams[num_streams];
    for (int i = 0; i < num_streams; i++) {
        checkCudaErrors(hipStreamCreate(&streams[i]));
    }


    // allocate flag
    int *flag = new int[FLAG_LENGTH];
    int *g_flag;
    for (int i = 0; i < FLAG_LENGTH; ++i) {
        flag[i] = 0;
    }
    checkCudaErrors(hipMalloc((void **)&g_flag, sizeof(int) * FLAG_LENGTH));
    checkCudaErrors(hipMemcpy(g_flag, flag, sizeof(int) * FLAG_LENGTH, hipMemcpyHostToDevice));

    int *flag_ = new int[FLAG_LENGTH];
    int *g_flag_;
    for (int i = 0; i < FLAG_LENGTH; ++i) {
        flag_[i] = 0;
    }
    checkCudaErrors(hipMalloc((void **)&g_flag_, sizeof(int) * FLAG_LENGTH));
    checkCudaErrors(hipMemcpy(g_flag_, flag_, sizeof(int) * FLAG_LENGTH, hipMemcpyHostToDevice));

    //prepare parm for kernel 1
    int *workers = new int[80];
    for(int i=0;i<80;i++)
    {
    workers[i]=0;
    }
    int *g_worker;
    checkCudaErrors(hipMalloc((void **)&g_worker, sizeof(int) * 80));
    checkCudaErrors(hipMemcpy( g_worker,workers, sizeof(int) * 80, hipMemcpyHostToDevice));

    int *g_worker2;
    checkCudaErrors(hipMalloc((void **)&g_worker2, sizeof(int) * 80));
    checkCudaErrors(hipMemcpy( g_worker2,workers, sizeof(int) * 80, hipMemcpyHostToDevice));


    float *placeholder0 = new float[802816];
    for(int i=0;i<802816;i++)
    {
    placeholder0[i]=1;
    }
    float *g_ph0;
    checkCudaErrors(hipMalloc((void **)&g_ph0, sizeof(float) * 802816));
    checkCudaErrors(hipMemcpy(g_ph0, placeholder0, sizeof(float) * 802816, hipMemcpyHostToDevice));

    float *placeholder1 = new float[2359296];
    for(int i=0;i<2359296;i++)
    {
    placeholder1[i]=2;
    }
    float *g_ph1;
    checkCudaErrors(hipMalloc((void **)&g_ph1, sizeof(float) * 2359296));
    //checkCudaErrors(cudaMemcpy(g_ph1, placeholder1, sizeof(float) * 2359296, cudaMemcpyHostToDevice));

    float *placeholder2 = new float[802816];
    for(int i=0;i<802816;i++)
    {
    placeholder2[i]=3;
    }
    float *g_ph2;
    checkCudaErrors(hipMalloc((void **)&g_ph2, sizeof(float) * 802816));
    checkCudaErrors(hipMemcpy(g_ph2, placeholder2, sizeof(float) * 802816, hipMemcpyHostToDevice));

    float *placeholder3 = new float[802816];
    for(int i=0;i<802816;i++)
    {
    placeholder3[i]=4;
    }
    float *g_ph3;
    hipMalloc((void **)&g_ph3, sizeof(float) * 802816);
    hipMemcpy(g_ph3, placeholder3, sizeof(float) * 802816, hipMemcpyHostToDevice);

    float *placeholder4 = new float[512];
    for(int i=0;i<512;i++)
    {
    placeholder4[i]=5;
    }
    float *g_ph4;
    hipMalloc((void **)&g_ph4, sizeof(float) * 512);
    hipMemcpy(g_ph4, placeholder4, sizeof(float) * 512, hipMemcpyHostToDevice);

    float *placeholder5 = new float[802816];
    for(int i=0;i<802816;i++)
    {
    placeholder5[i]=6;
    }
    float *g_ph5;
    hipMalloc((void **)&g_ph5, sizeof(float) * 802816);
    hipMemcpy(g_ph5, placeholder5, sizeof(float) * 802816, hipMemcpyHostToDevice);

    //prepare parm for kernel 2
    float *g_ph0_;
    checkCudaErrors(hipMalloc((void **)&g_ph0_, sizeof(float) * 802816));
    checkCudaErrors(hipMemcpy(g_ph0_, placeholder0, sizeof(float) * 802816, hipMemcpyHostToDevice));


    float *g_ph1_;
    checkCudaErrors(hipMalloc((void **)&g_ph1_, sizeof(float) * 2359296));
    //checkCudaErrors(cudaMemcpy(g_ph1_, placeholder1, sizeof(float) * 2359296, cudaMemcpyHostToDevice));


    float *g_ph2_;
    checkCudaErrors(hipMalloc((void **)&g_ph2_, sizeof(float) * 802816));
    checkCudaErrors(hipMemcpy(g_ph2_, placeholder2, sizeof(float) * 802816, hipMemcpyHostToDevice));


    float *g_ph3_;
    hipMalloc((void **)&g_ph3_, sizeof(float) * 802816);
    hipMemcpy(g_ph3_, placeholder3, sizeof(float) * 802816, hipMemcpyHostToDevice);


    float *g_ph4_;
    hipMalloc((void **)&g_ph4_, sizeof(float) * 512);
    hipMemcpy(g_ph4_, placeholder4, sizeof(float) * 512, hipMemcpyHostToDevice);


    float *g_ph5_;
    hipMalloc((void **)&g_ph5_, sizeof(float) * 802816);
    hipMemcpy(g_ph5_, placeholder5, sizeof(float) * 802816, hipMemcpyHostToDevice);



    dim3 Dim_block = dim3(LAUNCH_BLOCKX, LAUNCH_BLOCKY, LAUNCH_BLOCKZ);
    dim3 Dim_thread = dim3(LAUNCH_THREADX, LAUNCH_THREADY, LAUNCH_THREADZ);

    printf("hello?");
    // launch kernel

    fused_nn_conv2d_add_multiply_add_nn_relu_kernel0<<<Dim_block, Dim_thread, 0, streams[1]>>>(g_worker2,2, g_flag_, g_ph0_, g_ph1_, g_ph2_, g_ph3_, g_ph4_, g_ph5_);
    fused_nn_conv2d_add_multiply_add_nn_relu_kernel0<<<Dim_block, Dim_thread, 0, streams[0]>>>(g_worker,1, g_flag, g_ph0, g_ph1, g_ph2, g_ph3, g_ph4, g_ph5);

    hipDeviceSynchronize();
    printf("hello2?");
    checkCudaErrors(hipMemcpy(placeholder2, g_ph2,sizeof(float) * 802816, hipMemcpyDeviceToHost));
    printf("hello3?\n");
    /*
    for(int j=0;j<784;j++)
    {
    if(j%10==0)
    {
    printf("\n");
    }
    printf("%f  ",placeholder2[1024*j+j]);
    }
    */

    printf("\n");
    printf("kernel 1 \n");
    checkCudaErrors(hipMemcpy(workers,g_worker,sizeof(int) * 80, hipMemcpyDeviceToHost));
    for(int j=0;j<80;j++)
    {
    if(j%10==0&&j!=0)
    {
    printf("\n");
    }
    printf("%d  ",workers[j]);
    }

    printf("\n");
    printf("kernel 2 \n");
    checkCudaErrors(hipMemcpy(workers,g_worker2,sizeof(int) * 80, hipMemcpyDeviceToHost));
    for(int j=0;j<80;j++)
    {
    if(j%10==0&&j!=0)
    {
    printf("\n");
    }
    printf("%d  ",workers[j]);
    }
    printf("\n");
}