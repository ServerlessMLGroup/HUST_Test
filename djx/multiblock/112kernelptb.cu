#include <stdio.h>
#include <stdlib.h>
#include<hip/hip_runtime.h>

#include <iostream>
#include <sys/time.h>
#include <math.h>
#include <unistd.h>

#define LAUNCH_THREADX 56
#define LAUNCH_THREADY 1
#define LAUNCH_THREADZ 2

#define LAUNCH_BLOCKX 1
#define ORI_BLOCKX 1
#define LAUNCH_BLOCKY 1
#define ORI_BLOCKY 28
#define LAUNCH_BLOCKZ 512 * 5 // 5是额外部分，满足多层覆盖
#define ORI_BLOCKZ 32

#define SM_NUM 32
#define WORKER_NUM_PERSM 4

#define BLOCK_NUM LAUNCH_BLOCKZ * LAUNCH_BLOCKY * LAUNCH_BLOCKX
#define FLAG_LENGTH 65535
#define FLAG_BLOCK_BASE 0
#define FLAG_SM_BASE (FLAG_BLOCK_BASE + 1)
#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)
// nvcc -arch=native main.cu -o main

#define GPU_RETURN_STATUS(cmd) \
{ \
    hipError_t result = cmd; \
    if (result != hipSuccess) { \
        std::cout << #cmd " error, return code:" << result << " | " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); \
    } \
}

// These are the inline versions for all of the SDK helper functions
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    const char *errorStr = NULL;
    errorStr = hipGetErrorString(err);
    fprintf(stderr,
            "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, "
            "line %i.\n",
            err, errorStr, file, line);
    exit(EXIT_FAILURE);
  }
}

__device__ uint get_smid(void) {

    uint ret;

    asm("mov.u32 %0, %smid;" : "=r"(ret) );

    return ret;

}

// #if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 700)
// #define __shfl_sync(mask, var, lane, width) \
//         __shfl((var), (lane), (width))

// #define __shfl_down_sync(mask, var, offset, width) \
//         __shfl_down((var), (offset), (width))

// #define __shfl_up_sync(mask, var, offset, width) \
//         __shfl_up((var), (offset), (width))
// #endif

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_6_kernel0(int *worker,int number,int *flag,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2){
    int* sm_flag = flag;
    __shared__ int basicoffset;
    int offset;
    int smid;
    //judge whether to continue work,which work to fetch
    if(threadIdx.x+threadIdx.y+threadIdx.z == 0)
    {
       basicoffset=-1;
       smid = get_smid();

       //judge whther sm id is right
       if((smid < number*SM_NUM)&&(smid >= (number-1)*SM_NUM))
       {
            //judge whether worker is enough
            //get the basic offset for the block
            int blocknumber=atomicAdd(sm_flag + smid, 1);
            if(blocknumber< WORKER_NUM_PERSM)
            {
                basicoffset = WORKER_NUM_PERSM*(smid-(number-1)*SM_NUM) + blocknumber;
                atomicAdd(worker + smid, 1);
                //printf("smid %d\n", smid);
            }
       }
    }
    __syncthreads();
    if (basicoffset < 0) return ;
    //every thread has its own offset
    offset = basicoffset;
    // if ((threadIdx.x + threadIdx.y + threadIdx.z) == 0 && (number == 1)) {
    //     printf("smid %d\n", smid);
    // }

    while(offset < (ORI_BLOCKX * ORI_BLOCKY * ORI_BLOCKZ)) {
        int vx = (offset)/(ORI_BLOCKY * ORI_BLOCKZ);
        int vy = (offset - (vx * ORI_BLOCKY * ORI_BLOCKZ)) / ORI_BLOCKZ;
        int vz = offset - (vx * ORI_BLOCKY * ORI_BLOCKZ) - vy * ORI_BLOCKZ;
        offset += SM_NUM * WORKER_NUM_PERSM;
    }
}

int main(int argc, char *argv[]) {
    // init device
    if (argc < 2) {
        printf("args num error! argc:%d", argc);
        exit(1);
    }
    int gpu_no = atoi(argv[1]);
    checkCudaErrors(hipSetDevice(gpu_no));

    // allocate stream
    int num_streams = 2;
    hipStream_t streams[num_streams];
    for (int i = 0; i < num_streams; i++) {
        checkCudaErrors(hipStreamCreate(&streams[i]));
    }


    // allocate flag
    int *flag = new int[FLAG_LENGTH];
    int *g_flag;
    for (int i = 0; i < FLAG_LENGTH; ++i) {
        flag[i] = 0;
    }
    checkCudaErrors(hipMalloc((void **)&g_flag, sizeof(int) * FLAG_LENGTH));
    checkCudaErrors(hipMemcpy(g_flag, flag, sizeof(int) * FLAG_LENGTH, hipMemcpyHostToDevice));

    int *flag_ = new int[FLAG_LENGTH];
    int *g_flag_;
    for (int i = 0; i < FLAG_LENGTH; ++i) {
        flag_[i] = 0;
    }
    checkCudaErrors(hipMalloc((void **)&g_flag_, sizeof(int) * FLAG_LENGTH));
    checkCudaErrors(hipMemcpy(g_flag_, flag_, sizeof(int) * FLAG_LENGTH, hipMemcpyHostToDevice));

    //prepare parm for kernel 1
    int *workers = new int[80];
    for(int i=0;i<80;i++)
    {
    workers[i]=0;
    }
    int *g_worker;
    checkCudaErrors(hipMalloc((void **)&g_worker, sizeof(int) * 80));
    checkCudaErrors(hipMemcpy( g_worker,workers, sizeof(int) * 80, hipMemcpyHostToDevice));


    float *placeholder0 = new float[6422528];
    for(int i=0;i<6422528;i++)
    {
    placeholder0[i]=1;
    }
    float *g_ph0;
    checkCudaErrors(hipMalloc((void **)&g_ph0, sizeof(float) * 6422528));
    checkCudaErrors(hipMemcpy(g_ph0, placeholder0, sizeof(float) * 6422528, hipMemcpyHostToDevice));

    float *placeholder1 = new float[36864];
    for(int i=0;i<36864;i++)
    {
    placeholder1[i]=0;
    }
    float *g_ph1;
    checkCudaErrors(hipMalloc((void **)&g_ph1, sizeof(float) * 36864));
    checkCudaErrors(hipMemcpy(g_ph1, placeholder1, sizeof(float) * 36864, hipMemcpyHostToDevice));

    float *placeholder2 = new float[6422528];
    for(int i=0;i<6422528;i++)
    {
    placeholder2[i]=3;
    }
    float *g_ph2;
    checkCudaErrors(hipMalloc((void **)&g_ph2, sizeof(float) * 6422528));
    checkCudaErrors(hipMemcpy(g_ph2, placeholder2, sizeof(float) * 6422528, hipMemcpyHostToDevice));


    float *placeholder3 = new float[64];
    for(int i=0;i<64;i++)
    {
    placeholder3[i]=4;
    }
    float *g_ph3;
    hipMalloc((void **)&g_ph3, sizeof(float) * 64);
    hipMemcpy(g_ph3, placeholder3, sizeof(float) * 64, hipMemcpyHostToDevice);


    dim3 Dim_block = dim3(LAUNCH_BLOCKX, LAUNCH_BLOCKY, LAUNCH_BLOCKZ);
    dim3 Dim_thread = dim3(LAUNCH_THREADX, LAUNCH_THREADY, LAUNCH_THREADZ);

    printf("hello?");
    // launch kernel
    fused_nn_conv2d_add_nn_relu_6_kernel0<<<Dim_block, Dim_thread, 0, streams[0]>>>(g_worker,1, g_flag, g_ph0, g_ph1, g_ph2, g_ph3);

    hipDeviceSynchronize();

    checkCudaErrors(hipMemcpy(placeholder2, g_ph2,sizeof(float) * 6422528, hipMemcpyDeviceToHost));

    printf("hello3?\n");
    for(int j=0;j<784;j++)
    {
    if(j%10==0)
    {
    printf("\n");
    }
    printf("%f  ",placeholder2[1024*j+j]);
    }

    printf("\n");
    checkCudaErrors(hipMemcpy(workers,g_worker,sizeof(int) * 80, hipMemcpyDeviceToHost));
    for(int j=0;j<80;j++)
    {
    if(j%10==0&&j!=0)
    {
    printf("\n");
    }
    printf("%d  ",workers[j]);
    }
    printf("\n");
}