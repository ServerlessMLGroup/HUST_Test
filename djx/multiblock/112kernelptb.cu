#include <stdio.h>
#include <stdlib.h>
#include<hip/hip_runtime.h>

#include <iostream>
#include <sys/time.h>
#include <math.h>
#include <unistd.h>

#define LAUNCH_THREADX 56
#define LAUNCH_THREADY 1
#define LAUNCH_THREADZ 2

#define LAUNCH_BLOCKX 1
#define ORI_BLOCKX 1
#define LAUNCH_BLOCKY 1
#define ORI_BLOCKY 28
#define LAUNCH_BLOCKZ 512 * 5 // 5是额外部分，满足多层覆盖
#define ORI_BLOCKZ 32

#define SM_NUM 32
#define WORKER_NUM_PERSM 1

#define BLOCK_NUM LAUNCH_BLOCKZ * LAUNCH_BLOCKY * LAUNCH_BLOCKX
#define FLAG_LENGTH 65535
#define FLAG_BLOCK_BASE 0
#define FLAG_SM_BASE (FLAG_BLOCK_BASE + 1)
#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)
// nvcc -arch=native main.cu -o main

#define GPU_RETURN_STATUS(cmd) \
{ \
    hipError_t result = cmd; \
    if (result != hipSuccess) { \
        std::cout << #cmd " error, return code:" << result << " | " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); \
    } \
}

// These are the inline versions for all of the SDK helper functions
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    const char *errorStr = NULL;
    errorStr = hipGetErrorString(err);
    fprintf(stderr,
            "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, "
            "line %i.\n",
            err, errorStr, file, line);
    exit(EXIT_FAILURE);
  }
}

__device__ uint get_smid(void) {

    uint ret;

    asm("mov.u32 %0, %smid;" : "=r"(ret) );

    return ret;

}

// #if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 700)
// #define __shfl_sync(mask, var, lane, width) \
//         __shfl((var), (lane), (width))

// #define __shfl_down_sync(mask, var, offset, width) \
//         __shfl_down((var), (offset), (width))

// #define __shfl_up_sync(mask, var, offset, width) \
//         __shfl_up((var), (offset), (width))
// #endif

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_6_kernel0(int *worker,int number,int *flag,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2){
    int* sm_flag = flag;
    __shared__ int basicoffset;
    int offset;
    int smid;
    //judge whether to continue work,which work to fetch
    if(threadIdx.x+threadIdx.y+threadIdx.z == 0)
    {
       basicoffset=-1;
       smid = get_smid();

       //judge whther sm id is right
       if((smid < number*SM_NUM)&&(smid >= (number-1)*SM_NUM))
       {
            //judge whether worker is enough
            //get the basic offset for the block
            int blocknumber=atomicAdd(sm_flag + smid, 1);
            if(blocknumber< WORKER_NUM_PERSM)
            {
                basicoffset = WORKER_NUM_PERSM*(smid-(number-1)*SM_NUM) + blocknumber;
                atomicAdd(worker + smid, 1);
                //printf("smid %d\n", smid);
            }
       }
    }
    __syncthreads();
    if (basicoffset < 0) return ;
    //every thread has its own offset
    offset = basicoffset;
    // if ((threadIdx.x + threadIdx.y + threadIdx.z) == 0 && (number == 1)) {
    //     printf("smid %d\n", smid);
    // }

    while(offset < (ORI_BLOCKX * ORI_BLOCKY * ORI_BLOCKZ)) {
        int vx = (offset)/(ORI_BLOCKY * ORI_BLOCKZ);
        int vy = (offset - (vx * ORI_BLOCKY * ORI_BLOCKZ)) / ORI_BLOCKZ;
        int vz = offset - (vx * ORI_BLOCKY * ORI_BLOCKZ) - vy * ORI_BLOCKZ;
        offset += SM_NUM * WORKER_NUM_PERSM;
        float compute[64];
  __shared__ float pad_temp_shared[448];
  __shared__ float placeholder_shared[256];
  compute[(0)] = 0.000000e+00f;
  compute[(2)] = 0.000000e+00f;
  compute[(4)] = 0.000000e+00f;
  compute[(6)] = 0.000000e+00f;
  compute[(8)] = 0.000000e+00f;
  compute[(10)] = 0.000000e+00f;
  compute[(12)] = 0.000000e+00f;
  compute[(14)] = 0.000000e+00f;
  compute[(16)] = 0.000000e+00f;
  compute[(18)] = 0.000000e+00f;
  compute[(20)] = 0.000000e+00f;
  compute[(22)] = 0.000000e+00f;
  compute[(24)] = 0.000000e+00f;
  compute[(26)] = 0.000000e+00f;
  compute[(28)] = 0.000000e+00f;
  compute[(30)] = 0.000000e+00f;
  compute[(32)] = 0.000000e+00f;
  compute[(34)] = 0.000000e+00f;
  compute[(36)] = 0.000000e+00f;
  compute[(38)] = 0.000000e+00f;
  compute[(40)] = 0.000000e+00f;
  compute[(42)] = 0.000000e+00f;
  compute[(44)] = 0.000000e+00f;
  compute[(46)] = 0.000000e+00f;
  compute[(48)] = 0.000000e+00f;
  compute[(50)] = 0.000000e+00f;
  compute[(52)] = 0.000000e+00f;
  compute[(54)] = 0.000000e+00f;
  compute[(56)] = 0.000000e+00f;
  compute[(58)] = 0.000000e+00f;
  compute[(60)] = 0.000000e+00f;
  compute[(62)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  compute[(3)] = 0.000000e+00f;
  compute[(5)] = 0.000000e+00f;
  compute[(7)] = 0.000000e+00f;
  compute[(9)] = 0.000000e+00f;
  compute[(11)] = 0.000000e+00f;
  compute[(13)] = 0.000000e+00f;
  compute[(15)] = 0.000000e+00f;
  compute[(17)] = 0.000000e+00f;
  compute[(19)] = 0.000000e+00f;
  compute[(21)] = 0.000000e+00f;
  compute[(23)] = 0.000000e+00f;
  compute[(25)] = 0.000000e+00f;
  compute[(27)] = 0.000000e+00f;
  compute[(29)] = 0.000000e+00f;
  compute[(31)] = 0.000000e+00f;
  compute[(33)] = 0.000000e+00f;
  compute[(35)] = 0.000000e+00f;
  compute[(37)] = 0.000000e+00f;
  compute[(39)] = 0.000000e+00f;
  compute[(41)] = 0.000000e+00f;
  compute[(43)] = 0.000000e+00f;
  compute[(45)] = 0.000000e+00f;
  compute[(47)] = 0.000000e+00f;
  compute[(49)] = 0.000000e+00f;
  compute[(51)] = 0.000000e+00f;
  compute[(53)] = 0.000000e+00f;
  compute[(55)] = 0.000000e+00f;
  compute[(57)] = 0.000000e+00f;
  compute[(59)] = 0.000000e+00f;
  compute[(61)] = 0.000000e+00f;
  compute[(63)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 16; ++rc_outer) {
    for (int ry_outer = 0; ry_outer < 3; ++ry_outer) {
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 224) + (((int)threadIdx.x) * 4)))] = ((((1 <= (((((int)vy) * 2) + ((((int)threadIdx.x) % 28) / 14)) + ry_outer)) && ((((((int)vy) * 2) + ((((int)threadIdx.x) % 28) / 14)) + ry_outer) < 57)) && (1 <= (((int)threadIdx.x) % 14))) ? placeholder[(((((((((((int)vz) * 200704) + (rc_outer * 12544)) + (((int)threadIdx.z) * 6272)) + ((((int)threadIdx.x) / 28) * 3136)) + (((int)vy) * 112)) + (ry_outer * 56)) + ((((int)threadIdx.x) % 28) * 4)) - 57))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 224) + (((int)threadIdx.x) * 4)) + 1))] = ((((1 <= (((((int)vy) * 2) + ((((((int)threadIdx.x) * 4) + 1) % 112) / 56)) + ry_outer)) && ((((((int)vy) * 2) + ((((((int)threadIdx.x) * 4) + 1) % 112) / 56)) + ry_outer) < 57)) && (1 <= (((((int)threadIdx.x) * 4) + 1) % 56))) ? placeholder[(((((((((((int)vz) * 200704) + (rc_outer * 12544)) + (((int)threadIdx.z) * 6272)) + ((((((int)threadIdx.x) * 4) + 1) / 112) * 3136)) + (((int)vy) * 112)) + (ry_outer * 56)) + (((((int)threadIdx.x) * 4) + 1) % 112)) - 57))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 224) + (((int)threadIdx.x) * 4)) + 2))] = ((((1 <= (((((int)vy) * 2) + ((((((int)threadIdx.x) * 4) + 2) % 112) / 56)) + ry_outer)) && ((((((int)vy) * 2) + ((((((int)threadIdx.x) * 4) + 2) % 112) / 56)) + ry_outer) < 57)) && (1 <= (((((int)threadIdx.x) * 4) + 2) % 56))) ? placeholder[(((((((((((int)vz) * 200704) + (rc_outer * 12544)) + (((int)threadIdx.z) * 6272)) + ((((((int)threadIdx.x) * 4) + 2) / 112) * 3136)) + (((int)vy) * 112)) + (ry_outer * 56)) + (((((int)threadIdx.x) * 4) + 2) % 112)) - 57))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 224) + (((int)threadIdx.x) * 4)) + 3))] = ((((1 <= (((((int)vy) * 2) + ((((((int)threadIdx.x) * 4) + 3) % 112) / 56)) + ry_outer)) && ((((((int)vy) * 2) + ((((((int)threadIdx.x) * 4) + 3) % 112) / 56)) + ry_outer) < 57)) && (1 <= (((((int)threadIdx.x) * 4) + 3) % 56))) ? placeholder[(((((((((((int)vz) * 200704) + (rc_outer * 12544)) + (((int)threadIdx.z) * 6272)) + ((((((int)threadIdx.x) * 4) + 3) / 112) * 3136)) + (((int)vy) * 112)) + (ry_outer * 56)) + (((((int)threadIdx.x) * 4) + 3) % 112)) - 57))] : 0.000000e+00f);
      if (((((int)threadIdx.z) * 32) + ((((int)threadIdx.x) * 3) >> 2)) < 64) {
        if (((((int)threadIdx.z) * 128) + (((int)threadIdx.x) * 3)) < 256) {
          if (((int)threadIdx.x) < 43) {
            placeholder_shared[(((((int)threadIdx.z) * 128) + (((int)threadIdx.x) * 3)))] = placeholder1[((((((((int)threadIdx.z) * 18432) + (((((int)threadIdx.x) * 3) >> 2) * 576)) + (rc_outer * 36)) + (((((int)threadIdx.x) * 3) & 3) * 9)) + (ry_outer * 3)))];
          }
        }
      }
      if (((((int)threadIdx.z) * 32) + (((((int)threadIdx.x) * 3) + 1) >> 2)) < 64) {
        if (((((int)threadIdx.z) * 128) + (((int)threadIdx.x) * 3)) < 255) {
          if (((int)threadIdx.x) < 43) {
            placeholder_shared[((((((int)threadIdx.z) * 128) + (((int)threadIdx.x) * 3)) + 1))] = placeholder1[((((((((int)threadIdx.z) * 18432) + ((((((int)threadIdx.x) * 3) + 1) >> 2) * 576)) + (rc_outer * 36)) + ((((((int)threadIdx.x) * 3) + 1) & 3) * 9)) + (ry_outer * 3)))];
          }
        }
      }
      if (((((int)threadIdx.z) * 32) + (((((int)threadIdx.x) * 3) + 2) >> 2)) < 64) {
        if (((((int)threadIdx.z) * 128) + (((int)threadIdx.x) * 3)) < 254) {
          if (((int)threadIdx.x) < 42) {
            placeholder_shared[((((((int)threadIdx.z) * 128) + (((int)threadIdx.x) * 3)) + 2))] = placeholder1[((((((((int)threadIdx.z) * 18432) + ((((((int)threadIdx.x) * 3) + 2) >> 2) * 576)) + (rc_outer * 36)) + ((((((int)threadIdx.x) * 3) + 2) & 3) * 9)) + (ry_outer * 3)))];
          }
        }
      }
      __syncthreads();
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 4))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 8))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 16))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 24))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 32))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 40))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 48))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 56))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 64))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 72))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 80))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 88))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 96))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 104))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 112))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 120))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 128))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 136))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 144))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 152))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 160))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 168))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 176))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 184))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 192))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 200))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 208))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 216))]));
      compute[(56)] = (compute[(56)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 224))]));
      compute[(58)] = (compute[(58)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 232))]));
      compute[(60)] = (compute[(60)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 240))]));
      compute[(62)] = (compute[(62)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 248))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[((((int)threadIdx.z) * 4))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 8))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 16))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 24))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 32))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 40))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 48))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 56))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 64))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 72))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 80))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 88))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 96))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 104))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 112))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 120))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 128))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 136))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 144))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 152))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 160))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 168))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 176))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 184))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 192))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 200))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 208))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 216))]));
      compute[(57)] = (compute[(57)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 224))]));
      compute[(59)] = (compute[(59)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 232))]));
      compute[(61)] = (compute[(61)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 240))]));
      compute[(63)] = (compute[(63)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 248))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 1))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 9))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 17))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 25))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 33))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 41))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 49))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 57))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 65))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 73))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 81))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 89))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 97))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 105))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 113))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 121))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 129))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 137))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 145))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 153))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 161))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 169))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 177))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 185))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 193))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 201))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 209))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 217))]));
      compute[(56)] = (compute[(56)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 225))]));
      compute[(58)] = (compute[(58)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 233))]));
      compute[(60)] = (compute[(60)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 241))]));
      compute[(62)] = (compute[(62)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 249))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 1))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 9))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 17))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 25))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 33))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 41))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 49))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 57))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 65))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 73))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 81))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 89))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 97))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 105))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 113))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 121))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 129))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 137))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 145))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 153))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 161))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 169))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 177))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 185))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 193))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 201))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 209))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 217))]));
      compute[(57)] = (compute[(57)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 225))]));
      compute[(59)] = (compute[(59)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 233))]));
      compute[(61)] = (compute[(61)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 241))]));
      compute[(63)] = (compute[(63)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 249))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 2))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 10))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 18))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 26))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 34))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 42))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 50))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 58))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 66))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 74))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 82))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 90))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 98))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 106))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 114))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 122))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 130))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 138))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 146))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 154))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 162))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 170))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 178))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 186))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 194))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 202))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 210))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 218))]));
      compute[(56)] = (compute[(56)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 226))]));
      compute[(58)] = (compute[(58)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 234))]));
      compute[(60)] = (compute[(60)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 242))]));
      compute[(62)] = (compute[(62)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 250))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 2))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 10))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 18))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 26))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 34))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 42))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 50))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 58))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 66))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 74))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 82))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 90))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 98))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 106))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 114))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 122))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 130))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 138))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 146))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 154))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 162))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 170))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 178))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 186))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 194))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 202))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 210))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 218))]));
      compute[(57)] = (compute[(57)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 226))]));
      compute[(59)] = (compute[(59)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 234))]));
      compute[(61)] = (compute[(61)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 242))]));
      compute[(63)] = (compute[(63)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 250))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 3))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 11))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 19))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 27))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 35))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 43))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 51))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 59))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 67))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 75))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 83))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 91))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 99))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 107))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 115))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 123))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 131))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 139))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 147))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 155))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 163))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 171))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 179))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 187))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 195))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 203))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 211))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 219))]));
      compute[(56)] = (compute[(56)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 227))]));
      compute[(58)] = (compute[(58)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 235))]));
      compute[(60)] = (compute[(60)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 243))]));
      compute[(62)] = (compute[(62)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 251))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 3))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 11))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 19))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 27))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 35))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 43))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 51))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 59))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 67))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 75))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 83))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 91))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 99))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 107))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 115))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 123))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 131))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 139))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 147))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 155))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 163))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 171))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 179))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 187))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 195))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 203))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 211))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 219))]));
      compute[(57)] = (compute[(57)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 227))]));
      compute[(59)] = (compute[(59)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 235))]));
      compute[(61)] = (compute[(61)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 243))]));
      compute[(63)] = (compute[(63)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 251))]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 224) + (((int)threadIdx.x) * 4)))] = (((1 <= (((((int)vy) * 2) + ((((int)threadIdx.x) % 28) / 14)) + ry_outer)) && ((((((int)vy) * 2) + ((((int)threadIdx.x) % 28) / 14)) + ry_outer) < 57)) ? placeholder[(((((((((((int)vz) * 200704) + (rc_outer * 12544)) + (((int)threadIdx.z) * 6272)) + ((((int)threadIdx.x) / 28) * 3136)) + (((int)vy) * 112)) + (ry_outer * 56)) + ((((int)threadIdx.x) % 28) * 4)) - 56))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 224) + (((int)threadIdx.x) * 4)) + 1))] = (((1 <= (((((int)vy) * 2) + ((((((int)threadIdx.x) * 4) + 1) % 112) / 56)) + ry_outer)) && ((((((int)vy) * 2) + ((((((int)threadIdx.x) * 4) + 1) % 112) / 56)) + ry_outer) < 57)) ? placeholder[(((((((((((int)vz) * 200704) + (rc_outer * 12544)) + (((int)threadIdx.z) * 6272)) + ((((((int)threadIdx.x) * 4) + 1) / 112) * 3136)) + (((int)vy) * 112)) + (ry_outer * 56)) + (((((int)threadIdx.x) * 4) + 1) % 112)) - 56))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 224) + (((int)threadIdx.x) * 4)) + 2))] = (((1 <= (((((int)vy) * 2) + ((((((int)threadIdx.x) * 4) + 2) % 112) / 56)) + ry_outer)) && ((((((int)vy) * 2) + ((((((int)threadIdx.x) * 4) + 2) % 112) / 56)) + ry_outer) < 57)) ? placeholder[(((((((((((int)vz) * 200704) + (rc_outer * 12544)) + (((int)threadIdx.z) * 6272)) + ((((((int)threadIdx.x) * 4) + 2) / 112) * 3136)) + (((int)vy) * 112)) + (ry_outer * 56)) + (((((int)threadIdx.x) * 4) + 2) % 112)) - 56))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 224) + (((int)threadIdx.x) * 4)) + 3))] = (((1 <= (((((int)vy) * 2) + ((((((int)threadIdx.x) * 4) + 3) % 112) / 56)) + ry_outer)) && ((((((int)vy) * 2) + ((((((int)threadIdx.x) * 4) + 3) % 112) / 56)) + ry_outer) < 57)) ? placeholder[(((((((((((int)vz) * 200704) + (rc_outer * 12544)) + (((int)threadIdx.z) * 6272)) + ((((((int)threadIdx.x) * 4) + 3) / 112) * 3136)) + (((int)vy) * 112)) + (ry_outer * 56)) + (((((int)threadIdx.x) * 4) + 3) % 112)) - 56))] : 0.000000e+00f);
      if (((((int)threadIdx.z) * 32) + ((((int)threadIdx.x) * 3) >> 2)) < 64) {
        if (((((int)threadIdx.z) * 128) + (((int)threadIdx.x) * 3)) < 256) {
          if (((int)threadIdx.x) < 43) {
            placeholder_shared[(((((int)threadIdx.z) * 128) + (((int)threadIdx.x) * 3)))] = placeholder1[(((((((((int)threadIdx.z) * 18432) + (((((int)threadIdx.x) * 3) >> 2) * 576)) + (rc_outer * 36)) + (((((int)threadIdx.x) * 3) & 3) * 9)) + (ry_outer * 3)) + 1))];
          }
        }
      }
      if (((((int)threadIdx.z) * 32) + (((((int)threadIdx.x) * 3) + 1) >> 2)) < 64) {
        if (((((int)threadIdx.z) * 128) + (((int)threadIdx.x) * 3)) < 255) {
          if (((int)threadIdx.x) < 43) {
            placeholder_shared[((((((int)threadIdx.z) * 128) + (((int)threadIdx.x) * 3)) + 1))] = placeholder1[(((((((((int)threadIdx.z) * 18432) + ((((((int)threadIdx.x) * 3) + 1) >> 2) * 576)) + (rc_outer * 36)) + ((((((int)threadIdx.x) * 3) + 1) & 3) * 9)) + (ry_outer * 3)) + 1))];
          }
        }
      }
      if (((((int)threadIdx.z) * 32) + (((((int)threadIdx.x) * 3) + 2) >> 2)) < 64) {
        if (((((int)threadIdx.z) * 128) + (((int)threadIdx.x) * 3)) < 254) {
          if (((int)threadIdx.x) < 42) {
            placeholder_shared[((((((int)threadIdx.z) * 128) + (((int)threadIdx.x) * 3)) + 2))] = placeholder1[(((((((((int)threadIdx.z) * 18432) + ((((((int)threadIdx.x) * 3) + 2) >> 2) * 576)) + (rc_outer * 36)) + ((((((int)threadIdx.x) * 3) + 2) & 3) * 9)) + (ry_outer * 3)) + 1))];
          }
        }
      }
      __syncthreads();
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 4))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 8))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 16))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 24))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 32))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 40))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 48))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 56))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 64))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 72))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 80))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 88))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 96))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 104))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 112))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 120))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 128))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 136))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 144))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 152))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 160))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 168))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 176))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 184))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 192))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 200))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 208))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 216))]));
      compute[(56)] = (compute[(56)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 224))]));
      compute[(58)] = (compute[(58)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 232))]));
      compute[(60)] = (compute[(60)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 240))]));
      compute[(62)] = (compute[(62)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 248))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[((((int)threadIdx.z) * 4))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 8))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 16))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 24))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 32))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 40))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 48))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 56))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 64))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 72))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 80))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 88))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 96))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 104))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 112))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 120))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 128))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 136))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 144))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 152))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 160))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 168))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 176))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 184))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 192))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 200))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 208))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 216))]));
      compute[(57)] = (compute[(57)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 224))]));
      compute[(59)] = (compute[(59)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 232))]));
      compute[(61)] = (compute[(61)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 240))]));
      compute[(63)] = (compute[(63)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 248))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 1))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 9))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 17))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 25))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 33))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 41))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 49))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 57))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 65))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 73))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 81))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 89))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 97))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 105))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 113))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 121))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 129))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 137))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 145))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 153))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 161))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 169))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 177))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 185))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 193))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 201))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 209))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 217))]));
      compute[(56)] = (compute[(56)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 225))]));
      compute[(58)] = (compute[(58)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 233))]));
      compute[(60)] = (compute[(60)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 241))]));
      compute[(62)] = (compute[(62)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 249))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 1))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 9))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 17))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 25))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 33))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 41))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 49))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 57))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 65))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 73))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 81))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 89))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 97))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 105))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 113))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 121))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 129))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 137))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 145))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 153))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 161))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 169))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 177))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 185))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 193))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 201))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 209))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 217))]));
      compute[(57)] = (compute[(57)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 225))]));
      compute[(59)] = (compute[(59)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 233))]));
      compute[(61)] = (compute[(61)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 241))]));
      compute[(63)] = (compute[(63)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 249))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 2))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 10))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 18))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 26))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 34))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 42))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 50))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 58))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 66))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 74))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 82))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 90))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 98))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 106))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 114))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 122))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 130))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 138))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 146))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 154))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 162))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 170))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 178))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 186))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 194))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 202))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 210))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 218))]));
      compute[(56)] = (compute[(56)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 226))]));
      compute[(58)] = (compute[(58)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 234))]));
      compute[(60)] = (compute[(60)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 242))]));
      compute[(62)] = (compute[(62)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 250))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 2))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 10))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 18))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 26))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 34))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 42))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 50))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 58))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 66))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 74))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 82))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 90))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 98))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 106))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 114))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 122))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 130))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 138))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 146))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 154))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 162))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 170))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 178))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 186))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 194))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 202))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 210))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 218))]));
      compute[(57)] = (compute[(57)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 226))]));
      compute[(59)] = (compute[(59)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 234))]));
      compute[(61)] = (compute[(61)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 242))]));
      compute[(63)] = (compute[(63)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 250))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 3))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 11))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 19))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 27))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 35))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 43))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 51))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 59))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 67))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 75))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 83))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 91))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 99))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 107))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 115))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 123))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 131))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 139))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 147))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 155))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 163))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 171))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 179))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 187))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 195))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 203))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 211))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 219))]));
      compute[(56)] = (compute[(56)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 227))]));
      compute[(58)] = (compute[(58)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 235))]));
      compute[(60)] = (compute[(60)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 243))]));
      compute[(62)] = (compute[(62)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 251))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 3))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 11))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 19))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 27))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 35))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 43))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 51))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 59))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 67))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 75))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 83))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 91))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 99))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 107))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 115))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 123))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 131))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 139))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 147))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 155))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 163))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 171))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 179))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 187))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 195))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 203))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 211))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 219))]));
      compute[(57)] = (compute[(57)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 227))]));
      compute[(59)] = (compute[(59)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 235))]));
      compute[(61)] = (compute[(61)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 243))]));
      compute[(63)] = (compute[(63)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 251))]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 224) + (((int)threadIdx.x) * 4)))] = (((1 <= (((((int)vy) * 2) + ((((int)threadIdx.x) % 28) / 14)) + ry_outer)) && ((((((int)vy) * 2) + ((((int)threadIdx.x) % 28) / 14)) + ry_outer) < 57)) ? placeholder[(((((((((((int)vz) * 200704) + (rc_outer * 12544)) + (((int)threadIdx.z) * 6272)) + ((((int)threadIdx.x) / 28) * 3136)) + (((int)vy) * 112)) + (ry_outer * 56)) + ((((int)threadIdx.x) % 28) * 4)) - 55))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 224) + (((int)threadIdx.x) * 4)) + 1))] = ((((1 <= (((((int)vy) * 2) + ((((((int)threadIdx.x) * 4) + 1) % 112) / 56)) + ry_outer)) && ((((((int)vy) * 2) + ((((((int)threadIdx.x) * 4) + 1) % 112) / 56)) + ry_outer) < 57)) && ((((((int)threadIdx.x) * 4) + 1) % 56) < 55)) ? placeholder[(((((((((((int)vz) * 200704) + (rc_outer * 12544)) + (((int)threadIdx.z) * 6272)) + ((((((int)threadIdx.x) * 4) + 1) / 112) * 3136)) + (((int)vy) * 112)) + (ry_outer * 56)) + (((((int)threadIdx.x) * 4) + 1) % 112)) - 55))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 224) + (((int)threadIdx.x) * 4)) + 2))] = ((((1 <= (((((int)vy) * 2) + ((((((int)threadIdx.x) * 4) + 2) % 112) / 56)) + ry_outer)) && ((((((int)vy) * 2) + ((((((int)threadIdx.x) * 4) + 2) % 112) / 56)) + ry_outer) < 57)) && ((((((int)threadIdx.x) * 4) + 2) % 56) < 55)) ? placeholder[(((((((((((int)vz) * 200704) + (rc_outer * 12544)) + (((int)threadIdx.z) * 6272)) + ((((((int)threadIdx.x) * 4) + 2) / 112) * 3136)) + (((int)vy) * 112)) + (ry_outer * 56)) + (((((int)threadIdx.x) * 4) + 2) % 112)) - 55))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 224) + (((int)threadIdx.x) * 4)) + 3))] = ((((1 <= (((((int)vy) * 2) + ((((((int)threadIdx.x) * 4) + 3) % 112) / 56)) + ry_outer)) && ((((((int)vy) * 2) + ((((((int)threadIdx.x) * 4) + 3) % 112) / 56)) + ry_outer) < 57)) && ((((((int)threadIdx.x) * 4) + 3) % 56) < 55)) ? placeholder[(((((((((((int)vz) * 200704) + (rc_outer * 12544)) + (((int)threadIdx.z) * 6272)) + ((((((int)threadIdx.x) * 4) + 3) / 112) * 3136)) + (((int)vy) * 112)) + (ry_outer * 56)) + (((((int)threadIdx.x) * 4) + 3) % 112)) - 55))] : 0.000000e+00f);
      if (((((int)threadIdx.z) * 32) + ((((int)threadIdx.x) * 3) >> 2)) < 64) {
        if (((((int)threadIdx.z) * 128) + (((int)threadIdx.x) * 3)) < 256) {
          if (((int)threadIdx.x) < 43) {
            placeholder_shared[(((((int)threadIdx.z) * 128) + (((int)threadIdx.x) * 3)))] = placeholder1[(((((((((int)threadIdx.z) * 18432) + (((((int)threadIdx.x) * 3) >> 2) * 576)) + (rc_outer * 36)) + (((((int)threadIdx.x) * 3) & 3) * 9)) + (ry_outer * 3)) + 2))];
          }
        }
      }
      if (((((int)threadIdx.z) * 32) + (((((int)threadIdx.x) * 3) + 1) >> 2)) < 64) {
        if (((((int)threadIdx.z) * 128) + (((int)threadIdx.x) * 3)) < 255) {
          if (((int)threadIdx.x) < 43) {
            placeholder_shared[((((((int)threadIdx.z) * 128) + (((int)threadIdx.x) * 3)) + 1))] = placeholder1[(((((((((int)threadIdx.z) * 18432) + ((((((int)threadIdx.x) * 3) + 1) >> 2) * 576)) + (rc_outer * 36)) + ((((((int)threadIdx.x) * 3) + 1) & 3) * 9)) + (ry_outer * 3)) + 2))];
          }
        }
      }
      if (((((int)threadIdx.z) * 32) + (((((int)threadIdx.x) * 3) + 2) >> 2)) < 64) {
        if (((((int)threadIdx.z) * 128) + (((int)threadIdx.x) * 3)) < 254) {
          if (((int)threadIdx.x) < 42) {
            placeholder_shared[((((((int)threadIdx.z) * 128) + (((int)threadIdx.x) * 3)) + 2))] = placeholder1[(((((((((int)threadIdx.z) * 18432) + ((((((int)threadIdx.x) * 3) + 2) >> 2) * 576)) + (rc_outer * 36)) + ((((((int)threadIdx.x) * 3) + 2) & 3) * 9)) + (ry_outer * 3)) + 2))];
          }
        }
      }
      __syncthreads();
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 4))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 8))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 16))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 24))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 32))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 40))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 48))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 56))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 64))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 72))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 80))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 88))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 96))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 104))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 112))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 120))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 128))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 136))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 144))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 152))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 160))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 168))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 176))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 184))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 192))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 200))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 208))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 216))]));
      compute[(56)] = (compute[(56)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 224))]));
      compute[(58)] = (compute[(58)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 232))]));
      compute[(60)] = (compute[(60)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 240))]));
      compute[(62)] = (compute[(62)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 248))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[((((int)threadIdx.z) * 4))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 8))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 16))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 24))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 32))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 40))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 48))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 56))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 64))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 72))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 80))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 88))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 96))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 104))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 112))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 120))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 128))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 136))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 144))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 152))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 160))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 168))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 176))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 184))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 192))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 200))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 208))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 216))]));
      compute[(57)] = (compute[(57)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 224))]));
      compute[(59)] = (compute[(59)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 232))]));
      compute[(61)] = (compute[(61)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 240))]));
      compute[(63)] = (compute[(63)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 248))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 1))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 9))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 17))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 25))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 33))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 41))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 49))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 57))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 65))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 73))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 81))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 89))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 97))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 105))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 113))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 121))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 129))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 137))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 145))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 153))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 161))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 169))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 177))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 185))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 193))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 201))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 209))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 217))]));
      compute[(56)] = (compute[(56)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 225))]));
      compute[(58)] = (compute[(58)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 233))]));
      compute[(60)] = (compute[(60)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 241))]));
      compute[(62)] = (compute[(62)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 249))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 1))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 9))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 17))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 25))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 33))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 41))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 49))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 57))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 65))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 73))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 81))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 89))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 97))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 105))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 113))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 121))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 129))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 137))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 145))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 153))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 161))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 169))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 177))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 185))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 193))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 201))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 209))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 217))]));
      compute[(57)] = (compute[(57)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 225))]));
      compute[(59)] = (compute[(59)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 233))]));
      compute[(61)] = (compute[(61)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 241))]));
      compute[(63)] = (compute[(63)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 249))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 2))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 10))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 18))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 26))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 34))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 42))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 50))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 58))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 66))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 74))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 82))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 90))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 98))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 106))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 114))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 122))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 130))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 138))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 146))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 154))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 162))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 170))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 178))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 186))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 194))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 202))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 210))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 218))]));
      compute[(56)] = (compute[(56)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 226))]));
      compute[(58)] = (compute[(58)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 234))]));
      compute[(60)] = (compute[(60)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 242))]));
      compute[(62)] = (compute[(62)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 250))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 2))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 10))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 18))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 26))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 34))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 42))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 50))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 58))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 66))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 74))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 82))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 90))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 98))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 106))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 114))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 122))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 130))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 138))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 146))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 154))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 162))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 170))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 178))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 186))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 194))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 202))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 210))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 218))]));
      compute[(57)] = (compute[(57)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 226))]));
      compute[(59)] = (compute[(59)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 234))]));
      compute[(61)] = (compute[(61)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 242))]));
      compute[(63)] = (compute[(63)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 250))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 3))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 11))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 19))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 27))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 35))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 43))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 51))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 59))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 67))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 75))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 83))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 91))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 99))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 107))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 115))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 123))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 131))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 139))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 147))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 155))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 163))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 171))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 179))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 187))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 195))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 203))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 211))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 219))]));
      compute[(56)] = (compute[(56)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 227))]));
      compute[(58)] = (compute[(58)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 235))]));
      compute[(60)] = (compute[(60)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 243))]));
      compute[(62)] = (compute[(62)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 251))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 3))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 11))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 19))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 27))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 35))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 43))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 51))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 59))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 67))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 75))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 83))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 91))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 99))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 107))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 115))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 123))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 131))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 139))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 147))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 155))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 163))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 171))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 179))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 187))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 195))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 203))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 211))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 219))]));
      compute[(57)] = (compute[(57)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 227))]));
      compute[(59)] = (compute[(59)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 235))]));
      compute[(61)] = (compute[(61)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 243))]));
      compute[(63)] = (compute[(63)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 4) + 251))]));
    }
  }
  T_relu[(((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)))] = max((compute[(0)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 6272))] = max((compute[(2)] + placeholder2[((((int)threadIdx.z) + 2))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 12544))] = max((compute[(4)] + placeholder2[((((int)threadIdx.z) + 4))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 18816))] = max((compute[(6)] + placeholder2[((((int)threadIdx.z) + 6))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 25088))] = max((compute[(8)] + placeholder2[((((int)threadIdx.z) + 8))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 31360))] = max((compute[(10)] + placeholder2[((((int)threadIdx.z) + 10))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 37632))] = max((compute[(12)] + placeholder2[((((int)threadIdx.z) + 12))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 43904))] = max((compute[(14)] + placeholder2[((((int)threadIdx.z) + 14))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 50176))] = max((compute[(16)] + placeholder2[((((int)threadIdx.z) + 16))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 56448))] = max((compute[(18)] + placeholder2[((((int)threadIdx.z) + 18))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 62720))] = max((compute[(20)] + placeholder2[((((int)threadIdx.z) + 20))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 68992))] = max((compute[(22)] + placeholder2[((((int)threadIdx.z) + 22))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 75264))] = max((compute[(24)] + placeholder2[((((int)threadIdx.z) + 24))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 81536))] = max((compute[(26)] + placeholder2[((((int)threadIdx.z) + 26))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 87808))] = max((compute[(28)] + placeholder2[((((int)threadIdx.z) + 28))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 94080))] = max((compute[(30)] + placeholder2[((((int)threadIdx.z) + 30))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 100352))] = max((compute[(32)] + placeholder2[((((int)threadIdx.z) + 32))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 106624))] = max((compute[(34)] + placeholder2[((((int)threadIdx.z) + 34))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 112896))] = max((compute[(36)] + placeholder2[((((int)threadIdx.z) + 36))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 119168))] = max((compute[(38)] + placeholder2[((((int)threadIdx.z) + 38))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 125440))] = max((compute[(40)] + placeholder2[((((int)threadIdx.z) + 40))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 131712))] = max((compute[(42)] + placeholder2[((((int)threadIdx.z) + 42))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 137984))] = max((compute[(44)] + placeholder2[((((int)threadIdx.z) + 44))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 144256))] = max((compute[(46)] + placeholder2[((((int)threadIdx.z) + 46))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 150528))] = max((compute[(48)] + placeholder2[((((int)threadIdx.z) + 48))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 156800))] = max((compute[(50)] + placeholder2[((((int)threadIdx.z) + 50))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 163072))] = max((compute[(52)] + placeholder2[((((int)threadIdx.z) + 52))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 169344))] = max((compute[(54)] + placeholder2[((((int)threadIdx.z) + 54))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 175616))] = max((compute[(56)] + placeholder2[((((int)threadIdx.z) + 56))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 181888))] = max((compute[(58)] + placeholder2[((((int)threadIdx.z) + 58))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 188160))] = max((compute[(60)] + placeholder2[((((int)threadIdx.z) + 60))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 194432))] = max((compute[(62)] + placeholder2[((((int)threadIdx.z) + 62))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 56))] = max((compute[(1)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 6328))] = max((compute[(3)] + placeholder2[((((int)threadIdx.z) + 2))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 12600))] = max((compute[(5)] + placeholder2[((((int)threadIdx.z) + 4))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 18872))] = max((compute[(7)] + placeholder2[((((int)threadIdx.z) + 6))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 25144))] = max((compute[(9)] + placeholder2[((((int)threadIdx.z) + 8))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 31416))] = max((compute[(11)] + placeholder2[((((int)threadIdx.z) + 10))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 37688))] = max((compute[(13)] + placeholder2[((((int)threadIdx.z) + 12))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 43960))] = max((compute[(15)] + placeholder2[((((int)threadIdx.z) + 14))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 50232))] = max((compute[(17)] + placeholder2[((((int)threadIdx.z) + 16))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 56504))] = max((compute[(19)] + placeholder2[((((int)threadIdx.z) + 18))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 62776))] = max((compute[(21)] + placeholder2[((((int)threadIdx.z) + 20))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 69048))] = max((compute[(23)] + placeholder2[((((int)threadIdx.z) + 22))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 75320))] = max((compute[(25)] + placeholder2[((((int)threadIdx.z) + 24))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 81592))] = max((compute[(27)] + placeholder2[((((int)threadIdx.z) + 26))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 87864))] = max((compute[(29)] + placeholder2[((((int)threadIdx.z) + 28))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 94136))] = max((compute[(31)] + placeholder2[((((int)threadIdx.z) + 30))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 100408))] = max((compute[(33)] + placeholder2[((((int)threadIdx.z) + 32))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 106680))] = max((compute[(35)] + placeholder2[((((int)threadIdx.z) + 34))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 112952))] = max((compute[(37)] + placeholder2[((((int)threadIdx.z) + 36))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 119224))] = max((compute[(39)] + placeholder2[((((int)threadIdx.z) + 38))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 125496))] = max((compute[(41)] + placeholder2[((((int)threadIdx.z) + 40))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 131768))] = max((compute[(43)] + placeholder2[((((int)threadIdx.z) + 42))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 138040))] = max((compute[(45)] + placeholder2[((((int)threadIdx.z) + 44))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 144312))] = max((compute[(47)] + placeholder2[((((int)threadIdx.z) + 46))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 150584))] = max((compute[(49)] + placeholder2[((((int)threadIdx.z) + 48))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 156856))] = max((compute[(51)] + placeholder2[((((int)threadIdx.z) + 50))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 163128))] = max((compute[(53)] + placeholder2[((((int)threadIdx.z) + 52))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 169400))] = max((compute[(55)] + placeholder2[((((int)threadIdx.z) + 54))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 175672))] = max((compute[(57)] + placeholder2[((((int)threadIdx.z) + 56))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 181944))] = max((compute[(59)] + placeholder2[((((int)threadIdx.z) + 58))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 188216))] = max((compute[(61)] + placeholder2[((((int)threadIdx.z) + 60))]), 0.000000e+00f);
  T_relu[((((((((int)vz) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)vy) * 112)) + ((int)threadIdx.x)) + 194488))] = max((compute[(63)] + placeholder2[((((int)threadIdx.z) + 62))]), 0.000000e+00f);
    }
}

int main(int argc, char *argv[]) {
    // init device
    if (argc < 2) {
        printf("args num error! argc:%d", argc);
        exit(1);
    }
    int gpu_no = atoi(argv[1]);
    checkCudaErrors(hipSetDevice(gpu_no));

    // allocate stream
    int num_streams = 2;
    hipStream_t streams[num_streams];
    for (int i = 0; i < num_streams; i++) {
        checkCudaErrors(hipStreamCreate(&streams[i]));
    }


    // allocate flag
    int *flag = new int[FLAG_LENGTH];
    int *g_flag;
    for (int i = 0; i < FLAG_LENGTH; ++i) {
        flag[i] = 0;
    }
    checkCudaErrors(hipMalloc((void **)&g_flag, sizeof(int) * FLAG_LENGTH));
    checkCudaErrors(hipMemcpy(g_flag, flag, sizeof(int) * FLAG_LENGTH, hipMemcpyHostToDevice));

    int *flag_ = new int[FLAG_LENGTH];
    int *g_flag_;
    for (int i = 0; i < FLAG_LENGTH; ++i) {
        flag_[i] = 0;
    }
    checkCudaErrors(hipMalloc((void **)&g_flag_, sizeof(int) * FLAG_LENGTH));
    checkCudaErrors(hipMemcpy(g_flag_, flag_, sizeof(int) * FLAG_LENGTH, hipMemcpyHostToDevice));

    //prepare parm for kernel 1
    int *workers = new int[80];
    for(int i=0;i<80;i++)
    {
    workers[i]=0;
    }
    int *g_worker;
    checkCudaErrors(hipMalloc((void **)&g_worker, sizeof(int) * 80));
    checkCudaErrors(hipMemcpy( g_worker,workers, sizeof(int) * 80, hipMemcpyHostToDevice));


    float *placeholder0 = new float[6422528];
    for(int i=0;i<6422528;i++)
    {
    placeholder0[i]=1;
    }
    float *g_ph0;
    checkCudaErrors(hipMalloc((void **)&g_ph0, sizeof(float) * 6422528));
    checkCudaErrors(hipMemcpy(g_ph0, placeholder0, sizeof(float) * 6422528, hipMemcpyHostToDevice));

    float *placeholder1 = new float[36864];
    for(int i=0;i<36864;i++)
    {
    placeholder1[i]=0;
    }
    float *g_ph1;
    checkCudaErrors(hipMalloc((void **)&g_ph1, sizeof(float) * 36864));
    checkCudaErrors(hipMemcpy(g_ph1, placeholder1, sizeof(float) * 36864, hipMemcpyHostToDevice));

    float *placeholder2 = new float[6422528];
    for(int i=0;i<6422528;i++)
    {
    placeholder2[i]=3;
    }
    float *g_ph2;
    checkCudaErrors(hipMalloc((void **)&g_ph2, sizeof(float) * 6422528));
    checkCudaErrors(hipMemcpy(g_ph2, placeholder2, sizeof(float) * 6422528, hipMemcpyHostToDevice));


    float *placeholder3 = new float[64];
    for(int i=0;i<64;i++)
    {
    placeholder3[i]=4;
    }
    float *g_ph3;
    hipMalloc((void **)&g_ph3, sizeof(float) * 64);
    hipMemcpy(g_ph3, placeholder3, sizeof(float) * 64, hipMemcpyHostToDevice);


    dim3 Dim_block = dim3(LAUNCH_BLOCKX, LAUNCH_BLOCKY, LAUNCH_BLOCKZ);
    dim3 Dim_thread = dim3(LAUNCH_THREADX, LAUNCH_THREADY, LAUNCH_THREADZ);

    printf("hello?");
    // launch kernel
    fused_nn_conv2d_add_nn_relu_6_kernel0<<<Dim_block, Dim_thread, 0, streams[0]>>>(g_worker,1, g_flag, g_ph0, g_ph1, g_ph2, g_ph3);

    hipDeviceSynchronize();

    checkCudaErrors(hipMemcpy(placeholder2, g_ph2,sizeof(float) * 6422528, hipMemcpyDeviceToHost));

    printf("hello3?\n");
    for(int j=0;j<784;j++)
    {
    if(j%10==0)
    {
    printf("\n");
    }
    printf("%f  ",placeholder2[1024*j+j]);
    }

    printf("\n");
    checkCudaErrors(hipMemcpy(workers,g_worker,sizeof(int) * 80, hipMemcpyDeviceToHost));
    for(int j=0;j<80;j++)
    {
    if(j%10==0&&j!=0)
    {
    printf("\n");
    }
    printf("%d  ",workers[j]);
    }
    printf("\n");
}