#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include <iostream>
#include <sys/time.h>
#include <math.h>


#define LAUNCH_THREADX 7
#define LAUNCH_THREADY 1
#define LAUNCH_THREADZ 8

#define LAUNCH_BLOCKX 1
#define ORI_BLOCKX 1
#define LAUNCH_BLOCKY 1
#define ORI_BLOCKY 1
#define LAUNCH_BLOCKZ 512 * 5 // 5是额外部分，满足多层覆盖
#define ORI_BLOCKZ 256

#define SM_NUM 32
#define WORKER_NUM_PERSM 8

#define BLOCK_NUM LAUNCH_BLOCKZ * LAUNCH_BLOCKY * LAUNCH_BLOCKX
#define FLAG_LENGTH 65535
#define FLAG_BLOCK_BASE 0
#define FLAG_SM_BASE (FLAG_BLOCK_BASE + 1)
#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)
// nvcc -arch=native main.cu -o main

#define GPU_RETURN_STATUS(cmd) \
{ \
    hipError_t result = cmd; \
    if (result != hipSuccess) { \
        std::cout << #cmd " error, return code:" << result << " | " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); \
    } \
}

// These are the inline versions for all of the SDK helper functions
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    const char *errorStr = NULL;
    errorStr = hipGetErrorString(err);
    fprintf(stderr,
            "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, "
            "line %i.\n",
            err, errorStr, file, line);
    exit(EXIT_FAILURE);
  }
}

__device__ uint get_smid(void) {

    uint ret;

    asm("mov.u32 %0, %smid;" : "=r"(ret) );

    return ret;

}

// #if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 700)
// #define __shfl_sync(mask, var, lane, width) \
//         __shfl((var), (lane), (width))

// #define __shfl_down_sync(mask, var, offset, width) \
//         __shfl_down((var), (offset), (width))

// #define __shfl_up_sync(mask, var, offset, width) \
//         __shfl_up((var), (offset), (width))
// #endif

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_1_kernel0(int *worker,int number,int *flag, float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2){
    int* sm_flag = flag;
    __shared__ int basicoffset;
    int offset;
    int smid;
    //judge whether to continue work,which work to fetch
    if(threadIdx.x+threadIdx.y+threadIdx.z == 0)
    {
       basicoffset=-1;
       smid = get_smid();

       //judge whther sm id is right
       if((smid < number*SM_NUM)&&(smid >= (number-1)*SM_NUM))
       {
            //judge whether worker is enough
            //get the basic offset for the block
            int blocknumber=atomicAdd(sm_flag + smid, 1);
            if(blocknumber< WORKER_NUM_PERSM)
            {
                basicoffset = WORKER_NUM_PERSM*(smid-(number-1)*SM_NUM) + blocknumber;
                atomicAdd(worker + smid, 1);
                //printf("smid %d\n", smid);
            }
       }
    }
    __syncthreads();
    if (basicoffset < 0) return ;
    //every thread has its own offset
    offset = basicoffset;
    // if ((threadIdx.x + threadIdx.y + threadIdx.z) == 0 && (number == 1)) {
    //     printf("smid %d\n", smid);
    // }

    while(offset < (ORI_BLOCKX * ORI_BLOCKY * ORI_BLOCKZ)) {
        int vx = (offset)/(ORI_BLOCKY * ORI_BLOCKZ);
        int vy = (offset - (vx * ORI_BLOCKY * ORI_BLOCKZ)) / ORI_BLOCKZ;
        int vz = offset - (vx * ORI_BLOCKY * ORI_BLOCKZ) - vy * ORI_BLOCKZ;
        offset += SM_NUM * WORKER_NUM_PERSM;
        float compute[56];
  __shared__ float pad_temp_shared[1352];
  __shared__ float placeholder_shared[512];
  compute[(0)] = 0.000000e+00f;
  compute[(14)] = 0.000000e+00f;
  compute[(28)] = 0.000000e+00f;
  compute[(42)] = 0.000000e+00f;
  compute[(2)] = 0.000000e+00f;
  compute[(16)] = 0.000000e+00f;
  compute[(30)] = 0.000000e+00f;
  compute[(44)] = 0.000000e+00f;
  compute[(4)] = 0.000000e+00f;
  compute[(18)] = 0.000000e+00f;
  compute[(32)] = 0.000000e+00f;
  compute[(46)] = 0.000000e+00f;
  compute[(6)] = 0.000000e+00f;
  compute[(20)] = 0.000000e+00f;
  compute[(34)] = 0.000000e+00f;
  compute[(48)] = 0.000000e+00f;
  compute[(8)] = 0.000000e+00f;
  compute[(22)] = 0.000000e+00f;
  compute[(36)] = 0.000000e+00f;
  compute[(50)] = 0.000000e+00f;
  compute[(10)] = 0.000000e+00f;
  compute[(24)] = 0.000000e+00f;
  compute[(38)] = 0.000000e+00f;
  compute[(52)] = 0.000000e+00f;
  compute[(12)] = 0.000000e+00f;
  compute[(26)] = 0.000000e+00f;
  compute[(40)] = 0.000000e+00f;
  compute[(54)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  compute[(15)] = 0.000000e+00f;
  compute[(29)] = 0.000000e+00f;
  compute[(43)] = 0.000000e+00f;
  compute[(3)] = 0.000000e+00f;
  compute[(17)] = 0.000000e+00f;
  compute[(31)] = 0.000000e+00f;
  compute[(45)] = 0.000000e+00f;
  compute[(5)] = 0.000000e+00f;
  compute[(19)] = 0.000000e+00f;
  compute[(33)] = 0.000000e+00f;
  compute[(47)] = 0.000000e+00f;
  compute[(7)] = 0.000000e+00f;
  compute[(21)] = 0.000000e+00f;
  compute[(35)] = 0.000000e+00f;
  compute[(49)] = 0.000000e+00f;
  compute[(9)] = 0.000000e+00f;
  compute[(23)] = 0.000000e+00f;
  compute[(37)] = 0.000000e+00f;
  compute[(51)] = 0.000000e+00f;
  compute[(11)] = 0.000000e+00f;
  compute[(25)] = 0.000000e+00f;
  compute[(39)] = 0.000000e+00f;
  compute[(53)] = 0.000000e+00f;
  compute[(13)] = 0.000000e+00f;
  compute[(27)] = 0.000000e+00f;
  compute[(41)] = 0.000000e+00f;
  compute[(55)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 32; ++rc_outer) {
    for (int ry_outer = 0; ry_outer < 3; ++ry_outer) {
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)))] = (((1 <= (((((int)threadIdx.x) * 25) / 13) + ry_outer)) && (1 <= ((((int)threadIdx.x) * 25) % 13))) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((((int)threadIdx.x) * 25) / 13) * 14)) + (ry_outer * 14)) + ((((int)threadIdx.x) * 25) % 13)) - 15))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 1))] = (((1 <= ((((((int)threadIdx.x) * 25) + 1) / 13) + ry_outer)) && (1 <= (((((int)threadIdx.x) * 25) + 1) % 13))) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 1) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 1) % 13)) - 15))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 2))] = (((1 <= ((((((int)threadIdx.x) * 25) + 2) / 13) + ry_outer)) && (1 <= (((((int)threadIdx.x) * 25) + 2) % 13))) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 2) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 2) % 13)) - 15))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 3))] = (((1 <= ((((((int)threadIdx.x) * 25) + 3) / 13) + ry_outer)) && (1 <= (((((int)threadIdx.x) * 25) + 3) % 13))) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 3) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 3) % 13)) - 15))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 4))] = (((1 <= ((((((int)threadIdx.x) * 25) + 4) / 13) + ry_outer)) && (1 <= (((((int)threadIdx.x) * 25) + 4) % 13))) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 4) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 4) % 13)) - 15))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 5))] = (((1 <= ((((((int)threadIdx.x) * 25) + 5) / 13) + ry_outer)) && (1 <= (((((int)threadIdx.x) * 25) + 5) % 13))) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 5) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 5) % 13)) - 15))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 6))] = (((1 <= ((((((int)threadIdx.x) * 25) + 6) / 13) + ry_outer)) && (1 <= (((((int)threadIdx.x) * 25) + 6) % 13))) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 6) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 6) % 13)) - 15))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 7))] = (((1 <= ((((((int)threadIdx.x) * 25) + 7) / 13) + ry_outer)) && (1 <= (((((int)threadIdx.x) * 25) + 7) % 13))) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 7) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 7) % 13)) - 15))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 8))] = (((1 <= ((((((int)threadIdx.x) * 25) + 8) / 13) + ry_outer)) && (1 <= (((((int)threadIdx.x) * 25) + 8) % 13))) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 8) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 8) % 13)) - 15))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 9))] = (((1 <= ((((((int)threadIdx.x) * 25) + 9) / 13) + ry_outer)) && (1 <= (((((int)threadIdx.x) * 25) + 9) % 13))) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 9) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 9) % 13)) - 15))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 10))] = (((1 <= ((((((int)threadIdx.x) * 25) + 10) / 13) + ry_outer)) && (1 <= (((((int)threadIdx.x) * 25) + 10) % 13))) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 10) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 10) % 13)) - 15))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 11))] = (((1 <= ((((((int)threadIdx.x) * 25) + 11) / 13) + ry_outer)) && (1 <= (((((int)threadIdx.x) * 25) + 11) % 13))) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 11) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 11) % 13)) - 15))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 12))] = (((1 <= ((((((int)threadIdx.x) * 25) + 12) / 13) + ry_outer)) && (1 <= (((((int)threadIdx.x) * 25) + 12) % 13))) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 12) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 12) % 13)) - 15))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 13))] = ((1 <= ((((int)threadIdx.x) * 25) % 13)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((((int)threadIdx.x) * 25) / 13) * 14)) + (ry_outer * 14)) + ((((int)threadIdx.x) * 25) % 13)) - 1))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 14))] = ((1 <= (((((int)threadIdx.x) * 25) + 1) % 13)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 14) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 1) % 13)) - 15))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 15))] = ((1 <= (((((int)threadIdx.x) * 25) + 2) % 13)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 15) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 2) % 13)) - 15))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 16))] = ((1 <= (((((int)threadIdx.x) * 25) + 3) % 13)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 16) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 3) % 13)) - 15))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 17))] = ((1 <= (((((int)threadIdx.x) * 25) + 4) % 13)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 17) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 4) % 13)) - 15))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 18))] = ((1 <= (((((int)threadIdx.x) * 25) + 5) % 13)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 18) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 5) % 13)) - 15))] : 0.000000e+00f);
      if (((((((int)threadIdx.x) * 25) + 19) / 169) + ((int)threadIdx.z)) < 8) {
        if (((((int)threadIdx.z) * 13) + (((((int)threadIdx.x) * 25) + 19) / 13)) < 104) {
          if (((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) < 1333) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 19))] = ((1 <= (((((int)threadIdx.x) * 25) + 6) % 13)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 19) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 6) % 13)) - 15))] : 0.000000e+00f);
            }
          }
        }
      }
      if (((((((int)threadIdx.x) * 25) + 20) / 169) + ((int)threadIdx.z)) < 8) {
        if (((((int)threadIdx.z) * 13) + (((((int)threadIdx.x) * 25) + 20) / 13)) < 104) {
          if (((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) < 1332) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 20))] = ((1 <= (((((int)threadIdx.x) * 25) + 7) % 13)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 20) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 7) % 13)) - 15))] : 0.000000e+00f);
            }
          }
        }
      }
      if (((((((int)threadIdx.x) * 25) + 21) / 169) + ((int)threadIdx.z)) < 8) {
        if (((((int)threadIdx.z) * 13) + (((((int)threadIdx.x) * 25) + 21) / 13)) < 104) {
          if (((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) < 1331) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 21))] = ((1 <= (((((int)threadIdx.x) * 25) + 8) % 13)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 21) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 8) % 13)) - 15))] : 0.000000e+00f);
            }
          }
        }
      }
      if (((((((int)threadIdx.x) * 25) + 22) / 169) + ((int)threadIdx.z)) < 8) {
        if (((((int)threadIdx.z) * 13) + (((((int)threadIdx.x) * 25) + 22) / 13)) < 104) {
          if (((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) < 1330) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 22))] = ((1 <= (((((int)threadIdx.x) * 25) + 9) % 13)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 22) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 9) % 13)) - 15))] : 0.000000e+00f);
            }
          }
        }
      }
      if (((((((int)threadIdx.x) * 25) + 23) / 169) + ((int)threadIdx.z)) < 8) {
        if (((((int)threadIdx.z) * 13) + (((((int)threadIdx.x) * 25) + 23) / 13)) < 104) {
          if (((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) < 1329) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 23))] = ((1 <= (((((int)threadIdx.x) * 25) + 10) % 13)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 23) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 10) % 13)) - 15))] : 0.000000e+00f);
            }
          }
        }
      }
      if (((((((int)threadIdx.x) * 25) + 24) / 169) + ((int)threadIdx.z)) < 8) {
        if (((((int)threadIdx.z) * 13) + (((((int)threadIdx.x) * 25) + 24) / 13)) < 104) {
          if (((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) < 1328) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 24))] = ((1 <= (((((int)threadIdx.x) * 25) + 11) % 13)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 24) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 11) % 13)) - 15))] : 0.000000e+00f);
            }
          }
        }
      }
      placeholder_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)))] = placeholder1[((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + (((((int)threadIdx.x) * 10) >> 3) * 2304)) + (rc_outer * 72)) + (((((int)threadIdx.x) * 10) & 7) * 9)) + (ry_outer * 3)))];
      placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) + 1))] = placeholder1[((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 10) + 1) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 10) + 1) & 7) * 9)) + (ry_outer * 3)))];
      placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) + 2))] = placeholder1[((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 10) + 2) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 10) + 2) & 7) * 9)) + (ry_outer * 3)))];
      placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) + 3))] = placeholder1[((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 10) + 3) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 10) + 3) & 7) * 9)) + (ry_outer * 3)))];
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 10) + 4) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) < 508) {
          if (((int)threadIdx.x) < 6) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) + 4))] = placeholder1[((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 10) + 4) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 10) + 4) & 7) * 9)) + (ry_outer * 3)))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 10) + 5) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) < 507) {
          if (((int)threadIdx.x) < 6) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) + 5))] = placeholder1[((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 10) + 5) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 10) + 5) & 7) * 9)) + (ry_outer * 3)))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 10) + 6) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) < 506) {
          if (((int)threadIdx.x) < 6) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) + 6))] = placeholder1[((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 10) + 6) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 10) + 6) & 7) * 9)) + (ry_outer * 3)))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 10) + 7) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) < 505) {
          if (((int)threadIdx.x) < 6) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) + 7))] = placeholder1[((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 10) + 7) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 10) + 7) & 7) * 9)) + (ry_outer * 3)))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + ((((int)threadIdx.x) * 10) >> 3)) < 63) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) < 504) {
          if (((int)threadIdx.x) < 6) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) + 8))] = placeholder1[(((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + (((((int)threadIdx.x) * 10) >> 3) * 2304)) + (rc_outer * 72)) + (((((int)threadIdx.x) * 10) & 7) * 9)) + (ry_outer * 3)) + 2304))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 10) + 9) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) < 503) {
          if (((int)threadIdx.x) < 6) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) + 9))] = placeholder1[((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 10) + 9) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 10) + 1) & 7) * 9)) + (ry_outer * 3)))];
          }
        }
      }
      __syncthreads();
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[((((int)threadIdx.z) * 16))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 128))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 256))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 384))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 26))] * placeholder_shared[((((int)threadIdx.z) * 16))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 26))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 128))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 26))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 256))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 26))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 384))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 52))] * placeholder_shared[((((int)threadIdx.z) * 16))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 52))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 128))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 52))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 256))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 52))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 384))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 78))] * placeholder_shared[((((int)threadIdx.z) * 16))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 78))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 128))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 78))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 256))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 78))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 384))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 104))] * placeholder_shared[((((int)threadIdx.z) * 16))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 104))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 128))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 104))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 256))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 104))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 384))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 130))] * placeholder_shared[((((int)threadIdx.z) * 16))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 130))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 128))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 130))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 256))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 130))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 384))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 156))] * placeholder_shared[((((int)threadIdx.z) * 16))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 156))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 128))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 156))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 256))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 156))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 384))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 8))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 136))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 264))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 392))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 26))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 8))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 26))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 136))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 26))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 264))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 26))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 392))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 52))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 8))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 52))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 136))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 52))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 264))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 52))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 392))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 78))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 8))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 78))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 136))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 78))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 264))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 78))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 392))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 104))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 8))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 104))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 136))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 104))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 264))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 104))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 392))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 130))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 8))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 130))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 136))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 130))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 264))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 130))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 392))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 156))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 8))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 156))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 136))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 156))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 264))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 156))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 392))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 169))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 1))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 169))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 129))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 169))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 257))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 169))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 385))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 195))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 1))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 195))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 129))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 195))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 257))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 195))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 385))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 221))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 1))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 221))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 129))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 221))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 257))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 221))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 385))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 247))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 1))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 247))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 129))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 247))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 257))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 247))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 385))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 273))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 1))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 273))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 129))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 273))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 257))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 273))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 385))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 299))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 1))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 299))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 129))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 299))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 257))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 299))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 385))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 325))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 1))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 325))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 129))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 325))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 257))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 325))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 385))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 169))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 9))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 169))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 137))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 169))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 265))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 169))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 393))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 195))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 9))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 195))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 137))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 195))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 265))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 195))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 393))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 221))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 9))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 221))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 137))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 221))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 265))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 221))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 393))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 247))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 9))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 247))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 137))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 247))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 265))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 247))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 393))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 273))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 9))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 273))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 137))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 273))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 265))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 273))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 393))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 299))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 9))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 299))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 137))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 299))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 265))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 299))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 393))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 325))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 9))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 325))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 137))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 325))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 265))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 325))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 393))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 338))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 2))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 338))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 130))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 338))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 258))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 338))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 386))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 2))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 130))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 258))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 386))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 390))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 2))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 390))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 130))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 390))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 258))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 390))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 386))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 416))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 2))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 416))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 130))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 416))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 258))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 416))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 386))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 442))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 2))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 442))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 130))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 442))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 258))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 442))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 386))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 468))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 2))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 468))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 130))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 468))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 258))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 468))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 386))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 494))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 2))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 494))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 130))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 494))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 258))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 494))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 386))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 338))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 10))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 338))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 138))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 338))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 266))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 338))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 394))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 10))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 138))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 266))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 394))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 390))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 10))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 390))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 138))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 390))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 266))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 390))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 394))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 416))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 10))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 416))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 138))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 416))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 266))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 416))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 394))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 442))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 10))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 442))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 138))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 442))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 266))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 442))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 394))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 468))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 10))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 468))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 138))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 468))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 266))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 468))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 394))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 494))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 10))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 494))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 138))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 494))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 266))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 494))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 394))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 507))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 3))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 507))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 131))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 507))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 259))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 507))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 387))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 533))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 3))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 533))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 131))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 533))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 259))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 533))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 387))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 559))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 3))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 559))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 131))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 559))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 259))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 559))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 387))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 585))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 3))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 585))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 131))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 585))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 259))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 585))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 387))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 611))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 3))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 611))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 131))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 611))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 259))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 611))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 387))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 637))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 3))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 637))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 131))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 637))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 259))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 637))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 387))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 663))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 3))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 663))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 131))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 663))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 259))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 663))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 387))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 507))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 11))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 507))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 139))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 507))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 267))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 507))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 395))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 533))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 11))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 533))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 139))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 533))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 267))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 533))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 395))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 559))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 11))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 559))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 139))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 559))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 267))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 559))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 395))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 585))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 11))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 585))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 139))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 585))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 267))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 585))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 395))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 611))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 11))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 611))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 139))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 611))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 267))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 611))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 395))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 637))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 11))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 637))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 139))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 637))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 267))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 637))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 395))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 663))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 11))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 663))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 139))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 663))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 267))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 663))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 395))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 676))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 4))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 676))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 132))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 676))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 260))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 676))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 388))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 702))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 4))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 702))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 132))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 702))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 260))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 702))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 388))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 4))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 132))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 260))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 388))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 754))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 4))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 754))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 132))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 754))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 260))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 754))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 388))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 780))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 4))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 780))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 132))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 780))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 260))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 780))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 388))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 806))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 4))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 806))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 132))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 806))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 260))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 806))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 388))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 832))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 4))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 832))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 132))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 832))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 260))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 832))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 388))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 676))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 12))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 676))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 140))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 676))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 268))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 676))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 396))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 702))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 12))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 702))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 140))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 702))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 268))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 702))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 396))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 12))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 140))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 268))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 396))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 754))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 12))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 754))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 140))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 754))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 268))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 754))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 396))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 780))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 12))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 780))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 140))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 780))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 268))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 780))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 396))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 806))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 12))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 806))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 140))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 806))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 268))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 806))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 396))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 832))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 12))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 832))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 140))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 832))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 268))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 832))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 396))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 845))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 5))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 845))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 133))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 845))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 261))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 845))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 389))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 871))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 5))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 871))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 133))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 871))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 261))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 871))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 389))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 897))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 5))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 897))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 133))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 897))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 261))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 897))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 389))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 923))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 5))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 923))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 133))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 923))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 261))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 923))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 389))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 949))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 5))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 949))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 133))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 949))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 261))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 949))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 389))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 975))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 5))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 975))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 133))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 975))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 261))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 975))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 389))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1001))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 5))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1001))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 133))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1001))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 261))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1001))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 389))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 845))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 13))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 845))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 141))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 845))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 269))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 845))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 397))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 871))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 13))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 871))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 141))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 871))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 269))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 871))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 397))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 897))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 13))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 897))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 141))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 897))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 269))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 897))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 397))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 923))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 13))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 923))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 141))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 923))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 269))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 923))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 397))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 949))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 13))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 949))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 141))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 949))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 269))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 949))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 397))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 975))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 13))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 975))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 141))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 975))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 269))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 975))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 397))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1001))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 13))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1001))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 141))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1001))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 269))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1001))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 397))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1014))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 6))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1014))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 134))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1014))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 262))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1014))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 390))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1040))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 6))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1040))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 134))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1040))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 262))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1040))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 390))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1066))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 6))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1066))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 134))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1066))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 262))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1066))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 390))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1092))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 6))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1092))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 134))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1092))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 262))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1092))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 390))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1118))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 6))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1118))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 134))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1118))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 262))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1118))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 390))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1144))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 6))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1144))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 134))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1144))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 262))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1144))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 390))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1170))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 6))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1170))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 134))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1170))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 262))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1170))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 390))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1014))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 14))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1014))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 142))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1014))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 270))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1014))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 398))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1040))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 14))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1040))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 142))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1040))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 270))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1040))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 398))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1066))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 14))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1066))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 142))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1066))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 270))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1066))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 398))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1092))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 14))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1092))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 142))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1092))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 270))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1092))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 398))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1118))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 14))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1118))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 142))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1118))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 270))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1118))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 398))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1144))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 14))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1144))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 142))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1144))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 270))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1144))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 398))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1170))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 14))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1170))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 142))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1170))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 270))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1170))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 398))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1183))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 7))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1183))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 135))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1183))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 263))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1183))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 391))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1209))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 7))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1209))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 135))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1209))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 263))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1209))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 391))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1235))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 7))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1235))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 135))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1235))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 263))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1235))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 391))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1261))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 7))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1261))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 135))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1261))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 263))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1261))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 391))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1287))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 7))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1287))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 135))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1287))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 263))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1287))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 391))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1313))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 7))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1313))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 135))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1313))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 263))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1313))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 391))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1339))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 7))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1339))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 135))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1339))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 263))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1339))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 391))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1183))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 15))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1183))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 143))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1183))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 271))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1183))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 399))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1209))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 15))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1209))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 143))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1209))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 271))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1209))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 399))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1235))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 15))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1235))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 143))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1235))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 271))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1235))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 399))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1261))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 15))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1261))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 143))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1261))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 271))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1261))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 399))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1287))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 15))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1287))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 143))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1287))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 271))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1287))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 399))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1313))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 15))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1313))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 143))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1313))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 271))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1313))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 399))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1339))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 15))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1339))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 143))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1339))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 271))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1339))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 399))]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)))] = ((1 <= (((((int)threadIdx.x) * 25) / 13) + ry_outer)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((((int)threadIdx.x) * 25) / 13) * 14)) + (ry_outer * 14)) + ((((int)threadIdx.x) * 25) % 13)) - 14))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 1))] = ((1 <= ((((((int)threadIdx.x) * 25) + 1) / 13) + ry_outer)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 1) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 1) % 13)) - 14))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 2))] = ((1 <= ((((((int)threadIdx.x) * 25) + 2) / 13) + ry_outer)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 2) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 2) % 13)) - 14))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 3))] = ((1 <= ((((((int)threadIdx.x) * 25) + 3) / 13) + ry_outer)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 3) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 3) % 13)) - 14))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 4))] = ((1 <= ((((((int)threadIdx.x) * 25) + 4) / 13) + ry_outer)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 4) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 4) % 13)) - 14))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 5))] = ((1 <= ((((((int)threadIdx.x) * 25) + 5) / 13) + ry_outer)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 5) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 5) % 13)) - 14))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 6))] = ((1 <= ((((((int)threadIdx.x) * 25) + 6) / 13) + ry_outer)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 6) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 6) % 13)) - 14))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 7))] = ((1 <= ((((((int)threadIdx.x) * 25) + 7) / 13) + ry_outer)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 7) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 7) % 13)) - 14))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 8))] = ((1 <= ((((((int)threadIdx.x) * 25) + 8) / 13) + ry_outer)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 8) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 8) % 13)) - 14))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 9))] = ((1 <= ((((((int)threadIdx.x) * 25) + 9) / 13) + ry_outer)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 9) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 9) % 13)) - 14))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 10))] = ((1 <= ((((((int)threadIdx.x) * 25) + 10) / 13) + ry_outer)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 10) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 10) % 13)) - 14))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 11))] = ((1 <= ((((((int)threadIdx.x) * 25) + 11) / 13) + ry_outer)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 11) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 11) % 13)) - 14))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 12))] = ((1 <= ((((((int)threadIdx.x) * 25) + 12) / 13) + ry_outer)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 12) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 12) % 13)) - 14))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 13))] = placeholder[((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((((int)threadIdx.x) * 25) / 13) * 14)) + (ry_outer * 14)) + ((((int)threadIdx.x) * 25) % 13)))];
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 14))] = placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 14) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 1) % 13)) - 14))];
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 15))] = placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 15) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 2) % 13)) - 14))];
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 16))] = placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 16) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 3) % 13)) - 14))];
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 17))] = placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 17) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 4) % 13)) - 14))];
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 18))] = placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 18) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 5) % 13)) - 14))];
      if (((((((int)threadIdx.x) * 25) + 19) / 169) + ((int)threadIdx.z)) < 8) {
        if (((((int)threadIdx.z) * 13) + (((((int)threadIdx.x) * 25) + 19) / 13)) < 104) {
          if (((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) < 1333) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 19))] = placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 19) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 6) % 13)) - 14))];
            }
          }
        }
      }
      if (((((((int)threadIdx.x) * 25) + 20) / 169) + ((int)threadIdx.z)) < 8) {
        if (((((int)threadIdx.z) * 13) + (((((int)threadIdx.x) * 25) + 20) / 13)) < 104) {
          if (((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) < 1332) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 20))] = placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 20) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 7) % 13)) - 14))];
            }
          }
        }
      }
      if (((((((int)threadIdx.x) * 25) + 21) / 169) + ((int)threadIdx.z)) < 8) {
        if (((((int)threadIdx.z) * 13) + (((((int)threadIdx.x) * 25) + 21) / 13)) < 104) {
          if (((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) < 1331) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 21))] = placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 21) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 8) % 13)) - 14))];
            }
          }
        }
      }
      if (((((((int)threadIdx.x) * 25) + 22) / 169) + ((int)threadIdx.z)) < 8) {
        if (((((int)threadIdx.z) * 13) + (((((int)threadIdx.x) * 25) + 22) / 13)) < 104) {
          if (((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) < 1330) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 22))] = placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 22) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 9) % 13)) - 14))];
            }
          }
        }
      }
      if (((((((int)threadIdx.x) * 25) + 23) / 169) + ((int)threadIdx.z)) < 8) {
        if (((((int)threadIdx.z) * 13) + (((((int)threadIdx.x) * 25) + 23) / 13)) < 104) {
          if (((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) < 1329) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 23))] = placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 23) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 10) % 13)) - 14))];
            }
          }
        }
      }
      if (((((((int)threadIdx.x) * 25) + 24) / 169) + ((int)threadIdx.z)) < 8) {
        if (((((int)threadIdx.z) * 13) + (((((int)threadIdx.x) * 25) + 24) / 13)) < 104) {
          if (((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) < 1328) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 24))] = placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 24) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 11) % 13)) - 14))];
            }
          }
        }
      }
      placeholder_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)))] = placeholder1[(((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + (((((int)threadIdx.x) * 10) >> 3) * 2304)) + (rc_outer * 72)) + (((((int)threadIdx.x) * 10) & 7) * 9)) + (ry_outer * 3)) + 1))];
      placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) + 1))] = placeholder1[(((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 10) + 1) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 10) + 1) & 7) * 9)) + (ry_outer * 3)) + 1))];
      placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) + 2))] = placeholder1[(((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 10) + 2) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 10) + 2) & 7) * 9)) + (ry_outer * 3)) + 1))];
      placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) + 3))] = placeholder1[(((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 10) + 3) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 10) + 3) & 7) * 9)) + (ry_outer * 3)) + 1))];
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 10) + 4) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) < 508) {
          if (((int)threadIdx.x) < 6) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) + 4))] = placeholder1[(((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 10) + 4) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 10) + 4) & 7) * 9)) + (ry_outer * 3)) + 1))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 10) + 5) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) < 507) {
          if (((int)threadIdx.x) < 6) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) + 5))] = placeholder1[(((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 10) + 5) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 10) + 5) & 7) * 9)) + (ry_outer * 3)) + 1))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 10) + 6) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) < 506) {
          if (((int)threadIdx.x) < 6) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) + 6))] = placeholder1[(((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 10) + 6) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 10) + 6) & 7) * 9)) + (ry_outer * 3)) + 1))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 10) + 7) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) < 505) {
          if (((int)threadIdx.x) < 6) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) + 7))] = placeholder1[(((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 10) + 7) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 10) + 7) & 7) * 9)) + (ry_outer * 3)) + 1))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + ((((int)threadIdx.x) * 10) >> 3)) < 63) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) < 504) {
          if (((int)threadIdx.x) < 6) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) + 8))] = placeholder1[(((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + (((((int)threadIdx.x) * 10) >> 3) * 2304)) + (rc_outer * 72)) + (((((int)threadIdx.x) * 10) & 7) * 9)) + (ry_outer * 3)) + 2305))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 10) + 9) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) < 503) {
          if (((int)threadIdx.x) < 6) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) + 9))] = placeholder1[(((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 10) + 9) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 10) + 1) & 7) * 9)) + (ry_outer * 3)) + 1))];
          }
        }
      }
      __syncthreads();
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[((((int)threadIdx.z) * 16))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 128))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 256))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 384))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 26))] * placeholder_shared[((((int)threadIdx.z) * 16))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 26))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 128))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 26))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 256))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 26))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 384))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 52))] * placeholder_shared[((((int)threadIdx.z) * 16))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 52))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 128))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 52))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 256))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 52))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 384))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 78))] * placeholder_shared[((((int)threadIdx.z) * 16))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 78))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 128))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 78))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 256))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 78))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 384))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 104))] * placeholder_shared[((((int)threadIdx.z) * 16))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 104))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 128))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 104))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 256))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 104))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 384))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 130))] * placeholder_shared[((((int)threadIdx.z) * 16))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 130))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 128))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 130))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 256))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 130))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 384))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 156))] * placeholder_shared[((((int)threadIdx.z) * 16))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 156))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 128))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 156))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 256))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 156))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 384))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 8))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 136))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 264))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 392))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 26))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 8))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 26))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 136))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 26))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 264))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 26))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 392))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 52))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 8))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 52))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 136))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 52))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 264))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 52))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 392))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 78))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 8))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 78))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 136))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 78))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 264))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 78))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 392))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 104))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 8))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 104))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 136))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 104))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 264))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 104))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 392))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 130))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 8))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 130))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 136))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 130))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 264))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 130))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 392))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 156))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 8))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 156))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 136))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 156))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 264))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 156))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 392))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 169))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 1))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 169))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 129))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 169))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 257))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 169))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 385))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 195))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 1))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 195))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 129))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 195))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 257))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 195))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 385))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 221))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 1))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 221))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 129))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 221))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 257))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 221))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 385))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 247))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 1))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 247))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 129))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 247))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 257))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 247))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 385))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 273))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 1))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 273))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 129))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 273))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 257))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 273))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 385))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 299))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 1))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 299))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 129))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 299))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 257))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 299))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 385))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 325))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 1))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 325))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 129))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 325))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 257))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 325))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 385))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 169))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 9))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 169))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 137))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 169))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 265))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 169))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 393))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 195))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 9))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 195))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 137))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 195))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 265))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 195))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 393))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 221))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 9))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 221))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 137))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 221))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 265))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 221))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 393))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 247))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 9))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 247))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 137))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 247))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 265))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 247))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 393))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 273))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 9))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 273))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 137))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 273))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 265))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 273))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 393))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 299))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 9))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 299))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 137))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 299))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 265))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 299))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 393))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 325))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 9))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 325))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 137))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 325))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 265))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 325))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 393))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 338))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 2))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 338))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 130))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 338))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 258))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 338))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 386))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 2))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 130))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 258))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 386))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 390))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 2))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 390))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 130))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 390))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 258))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 390))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 386))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 416))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 2))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 416))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 130))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 416))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 258))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 416))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 386))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 442))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 2))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 442))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 130))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 442))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 258))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 442))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 386))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 468))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 2))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 468))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 130))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 468))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 258))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 468))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 386))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 494))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 2))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 494))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 130))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 494))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 258))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 494))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 386))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 338))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 10))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 338))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 138))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 338))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 266))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 338))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 394))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 10))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 138))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 266))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 394))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 390))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 10))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 390))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 138))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 390))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 266))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 390))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 394))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 416))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 10))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 416))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 138))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 416))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 266))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 416))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 394))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 442))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 10))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 442))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 138))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 442))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 266))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 442))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 394))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 468))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 10))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 468))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 138))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 468))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 266))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 468))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 394))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 494))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 10))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 494))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 138))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 494))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 266))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 494))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 394))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 507))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 3))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 507))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 131))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 507))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 259))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 507))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 387))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 533))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 3))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 533))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 131))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 533))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 259))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 533))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 387))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 559))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 3))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 559))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 131))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 559))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 259))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 559))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 387))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 585))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 3))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 585))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 131))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 585))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 259))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 585))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 387))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 611))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 3))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 611))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 131))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 611))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 259))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 611))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 387))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 637))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 3))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 637))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 131))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 637))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 259))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 637))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 387))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 663))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 3))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 663))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 131))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 663))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 259))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 663))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 387))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 507))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 11))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 507))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 139))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 507))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 267))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 507))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 395))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 533))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 11))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 533))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 139))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 533))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 267))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 533))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 395))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 559))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 11))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 559))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 139))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 559))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 267))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 559))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 395))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 585))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 11))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 585))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 139))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 585))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 267))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 585))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 395))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 611))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 11))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 611))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 139))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 611))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 267))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 611))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 395))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 637))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 11))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 637))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 139))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 637))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 267))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 637))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 395))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 663))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 11))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 663))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 139))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 663))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 267))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 663))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 395))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 676))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 4))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 676))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 132))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 676))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 260))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 676))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 388))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 702))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 4))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 702))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 132))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 702))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 260))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 702))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 388))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 4))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 132))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 260))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 388))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 754))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 4))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 754))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 132))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 754))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 260))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 754))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 388))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 780))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 4))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 780))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 132))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 780))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 260))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 780))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 388))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 806))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 4))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 806))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 132))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 806))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 260))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 806))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 388))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 832))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 4))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 832))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 132))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 832))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 260))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 832))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 388))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 676))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 12))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 676))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 140))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 676))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 268))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 676))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 396))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 702))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 12))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 702))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 140))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 702))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 268))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 702))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 396))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 12))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 140))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 268))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 396))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 754))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 12))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 754))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 140))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 754))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 268))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 754))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 396))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 780))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 12))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 780))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 140))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 780))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 268))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 780))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 396))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 806))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 12))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 806))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 140))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 806))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 268))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 806))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 396))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 832))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 12))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 832))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 140))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 832))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 268))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 832))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 396))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 845))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 5))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 845))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 133))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 845))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 261))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 845))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 389))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 871))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 5))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 871))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 133))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 871))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 261))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 871))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 389))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 897))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 5))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 897))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 133))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 897))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 261))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 897))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 389))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 923))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 5))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 923))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 133))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 923))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 261))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 923))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 389))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 949))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 5))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 949))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 133))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 949))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 261))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 949))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 389))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 975))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 5))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 975))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 133))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 975))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 261))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 975))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 389))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1001))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 5))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1001))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 133))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1001))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 261))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1001))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 389))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 845))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 13))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 845))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 141))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 845))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 269))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 845))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 397))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 871))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 13))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 871))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 141))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 871))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 269))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 871))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 397))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 897))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 13))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 897))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 141))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 897))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 269))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 897))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 397))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 923))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 13))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 923))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 141))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 923))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 269))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 923))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 397))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 949))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 13))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 949))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 141))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 949))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 269))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 949))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 397))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 975))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 13))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 975))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 141))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 975))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 269))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 975))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 397))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1001))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 13))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1001))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 141))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1001))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 269))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1001))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 397))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1014))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 6))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1014))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 134))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1014))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 262))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1014))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 390))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1040))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 6))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1040))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 134))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1040))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 262))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1040))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 390))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1066))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 6))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1066))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 134))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1066))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 262))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1066))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 390))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1092))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 6))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1092))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 134))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1092))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 262))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1092))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 390))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1118))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 6))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1118))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 134))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1118))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 262))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1118))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 390))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1144))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 6))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1144))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 134))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1144))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 262))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1144))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 390))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1170))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 6))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1170))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 134))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1170))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 262))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1170))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 390))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1014))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 14))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1014))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 142))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1014))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 270))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1014))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 398))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1040))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 14))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1040))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 142))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1040))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 270))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1040))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 398))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1066))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 14))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1066))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 142))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1066))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 270))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1066))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 398))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1092))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 14))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1092))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 142))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1092))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 270))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1092))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 398))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1118))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 14))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1118))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 142))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1118))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 270))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1118))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 398))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1144))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 14))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1144))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 142))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1144))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 270))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1144))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 398))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1170))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 14))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1170))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 142))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1170))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 270))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1170))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 398))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1183))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 7))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1183))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 135))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1183))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 263))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1183))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 391))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1209))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 7))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1209))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 135))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1209))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 263))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1209))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 391))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1235))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 7))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1235))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 135))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1235))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 263))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1235))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 391))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1261))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 7))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1261))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 135))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1261))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 263))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1261))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 391))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1287))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 7))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1287))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 135))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1287))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 263))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1287))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 391))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1313))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 7))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1313))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 135))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1313))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 263))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1313))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 391))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1339))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 7))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1339))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 135))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1339))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 263))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1339))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 391))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1183))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 15))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1183))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 143))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1183))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 271))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1183))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 399))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1209))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 15))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1209))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 143))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1209))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 271))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1209))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 399))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1235))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 15))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1235))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 143))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1235))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 271))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1235))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 399))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1261))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 15))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1261))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 143))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1261))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 271))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1261))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 399))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1287))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 15))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1287))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 143))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1287))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 271))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1287))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 399))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1313))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 15))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1313))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 143))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1313))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 271))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1313))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 399))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1339))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 15))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1339))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 143))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1339))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 271))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1339))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 399))]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)))] = ((1 <= (((((int)threadIdx.x) * 25) / 13) + ry_outer)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((((int)threadIdx.x) * 25) / 13) * 14)) + (ry_outer * 14)) + ((((int)threadIdx.x) * 25) % 13)) - 13))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 1))] = ((1 <= ((((((int)threadIdx.x) * 25) + 1) / 13) + ry_outer)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 1) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 1) % 13)) - 13))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 2))] = ((1 <= ((((((int)threadIdx.x) * 25) + 2) / 13) + ry_outer)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 2) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 2) % 13)) - 13))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 3))] = ((1 <= ((((((int)threadIdx.x) * 25) + 3) / 13) + ry_outer)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 3) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 3) % 13)) - 13))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 4))] = ((1 <= ((((((int)threadIdx.x) * 25) + 4) / 13) + ry_outer)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 4) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 4) % 13)) - 13))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 5))] = ((1 <= ((((((int)threadIdx.x) * 25) + 5) / 13) + ry_outer)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 5) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 5) % 13)) - 13))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 6))] = ((1 <= ((((((int)threadIdx.x) * 25) + 6) / 13) + ry_outer)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 6) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 6) % 13)) - 13))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 7))] = ((1 <= ((((((int)threadIdx.x) * 25) + 7) / 13) + ry_outer)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 7) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 7) % 13)) - 13))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 8))] = ((1 <= ((((((int)threadIdx.x) * 25) + 8) / 13) + ry_outer)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 8) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 8) % 13)) - 13))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 9))] = ((1 <= ((((((int)threadIdx.x) * 25) + 9) / 13) + ry_outer)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 9) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 9) % 13)) - 13))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 10))] = ((1 <= ((((((int)threadIdx.x) * 25) + 10) / 13) + ry_outer)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 10) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 10) % 13)) - 13))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 11))] = ((1 <= ((((((int)threadIdx.x) * 25) + 11) / 13) + ry_outer)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 11) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 11) % 13)) - 13))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 12))] = ((1 <= ((((((int)threadIdx.x) * 25) + 12) / 13) + ry_outer)) ? placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 12) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 12) % 13)) - 13))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 13))] = placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((((int)threadIdx.x) * 25) / 13) * 14)) + (ry_outer * 14)) + ((((int)threadIdx.x) * 25) % 13)) + 1))];
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 14))] = placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 14) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 1) % 13)) - 13))];
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 15))] = placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 15) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 2) % 13)) - 13))];
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 16))] = placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 16) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 3) % 13)) - 13))];
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 17))] = placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 17) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 4) % 13)) - 13))];
      pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 18))] = placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 18) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 5) % 13)) - 13))];
      if (((((((int)threadIdx.x) * 25) + 19) / 169) + ((int)threadIdx.z)) < 8) {
        if (((((int)threadIdx.z) * 13) + (((((int)threadIdx.x) * 25) + 19) / 13)) < 104) {
          if (((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) < 1333) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 19))] = placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 19) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 6) % 13)) - 13))];
            }
          }
        }
      }
      if (((((((int)threadIdx.x) * 25) + 20) / 169) + ((int)threadIdx.z)) < 8) {
        if (((((int)threadIdx.z) * 13) + (((((int)threadIdx.x) * 25) + 20) / 13)) < 104) {
          if (((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) < 1332) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 20))] = placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 20) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 7) % 13)) - 13))];
            }
          }
        }
      }
      if (((((((int)threadIdx.x) * 25) + 21) / 169) + ((int)threadIdx.z)) < 8) {
        if (((((int)threadIdx.z) * 13) + (((((int)threadIdx.x) * 25) + 21) / 13)) < 104) {
          if (((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) < 1331) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 21))] = placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 21) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 8) % 13)) - 13))];
            }
          }
        }
      }
      if (((((((int)threadIdx.x) * 25) + 22) / 169) + ((int)threadIdx.z)) < 8) {
        if (((((int)threadIdx.z) * 13) + (((((int)threadIdx.x) * 25) + 22) / 13)) < 104) {
          if (((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) < 1330) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 22))] = placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 22) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 9) % 13)) - 13))];
            }
          }
        }
      }
      if (((((((int)threadIdx.x) * 25) + 23) / 169) + ((int)threadIdx.z)) < 8) {
        if (((((int)threadIdx.z) * 13) + (((((int)threadIdx.x) * 25) + 23) / 13)) < 104) {
          if (((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) < 1329) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 23))] = placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 23) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 10) % 13)) - 13))];
            }
          }
        }
      }
      if (((((((int)threadIdx.x) * 25) + 24) / 169) + ((int)threadIdx.z)) < 8) {
        if (((((int)threadIdx.z) * 13) + (((((int)threadIdx.x) * 25) + 24) / 13)) < 104) {
          if (((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) < 1328) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[((((((int)threadIdx.z) * 169) + (((int)threadIdx.x) * 25)) + 24))] = placeholder[(((((((((((int)vz) >> 3) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + ((((((int)threadIdx.x) * 25) + 24) / 13) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.x) * 25) + 11) % 13)) - 13))];
            }
          }
        }
      }
      placeholder_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)))] = placeholder1[(((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + (((((int)threadIdx.x) * 10) >> 3) * 2304)) + (rc_outer * 72)) + (((((int)threadIdx.x) * 10) & 7) * 9)) + (ry_outer * 3)) + 2))];
      placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) + 1))] = placeholder1[(((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 10) + 1) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 10) + 1) & 7) * 9)) + (ry_outer * 3)) + 2))];
      placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) + 2))] = placeholder1[(((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 10) + 2) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 10) + 2) & 7) * 9)) + (ry_outer * 3)) + 2))];
      placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) + 3))] = placeholder1[(((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 10) + 3) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 10) + 3) & 7) * 9)) + (ry_outer * 3)) + 2))];
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 10) + 4) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) < 508) {
          if (((int)threadIdx.x) < 6) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) + 4))] = placeholder1[(((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 10) + 4) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 10) + 4) & 7) * 9)) + (ry_outer * 3)) + 2))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 10) + 5) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) < 507) {
          if (((int)threadIdx.x) < 6) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) + 5))] = placeholder1[(((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 10) + 5) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 10) + 5) & 7) * 9)) + (ry_outer * 3)) + 2))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 10) + 6) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) < 506) {
          if (((int)threadIdx.x) < 6) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) + 6))] = placeholder1[(((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 10) + 6) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 10) + 6) & 7) * 9)) + (ry_outer * 3)) + 2))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 10) + 7) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) < 505) {
          if (((int)threadIdx.x) < 6) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) + 7))] = placeholder1[(((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 10) + 7) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 10) + 7) & 7) * 9)) + (ry_outer * 3)) + 2))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + ((((int)threadIdx.x) * 10) >> 3)) < 63) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) < 504) {
          if (((int)threadIdx.x) < 6) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) + 8))] = placeholder1[(((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + (((((int)threadIdx.x) * 10) >> 3) * 2304)) + (rc_outer * 72)) + (((((int)threadIdx.x) * 10) & 7) * 9)) + (ry_outer * 3)) + 2306))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 10) + 9) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) < 503) {
          if (((int)threadIdx.x) < 6) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 10)) + 9))] = placeholder1[(((((((((((int)vz) & 7) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 10) + 9) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 10) + 1) & 7) * 9)) + (ry_outer * 3)) + 2))];
          }
        }
      }
      __syncthreads();
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[((((int)threadIdx.z) * 16))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 128))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 256))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 384))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 26))] * placeholder_shared[((((int)threadIdx.z) * 16))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 26))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 128))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 26))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 256))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 26))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 384))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 52))] * placeholder_shared[((((int)threadIdx.z) * 16))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 52))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 128))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 52))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 256))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 52))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 384))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 78))] * placeholder_shared[((((int)threadIdx.z) * 16))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 78))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 128))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 78))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 256))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 78))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 384))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 104))] * placeholder_shared[((((int)threadIdx.z) * 16))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 104))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 128))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 104))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 256))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 104))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 384))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 130))] * placeholder_shared[((((int)threadIdx.z) * 16))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 130))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 128))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 130))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 256))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 130))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 384))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 156))] * placeholder_shared[((((int)threadIdx.z) * 16))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 156))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 128))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 156))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 256))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 156))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 384))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 8))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 136))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 264))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 392))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 26))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 8))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 26))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 136))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 26))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 264))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 26))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 392))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 52))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 8))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 52))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 136))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 52))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 264))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 52))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 392))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 78))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 8))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 78))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 136))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 78))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 264))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 78))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 392))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 104))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 8))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 104))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 136))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 104))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 264))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 104))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 392))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 130))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 8))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 130))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 136))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 130))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 264))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 130))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 392))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 156))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 8))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 156))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 136))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 156))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 264))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 156))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 392))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 169))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 1))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 169))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 129))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 169))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 257))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 169))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 385))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 195))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 1))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 195))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 129))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 195))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 257))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 195))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 385))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 221))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 1))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 221))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 129))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 221))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 257))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 221))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 385))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 247))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 1))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 247))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 129))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 247))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 257))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 247))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 385))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 273))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 1))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 273))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 129))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 273))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 257))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 273))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 385))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 299))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 1))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 299))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 129))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 299))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 257))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 299))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 385))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 325))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 1))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 325))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 129))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 325))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 257))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 325))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 385))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 169))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 9))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 169))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 137))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 169))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 265))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 169))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 393))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 195))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 9))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 195))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 137))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 195))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 265))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 195))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 393))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 221))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 9))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 221))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 137))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 221))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 265))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 221))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 393))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 247))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 9))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 247))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 137))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 247))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 265))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 247))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 393))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 273))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 9))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 273))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 137))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 273))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 265))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 273))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 393))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 299))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 9))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 299))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 137))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 299))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 265))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 299))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 393))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 325))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 9))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 325))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 137))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 325))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 265))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 325))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 393))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 338))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 2))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 338))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 130))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 338))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 258))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 338))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 386))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 2))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 130))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 258))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 386))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 390))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 2))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 390))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 130))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 390))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 258))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 390))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 386))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 416))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 2))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 416))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 130))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 416))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 258))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 416))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 386))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 442))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 2))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 442))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 130))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 442))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 258))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 442))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 386))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 468))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 2))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 468))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 130))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 468))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 258))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 468))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 386))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 494))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 2))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 494))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 130))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 494))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 258))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 494))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 386))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 338))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 10))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 338))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 138))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 338))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 266))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 338))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 394))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 10))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 138))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 266))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 394))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 390))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 10))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 390))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 138))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 390))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 266))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 390))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 394))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 416))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 10))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 416))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 138))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 416))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 266))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 416))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 394))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 442))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 10))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 442))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 138))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 442))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 266))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 442))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 394))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 468))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 10))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 468))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 138))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 468))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 266))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 468))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 394))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 494))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 10))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 494))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 138))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 494))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 266))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 494))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 394))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 507))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 3))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 507))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 131))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 507))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 259))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 507))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 387))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 533))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 3))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 533))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 131))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 533))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 259))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 533))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 387))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 559))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 3))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 559))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 131))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 559))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 259))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 559))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 387))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 585))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 3))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 585))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 131))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 585))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 259))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 585))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 387))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 611))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 3))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 611))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 131))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 611))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 259))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 611))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 387))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 637))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 3))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 637))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 131))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 637))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 259))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 637))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 387))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 663))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 3))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 663))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 131))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 663))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 259))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 663))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 387))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 507))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 11))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 507))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 139))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 507))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 267))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 507))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 395))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 533))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 11))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 533))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 139))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 533))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 267))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 533))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 395))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 559))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 11))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 559))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 139))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 559))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 267))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 559))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 395))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 585))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 11))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 585))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 139))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 585))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 267))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 585))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 395))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 611))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 11))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 611))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 139))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 611))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 267))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 611))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 395))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 637))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 11))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 637))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 139))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 637))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 267))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 637))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 395))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 663))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 11))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 663))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 139))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 663))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 267))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 663))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 395))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 676))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 4))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 676))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 132))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 676))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 260))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 676))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 388))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 702))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 4))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 702))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 132))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 702))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 260))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 702))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 388))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 4))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 132))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 260))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 388))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 754))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 4))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 754))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 132))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 754))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 260))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 754))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 388))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 780))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 4))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 780))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 132))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 780))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 260))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 780))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 388))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 806))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 4))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 806))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 132))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 806))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 260))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 806))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 388))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 832))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 4))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 832))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 132))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 832))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 260))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 832))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 388))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 676))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 12))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 676))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 140))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 676))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 268))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 676))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 396))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 702))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 12))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 702))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 140))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 702))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 268))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 702))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 396))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 12))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 140))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 268))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 396))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 754))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 12))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 754))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 140))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 754))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 268))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 754))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 396))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 780))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 12))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 780))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 140))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 780))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 268))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 780))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 396))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 806))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 12))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 806))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 140))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 806))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 268))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 806))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 396))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 832))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 12))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 832))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 140))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 832))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 268))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 832))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 396))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 845))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 5))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 845))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 133))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 845))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 261))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 845))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 389))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 871))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 5))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 871))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 133))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 871))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 261))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 871))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 389))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 897))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 5))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 897))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 133))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 897))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 261))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 897))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 389))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 923))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 5))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 923))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 133))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 923))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 261))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 923))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 389))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 949))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 5))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 949))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 133))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 949))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 261))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 949))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 389))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 975))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 5))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 975))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 133))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 975))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 261))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 975))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 389))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1001))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 5))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1001))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 133))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1001))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 261))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1001))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 389))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 845))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 13))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 845))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 141))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 845))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 269))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 845))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 397))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 871))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 13))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 871))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 141))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 871))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 269))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 871))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 397))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 897))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 13))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 897))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 141))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 897))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 269))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 897))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 397))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 923))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 13))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 923))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 141))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 923))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 269))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 923))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 397))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 949))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 13))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 949))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 141))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 949))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 269))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 949))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 397))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 975))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 13))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 975))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 141))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 975))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 269))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 975))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 397))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1001))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 13))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1001))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 141))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1001))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 269))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1001))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 397))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1014))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 6))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1014))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 134))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1014))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 262))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1014))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 390))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1040))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 6))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1040))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 134))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1040))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 262))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1040))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 390))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1066))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 6))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1066))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 134))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1066))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 262))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1066))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 390))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1092))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 6))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1092))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 134))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1092))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 262))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1092))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 390))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1118))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 6))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1118))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 134))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1118))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 262))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1118))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 390))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1144))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 6))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1144))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 134))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1144))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 262))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1144))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 390))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1170))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 6))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1170))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 134))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1170))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 262))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1170))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 390))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1014))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 14))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1014))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 142))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1014))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 270))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1014))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 398))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1040))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 14))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1040))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 142))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1040))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 270))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1040))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 398))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1066))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 14))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1066))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 142))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1066))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 270))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1066))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 398))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1092))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 14))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1092))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 142))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1092))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 270))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1092))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 398))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1118))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 14))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1118))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 142))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1118))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 270))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1118))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 398))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1144))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 14))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1144))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 142))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1144))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 270))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1144))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 398))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1170))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 14))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1170))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 142))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1170))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 270))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1170))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 398))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1183))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 7))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1183))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 135))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1183))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 263))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1183))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 391))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1209))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 7))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1209))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 135))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1209))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 263))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1209))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 391))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1235))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 7))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1235))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 135))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1235))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 263))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1235))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 391))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1261))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 7))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1261))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 135))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1261))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 263))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1261))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 391))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1287))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 7))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1287))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 135))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1287))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 263))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1287))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 391))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1313))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 7))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1313))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 135))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1313))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 263))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1313))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 391))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1339))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 7))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1339))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 135))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1339))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 263))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1339))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 391))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1183))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 15))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1183))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 143))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1183))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 271))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1183))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 399))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1209))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 15))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1209))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 143))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1209))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 271))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1209))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 399))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1235))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 15))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1235))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 143))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1235))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 271))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1235))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 399))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1261))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 15))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1261))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 143))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1261))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 271))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1261))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 399))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1287))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 15))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1287))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 143))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1287))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 271))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1287))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 399))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1313))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 15))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1313))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 143))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1313))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 271))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1313))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 399))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1339))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 15))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1339))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 143))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1339))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 271))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1339))] * placeholder_shared[(((((int)threadIdx.z) * 16) + 399))]));
    }
  }
  T_relu[((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)))] = max((compute[(0)] + placeholder2[((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 784))] = max((compute[(14)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 16))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 1568))] = max((compute[(28)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 32))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 2352))] = max((compute[(42)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 48))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 7))] = max((compute[(2)] + placeholder2[((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 791))] = max((compute[(16)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 16))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 1575))] = max((compute[(30)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 32))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 2359))] = max((compute[(44)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 48))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 14))] = max((compute[(4)] + placeholder2[((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 798))] = max((compute[(18)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 16))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 1582))] = max((compute[(32)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 32))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 2366))] = max((compute[(46)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 48))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 21))] = max((compute[(6)] + placeholder2[((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 805))] = max((compute[(20)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 16))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 1589))] = max((compute[(34)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 32))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 2373))] = max((compute[(48)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 48))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 28))] = max((compute[(8)] + placeholder2[((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 812))] = max((compute[(22)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 16))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 1596))] = max((compute[(36)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 32))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 2380))] = max((compute[(50)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 48))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 35))] = max((compute[(10)] + placeholder2[((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 819))] = max((compute[(24)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 16))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 1603))] = max((compute[(38)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 32))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 2387))] = max((compute[(52)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 48))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 42))] = max((compute[(12)] + placeholder2[((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 826))] = max((compute[(26)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 16))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 1610))] = max((compute[(40)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 32))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 2394))] = max((compute[(54)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 48))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 49))] = max((compute[(1)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 1))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 833))] = max((compute[(15)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 17))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 1617))] = max((compute[(29)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 33))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 2401))] = max((compute[(43)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 49))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 56))] = max((compute[(3)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 1))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 840))] = max((compute[(17)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 17))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 1624))] = max((compute[(31)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 33))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 2408))] = max((compute[(45)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 49))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 63))] = max((compute[(5)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 1))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 847))] = max((compute[(19)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 17))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 1631))] = max((compute[(33)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 33))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 2415))] = max((compute[(47)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 49))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 70))] = max((compute[(7)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 1))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 854))] = max((compute[(21)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 17))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 1638))] = max((compute[(35)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 33))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 2422))] = max((compute[(49)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 49))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 77))] = max((compute[(9)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 1))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 861))] = max((compute[(23)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 17))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 1645))] = max((compute[(37)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 33))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 2429))] = max((compute[(51)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 49))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 84))] = max((compute[(11)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 1))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 868))] = max((compute[(25)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 17))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 1652))] = max((compute[(39)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 33))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 2436))] = max((compute[(53)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 49))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 91))] = max((compute[(13)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 1))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 875))] = max((compute[(27)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 17))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 1659))] = max((compute[(41)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 33))]), 0.000000e+00f);
  T_relu[(((((((int)vz) * 3136) + (((int)threadIdx.z) * 98)) + ((int)threadIdx.x)) + 2443))] = max((compute[(55)] + placeholder2[(((((((int)vz) & 7) * 64) + (((int)threadIdx.z) * 2)) + 49))]), 0.000000e+00f);

    }
}

int main(int argc, char *argv[]) {
    // init device
    if (argc < 2) {
        printf("args num error! argc:%d", argc);
        exit(1);
    }
    int gpu_no = atoi(argv[1]);
    checkCudaErrors(hipSetDevice(gpu_no));

    // allocate stream
    int num_streams = 2;
    hipStream_t streams[num_streams];
    for (int i = 0; i < num_streams; i++) {
        checkCudaErrors(hipStreamCreate(&streams[i]));
    }


    // allocate flag
    int *flag = new int[FLAG_LENGTH];
    int *g_flag;
    for (int i = 0; i < FLAG_LENGTH; ++i) {
        flag[i] = 0;
    }
    checkCudaErrors(hipMalloc((void **)&g_flag, sizeof(int) * FLAG_LENGTH));
    checkCudaErrors(hipMemcpy(g_flag, flag, sizeof(int) * FLAG_LENGTH, hipMemcpyHostToDevice));

    int *flag_ = new int[FLAG_LENGTH];
    int *g_flag_;
    for (int i = 0; i < FLAG_LENGTH; ++i) {
        flag_[i] = 0;
    }
    checkCudaErrors(hipMalloc((void **)&g_flag_, sizeof(int) * FLAG_LENGTH));
    checkCudaErrors(hipMemcpy(g_flag_, flag_, sizeof(int) * FLAG_LENGTH, hipMemcpyHostToDevice));

    //prepare parm for kernel 1
    int *workers = new int[80];
    for(int i=0;i<80;i++)
    {
    workers[i]=0;
    }
    int *g_worker;
    checkCudaErrors(hipMalloc((void **)&g_worker, sizeof(int) * 80));
    checkCudaErrors(hipMemcpy( g_worker,workers, sizeof(int) * 80, hipMemcpyHostToDevice));


    float *placeholder0 = new float[1605632];
    for(int i=0;i<1605632;i++)
    {
    placeholder0[i]=1;
    }
    float *g_ph0;
    checkCudaErrors(hipMalloc((void **)&g_ph0, sizeof(float) * 1605632));
    checkCudaErrors(hipMemcpy(g_ph0, placeholder0, sizeof(float) * 1605632, hipMemcpyHostToDevice));

    float *placeholder1 = new float[1179648];
    for(int i=0;i<1179648;i++)
    {
    placeholder1[i]=0;
    }
    float *g_ph1;
    checkCudaErrors(hipMalloc((void **)&g_ph1, sizeof(float) * 1179648));
    checkCudaErrors(hipMemcpy(g_ph1, placeholder1, sizeof(float) * 1179648, hipMemcpyHostToDevice));

    float *placeholder2 = new float[802816];
    for(int i=0;i<802816;i++)
    {
    placeholder2[i]=3;
    }
    float *g_ph2;
    checkCudaErrors(hipMalloc((void **)&g_ph2, sizeof(float) * 802816));
    checkCudaErrors(hipMemcpy(g_ph2, placeholder2, sizeof(float) * 802816, hipMemcpyHostToDevice));


    float *placeholder3 = new float[512];
    for(int i=0;i<512;i++)
    {
    placeholder3[i]=5;
    }
    float *g_ph3;
    hipMalloc((void **)&g_ph3, sizeof(float) * 512);
    hipMemcpy(g_ph3, placeholder3, sizeof(float) * 512, hipMemcpyHostToDevice);


    dim3 Dim_block = dim3(LAUNCH_BLOCKX, LAUNCH_BLOCKY, LAUNCH_BLOCKZ);
    dim3 Dim_thread = dim3(LAUNCH_THREADX, LAUNCH_THREADY, LAUNCH_THREADZ);

    printf("hello?");
    // launch kernel
    fused_nn_conv2d_add_multiply_add_nn_relu_kernel0<<<Dim_block, Dim_thread, 0, streams[0]>>>(g_worker,1, g_flag, g_ph0, g_ph1, g_ph2, g_ph3);

    hipDeviceSynchronize();

    checkCudaErrors(hipMemcpy(placeholder2, g_ph2,sizeof(float) * 802816, hipMemcpyDeviceToHost));
    printf("hello3?\n");
    for(int j=0;j<784;j++)
    {
    if(j%10==0)
    {
    printf("\n");
    }
    printf("%f  ",placeholder2[1024*j+j]);
    }

    printf("\n");
    checkCudaErrors(hipMemcpy(workers,g_worker,sizeof(int) * 80, hipMemcpyDeviceToHost));
    for(int j=0;j<80;j++)
    {
    if(j%10==0&&j!=0)
    {
    printf("\n");
    }
    printf("%d  ",workers[j]);
    }
    printf("\n");
}