#include <stdio.h>
#include <stdlib.h>
#include<hip/hip_runtime.h>

#define LAUNCH_THREADX 7
#define LAUNCH_THREADY 1
#define LAUNCH_THREADZ 4
#define LAUNCH_BLOCKX 1
#define ORI_BLOCKX 1
#define LAUNCH_BLOCKY 1
#define ORI_BLOCKY 1
#define ORI_BLOCKZ 512
#define LAUNCH_BLOCKZ ORI_BLOCKZ * 5 // 5是额外部分，满足多层覆盖
#define SM_NUM 32
#define WORKER_NUM_PERSM 24
#define BLOCK_NUM LAUNCH_BLOCKZ * LAUNCH_BLOCKY * LAUNCH_BLOCKX
#define FLAG_LENGTH 65535
#define FLAG_BLOCK_BASE 0
#define FLAG_SM_BASE (FLAG_BLOCK_BASE + BLOCK_NUM)
#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)
// nvcc -arch=native smtest.cu -o smtest

#define GPU_RETURN_STATUS(cmd) \
{ \
    hipError_t result = cmd; \
    if (result != hipSuccess) { \
        std::cout << #cmd " error, return code:" << result << " | " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); \
    } \
}

// These are the inline versions for all of the SDK helper functions
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    const char *errorStr = NULL;
    errorStr = hipGetErrorString(err);
    fprintf(stderr,
            "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, "
            "line %i.\n",
            err, errorStr, file, line);
    exit(EXIT_FAILURE);
  }
}

__device__ uint get_smid(void) {

    uint ret;
  
    asm("mov.u32 %0, %smid;" : "=r"(ret) );
  
    return ret;
  
}

__global__ void workload(int *flag) {
    int n1 = 15.6, n2 = 64.9, n3 = 134.7;
    int smid = get_smid();
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0) atomicAdd(flag + smid, 1);
    for (int i = 0; i < 50000; i++) {
        n1=sinf(n1);
        n2=n3/n2;
    }
    __syncthreads();
}


int main(int argc, char *argv[]) {
    // init device
    if (argc < 2) {
        printf("args num error! argc:%d", argc);
        exit(1);
    }
    int gpu_no = atoi(argv[1]);
    checkCudaErrors(hipSetDevice(gpu_no));

    // allocate stream
    int num_streams = 2;
    hipStream_t streams[num_streams];
    for (int i = 0; i < num_streams; i++) {
        checkCudaErrors(hipStreamCreate(&streams[i]));
    }
    // allocate flag
    int *flag = new int[FLAG_LENGTH];
    int *g_flag;
    for (int i = 0; i < FLAG_LENGTH; ++i) {
        flag[i] = 0;
    }
    checkCudaErrors(hipMalloc((void **)&g_flag, sizeof(int) * FLAG_LENGTH));
    checkCudaErrors(hipMemcpy(g_flag, flag, sizeof(int) * FLAG_LENGTH, hipMemcpyHostToDevice));


    dim3 Dim_block = dim3(LAUNCH_BLOCKX, LAUNCH_BLOCKY, LAUNCH_BLOCKZ);
    dim3 Dim_thread = dim3(LAUNCH_THREADX, LAUNCH_THREADY, LAUNCH_THREADZ);
    // launch kernel
    workload<<<Dim_block, Dim_thread, 0, streams[0]>>>(g_flag);

    hipDeviceSynchronize();
    checkCudaErrors(hipMemcpy(flag, g_flag, sizeof(int) * FLAG_LENGTH, hipMemcpyDeviceToHost));
    for (int i = 0; i < 100; ++i) {
        printf("smid-num %d-%d\n", i, flag[i]);
    }

    
}