
#include <hip/hip_runtime.h>
__global__ void fused_nn_conv2d_add_nn_relu_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
    float compute[1];
    __shared__ float pad_temp_shared[180];
    __shared__ float placeholder_shared[576];
    compute[(0)] = 0.000000e+00f;
    for (int rc_outer = 0; rc_outer < 64; ++rc_outer) {
      __syncthreads();
      if (((((int)threadIdx.z) * 12) + (((int)threadIdx.x) * 2)) < 180) {
        if (((int)threadIdx.x) < 6) {
          pad_temp_shared[(((((int)threadIdx.z) * 12) + (((int)threadIdx.x) * 2)))] = (((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.z) * 12) + (((int)threadIdx.x) * 2)) % 45) / 15))) && (1 <= (((((int)threadIdx.z) * 12) + (((int)threadIdx.x) * 2)) % 15))) ? placeholder[(((((((rc_outer * 784) + ((((((int)threadIdx.z) * 12) + (((int)threadIdx.x) * 2)) / 45) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.z) * 12) + (((int)threadIdx.x) * 2)) % 45) / 15) * 14)) + (((((int)threadIdx.z) * 12) + (((int)threadIdx.x) * 2)) % 15)) - 15))] : 0.000000e+00f);
        }
      }
      if (((((int)threadIdx.z) * 12) + (((int)threadIdx.x) * 2)) < 179) {
        if (((int)threadIdx.x) < 6) {
          pad_temp_shared[((((((int)threadIdx.z) * 12) + (((int)threadIdx.x) * 2)) + 1))] = (((1 <= ((((int)blockIdx.y) * 2) + (((((((int)threadIdx.z) * 12) + (((int)threadIdx.x) * 2)) + 1) % 45) / 15))) && (1 <= ((((((int)threadIdx.z) * 12) + (((int)threadIdx.x) * 2)) + 1) % 15))) ? placeholder[(((((((rc_outer * 784) + (((((((int)threadIdx.z) * 12) + (((int)threadIdx.x) * 2)) + 1) / 45) * 196)) + (((int)blockIdx.y) * 28)) + ((((((((int)threadIdx.z) * 12) + (((int)threadIdx.x) * 2)) + 1) % 45) / 15) * 14)) + ((((((int)threadIdx.z) * 12) + (((int)threadIdx.x) * 2)) + 1) % 15)) - 15))] : 0.000000e+00f);
        }
      }
      if (((((int)threadIdx.x) / 6) + ((int)threadIdx.z)) < 16) {
        if (((((int)threadIdx.z) * 4) + ((((int)threadIdx.x) * 2) / 3)) < 64) {
          if (((((int)threadIdx.z) * 12) + (((int)threadIdx.x) * 2)) < 192) {
            if (((((int)threadIdx.z) * 36) + (((int)threadIdx.x) * 6)) < 576) {
              if (((int)threadIdx.x) < 6) {
                placeholder_shared[(((((int)threadIdx.z) * 36) + (((int)threadIdx.x) * 6)))] = placeholder1[(((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 36)) + (((int)threadIdx.x) * 6)))];
              }
            }
          }
        }
      }
      if (((((int)threadIdx.x) / 6) + ((int)threadIdx.z)) < 16) {
        if (((((int)threadIdx.z) * 4) + ((((int)threadIdx.x) * 2) / 3)) < 64) {
          if (((((int)threadIdx.z) * 12) + (((int)threadIdx.x) * 2)) < 192) {
            if (((((int)threadIdx.z) * 36) + (((int)threadIdx.x) * 6)) < 575) {
              if (((int)threadIdx.x) < 6) {
                placeholder_shared[((((((int)threadIdx.z) * 36) + (((int)threadIdx.x) * 6)) + 1))] = placeholder1[((((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 36)) + (((int)threadIdx.x) * 6)) + 1))];
              }
            }
          }
        }
      }
      if (((((int)threadIdx.x) / 6) + ((int)threadIdx.z)) < 16) {
        if (((((int)threadIdx.z) * 4) + ((((int)threadIdx.x) * 2) / 3)) < 64) {
          if (((((int)threadIdx.z) * 12) + (((int)threadIdx.x) * 2)) < 192) {
            if (((((int)threadIdx.z) * 36) + (((int)threadIdx.x) * 6)) < 574) {
              if (((int)threadIdx.x) < 6) {
                placeholder_shared[((((((int)threadIdx.z) * 36) + (((int)threadIdx.x) * 6)) + 2))] = placeholder1[((((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 36)) + (((int)threadIdx.x) * 6)) + 2))];
              }
            }
          }
        }
      }
      if (((((((int)threadIdx.x) * 2) + 1) / 12) + ((int)threadIdx.z)) < 16) {
        if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 2) + 1) / 3)) < 64) {
          if (((((int)threadIdx.z) * 12) + (((int)threadIdx.x) * 2)) < 191) {
            if (((((int)threadIdx.z) * 36) + (((int)threadIdx.x) * 6)) < 573) {
              if (((int)threadIdx.x) < 6) {
                placeholder_shared[((((((int)threadIdx.z) * 36) + (((int)threadIdx.x) * 6)) + 3))] = placeholder1[((((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 36)) + (((int)threadIdx.x) * 6)) + 3))];
              }
            }
          }
        }
      }
      if (((((((int)threadIdx.x) * 2) + 1) / 12) + ((int)threadIdx.z)) < 16) {
        if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 2) + 1) / 3)) < 64) {
          if (((((int)threadIdx.z) * 12) + (((int)threadIdx.x) * 2)) < 191) {
            if (((((int)threadIdx.z) * 36) + (((int)threadIdx.x) * 6)) < 572) {
              if (((int)threadIdx.x) < 6) {
                placeholder_shared[((((((int)threadIdx.z) * 36) + (((int)threadIdx.x) * 6)) + 4))] = placeholder1[((((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 36)) + (((int)threadIdx.x) * 6)) + 4))];
              }
            }
          }
        }
      }
      if (((((((int)threadIdx.x) * 2) + 1) / 12) + ((int)threadIdx.z)) < 16) {
        if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 2) + 1) / 3)) < 64) {
          if (((((int)threadIdx.z) * 12) + (((int)threadIdx.x) * 2)) < 191) {
            if (((((int)threadIdx.z) * 36) + (((int)threadIdx.x) * 6)) < 571) {
              if (((int)threadIdx.x) < 6) {
                placeholder_shared[((((((int)threadIdx.z) * 36) + (((int)threadIdx.x) * 6)) + 5))] = placeholder1[((((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 36)) + (((int)threadIdx.x) * 6)) + 5))];
              }
            }
          }
        }
      }
      __syncthreads();
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[((((int)threadIdx.z) * 36))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 1))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 2))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 2))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 15))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 3))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 16))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 4))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 17))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 5))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 30))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 6))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 31))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 7))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 32))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 8))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 45))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 9))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 46))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 10))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 47))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 11))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 60))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 12))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 61))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 13))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 62))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 14))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 75))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 15))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 76))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 16))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 77))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 17))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 90))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 18))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 91))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 19))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 92))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 20))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 105))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 21))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 106))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 22))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 107))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 23))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 120))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 24))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 121))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 25))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 122))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 26))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 135))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 27))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 136))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 28))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 137))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 29))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 150))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 30))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 151))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 31))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 152))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 32))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 165))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 33))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 166))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 34))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 167))] * placeholder_shared[(((((int)threadIdx.z) * 36) + 35))]));
    }
    T_relu[(((((((int)blockIdx.z) * 784) + (((int)threadIdx.z) * 49)) + (((int)blockIdx.y) * 7)) + ((int)threadIdx.x)))] = max((compute[(0)] + placeholder2[(((((int)blockIdx.z) * 16) + ((int)threadIdx.z)))]), 0.000000e+00f);
  }