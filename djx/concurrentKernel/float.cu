
#include <hip/hip_runtime.h>
extern "C" __device__ uint get_smid(void) {

    uint ret;
  
    asm("mov.u32 %0, %smid;" : "=r"(ret) );
  
    return ret;
  
}
extern "C" __global__ void matrixMulCpu(int *sm)
{   sm[blockIdx.z] = get_smid();
    float sum = 0.0f;
    for (int time = 0; time <= 100000000; ++time) {
        for(float i = 0.0f; i < 10000000.0f; i+=1.0f)
        {
            for(float j = 0.0f; j < 10000000.0f; j+=1.0f)
            {
                for(float l = 0.0f; l < 1000000.0f; l+=1.0f)
                {
                    sum += (i * j + l) * (i * l + j) * blockIdx.z * blockIdx.z;
                    sum -= (i * j) * (i * l) * blockIdx.z * blockIdx.z;
                }
                sum = 0.0f;
            }
        }
    }
}