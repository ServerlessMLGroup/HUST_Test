#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "kernel.cu"

#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)

// These are the inline versions for all of the SDK helper functions
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    const char *errorStr = NULL;
    hipDrvGetErrorString(err, &errorStr);
    fprintf(stderr,
            "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, "
            "line %i.\n",
            err, errorStr, file, line);
    exit(EXIT_FAILURE);
  }
}

int main(int argc, char **argv) {
    int cuda_device = 0;
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDevice(&cuda_device));
    int nstreams = 1;
  
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));
  
    if ((deviceProp.concurrentKernels == 0)) {
      printf("> GPU does not support concurrent kernel execution\n");
      printf("  CUDA kernel runs will be serialized\n");
    }
  
    printf("> Detected Compute SM %d.%d hardware with %d multi-processors\n",
           deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);
    
    // allocate host memory
    float *arg52 = 0; 
    checkCudaErrors(hipHostMalloc((void **)&arg52, 50176 * sizeof(float)));
    
    float *arg53 = 0; 
    checkCudaErrors(hipHostMalloc((void **)&arg53, 1179648 * sizeof(float)));

    float *arg55 = 0; 
    checkCudaErrors(hipHostMalloc((void **)&arg55, 25088 * sizeof(float)));

    float *arg54 = 0; 
    checkCudaErrors(hipHostMalloc((void **)&arg54, 512 * sizeof(float)));


    float *d_arg52 = 0; 
    checkCudaErrors(hipMalloc((void **)&arg52, 50176 * sizeof(float)));
    
    float *d_arg53 = 0; 
    checkCudaErrors(hipMalloc((void **)&arg53, 1179648 * sizeof(float)));

    float *d_arg55 = 0; 
    checkCudaErrors(hipMalloc((void **)&arg55, 25088 * sizeof(float)));

    float *d_arg54 = 0; 
    checkCudaErrors(hipMalloc((void **)&arg54, 512 * sizeof(float)));

    // allocate and initialize an array of stream handles
    hipStream_t *streams =
     (hipStream_t *)malloc(nstreams * sizeof(hipStream_t));
    
    for (int i = 0; i < nstreams; i++) {
        checkCudaErrors(hipStreamCreate(&(streams[i])));
    }

    // create CUDA event handles
    hipEvent_t start_event, stop_event;
    checkCudaErrors(hipEventCreate(&start_event));
    checkCudaErrors(hipEventCreate(&stop_event));

    checkCudaErrors(hipEventRecord(start_event, streams[0]));

    // Record the stop event
    checkCudaErrors(hipEventRecord(stop_event, streams[0]));

    // Wait for the stop event to complete
    checkCudaErrors(hipEventSynchronize(stop_event));


}