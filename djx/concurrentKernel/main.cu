#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "kernel.cu"

#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)

// These are the inline versions for all of the SDK helper functions
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    const char *errorStr = NULL;
    hipDrvGetErrorString(err, &errorStr);
    fprintf(stderr,
            "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, "
            "line %i.\n",
            err, errorStr, file, line);
    exit(EXIT_FAILURE);
  }
}

int main(int argc, char **argv) {
    int cuda_device = 0;
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDevice(&cuda_device));
    int nstreams = 1;
  
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));
  
    if ((deviceProp.concurrentKernels == 0)) {
      printf("> GPU does not support concurrent kernel execution\n");
      printf("  CUDA kernel runs will be serialized\n");
    }
  
    printf("> Detected Compute SM %d.%d hardware with %d multi-processors\n",
           deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);
    
    // allocate host memory
    std::vector<hipDeviceptr_t*> args;
    size_t storage_size = 50176 * sizeof(float);
    hipDeviceptr_t device_ptr;
    std::vector<char> temp;
    temp.resize(storage_size, 0);
    checkCudaErrors(hipMalloc((hipDeviceptr_t*)&device_ptr, storage_size)); // 52
    checkCudaErrors(hipMemcpyHtoD(device_ptr, temp.data(), storage_size)); 
    args.push_back(&device_ptr);

    storage_size = 1179648 * sizeof(float);
    temp.resize(storage_size, 0);
    checkCudaErrors(hipMalloc((hipDeviceptr_t*)&device_ptr, storage_size)); // 53
    checkCudaErrors(hipMemcpyHtoD(device_ptr, temp.data(), storage_size));
    args.push_back(&device_ptr);

    storage_size = 25088 * sizeof(float);
    temp.resize(storage_size, 0);
    checkCudaErrors(hipMalloc((hipDeviceptr_t*)&device_ptr, storage_size)); // 55, ouput
    checkCudaErrors(hipMemcpyHtoD(device_ptr, temp.data(), storage_size));
    args.push_back(&device_ptr);

    storage_size = 512 * sizeof(float);
    temp.resize(storage_size, 0);
    checkCudaErrors(hipMalloc((hipDeviceptr_t*)&device_ptr, storage_size)); // 54
    checkCudaErrors(hipMemcpyHtoD(device_ptr, temp.data(), storage_size));
    args.push_back(&device_ptr);

    std::vector<float> input52(50176);
    for (size_t i = 0; i < 50176; i++)
        input52[i] = 10.0;
    std::vector<float> input53(1179648);
    for (size_t i = 0; i < 1179648; i++)
        input53[i] = 10.0;
    std::vector<float> input54(512);
    for (size_t i = 0; i < 512; i++)
        input54[i] = 10.0;
    
    checkCudaErrors(hipMemcpyHtoD(
      (hipDeviceptr_t)args[0], (void*)input52.data()
    ))
    checkCudaErrors(hipMemcpyHtoD(
      (hipDeviceptr_t)args[1], (void*)input53.data()
    ))
    checkCudaErrors(hipMemcpyHtoD(
      (hipDeviceptr_t)args[3], (void*)input54.data()
    ))

    fused_nn_conv2d_add_nn_relu_kernel0<<<224, 112, 0, 0>>>(args[0], args[1], args[2], args[3]);
    
    // float *arg53 = 0; 
    // checkCudaErrors(hipHostMalloc((void **)&arg53, 1179648 * sizeof(float)));

    // float *arg55 = 0; 
    // checkCudaErrors(hipHostMalloc((void **)&arg55, 25088 * sizeof(float)));

    // float *arg54 = 0; 
    // checkCudaErrors(hipHostMalloc((void **)&arg54, 512 * sizeof(float)));


    // float *d_arg52 = 0; 
    // checkCudaErrors(hipMalloc((void **)&arg52, 50176 * sizeof(float)));
    
    // float *d_arg53 = 0; 
    // checkCudaErrors(hipMalloc((void **)&arg53, 1179648 * sizeof(float)));

    // float *d_arg55 = 0; 
    // checkCudaErrors(hipMalloc((void **)&arg55, 25088 * sizeof(float)));

    // float *d_arg54 = 0; 
    // checkCudaErrors(hipMalloc((void **)&arg54, 512 * sizeof(float)));

    // allocate and initialize an array of stream handles


    // *****
    // hipStream_t *streams =
    //  (hipStream_t *)malloc(nstreams * sizeof(hipStream_t));
    
    // for (int i = 0; i < nstreams; i++) {
    //     checkCudaErrors(hipStreamCreate(&(streams[i])));
    // }

    // // create CUDA event handles
    // hipEvent_t start_event, stop_event;
    // checkCudaErrors(hipEventCreate(&start_event));
    // checkCudaErrors(hipEventCreate(&stop_event));

    // checkCudaErrors(hipEventRecord(start_event, streams[0]));

    // // Record the stop event
    // checkCudaErrors(hipEventRecord(stop_event, streams[0]));

    // // Wait for the stop event to complete
    // checkCudaErrors(hipEventSynchronize(stop_event));


}