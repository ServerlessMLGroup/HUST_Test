#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)
// nvcc -arch=native ex.cu -o ex_sleep
// 1:540
#define GPU_RETURN_STATUS(cmd) \
{ \
    hipError_t result = cmd; \
    if (result != hipSuccess) { \
        std::cout << #cmd " error, return code:" << result << " | " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); \
    } \
}

// These are the inline versions for all of the SDK helper functions
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    const char *errorStr = NULL;
    errorStr = hipGetErrorString(err);
    fprintf(stderr,
            "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, "
            "line %i.\n",
            err, errorStr, file, line);
    exit(EXIT_FAILURE);
  }
}

__device__ uint get_smid(void) {

    uint ret;
  
    asm("mov.u32 %0, %smid;" : "=r"(ret) );
  
    return ret;
  
}

__device__ uint get_warpid(void) {

	    uint ret;
	      
	        asm("mov.u32 %0, %warpid;" : "=r"(ret) );
		  
		    return ret;
		      
}

__global__ void kernel(float n1, float n2, float n3, long long unsigned ** times, int stop, int* flag, long long unsigned * sleep_sm) {
	unsigned long long mclk; 
	//if (threadIdx.x %16 == 0) {
		asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk));
		times[blockIdx.x][threadIdx.x] = (mclk) / 1000; // us
	//}
        //__syncthreads();
	//for (int i = 0; i < 1000; i++) {
	//	n1=sinf(n1);
	//	n2=n3/n2;
	//}
	 __syncthreads();
	for (int i = 0; i < stop; i++) {
		n1=sinf(n1);
		n2=n3/n2;
	}
        __syncthreads();
	// flag[0] = 1在此在ms级别无变化
	//if (threadIdx.x %16 == 0) {
		unsigned long long mclk2;
		asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk2));
		times[blockIdx.x + 80][threadIdx.x] = (mclk2) / 1000; // us
	//}
    // flag[0] = 1;
    // __syncthreads();
    // flag[0] = 1;
}

__global__ void kernel_(float n1, float n2, float n3, long long unsigned * times, int stop, int* flag, long long unsigned * sleep_sm) {

	unsigned long long mclk; 
	if (threadIdx.x == 0) {
		asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk));
		times[blockIdx.x] = (mclk) / 1000; // us
	}
        __syncthreads();
	for (int i = 0; i < 1000; i++) {
		n1=sinf(n1);
		n2=n3/n2;
	}
	for (int i = 0; i < stop; i++) {
		n1=sinf(n1);
		n2=n3/n2;
	} 
	__syncthreads();
	if (threadIdx.x == 0) {
		unsigned long long mclk2;
		asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk2));
		times[blockIdx.x + 10] = (mclk2) / 1000; // us
	}
}

void run_kernel(int a_blocks, int b_blocks, int a_threads, int b_threads) {
	int num_streams = 2;
	hipStream_t streams[num_streams];
	for (int i = 0; i < num_streams; i++) {
		hipStreamCreate(&streams[i]);
	}
	
    // allocate resource
	long long unsigned ** h_sm_ids = (long long unsigned **)malloc(sizeof(long long unsigned *)* a_blocks * 2);
        long long unsigned * h_data = (long long unsigned *)malloc(sizeof(long long unsigned )* a_blocks * 2*512);
	//for(int i=0;i<a_blocks * 2;i++){
	//	h_sm_ids[i] = h_data + i*32 ;
	//}
	long long unsigned ** d_sm_ids ;
	long long unsigned * d_data ;
	hipMalloc((void **)&d_sm_ids, a_blocks * sizeof(long long unsigned*) * 2);
	hipMalloc((void **)&d_data, sizeof(long long unsigned) * 512 * a_blocks * 2);
	for(int i=0;i<a_blocks * 2;i++){
		 h_sm_ids[i] = d_data + i*512 ;
         }
        hipMemcpy((void*)(d_sm_ids), (void*)(h_sm_ids), a_blocks * sizeof(long long unsigned*) * 2, hipMemcpyHostToDevice);

	//size_t pitch = 32;
	//hipMalloc(&d_sm_ids, a_blocks * sizeof(long long unsigned) * 2);
	//hipMallocPitch(&d_sm_ids, &pitch, sizeof(long long unsigned) * 32, a_blocks);


	long long unsigned *h_sm_ids2 = new long long unsigned[b_blocks];
	long long unsigned *d_sm_ids2;
	hipMalloc(&d_sm_ids2, b_blocks * sizeof(long long unsigned));

    // allocate flag
    int *flag;
    int *g_flag;
    flag = (int*) malloc(1 * sizeof(int));
    flag[0] = 0;
    hipMalloc((void **)&g_flag, sizeof(int) * 1);
    hipMemcpy(g_flag, flag, sizeof(int) * 1, hipMemcpyHostToDevice);

	// allocate sleep_time
	long long unsigned *h_sleep_time = new long long unsigned[b_blocks];
	long long unsigned *d_sleep_time;
	hipMalloc(&d_sleep_time, b_blocks * sizeof(long long unsigned));

	// allocate kernel_sleep sm
	long long unsigned *h_sleep_sm = new long long unsigned[b_blocks];
	long long unsigned *d_sleep_sm;
	hipMalloc(&d_sleep_sm, b_blocks * sizeof(long long unsigned));


    // allocate warm flag
    int *flag_warm;
    int *g_flag_warm;
    flag_warm = (int*) malloc(1 * sizeof(int));
    flag_warm[0] = 0;
    hipMalloc((void **)&g_flag_warm, sizeof(int) * 1);
    hipMemcpy(g_flag_warm, flag_warm, sizeof(int) * 1, hipMemcpyHostToDevice);

    // cuda launch kernel
	dim3 Dba = dim3(a_threads,1,1);
	dim3 Dga = dim3(a_blocks,1,1);
	dim3 Dbb = dim3(b_threads,1,1);
	dim3 Dgb = dim3(b_blocks,1,1);
    // warm-up
    for (int i = 0; i < 100; ++i) {
        kernel <<<Dga, Dba, 0, streams[0]>>>(15.6, 64.9, 134.7, d_sm_ids, 8000, g_flag_warm, d_sleep_sm);
    }
	hipDeviceSynchronize();
    // test kernel
	//kernel_ <<<Dga, Dba, 0, streams[0]>>>(15.6, 64.9, 134.7, d_sm_ids, 8000, g_flag, d_sleep_sm);
    // sleep until kernel finish
	//kernel_sleep <<<Dgb, Dbb, 0, streams[1]>>>(15.6, 64.9, 134.7, d_sm_ids2, 50000, g_flag, d_sleep_time, d_sleep_sm);
	
	hipDeviceSynchronize();
	
	hipMemcpy((void*)(h_data),(void*) (d_data), a_blocks * sizeof(long long unsigned) *512* 2, hipMemcpyDeviceToHost);
	//hipMemcpy2D(h_sm_ids,sizeof(long long unsigned) * 32, d_sm_ids,pitch, sizeof(long long unsigned) * 32, a_blocks * 2 , hipMemcpyDeviceToHost);
	hipMemcpy(h_sm_ids2, d_sm_ids2, b_blocks * sizeof(long long unsigned), hipMemcpyDeviceToHost);

	hipMemcpy(h_sleep_time, d_sleep_time, b_blocks * sizeof(long long unsigned), hipMemcpyDeviceToHost);
	hipMemcpy(h_sleep_sm, d_sleep_sm, b_blocks * sizeof(long long unsigned), hipMemcpyDeviceToHost);

    long long unsigned maxm = 0, minm = 2668828023469159, max2 = 0, min_2= 2668828023469159, max_2 = 0,max_thread = 0, min_thread = 2668828023469159;
	long long unsigned maxm_e = 0, minm_e = 2668828023469159;
    printf("---1---\n");
	for (int i = 0; i < a_blocks; i++) {
		
		printf("block%d: %llu - %llu  %llu \n", i, h_data[i*512], h_data[i*512 + a_blocks*512] , h_data[i*512 + a_blocks*512]- h_data[i*512]);
        // printf("block-%d : %llu\n", i, h_sleep_sm[i]);
        //maxm = max(maxm, h_sm_ids[i][0]);
       // minm = min(minm, h_sm_ids[i][0]);
	//	maxm_e = max(maxm_e, h_sm_ids[i + a_blocks][0]);
        //minm_e = min(minm_e, h_sm_ids[i + a_blocks][0]);
	max_thread = 0;
	max2 = 0;
	min_thread = 2668828023469159;
	//min_2 = 2668828023469159;
	max_2 = max(max_2,  h_data[i*512 + a_blocks*512]- h_data[i*512]);
	min_2 = min(min_2,  h_data[i*512 + a_blocks*512]- h_data[i*512]);
		for(int j=0;j<512;j++){
			max2 = max(max2, h_data[i*512+a_blocks*512+j]- h_data[i*512+j]);
			max_thread = max(max_thread, h_data[i*512+a_blocks*512+j]);
			min_thread = min(min_thread, h_data[i*512+j]);

		}
		printf("***block%d: %llu - %llu  %llu  %llu \n", i, min_thread, max_thread , max2, max_thread-min_thread );
	}
    printf("START_TIMING:max-%llu, min-%llu\n", maxm, minm);
	printf("END_TIMING:max-%llu, min-%llu\n", maxm_e, minm_e);
	printf("DURATION:%llu, %llu\n", max_2, max_2-min_2);
	// printf("---2---\n");
	// for (int i = 0; i < b_blocks; i++) {
	// 	printf("%llu\n", h_sm_ids2[i]);
	// }
	// printf("---sleep_times---\n");
	// for (int i = 0; i < b_blocks; i++) {
	// 	printf("block-%d : %llu\n", i, h_sleep_time[i]);
	// }
	// printf("---sm---\n");
	// for (int i = 0; i < b_blocks; ++i) {
	// 	printf("block-%d : %llu\n", i, h_sleep_sm[i]);
	// }
	
	hipFree(d_sm_ids);
	hipFree(d_sm_ids2);

}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        printf("args num error! argc:%d", argc);
        exit(1);
    }
    int gpu_no = atoi(argv[1]);
    checkCudaErrors(hipSetDevice(gpu_no));
	run_kernel(80, 80, 512, 512);

	return 0;
}

