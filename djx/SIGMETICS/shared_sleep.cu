#include <stdio.h>
#include <stdlib.h>
#include<hip/hip_runtime.h>

#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)
// nvcc -arch=native ex.cu -o ex_sleep

#define GPU_RETURN_STATUS(cmd) \
{ \
    hipError_t result = cmd; \
    if (result != hipSuccess) { \
        std::cout << #cmd " error, return code:" << result << " | " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); \
    } \
}

// These are the inline versions for all of the SDK helper functions
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    const char *errorStr = NULL;
    errorStr = hipGetErrorString(err);
    fprintf(stderr,
            "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, "
            "line %i.\n",
            err, errorStr, file, line);
    exit(EXIT_FAILURE);
  }
}

__device__ uint get_smid(void) {

    uint ret;
  
    asm("mov.u32 %0, %smid;" : "=r"(ret) );
  
    return ret;
  
}

__global__ void kernel(float n1, float n2, float n3, long long unsigned *times, int stop, int* flag) {
	unsigned long long mclk; 
	if (threadIdx.x == 0) {
		asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk));
	}

	for (int i = 0; i < stop; i++) {
		n1=sinf(n1);
		n2=n3/n2;
	}
	__syncthreads();
	if (threadIdx.x == 0) {
		unsigned long long mclk2;
		asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk2));
		times[blockIdx.x] = (mclk2 - mclk) / 1000000;
	}
    flag[0] = 1;
}

__global__ void kernel_sleep(float n1, float n2, float n3, long long unsigned *times, int stop, int* flag) {
    #if __CUDA_ARCH__ >= 700
	while(flag[0] != 1)
		__nanosleep(1000000); // 1ms
	#else
	printf(">>> __CUDA_ARCH__ !\n");
	#endif
    unsigned long long mclk; 
	if (threadIdx.x == 0) {
		asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk));
	}
	for (int i = 0; i < stop; i++) {
		n1=sinf(n1);
		n2=n3/n2;
	}
	
	__syncthreads();
	
	if (threadIdx.x == 0) {
		unsigned long long mclk2;
		asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk2));
		times[blockIdx.x] = (mclk2 - mclk) / 1000000;
	}
}

void run_kernel(int a_blocks, int b_blocks, int a_threads, int b_threads) {
	int num_streams = 2;
	hipStream_t streams[num_streams];
	for (int i = 0; i < num_streams; i++) {
		hipStreamCreate(&streams[i]);
	}
	
    // allocate resource
	long long unsigned *h_sm_ids = new long long unsigned[a_blocks];
	long long unsigned *d_sm_ids;
	hipMalloc(&d_sm_ids, a_blocks * sizeof(long long unsigned));
	
	long long unsigned *h_sm_ids2 = new long long unsigned[b_blocks];
	long long unsigned *d_sm_ids2;
	hipMalloc(&d_sm_ids2, b_blocks * sizeof(long long unsigned));

    // allocate flag
    int *flag;
    int *g_flag;
    flag = (int*) malloc(1 * sizeof(int));
    flag[0] = 0;
    hipMalloc((void **)&g_flag, sizeof(int) * 1);
    hipMemcpy(g_flag, flag, sizeof(int) * 1, hipMemcpyHostToDevice);


    // allocate warm flag
    int *flag_warm;
    int *g_flag_warm;
    flag_warm = (int*) malloc(1 * sizeof(int));
    flag_warm[0] = 0;
    hipMalloc((void **)&g_flag_warm, sizeof(int) * 1);
    hipMemcpy(g_flag_warm, flag_warm, sizeof(int) * 1, hipMemcpyHostToDevice);

    // cuda launch kernel
	dim3 Dba = dim3(a_threads);
	dim3 Dga = dim3(a_blocks,1,1);
	dim3 Dbb = dim3(b_threads);
	dim3 Dgb = dim3(b_blocks,1,1);
    // warm-up
    for (int i = 0; i < 50; ++i) {
        kernel <<<Dga, Dba, 0, streams[0]>>>(15.6, 64.9, 134.7, d_sm_ids, 5000000, g_flag_warm);
    }
	hipDeviceSynchronize();
    // test kernel
	kernel <<<Dga, Dba, 0, streams[0]>>>(15.6, 64.9, 134.7, d_sm_ids, 5000000, g_flag);
    // sleep until kernel finish
	kernel_sleep <<<Dgb, Dbb, 0, streams[1]>>>(15.6, 64.9, 134.7, d_sm_ids2, 5000000, g_flag);
	
	hipDeviceSynchronize();
	
	hipMemcpy(h_sm_ids, d_sm_ids, a_blocks * sizeof(long long unsigned), hipMemcpyDeviceToHost);
	hipMemcpy(h_sm_ids2, d_sm_ids2, b_blocks * sizeof(long long unsigned), hipMemcpyDeviceToHost);

    printf("---1---\n");
	for (int i = 0; i < a_blocks; i++) {
		printf("%llu\n", h_sm_ids[i]);
	}
	printf("---2---\n");
	for (int i = 0; i < b_blocks; i++) {
		printf("%llu\n", h_sm_ids2[i]);
	}
	
	hipFree(d_sm_ids);
	hipFree(d_sm_ids2);

}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        printf("args num error! argc:%d", argc);
        exit(1);
    }
    int gpu_no = atoi(argv[1]);
    checkCudaErrors(hipSetDevice(gpu_no));
	run_kernel(80, 80, 512, 512);

	return 0;
}

