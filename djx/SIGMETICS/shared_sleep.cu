#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)
// nvcc -arch=native ex.cu -o ex_sleep

#define GPU_RETURN_STATUS(cmd) \
{ \
    hipError_t result = cmd; \
    if (result != hipSuccess) { \
        std::cout << #cmd " error, return code:" << result << " | " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); \
    } \
}

// These are the inline versions for all of the SDK helper functions
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    const char *errorStr = NULL;
    errorStr = hipGetErrorString(err);
    fprintf(stderr,
            "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, "
            "line %i.\n",
            err, errorStr, file, line);
    exit(EXIT_FAILURE);
  }
}

__device__ uint get_smid(void) {

    uint ret;
  
    asm("mov.u32 %0, %smid;" : "=r"(ret) );
  
    return ret;
  
}

__global__ void kernel(float n1, float n2, float n3, long long unsigned *times, int stop, int* flag) {
	unsigned long long mclk; 
	if (threadIdx.x == 0) {
		asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk));
		times[blockIdx.x] = mclk / 1000;
	}

	for (int i = 0; i < stop; i++) {
		n1=cosf(n1);
		n3=n2/n3;
	}
	__syncthreads();
	// flag[0] = 1在此在ms级别无变化
	if (threadIdx.x == 0) {
		unsigned long long mclk2;
		asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk2));
		times[blockIdx.x + 80] = mclk2 / 1000;
	}
	__syncthreads();
    flag[0] = 1;
}

__global__ void kernel_sleep(float n1, float n2, float n3, long long unsigned *times, int stop, int* flag, long long unsigned * sleep_time, long long unsigned * sleep_sm) {
	// if (threadIdx.x == 0) {
	// 	sleep_sm[blockIdx.x] = get_smid();
	// }
    #if __CUDA_ARCH__ >= 700
	while(flag[0] != 1) {
		// if (threadIdx.x == 0)
		// 	sleep_time[blockIdx.x]++;
		__nanosleep(10000); // 10us
	}
	#else
	printf(">>> __CUDA_ARCH__ !\n");
	#endif
	__syncthreads(); 
    unsigned long long mclk; 
	if (threadIdx.x == 0) {
		asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk));
		times[blockIdx.x] = mclk / 1000;
	}
	for (int i = 0; i < stop; i++) {
		n1=sinf(n1);
		n2=n3/n2;
	}
	
	__syncthreads();
	
	if (threadIdx.x == 0) {
		unsigned long long mclk2;
		asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk2));
		times[blockIdx.x + 80] = mclk2 / 1000;
	}
}

void run_kernel(int a_blocks, int b_blocks, int a_threads, int b_threads) {
	int num_streams = 2;
	hipStream_t streams[num_streams];
	for (int i = 0; i < num_streams; i++) {
		hipStreamCreate(&streams[i]);
	}
	
    // allocate resource
	long long unsigned *h_sm_ids = new long long unsigned[a_blocks * 2];
	long long unsigned *d_sm_ids;
	hipMalloc(&d_sm_ids, a_blocks * sizeof(long long unsigned) * 2);
	
	long long unsigned *h_sm_ids2 = new long long unsigned[b_blocks * 2];
	long long unsigned *d_sm_ids2;
	hipMalloc(&d_sm_ids2, b_blocks * sizeof(long long unsigned) * 2);

    // allocate flag
    int *flag;
    int *g_flag;
    flag = (int*) malloc(1 * sizeof(int));
    flag[0] = 0;
    hipMalloc((void **)&g_flag, sizeof(int) * 1);
    hipMemcpy(g_flag, flag, sizeof(int) * 1, hipMemcpyHostToDevice);

	// allocate sleep_time
	long long unsigned *h_sleep_time = new long long unsigned[b_blocks];
	long long unsigned *d_sleep_time;
	hipMalloc(&d_sleep_time, b_blocks * sizeof(long long unsigned));

	// allocate kernel_sleep sm
	long long unsigned *h_sleep_sm = new long long unsigned[b_blocks];
	long long unsigned *d_sleep_sm;
	hipMalloc(&d_sleep_sm, b_blocks * sizeof(long long unsigned));


    // allocate warm flag
    int *flag_warm;
    int *g_flag_warm;
    flag_warm = (int*) malloc(1 * sizeof(int));
    flag_warm[0] = 0;
    hipMalloc((void **)&g_flag_warm, sizeof(int) * 1);
    hipMemcpy(g_flag_warm, flag_warm, sizeof(int) * 1, hipMemcpyHostToDevice);

    // cuda launch kernel
	dim3 Dba = dim3(a_threads);
	dim3 Dga = dim3(a_blocks,1,1);
	dim3 Dbb = dim3(b_threads);
	dim3 Dgb = dim3(b_blocks,1,1);
    // warm-up
    for (int i = 0; i < 100; ++i) {
        kernel <<<Dga, Dba, 0, streams[0]>>>(15.6, 64.9, 134.7, d_sm_ids, 8000, g_flag_warm);
    }
	hipDeviceSynchronize();
    // test kernel
	kernel <<<Dga, Dba, 0, streams[0]>>>(15.6, 64.9, 134.7, d_sm_ids, 8000, g_flag);
    // sleep until kernel finish
	kernel_sleep <<<Dgb, Dbb, 0, streams[1]>>>(15.6, 64.9, 134.7, d_sm_ids2, 8000, g_flag, d_sleep_time, d_sleep_sm);
	
	hipDeviceSynchronize();
	
	hipMemcpy(h_sm_ids, d_sm_ids, a_blocks * sizeof(long long unsigned) * 2, hipMemcpyDeviceToHost);
	hipMemcpy(h_sm_ids2, d_sm_ids2, b_blocks * sizeof(long long unsigned) * 2, hipMemcpyDeviceToHost);

	hipMemcpy(h_sleep_time, d_sleep_time, b_blocks * sizeof(long long unsigned), hipMemcpyDeviceToHost);
	hipMemcpy(h_sleep_sm, d_sleep_sm, b_blocks * sizeof(long long unsigned), hipMemcpyDeviceToHost);

	long long unsigned maxm = 0, minm = 1768959725180341;
	long long unsigned maxm_e = 0, minm_e = 1768959725180341;
    printf("---1---\n");
	for (int i = 0; i < a_blocks; i++) {
        maxm = max(maxm, h_sm_ids[i]);
        minm = min(minm, h_sm_ids[i]);
		maxm_e = max(maxm_e, h_sm_ids[i + a_blocks]);
        minm_e = min(minm_e, h_sm_ids[i + a_blocks]);
	}
    printf("START_TIMING:max-%llu, min-%llu(us)\n", maxm, minm);
	printf("END_TIMING__:max-%llu, min-%llu(us)\n", maxm_e, minm_e);
	printf("DURATION:%llu(us)\n", maxm_e - maxm);
        
	maxm = 0; minm = 1768959725180341;
	maxm_e = 0; minm_e = 1768959725180341;
	printf("---2---\n");
	for (int i = 0; i < b_blocks; i++) {
	//	printf("blcok%d:%llu-%llu\n",i, h_sm_ids2[i + b_blocks]-h_sm_ids2[i]);
        maxm = max(maxm, h_sm_ids2[i]);
        minm = min(minm, h_sm_ids2[i]);
		maxm_e = max(maxm_e, h_sm_ids2[i + b_blocks]);
        minm_e = min(minm_e, h_sm_ids2[i + b_blocks]);
	}
    printf("START_TIMING:max-%llu, min-%llu(us)\n", maxm, minm);
	printf("END_TIMING__:max-%llu, min-%llu(us)\n", maxm_e, minm_e);
	printf("DURATION:%llu(us)\n", maxm_e - maxm);

	// printf("---sleep_times---\n");
	// for (int i = 0; i < b_blocks; i++) {
	// 	printf("block-%d : %llu\n", i, h_sleep_time[i]);
	// }
	// printf("---sleep_sm---\n");
	// for (int i = 0; i < b_blocks; ++i) {
	// 	printf("block-%d : %llu\n", i, h_sleep_sm[i]);
	// }
	
	hipFree(d_sm_ids);
	hipFree(d_sm_ids2);

}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        printf("args num error! argc:%d", argc);
        exit(1);
    }
    int gpu_no = atoi(argv[1]);
    checkCudaErrors(hipSetDevice(gpu_no));
	run_kernel(80, 80, 512, 512);

	return 0;
}

