#include <stdio.h>
#include <stdlib.h>
#include<hip/hip_runtime.h>

#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)
// nvcc -arch=native ex.cu -o ex_sleep

#define GPU_RETURN_STATUS(cmd) \
{ \
    hipError_t result = cmd; \
    if (result != hipSuccess) { \
        std::cout << #cmd " error, return code:" << result << " | " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); \
    } \
}

// These are the inline versions for all of the SDK helper functions
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    const char *errorStr = NULL;
    errorStr = hipGetErrorString(err);
    fprintf(stderr,
            "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, "
            "line %i.\n",
            err, errorStr, file, line);
    exit(EXIT_FAILURE);
  }
}

__global__ void kernel(float n1, float n2, float n3, long long unsigned *times, int stop) {
	unsigned long long mclk; 
	if (threadIdx.x == 0) {
		asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk));
	}

	for (int i = 0; i < stop; i++) {
		n1=sinf(n1);
		n2=n3/n2;
	}
	
	__syncthreads();
	
	if (threadIdx.x == 0) {
		unsigned long long mclk2;
		asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk2));
		times[blockIdx.x] = (mclk2 - mclk) / 1000000;
	}
}

__global__ void kernel_sleep(float n1, float n2, float n3, long long unsigned *times, int stop) {
	#if __CUDA_ARCH__ >= 700
	for (int i = 0; i < 1000; i++)
		__nanosleep(1000000); // 10ms
	#else
	printf(">>> __CUDA_ARCH__ !\n");
	#endif
	unsigned long long mclk; 
	if (threadIdx.x == 0) {
		asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk));
	}
	// #if __CUDA_ARCH__ >= 700
	// for (int i = 0; i < 1000; i++)
	// 	__nanosleep(100000000); // 100ms
	// #else
	// printf(">>> __CUDA_ARCH__ !\n");
	// #endif

	for (int i = 0; i < stop; i++) {
		n1=sinf(n1);
		n2=n3/n2;
	}
	
	__syncthreads();
	
	if (threadIdx.x == 0) {
		unsigned long long mclk2;
		asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk2));
		times[blockIdx.x] = (mclk2 - mclk) / 1000000;
	}
}

void run_kernel(int a_blocks, int b_blocks, int a_threads, int b_threads) {
	int num_streams = 2;
	hipStream_t streams[num_streams];
	for (int i = 0; i < num_streams; i++) {
		hipStreamCreate(&streams[i]);
	}
	
	long long unsigned *h_sm_ids = new long long unsigned[a_blocks];
	long long unsigned *d_sm_ids;
	hipMalloc(&d_sm_ids, a_blocks * sizeof(long long unsigned));
	
	long long unsigned *h_sm_ids2 = new long long unsigned[b_blocks];
	long long unsigned *d_sm_ids2;
	hipMalloc(&d_sm_ids2, b_blocks * sizeof(long long unsigned));

	dim3 Dba = dim3(a_threads);
	dim3 Dga = dim3(a_blocks,1,1);
	dim3 Dbb = dim3(b_threads);
	dim3 Dgb = dim3(b_blocks,1,1);
	kernel <<<Dga, Dba, 0, streams[0]>>>(15.6, 64.9, 134.7, d_sm_ids, 5000000);
	hipDeviceSynchronize();
	kernel <<<Dga, Dba, 0, streams[0]>>>(15.6, 64.9, 134.7, d_sm_ids, 5000000);
	kernel_sleep <<<Dgb, Dbb, 0, streams[1]>>>(15.6, 64.9, 134.7, d_sm_ids2, 5000000);
	
	hipDeviceSynchronize();
	
	hipMemcpy(h_sm_ids, d_sm_ids, a_blocks * sizeof(long long unsigned), hipMemcpyDeviceToHost);
	hipMemcpy(h_sm_ids2, d_sm_ids2, b_blocks * sizeof(long long unsigned), hipMemcpyDeviceToHost);

    printf("---1---\n");
	for (int i = 0; i < a_blocks; i++) {
		printf("%llu\n", h_sm_ids[i]);
	}
	printf("---2---\n");
	for (int i = 0; i < b_blocks; i++) {
		printf("%llu\n", h_sm_ids2[i]);
	}
	
	hipFree(d_sm_ids);
	hipFree(d_sm_ids2);

}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        printf("args num error! argc:%d", argc);
        exit(1);
    }
    int gpu_no = atoi(argv[1]);
    checkCudaErrors(hipSetDevice(gpu_no));
	run_kernel(80, 80, 512, 512);

	return 0;
}

