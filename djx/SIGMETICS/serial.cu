#include <stdio.h>
#include <stdlib.h>
#include<hip/hip_runtime.h>

#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)

#define GPU_RETURN_STATUS(cmd) \
{ \
    hipError_t result = cmd; \
    if (result != hipSuccess) { \
        std::cout << #cmd " error, return code:" << result << " | " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); \
    } \
}

// These are the inline versions for all of the SDK helper functions
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    const char *errorStr = NULL;
    errorStr = hipGetErrorString(err);
    fprintf(stderr,
            "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, "
            "line %i.\n",
            err, errorStr, file, line);
    exit(EXIT_FAILURE);
  }
}

__device__ uint get_smid(void) {

    uint ret;
  
    asm("mov.u32 %0, %smid;" : "=r"(ret) );
  
    return ret;
  
}

__global__ void kernel(float n1, float n2, float n3, long long unsigned *times, int stop, int* flag) {
	// if (threadIdx.x == 0) {
	// 	int sm = get_smid();
	// 	printf("kernel-sm:%d\n", sm);
	// }
	unsigned long long mclk; 
	if (threadIdx.x == 0) {
		asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk));
	}

	for (int i = 0; i < stop; i++) {
		n1=sinf(n1);
		n2=n3/n2;
	}
	__syncthreads();
	// flag[0] = 1在此在ms级别无变化
	if (threadIdx.x == 0) {
		unsigned long long mclk2;
		asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk2));
		times[blockIdx.x] = (mclk2 - mclk) / 1000000;
	}
    flag[0] = 1;
}

__global__ void kernel_sleep(float n1, float n2, float n3, long long unsigned *times, int stop, int* flag, long long unsigned * sleep_time, long long unsigned * sleep_sm) {
	if (threadIdx.x == 0) {
		sleep_sm[blockIdx.x] = get_smid();
	}
    #if __CUDA_ARCH__ >= 700
	while(flag[0] != 1) {
		if (threadIdx.x == 0)
			sleep_time[blockIdx.x]++;
		__nanosleep(1000000); // 1ms
	}
	#else
	printf(">>> __CUDA_ARCH__ !\n");
	#endif
	// __syncthreads(); // 如果加入这一行，那么稳定出现20ms计算延迟
    unsigned long long mclk; 
	if (threadIdx.x == 0) {
		asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk));
	}
	for (int i = 0; i < stop; i++) {
		n1=sinf(n1);
		n2=n3/n2;
	}
	
	__syncthreads();
	
	if (threadIdx.x == 0) {
		unsigned long long mclk2;
		asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk2));
		times[blockIdx.x] = (mclk2 - mclk) / 1000000;
	}
}

void run_kernel(int a_blocks, int b_blocks, int a_threads, int b_threads) {
	int num_streams = 2;
	hipStream_t streams[num_streams];
	for (int i = 0; i < num_streams; i++) {
		hipStreamCreate(&streams[i]);
	}
	
    hipEvent_t all_start_event, all_end_event;

    // allocate resource
	long long unsigned *h_sm_ids = new long long unsigned[a_blocks];
	long long unsigned *d_sm_ids;
	hipMalloc(&d_sm_ids, a_blocks * sizeof(long long unsigned));
	
	long long unsigned *h_sm_ids2 = new long long unsigned[b_blocks];
	long long unsigned *d_sm_ids2;
	hipMalloc(&d_sm_ids2, b_blocks * sizeof(long long unsigned));

    // allocate flag
    int *flag;
    int *g_flag;
    flag = (int*) malloc(1 * sizeof(int));
    flag[0] = 0;
    hipMalloc((void **)&g_flag, sizeof(int) * 1);
    hipMemcpy(g_flag, flag, sizeof(int) * 1, hipMemcpyHostToDevice);

	// allocate sleep_time
	long long unsigned *h_sleep_time = new long long unsigned[b_blocks];
	long long unsigned *d_sleep_time;
	hipMalloc(&d_sleep_time, b_blocks * sizeof(long long unsigned));

	// allocate kernel_sleep sm
	long long unsigned *h_sleep_sm = new long long unsigned[b_blocks];
	long long unsigned *d_sleep_sm;
	hipMalloc(&d_sleep_sm, b_blocks * sizeof(long long unsigned));


    // allocate warm flag
    int *flag_warm;
    int *g_flag_warm;
    flag_warm = (int*) malloc(1 * sizeof(int));
    flag_warm[0] = 0;
    hipMalloc((void **)&g_flag_warm, sizeof(int) * 1);
    hipMemcpy(g_flag_warm, flag_warm, sizeof(int) * 1, hipMemcpyHostToDevice);

    // cuda launch kernel
	dim3 Dba = dim3(a_threads);
	dim3 Dga = dim3(a_blocks,1,1);
	dim3 Dbb = dim3(b_threads);
	dim3 Dgb = dim3(b_blocks,1,1);
    checkCudaErrors(hipEventCreate(&all_start_event));
    checkCudaErrors(hipEventCreate(&all_end_event));
    // warm-up
    for (int i = 0; i < 10; ++i) {
        kernel <<<Dga, Dba, 0, streams[0]>>>(15.6, 64.9, 134.7, d_sm_ids, 5000000, g_flag_warm);
    }
	hipDeviceSynchronize();
    checkCudaErrors(hipEventRecord(all_start_event, 0));
    // 顺序执行，同一个stream下
	kernel <<<Dga, Dba, 0, streams[0]>>>(15.6, 64.9, 134.7, d_sm_ids, 5000000, g_flag);
	kernel_sleep <<<Dgb, Dbb, 0, streams[0]>>>(15.6, 64.9, 134.7, d_sm_ids2, 5000000, g_flag, d_sleep_time, d_sleep_sm);

    checkCudaErrors(hipEventRecord(all_end_event, 0));

    checkCudaErrors(hipEventSynchronize(all_end_event));

    float elapsed;
    checkCudaErrors(hipEventElapsedTime(&elapsed, all_start_event, all_end_event));
    printf("Total GPU Measured time for sample = %.3fms\n", elapsed); 
	
	hipMemcpy(h_sm_ids, d_sm_ids, a_blocks * sizeof(long long unsigned), hipMemcpyDeviceToHost);
	hipMemcpy(h_sm_ids2, d_sm_ids2, b_blocks * sizeof(long long unsigned), hipMemcpyDeviceToHost);

	hipMemcpy(h_sleep_time, d_sleep_time, b_blocks * sizeof(long long unsigned), hipMemcpyDeviceToHost);
	hipMemcpy(h_sleep_sm, d_sleep_sm, b_blocks * sizeof(long long unsigned), hipMemcpyDeviceToHost);

    printf("---1---\n");
	for (int i = 0; i < a_blocks; i++) {
		printf("%llu\n", h_sm_ids[i]);
	}
	printf("---2---\n");
	for (int i = 0; i < b_blocks; i++) {
		printf("%llu\n", h_sm_ids2[i]);
	}
	printf("---sleep_times---\n");
	for (int i = 0; i < b_blocks; i++) {
		printf("block-%d : %llu\n", i, h_sleep_time[i]);
	}
	printf("---sleep_sm---\n");
	for (int i = 0; i < b_blocks; ++i) {
		printf("block-%d : %llu\n", i, h_sleep_sm[i]);
	}
	
	hipFree(d_sm_ids);
	hipFree(d_sm_ids2);

}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        printf("args num error! argc:%d", argc);
        exit(1);
    }
    int gpu_no = atoi(argv[1]);
    checkCudaErrors(hipSetDevice(gpu_no));
	run_kernel(80, 80, 512, 512);

	return 0;
}

