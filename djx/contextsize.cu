#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <thread>
#include <random>
#include <ctime>
using namespace std;

enum class Unit{
    Byte, KB, MB, GB, TB, PB, EB
};


double convert(double size, Unit unit)
{
    double result = size;
    switch (unit)
    {
    case Unit::EB:
        result /= 1024;     // flow through
    case Unit::PB:
        result /= 1024;     // flow through
    case Unit::TB:
        result /= 1024;     // flow through
    case Unit::GB:
        result /= 1024;     // flow through
    case Unit::MB:
        result /= 1024;     // flow through
    case Unit::KB:
        result /= 1024;     // flow through
    case Unit::Byte:
        result /= 1;
    default:
        break;
    }
    return result;
}

void getMem() {
    size_t free, total;
    hipMemGetInfo(&free, &total);
    printf("Free mem = %.4f MB, Total = %.4f MB \n", convert(free, Unit::MB), convert(total, Unit::MB));
}

void getMembycu() {
    size_t free, total;
    int err = hipMemGetInfo(&free, &total);
    if (err) {
        cout<<"getMembycu error:"<<err<<endl;
    }
    else {
        printf("Free mem = %.4f MB, Total = %.4f MB \n", convert(free, Unit::MB), convert(total, Unit::MB));
    }
}


__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}


int main()
{
    hipSetDevice(0);
    hipCtx_t pctx;
    hipDevice_t dev;
    // getMem();
    /*
    cout<<"getDeviceCount"<<endl;
    int count;
    int err = hipGetDeviceCount(&count);
    if(err){
        cout<<"getDeviceCount error:"<<err<<endl;
        return 0;
    }
    cout<<"getDeviceCount Fininshed"<<endl;

    // cout<<"after new context1:"<<endl;
    */
    cout<<"hipCtxGetDevice"<<endl;
    int err = hipCtxGetDevice(&dev);
    if(err){
        cout<<"hipCtxGetDevice error:"<<err<<endl;
        return 0;
    }
    cout<<"basic memory"<<endl;


    cout<<"new context:"<<endl;
    err = hipCtxCreate(&pctx, hipDeviceScheduleYield, dev);
    if(err) {
        cout<<"hipCtxCreate error:"<<err<<endl;
        return 0;
    }
    getMem();


    cout<<"new context:"<<endl;
    err = hipCtxCreate(&pctx, hipDeviceScheduleYield, dev);
    if(err) {
        cout<<"hipCtxCreate error:"<<err<<endl;
        return 0;
    }
    getMem();


    cout<<"new context:"<<endl;
    err = hipCtxCreate(&pctx, hipDeviceScheduleYield, dev);
    if(err) {
        cout<<"hipCtxCreate error:"<<err<<endl;
        return 0;
    }
    getMem();

    return 0;
}
