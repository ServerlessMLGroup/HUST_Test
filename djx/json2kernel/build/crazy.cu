#include "hip/hip_runtime.h"
#include "log.h"
#include <bits/unique_ptr.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
//yy add
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <thread>
#include <math.h>
#include "unistd.h"
#include <thread>
#include <sys/time.h>
#include <unistd.h>

// #include <glog/logging.h>
//Notice
// To make some experiments, i(yy) make some changes here. Before changing, i copied all the code
// Just read the code at copymain.cpp. If some bad change were made, we can fix it by the copy
enum Status {
    Succ,
    Fail,
    NotFound,
    OutOfRange,
    Full
};
#define GPU_RETURN_STATUS(cmd) \
{ \
    hipError_t result = cmd; \
    if (result != hipSuccess) { \
        std::cout << #cmd " error, return code:" << result << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); \
    } \
}
#define RETURN_STATUS(cmd) \
{\
    Status s = cmd;\
    if (s != Status::Succ) {\
        std::cout << #cmd " error, " << __FILE__ << ":" << __LINE__ << std::endl; \
        return s;\
    }\
}


int main(int argc, char **argv) {
    if (argc < 2) {
        printf("args num error! argc:%d", argc);
    }
    int gpu_no = atoi(argv[1]);


    hipModule_t mod;
    GPU_RETURN_STATUS(hipModuleLoad(&mod, "/home/wuhao/HUST_Test/djx/json2kernel/resource/resnet18.ptx"));
    printf("load cuda kernels!\n");
    //yy add stream
    hipStream_t iofirststream;
    hipStreamCreateWithFlags(&iofirststream,0);
    hipStream_t iosecondstream;
    hipStreamCreateWithFlags(&iosecondstream,0);
    hipStream_t kefirststream;
    hipStreamCreateWithFlags(&kefirststream,0);
    hipStream_t kesecondstream;
    hipStreamCreateWithFlags(&kesecondstream,0);

    hipFunction_t kernel;
    GPU_RETURN_STATUS(
            hipModuleGetFunction(&kernel, mod, "fused_nn_conv2d_add_multiply_add_nn_relu_kernel0")
        );

    hipDeviceptr_t deviceptr0;
    hipDeviceptr_t deviceptr1;
    hipDeviceptr_t deviceptr2;
    hipDeviceptr_t deviceptr3;
    hipDeviceptr_t deviceptr4;
    hipDeviceptr_t deviceptr5;
    //check answer
    GPU_RETURN_STATUS(hipMalloc((hipDeviceptr_t*)&deviceptr0, sizeof(float)*802816));
    float *placeholder0 = new float[802816];
    for(int i=0;i<802816;i++)
    {
    placeholder0[i]=1;
    }
    GPU_RETURN_STATUS(hipMemcpyHtoDAsync((hipDeviceptr_t)deviceptr0,placeholder0, sizeof(float)*802816,iofirststream));

    GPU_RETURN_STATUS(hipMalloc((hipDeviceptr_t*)&deviceptr1, sizeof(float)*2359296));
    float *placeholder1 = new float[2359296];
    for(int i=0;i<2359296;i++)
    {
    placeholder1[i]=2;
    }
    GPU_RETURN_STATUS(hipMemcpyHtoDAsync((hipDeviceptr_t)deviceptr1,placeholder1, sizeof(float)*2359296,iofirststream));

    GPU_RETURN_STATUS(hipMalloc((hipDeviceptr_t*)&deviceptr2, sizeof(float)*802816));
    float *placeholder2 = new float[802816];
    for(int i=0;i<802816;i++)
    {
    placeholder2[i]=3;
    }
    GPU_RETURN_STATUS(hipMemcpyHtoDAsync((hipDeviceptr_t)deviceptr2,placeholder2, sizeof(float)*802816,iofirststream));

    GPU_RETURN_STATUS(hipMalloc((hipDeviceptr_t*)&deviceptr3, sizeof(float)*802816));
    float *placeholder3 = new float[802816];
    for(int i=0;i<802816;i++)
    {
    placeholder3[i]=4;
    }
    GPU_RETURN_STATUS(hipMemcpyHtoDAsync((hipDeviceptr_t)deviceptr3,placeholder3, sizeof(float)*802816,iofirststream));

    GPU_RETURN_STATUS(hipMalloc((hipDeviceptr_t*)&deviceptr4, sizeof(float)*512));
    float *placeholder4 = new float[512];
    for(int i=0;i<512;i++)
    {
    placeholder4[i]=5;
    }
    GPU_RETURN_STATUS(hipMemcpyHtoDAsync((hipDeviceptr_t)deviceptr4,placeholder4, sizeof(float)*512,iofirststream));

    GPU_RETURN_STATUS(hipMalloc((hipDeviceptr_t*)&deviceptr5, sizeof(float)*512));
    float *placeholder5 = new float[512];
    for(int i=0;i<512;i++)
    {
    placeholder5[i]=6;
    }
    GPU_RETURN_STATUS(hipMemcpyHtoDAsync((hipDeviceptr_t)deviceptr5,placeholder5, sizeof(float)*512,iofirststream));

    std::vector<hipDeviceptr_t*> extrarg;
    extrarg.push_back(&deviceptr0);
    extrarg.push_back(&deviceptr1);
    extrarg.push_back(&deviceptr2);
    extrarg.push_back(&deviceptr3);
    extrarg.push_back(&deviceptr4);
    extrarg.push_back(&deviceptr5);

    GPU_RETURN_STATUS(hipModuleLaunchKernel(kernel,
        1, 1, 512,
        7, 1, 4,
        0, kefirststream, (void **)extrarg.data(), 0 // raw_args1是json中指示的storage的下标
    ));

    hipStreamSynchronize(kefirststream);

    for(int j=0;j<784;j++)
    {
    if(j%10==0)
    {
    std::cout<<std::endl;
    }
    std::cout<<placeholder2[1024*j+j]<<" ";
    }

    printf("reset model!\n");
    model.reset();
    return 0;
}