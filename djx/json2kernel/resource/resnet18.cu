
#include <hip/hip_runtime.h>
extern "C" __global__ void fused_add_nn_relu_1_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ T_relu, float* __restrict__ placeholder, float* __restrict__ placeholder1) {
    int vx=blockIdx.x;
  int vy=blockIdx.y;
  int vz=blockIdx.z;
  int offset=0;

  if((blocknum[0]*blocknum[1]*blocknum[2])>blocksize[0])
  {
    offset=vx;
    while(offset<(blocknum[0]*blocknum[1]*blocknum[2]))
    {
    vz=(offset)/(blocknum[0]*blocknum[1]);
    vy= (offset-(vz*blocknum[0]*blocknum[1]))/blocknum[0];
    vx=offset - (vz*blocknum[0]*blocknum[1])-vy*blocknum[0];
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer < 7; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)vx) * 1024)) + ((int)threadIdx.x)) < 1605632) {
      T_relu[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)vx) * 1024)) + ((int)threadIdx.x)))] = max((placeholder[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)vx) * 1024)) + ((int)threadIdx.x)))] + placeholder1[((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)vx) * 1024)) + ((int)threadIdx.x)) % 50176) / 196))]), 0.000000e+00f);
    }
  }
    offset+=blocksize[0];
    }
  }
  else
  {
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer < 7; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)vx) * 1024)) + ((int)threadIdx.x)) < 1605632) {
      T_relu[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)vx) * 1024)) + ((int)threadIdx.x)))] = max((placeholder[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)vx) * 1024)) + ((int)threadIdx.x)))] + placeholder1[((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)vx) * 1024)) + ((int)threadIdx.x)) % 50176) / 196))]), 0.000000e+00f);
    }
  }
  }

}
extern "C" __global__ void fused_nn_conv2d_add_1_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {

}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_7_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {

}
extern "C" __global__ void fused_nn_max_pool2d_add_nn_relu_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ T_relu, float* __restrict__ placeholder1) {

}

extern "C" __global__ void fused_nn_conv2d_add_add_nn_relu_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {

}
extern "C" __global__ void fused_add_nn_relu_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ T_relu, float* __restrict__ placeholder, float* __restrict__ placeholder1) {

}
extern "C" __global__ void fused_nn_softmax_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ T_softmax_norm) {

}
extern "C" __global__ void fused_add_nn_relu_2_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ T_relu, float* __restrict__ placeholder, float* __restrict__ placeholder1) {

}
extern "C" __global__ void fused_nn_conv2d_1_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {

}

extern "C" __global__ void fused_nn_conv2d_add_add_nn_relu_1_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {

}
extern "C" __global__ void fused_nn_conv2d_add_nn_relu_2_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {

}
extern "C" __global__ void fused_nn_dense_add_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {

}
extern "C" __global__ void fused_add_10_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ T_add, float* __restrict__ placeholder, float* __restrict__ placeholder1) {

}
extern "C" __global__ void fused_nn_conv2d_add_nn_relu_1_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {

}
extern "C" __global__ void fused_nn_conv2d_add_nn_relu_4_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {

}
extern "C" __global__ void fused_nn_conv2d_add_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {

}
extern "C" __global__ void fused_nn_conv2d_add_add_nn_relu_2_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {

}
extern "C" __global__ void fused_nn_conv2d_add_2_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {

}
extern "C" __global__ void fused_nn_conv2d_3_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {

}
extern "C" __global__ void fused_nn_conv2d_add_nn_relu_6_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {

}
extern "C" __global__ void fused_nn_conv2d_add_3_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {

}
extern "C" __global__ void fused_nn_conv2d_add_nn_relu_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {

}
extern "C" __global__ void fused_nn_conv2d_2_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {

}
extern "C" __global__ void fused_nn_conv2d_add_multiply_add_nn_relu_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3, float* __restrict__ placeholder4) {

}
extern "C" __global__ void fused_nn_global_avg_pool2d_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ tensor) {

}
extern "C" __global__ void fused_nn_batch_flatten_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ tensor, float* __restrict__ placeholder) {

}
extern "C" __global__ void fused_nn_conv2d_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {

}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_5_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {

}
extern "C" __global__ void fused_add_nn_relu_3_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ T_relu, float* __restrict__ placeholder, float* __restrict__ placeholder1) {

}
extern "C" __global__ void fused_nn_conv2d_add_nn_relu_3_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {

}

























