
#include <hip/hip_runtime.h>
extern "C" __global__ void fused_add_nn_relu_1_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ T_relu, float* __restrict__ placeholder, float* __restrict__ placeholder1) {
    int vx=blockIdx.x;
  int vy=blockIdx.y;
  int vz=blockIdx.z;
  int offset=0;

  if((blocknum[0]*blocknum[1]*blocknum[2])>blocksize[0])
  {
    offset=vx;
    while(offset<(blocknum[0]*blocknum[1]*blocknum[2]))
    {
    vz=(offset)/(blocknum[0]*blocknum[1]);
    vy= (offset-(vz*blocknum[0]*blocknum[1]))/blocknum[0];
    vx=offset - (vz*blocknum[0]*blocknum[1])-vy*blocknum[0];
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer < 7; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)vx) * 1024)) + ((int)threadIdx.x)) < 1605632) {
      T_relu[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)vx) * 1024)) + ((int)threadIdx.x)))] = max((placeholder[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)vx) * 1024)) + ((int)threadIdx.x)))] + placeholder1[((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)vx) * 1024)) + ((int)threadIdx.x)) % 50176) / 196))]), 0.000000e+00f);
    }
  }
    offset+=blocksize[0];
    }
  }
  else
  {
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer < 7; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)vx) * 1024)) + ((int)threadIdx.x)) < 1605632) {
      T_relu[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)vx) * 1024)) + ((int)threadIdx.x)))] = max((placeholder[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)vx) * 1024)) + ((int)threadIdx.x)))] + placeholder1[((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)vx) * 1024)) + ((int)threadIdx.x)) % 50176) / 196))]), 0.000000e+00f);
    }
  }
  }

}
extern "C" __global__ void fused_nn_conv2d_add_1_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
     int vx=blockIdx.x;
  int vy=blockIdx.y;
  int vz=blockIdx.z;
  int offset=0;

  if((blocknum[0]*blocknum[1]*blocknum[2])>blocksize[0])
  {
    offset=vx;
    while(offset<(blocknum[0]*blocknum[1]*blocknum[2]))
    {
    vz=(offset)/(blocknum[0]*blocknum[1]);
    vy= (offset-(vz*blocknum[0]*blocknum[1]))/blocknum[0];
    vx=offset - (vz*blocknum[0]*blocknum[1])-vy*blocknum[0];
    float compute[56];
  __shared__ float pad_temp_shared[784];
  __shared__ float placeholder_shared[512];
  compute[(0)] = 0.000000e+00f;
  compute[(7)] = 0.000000e+00f;
  compute[(14)] = 0.000000e+00f;
  compute[(21)] = 0.000000e+00f;
  compute[(28)] = 0.000000e+00f;
  compute[(35)] = 0.000000e+00f;
  compute[(42)] = 0.000000e+00f;
  compute[(49)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  compute[(8)] = 0.000000e+00f;
  compute[(15)] = 0.000000e+00f;
  compute[(22)] = 0.000000e+00f;
  compute[(29)] = 0.000000e+00f;
  compute[(36)] = 0.000000e+00f;
  compute[(43)] = 0.000000e+00f;
  compute[(50)] = 0.000000e+00f;
  compute[(2)] = 0.000000e+00f;
  compute[(9)] = 0.000000e+00f;
  compute[(16)] = 0.000000e+00f;
  compute[(23)] = 0.000000e+00f;
  compute[(30)] = 0.000000e+00f;
  compute[(37)] = 0.000000e+00f;
  compute[(44)] = 0.000000e+00f;
  compute[(51)] = 0.000000e+00f;
  compute[(3)] = 0.000000e+00f;
  compute[(10)] = 0.000000e+00f;
  compute[(17)] = 0.000000e+00f;
  compute[(24)] = 0.000000e+00f;
  compute[(31)] = 0.000000e+00f;
  compute[(38)] = 0.000000e+00f;
  compute[(45)] = 0.000000e+00f;
  compute[(52)] = 0.000000e+00f;
  compute[(4)] = 0.000000e+00f;
  compute[(11)] = 0.000000e+00f;
  compute[(18)] = 0.000000e+00f;
  compute[(25)] = 0.000000e+00f;
  compute[(32)] = 0.000000e+00f;
  compute[(39)] = 0.000000e+00f;
  compute[(46)] = 0.000000e+00f;
  compute[(53)] = 0.000000e+00f;
  compute[(5)] = 0.000000e+00f;
  compute[(12)] = 0.000000e+00f;
  compute[(19)] = 0.000000e+00f;
  compute[(26)] = 0.000000e+00f;
  compute[(33)] = 0.000000e+00f;
  compute[(40)] = 0.000000e+00f;
  compute[(47)] = 0.000000e+00f;
  compute[(54)] = 0.000000e+00f;
  compute[(6)] = 0.000000e+00f;
  compute[(13)] = 0.000000e+00f;
  compute[(20)] = 0.000000e+00f;
  compute[(27)] = 0.000000e+00f;
  compute[(34)] = 0.000000e+00f;
  compute[(41)] = 0.000000e+00f;
  compute[(48)] = 0.000000e+00f;
  compute[(55)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 32; ++rc_outer) {
    for (int ry_outer = 0; ry_outer < 3; ++ry_outer) {
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)))] = ((((1 <= (((((int)vy) * 7) + (((int)threadIdx.x) >> 1)) + ry_outer)) && ((((((int)vy) * 7) + (((int)threadIdx.x) >> 1)) + ry_outer) < 15)) && (1 <= (((int)threadIdx.x) & 1))) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 15))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 1))] = ((((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 1) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 1) / 14)) + ry_outer) < 15)) && (1 <= (((((int)threadIdx.x) * 7) + 1) % 14))) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 14))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 2))] = ((((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 2) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 2) / 14)) + ry_outer) < 15)) && (1 <= (((((int)threadIdx.x) * 7) + 2) % 14))) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 13))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 3))] = ((((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 3) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 3) / 14)) + ry_outer) < 15)) && (1 <= (((((int)threadIdx.x) * 7) + 3) % 14))) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 12))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 4))] = ((((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 4) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 4) / 14)) + ry_outer) < 15)) && (1 <= (((((int)threadIdx.x) * 7) + 4) % 14))) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 11))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 5))] = ((((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 5) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 5) / 14)) + ry_outer) < 15)) && (1 <= (((((int)threadIdx.x) * 7) + 5) % 14))) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 10))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 6))] = ((((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 6) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 6) / 14)) + ry_outer) < 15)) && (1 <= (((((int)threadIdx.x) * 7) + 6) % 14))) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 9))] : 0.000000e+00f);
      if (((((int)threadIdx.z) * 8) + ((((int)threadIdx.x) * 5) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 512) {
          if (((int)threadIdx.x) < 13) {
            placeholder_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)))] = placeholder1[((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + (((((int)threadIdx.x) * 5) >> 3) * 2304)) + (rc_outer * 72)) + (((((int)threadIdx.x) * 5) & 7) * 9)) + (ry_outer * 3)))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 5) + 1) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 511) {
          if (((int)threadIdx.x) < 13) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) + 1))] = placeholder1[((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 5) + 1) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 5) + 1) & 7) * 9)) + (ry_outer * 3)))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 5) + 2) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 510) {
          if (((int)threadIdx.x) < 13) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) + 2))] = placeholder1[((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 5) + 2) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 5) + 2) & 7) * 9)) + (ry_outer * 3)))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 5) + 3) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 509) {
          if (((int)threadIdx.x) < 13) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) + 3))] = placeholder1[((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 5) + 3) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 5) + 3) & 7) * 9)) + (ry_outer * 3)))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 5) + 4) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 508) {
          if (((int)threadIdx.x) < 12) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) + 4))] = placeholder1[((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 5) + 4) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 5) + 4) & 7) * 9)) + (ry_outer * 3)))];
          }
        }
      }
      __syncthreads();
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)))] = (((1 <= (((((int)vy) * 7) + (((int)threadIdx.x) >> 1)) + ry_outer)) && ((((((int)vy) * 7) + (((int)threadIdx.x) >> 1)) + ry_outer) < 15)) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 14))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 1))] = (((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 1) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 1) / 14)) + ry_outer) < 15)) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 13))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 2))] = (((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 2) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 2) / 14)) + ry_outer) < 15)) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 12))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 3))] = (((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 3) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 3) / 14)) + ry_outer) < 15)) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 11))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 4))] = (((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 4) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 4) / 14)) + ry_outer) < 15)) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 10))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 5))] = (((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 5) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 5) / 14)) + ry_outer) < 15)) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 9))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 6))] = (((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 6) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 6) / 14)) + ry_outer) < 15)) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 8))] : 0.000000e+00f);
      if (((((int)threadIdx.z) * 8) + ((((int)threadIdx.x) * 5) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 512) {
          if (((int)threadIdx.x) < 13) {
            placeholder_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)))] = placeholder1[(((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + (((((int)threadIdx.x) * 5) >> 3) * 2304)) + (rc_outer * 72)) + (((((int)threadIdx.x) * 5) & 7) * 9)) + (ry_outer * 3)) + 1))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 5) + 1) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 511) {
          if (((int)threadIdx.x) < 13) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) + 1))] = placeholder1[(((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 5) + 1) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 5) + 1) & 7) * 9)) + (ry_outer * 3)) + 1))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 5) + 2) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 510) {
          if (((int)threadIdx.x) < 13) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) + 2))] = placeholder1[(((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 5) + 2) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 5) + 2) & 7) * 9)) + (ry_outer * 3)) + 1))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 5) + 3) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 509) {
          if (((int)threadIdx.x) < 13) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) + 3))] = placeholder1[(((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 5) + 3) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 5) + 3) & 7) * 9)) + (ry_outer * 3)) + 1))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 5) + 4) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 508) {
          if (((int)threadIdx.x) < 12) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) + 4))] = placeholder1[(((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 5) + 4) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 5) + 4) & 7) * 9)) + (ry_outer * 3)) + 1))];
          }
        }
      }
      __syncthreads();
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)))] = (((1 <= (((((int)vy) * 7) + (((int)threadIdx.x) >> 1)) + ry_outer)) && ((((((int)vy) * 7) + (((int)threadIdx.x) >> 1)) + ry_outer) < 15)) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 13))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 1))] = ((((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 1) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 1) / 14)) + ry_outer) < 15)) && ((((((int)threadIdx.x) * 7) + 1) % 14) < 13)) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 12))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 2))] = ((((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 2) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 2) / 14)) + ry_outer) < 15)) && ((((((int)threadIdx.x) * 7) + 2) % 14) < 13)) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 11))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 3))] = ((((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 3) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 3) / 14)) + ry_outer) < 15)) && ((((((int)threadIdx.x) * 7) + 3) % 14) < 13)) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 10))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 4))] = ((((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 4) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 4) / 14)) + ry_outer) < 15)) && ((((((int)threadIdx.x) * 7) + 4) % 14) < 13)) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 9))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 5))] = ((((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 5) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 5) / 14)) + ry_outer) < 15)) && ((((((int)threadIdx.x) * 7) + 5) % 14) < 13)) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 8))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 6))] = ((((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 6) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 6) / 14)) + ry_outer) < 15)) && ((((((int)threadIdx.x) * 7) + 6) % 14) < 13)) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 7))] : 0.000000e+00f);
      if (((((int)threadIdx.z) * 8) + ((((int)threadIdx.x) * 5) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 512) {
          if (((int)threadIdx.x) < 13) {
            placeholder_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)))] = placeholder1[(((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + (((((int)threadIdx.x) * 5) >> 3) * 2304)) + (rc_outer * 72)) + (((((int)threadIdx.x) * 5) & 7) * 9)) + (ry_outer * 3)) + 2))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 5) + 1) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 511) {
          if (((int)threadIdx.x) < 13) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) + 1))] = placeholder1[(((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 5) + 1) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 5) + 1) & 7) * 9)) + (ry_outer * 3)) + 2))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 5) + 2) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 510) {
          if (((int)threadIdx.x) < 13) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) + 2))] = placeholder1[(((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 5) + 2) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 5) + 2) & 7) * 9)) + (ry_outer * 3)) + 2))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 5) + 3) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 509) {
          if (((int)threadIdx.x) < 13) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) + 3))] = placeholder1[(((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 5) + 3) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 5) + 3) & 7) * 9)) + (ry_outer * 3)) + 2))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 5) + 4) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 508) {
          if (((int)threadIdx.x) < 12) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) + 4))] = placeholder1[(((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 5) + 4) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 5) + 4) & 7) * 9)) + (ry_outer * 3)) + 2))];
          }
        }
      }
      __syncthreads();
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
    }
  }
  T_add[(((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)))] = (compute[(0)] + placeholder2[(((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 1568))] = (compute[(7)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 1568))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 3136))] = (compute[(14)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 3136))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 4704))] = (compute[(21)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 4704))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 6272))] = (compute[(28)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 6272))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 7840))] = (compute[(35)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 7840))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 9408))] = (compute[(42)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 9408))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 10976))] = (compute[(49)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 10976))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 14))] = (compute[(1)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 14))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 1582))] = (compute[(8)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 1582))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 3150))] = (compute[(15)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 3150))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 4718))] = (compute[(22)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 4718))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 6286))] = (compute[(29)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 6286))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 7854))] = (compute[(36)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 7854))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 9422))] = (compute[(43)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 9422))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 10990))] = (compute[(50)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 10990))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 28))] = (compute[(2)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 28))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 1596))] = (compute[(9)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 1596))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 3164))] = (compute[(16)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 3164))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 4732))] = (compute[(23)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 4732))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 6300))] = (compute[(30)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 6300))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 7868))] = (compute[(37)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 7868))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 9436))] = (compute[(44)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 9436))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 11004))] = (compute[(51)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 11004))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 42))] = (compute[(3)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 42))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 1610))] = (compute[(10)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 1610))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 3178))] = (compute[(17)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 3178))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 4746))] = (compute[(24)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 4746))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 6314))] = (compute[(31)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 6314))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 7882))] = (compute[(38)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 7882))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 9450))] = (compute[(45)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 9450))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 11018))] = (compute[(52)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 11018))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 56))] = (compute[(4)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 56))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 1624))] = (compute[(11)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 1624))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 3192))] = (compute[(18)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 3192))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 4760))] = (compute[(25)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 4760))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 6328))] = (compute[(32)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 6328))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 7896))] = (compute[(39)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 7896))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 9464))] = (compute[(46)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 9464))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 11032))] = (compute[(53)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 11032))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 70))] = (compute[(5)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 70))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 1638))] = (compute[(12)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 1638))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 3206))] = (compute[(19)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 3206))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 4774))] = (compute[(26)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 4774))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 6342))] = (compute[(33)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 6342))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 7910))] = (compute[(40)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 7910))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 9478))] = (compute[(47)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 9478))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 11046))] = (compute[(54)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 11046))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 84))] = (compute[(6)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 84))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 1652))] = (compute[(13)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 1652))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 3220))] = (compute[(20)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 3220))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 4788))] = (compute[(27)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 4788))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 6356))] = (compute[(34)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 6356))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 7924))] = (compute[(41)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 7924))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 9492))] = (compute[(48)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 9492))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 11060))] = (compute[(55)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 11060))]);
    offset+=blocksize[0];
    }
  }
  else
  {
  float compute[56];
  __shared__ float pad_temp_shared[784];
  __shared__ float placeholder_shared[512];
  compute[(0)] = 0.000000e+00f;
  compute[(7)] = 0.000000e+00f;
  compute[(14)] = 0.000000e+00f;
  compute[(21)] = 0.000000e+00f;
  compute[(28)] = 0.000000e+00f;
  compute[(35)] = 0.000000e+00f;
  compute[(42)] = 0.000000e+00f;
  compute[(49)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  compute[(8)] = 0.000000e+00f;
  compute[(15)] = 0.000000e+00f;
  compute[(22)] = 0.000000e+00f;
  compute[(29)] = 0.000000e+00f;
  compute[(36)] = 0.000000e+00f;
  compute[(43)] = 0.000000e+00f;
  compute[(50)] = 0.000000e+00f;
  compute[(2)] = 0.000000e+00f;
  compute[(9)] = 0.000000e+00f;
  compute[(16)] = 0.000000e+00f;
  compute[(23)] = 0.000000e+00f;
  compute[(30)] = 0.000000e+00f;
  compute[(37)] = 0.000000e+00f;
  compute[(44)] = 0.000000e+00f;
  compute[(51)] = 0.000000e+00f;
  compute[(3)] = 0.000000e+00f;
  compute[(10)] = 0.000000e+00f;
  compute[(17)] = 0.000000e+00f;
  compute[(24)] = 0.000000e+00f;
  compute[(31)] = 0.000000e+00f;
  compute[(38)] = 0.000000e+00f;
  compute[(45)] = 0.000000e+00f;
  compute[(52)] = 0.000000e+00f;
  compute[(4)] = 0.000000e+00f;
  compute[(11)] = 0.000000e+00f;
  compute[(18)] = 0.000000e+00f;
  compute[(25)] = 0.000000e+00f;
  compute[(32)] = 0.000000e+00f;
  compute[(39)] = 0.000000e+00f;
  compute[(46)] = 0.000000e+00f;
  compute[(53)] = 0.000000e+00f;
  compute[(5)] = 0.000000e+00f;
  compute[(12)] = 0.000000e+00f;
  compute[(19)] = 0.000000e+00f;
  compute[(26)] = 0.000000e+00f;
  compute[(33)] = 0.000000e+00f;
  compute[(40)] = 0.000000e+00f;
  compute[(47)] = 0.000000e+00f;
  compute[(54)] = 0.000000e+00f;
  compute[(6)] = 0.000000e+00f;
  compute[(13)] = 0.000000e+00f;
  compute[(20)] = 0.000000e+00f;
  compute[(27)] = 0.000000e+00f;
  compute[(34)] = 0.000000e+00f;
  compute[(41)] = 0.000000e+00f;
  compute[(48)] = 0.000000e+00f;
  compute[(55)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 32; ++rc_outer) {
    for (int ry_outer = 0; ry_outer < 3; ++ry_outer) {
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)))] = ((((1 <= (((((int)vy) * 7) + (((int)threadIdx.x) >> 1)) + ry_outer)) && ((((((int)vy) * 7) + (((int)threadIdx.x) >> 1)) + ry_outer) < 15)) && (1 <= (((int)threadIdx.x) & 1))) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 15))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 1))] = ((((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 1) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 1) / 14)) + ry_outer) < 15)) && (1 <= (((((int)threadIdx.x) * 7) + 1) % 14))) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 14))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 2))] = ((((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 2) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 2) / 14)) + ry_outer) < 15)) && (1 <= (((((int)threadIdx.x) * 7) + 2) % 14))) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 13))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 3))] = ((((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 3) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 3) / 14)) + ry_outer) < 15)) && (1 <= (((((int)threadIdx.x) * 7) + 3) % 14))) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 12))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 4))] = ((((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 4) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 4) / 14)) + ry_outer) < 15)) && (1 <= (((((int)threadIdx.x) * 7) + 4) % 14))) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 11))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 5))] = ((((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 5) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 5) / 14)) + ry_outer) < 15)) && (1 <= (((((int)threadIdx.x) * 7) + 5) % 14))) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 10))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 6))] = ((((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 6) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 6) / 14)) + ry_outer) < 15)) && (1 <= (((((int)threadIdx.x) * 7) + 6) % 14))) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 9))] : 0.000000e+00f);
      if (((((int)threadIdx.z) * 8) + ((((int)threadIdx.x) * 5) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 512) {
          if (((int)threadIdx.x) < 13) {
            placeholder_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)))] = placeholder1[((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + (((((int)threadIdx.x) * 5) >> 3) * 2304)) + (rc_outer * 72)) + (((((int)threadIdx.x) * 5) & 7) * 9)) + (ry_outer * 3)))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 5) + 1) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 511) {
          if (((int)threadIdx.x) < 13) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) + 1))] = placeholder1[((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 5) + 1) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 5) + 1) & 7) * 9)) + (ry_outer * 3)))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 5) + 2) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 510) {
          if (((int)threadIdx.x) < 13) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) + 2))] = placeholder1[((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 5) + 2) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 5) + 2) & 7) * 9)) + (ry_outer * 3)))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 5) + 3) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 509) {
          if (((int)threadIdx.x) < 13) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) + 3))] = placeholder1[((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 5) + 3) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 5) + 3) & 7) * 9)) + (ry_outer * 3)))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 5) + 4) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 508) {
          if (((int)threadIdx.x) < 12) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) + 4))] = placeholder1[((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 5) + 4) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 5) + 4) & 7) * 9)) + (ry_outer * 3)))];
          }
        }
      }
      __syncthreads();
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)))] = (((1 <= (((((int)vy) * 7) + (((int)threadIdx.x) >> 1)) + ry_outer)) && ((((((int)vy) * 7) + (((int)threadIdx.x) >> 1)) + ry_outer) < 15)) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 14))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 1))] = (((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 1) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 1) / 14)) + ry_outer) < 15)) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 13))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 2))] = (((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 2) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 2) / 14)) + ry_outer) < 15)) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 12))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 3))] = (((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 3) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 3) / 14)) + ry_outer) < 15)) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 11))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 4))] = (((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 4) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 4) / 14)) + ry_outer) < 15)) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 10))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 5))] = (((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 5) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 5) / 14)) + ry_outer) < 15)) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 9))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 6))] = (((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 6) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 6) / 14)) + ry_outer) < 15)) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 8))] : 0.000000e+00f);
      if (((((int)threadIdx.z) * 8) + ((((int)threadIdx.x) * 5) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 512) {
          if (((int)threadIdx.x) < 13) {
            placeholder_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)))] = placeholder1[(((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + (((((int)threadIdx.x) * 5) >> 3) * 2304)) + (rc_outer * 72)) + (((((int)threadIdx.x) * 5) & 7) * 9)) + (ry_outer * 3)) + 1))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 5) + 1) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 511) {
          if (((int)threadIdx.x) < 13) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) + 1))] = placeholder1[(((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 5) + 1) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 5) + 1) & 7) * 9)) + (ry_outer * 3)) + 1))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 5) + 2) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 510) {
          if (((int)threadIdx.x) < 13) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) + 2))] = placeholder1[(((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 5) + 2) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 5) + 2) & 7) * 9)) + (ry_outer * 3)) + 1))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 5) + 3) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 509) {
          if (((int)threadIdx.x) < 13) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) + 3))] = placeholder1[(((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 5) + 3) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 5) + 3) & 7) * 9)) + (ry_outer * 3)) + 1))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 5) + 4) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 508) {
          if (((int)threadIdx.x) < 12) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) + 4))] = placeholder1[(((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 5) + 4) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 5) + 4) & 7) * 9)) + (ry_outer * 3)) + 1))];
          }
        }
      }
      __syncthreads();
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)))] = (((1 <= (((((int)vy) * 7) + (((int)threadIdx.x) >> 1)) + ry_outer)) && ((((((int)vy) * 7) + (((int)threadIdx.x) >> 1)) + ry_outer) < 15)) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 13))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 1))] = ((((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 1) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 1) / 14)) + ry_outer) < 15)) && ((((((int)threadIdx.x) * 7) + 1) % 14) < 13)) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 12))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 2))] = ((((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 2) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 2) / 14)) + ry_outer) < 15)) && ((((((int)threadIdx.x) * 7) + 2) % 14) < 13)) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 11))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 3))] = ((((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 3) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 3) / 14)) + ry_outer) < 15)) && ((((((int)threadIdx.x) * 7) + 3) % 14) < 13)) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 10))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 4))] = ((((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 4) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 4) / 14)) + ry_outer) < 15)) && ((((((int)threadIdx.x) * 7) + 4) % 14) < 13)) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 9))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 5))] = ((((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 5) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 5) / 14)) + ry_outer) < 15)) && ((((((int)threadIdx.x) * 7) + 5) % 14) < 13)) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 8))] : 0.000000e+00f);
      pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.x) * 7)) + 6))] = ((((1 <= (((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 6) / 14)) + ry_outer)) && ((((((int)vy) * 7) + (((((int)threadIdx.x) * 7) + 6) / 14)) + ry_outer) < 15)) && ((((((int)threadIdx.x) * 7) + 6) % 14) < 13)) ? placeholder[(((((((((((int)vz) >> 2) * 50176) + (rc_outer * 1568)) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + (ry_outer * 14)) + (((int)threadIdx.x) * 7)) - 7))] : 0.000000e+00f);
      if (((((int)threadIdx.z) * 8) + ((((int)threadIdx.x) * 5) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 512) {
          if (((int)threadIdx.x) < 13) {
            placeholder_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)))] = placeholder1[(((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + (((((int)threadIdx.x) * 5) >> 3) * 2304)) + (rc_outer * 72)) + (((((int)threadIdx.x) * 5) & 7) * 9)) + (ry_outer * 3)) + 2))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 5) + 1) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 511) {
          if (((int)threadIdx.x) < 13) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) + 1))] = placeholder1[(((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 5) + 1) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 5) + 1) & 7) * 9)) + (ry_outer * 3)) + 2))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 5) + 2) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 510) {
          if (((int)threadIdx.x) < 13) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) + 2))] = placeholder1[(((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 5) + 2) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 5) + 2) & 7) * 9)) + (ry_outer * 3)) + 2))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 5) + 3) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 509) {
          if (((int)threadIdx.x) < 13) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) + 3))] = placeholder1[(((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 5) + 3) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 5) + 3) & 7) * 9)) + (ry_outer * 3)) + 2))];
          }
        }
      }
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 5) + 4) >> 3)) < 64) {
        if (((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) < 508) {
          if (((int)threadIdx.x) < 12) {
            placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 5)) + 4))] = placeholder1[(((((((((((int)vz) & 3) * 147456) + (((int)threadIdx.z) * 18432)) + ((((((int)threadIdx.x) * 5) + 4) >> 3) * 2304)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 5) + 4) & 7) * 9)) + (ry_outer * 3)) + 2))];
          }
        }
      }
      __syncthreads();
      compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 28))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 42))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 56))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 70))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[((((int)threadIdx.z) * 8))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 64))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 128))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 192))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 256))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 320))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 384))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 84))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 448))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 98))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 112))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 126))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 140))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 154))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 168))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 1))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 65))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 129))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 193))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 257))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 321))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 385))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 182))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 449))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 196))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 210))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 224))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 238))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 252))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 266))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 2))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 66))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 130))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 194))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 258))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 322))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 386))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 280))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 450))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 294))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 308))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 322))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 336))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 350))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 364))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 3))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 67))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 131))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 195))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 259))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 323))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 387))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 378))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 451))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 392))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 406))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 420))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 434))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 448))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 462))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 4))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 68))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 132))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 196))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 260))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 324))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 388))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 476))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 452))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 490))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 504))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 518))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 532))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 546))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 560))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 5))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 69))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 133))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 197))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 261))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 325))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 389))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 574))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 453))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 588))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 602))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 616))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 630))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 644))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 658))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 6))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 70))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 134))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 198))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 262))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 326))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 390))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 672))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 454))]));
      compute[(0)] = (compute[(0)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(7)] = (compute[(7)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(14)] = (compute[(14)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(21)] = (compute[(21)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(28)] = (compute[(28)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(35)] = (compute[(35)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(42)] = (compute[(42)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(49)] = (compute[(49)] + (pad_temp_shared[((((int)threadIdx.x) + 686))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(1)] = (compute[(1)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(8)] = (compute[(8)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(15)] = (compute[(15)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(22)] = (compute[(22)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(29)] = (compute[(29)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(36)] = (compute[(36)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(43)] = (compute[(43)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(50)] = (compute[(50)] + (pad_temp_shared[((((int)threadIdx.x) + 700))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(9)] = (compute[(9)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(16)] = (compute[(16)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(23)] = (compute[(23)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(30)] = (compute[(30)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(37)] = (compute[(37)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(44)] = (compute[(44)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(51)] = (compute[(51)] + (pad_temp_shared[((((int)threadIdx.x) + 714))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(10)] = (compute[(10)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(17)] = (compute[(17)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(24)] = (compute[(24)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(31)] = (compute[(31)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(38)] = (compute[(38)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(45)] = (compute[(45)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(52)] = (compute[(52)] + (pad_temp_shared[((((int)threadIdx.x) + 728))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(4)] = (compute[(4)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(11)] = (compute[(11)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(18)] = (compute[(18)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(25)] = (compute[(25)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(32)] = (compute[(32)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(39)] = (compute[(39)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(46)] = (compute[(46)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(53)] = (compute[(53)] + (pad_temp_shared[((((int)threadIdx.x) + 742))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(5)] = (compute[(5)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(12)] = (compute[(12)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(19)] = (compute[(19)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(26)] = (compute[(26)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(33)] = (compute[(33)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(40)] = (compute[(40)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(47)] = (compute[(47)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(54)] = (compute[(54)] + (pad_temp_shared[((((int)threadIdx.x) + 756))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
      compute[(6)] = (compute[(6)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 7))]));
      compute[(13)] = (compute[(13)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 71))]));
      compute[(20)] = (compute[(20)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 135))]));
      compute[(27)] = (compute[(27)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 199))]));
      compute[(34)] = (compute[(34)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 263))]));
      compute[(41)] = (compute[(41)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 327))]));
      compute[(48)] = (compute[(48)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 391))]));
      compute[(55)] = (compute[(55)] + (pad_temp_shared[((((int)threadIdx.x) + 770))] * placeholder_shared[(((((int)threadIdx.z) * 8) + 455))]));
    }
  }
  T_add[(((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)))] = (compute[(0)] + placeholder2[(((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 1568))] = (compute[(7)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 1568))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 3136))] = (compute[(14)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 3136))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 4704))] = (compute[(21)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 4704))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 6272))] = (compute[(28)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 6272))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 7840))] = (compute[(35)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 7840))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 9408))] = (compute[(42)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 9408))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 10976))] = (compute[(49)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 10976))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 14))] = (compute[(1)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 14))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 1582))] = (compute[(8)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 1582))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 3150))] = (compute[(15)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 3150))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 4718))] = (compute[(22)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 4718))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 6286))] = (compute[(29)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 6286))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 7854))] = (compute[(36)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 7854))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 9422))] = (compute[(43)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 9422))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 10990))] = (compute[(50)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 10990))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 28))] = (compute[(2)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 28))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 1596))] = (compute[(9)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 1596))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 3164))] = (compute[(16)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 3164))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 4732))] = (compute[(23)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 4732))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 6300))] = (compute[(30)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 6300))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 7868))] = (compute[(37)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 7868))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 9436))] = (compute[(44)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 9436))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 11004))] = (compute[(51)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 11004))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 42))] = (compute[(3)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 42))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 1610))] = (compute[(10)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 1610))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 3178))] = (compute[(17)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 3178))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 4746))] = (compute[(24)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 4746))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 6314))] = (compute[(31)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 6314))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 7882))] = (compute[(38)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 7882))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 9450))] = (compute[(45)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 9450))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 11018))] = (compute[(52)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 11018))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 56))] = (compute[(4)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 56))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 1624))] = (compute[(11)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 1624))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 3192))] = (compute[(18)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 3192))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 4760))] = (compute[(25)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 4760))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 6328))] = (compute[(32)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 6328))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 7896))] = (compute[(39)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 7896))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 9464))] = (compute[(46)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 9464))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 11032))] = (compute[(53)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 11032))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 70))] = (compute[(5)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 70))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 1638))] = (compute[(12)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 1638))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 3206))] = (compute[(19)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 3206))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 4774))] = (compute[(26)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 4774))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 6342))] = (compute[(33)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 6342))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 7910))] = (compute[(40)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 7910))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 9478))] = (compute[(47)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 9478))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 11046))] = (compute[(54)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 11046))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 84))] = (compute[(6)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 84))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 1652))] = (compute[(13)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 1652))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 3220))] = (compute[(20)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 3220))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 4788))] = (compute[(27)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 4788))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 6356))] = (compute[(34)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 6356))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 7924))] = (compute[(41)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 7924))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 9492))] = (compute[(48)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 9492))]);
  T_add[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 11060))] = (compute[(55)] + placeholder2[((((((((int)vz) * 12544) + (((int)threadIdx.z) * 196)) + (((int)vy) * 98)) + ((int)threadIdx.x)) + 11060))]);
  }
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_7_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
    int vx=blockIdx.x;
  int vy=blockIdx.y;
  int vz=blockIdx.z;
  int offset=0;

  if((blocknum[0]*blocknum[1]*blocknum[2])>blocksize[0])
  {
    offset=vx;
    while(offset<(blocknum[0]*blocknum[1]*blocknum[2]))
    {
    vz=(offset)/(blocknum[0]*blocknum[1]);
    vy= (offset-(vz*blocknum[0]*blocknum[1]))/blocknum[0];
    vx=offset - (vz*blocknum[0]*blocknum[1])-vy*blocknum[0];
    float compute[16];
  __shared__ float pad_temp_shared[1155];
  __shared__ float placeholder_shared[96];
  for (int yy_init = 0; yy_init < 2; ++yy_init) {
    compute[(yy_init)] = 0.000000e+00f;
    compute[((yy_init + 4))] = 0.000000e+00f;
    compute[((yy_init + 8))] = 0.000000e+00f;
    compute[((yy_init + 12))] = 0.000000e+00f;
    compute[((yy_init + 2))] = 0.000000e+00f;
    compute[((yy_init + 6))] = 0.000000e+00f;
    compute[((yy_init + 10))] = 0.000000e+00f;
    compute[((yy_init + 14))] = 0.000000e+00f;
  }
  for (int ry_outer = 0; ry_outer < 7; ++ry_outer) {
    for (int rx_outer = 0; rx_outer < 7; ++rx_outer) {
      __syncthreads();
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 6; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
        if ((((((int)threadIdx.z) * 145) + (((int)threadIdx.x) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 1155) {
          if (((((int)threadIdx.x) * 6) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 145) {
            pad_temp_shared[((((((int)threadIdx.z) * 145) + (((int)threadIdx.x) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((3 <= (((((int)vy) * 8) + (((((((int)threadIdx.z) * 145) + (((int)threadIdx.x) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 385) / 55)) + ry_outer)) && ((((((int)vy) * 8) + (((((((int)threadIdx.z) * 145) + (((int)threadIdx.x) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 385) / 55)) + ry_outer) < 227)) && (3 <= (((((int)vx) * 56) + rx_outer) + ((((((int)threadIdx.z) * 145) + (((int)threadIdx.x) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 55)))) && ((((((int)vx) * 56) + rx_outer) + ((((((int)threadIdx.z) * 145) + (((int)threadIdx.x) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 55)) < 227)) ? placeholder[(((((((((((((int)vz) >> 1) * 150528) + (((((((int)threadIdx.z) * 145) + (((int)threadIdx.x) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 385) * 50176)) + (((int)vy) * 1792)) + ((((((((int)threadIdx.z) * 145) + (((int)threadIdx.x) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 385) / 55) * 224)) + (ry_outer * 224)) + (((int)vx) * 56)) + rx_outer) + ((((((int)threadIdx.z) * 145) + (((int)threadIdx.x) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 55)) - 675))] : 0.000000e+00f);
          }
        }
      }
      if (((((int)threadIdx.z) * 4) + (((int)threadIdx.x) / 3)) < 32) {
        if (((((int)threadIdx.z) * 12) + ((int)threadIdx.x)) < 96) {
          if (((int)threadIdx.x) < 12) {
            placeholder_shared[(((((int)threadIdx.z) * 12) + ((int)threadIdx.x)))] = placeholder1[(((((((((int)vz) & 1) * 4704) + (((int)threadIdx.z) * 588)) + (((int)threadIdx.x) * 49)) + (ry_outer * 7)) + rx_outer))];
          }
        }
      }
      __syncthreads();
      for (int rc_inner = 0; rc_inner < 3; ++rc_inner) {
        for (int yy = 0; yy < 2; ++yy) {
          compute[(yy)] = (compute[(yy)] + (pad_temp_shared[((((rc_inner * 385) + (yy * 110)) + (((int)threadIdx.x) * 2)))] * placeholder_shared[(((((int)threadIdx.z) * 3) + rc_inner))]));
          compute[((yy + 4))] = (compute[((yy + 4))] + (pad_temp_shared[((((rc_inner * 385) + (yy * 110)) + (((int)threadIdx.x) * 2)))] * placeholder_shared[((((((int)threadIdx.z) * 3) + rc_inner) + 24))]));
          compute[((yy + 8))] = (compute[((yy + 8))] + (pad_temp_shared[((((rc_inner * 385) + (yy * 110)) + (((int)threadIdx.x) * 2)))] * placeholder_shared[((((((int)threadIdx.z) * 3) + rc_inner) + 48))]));
          compute[((yy + 12))] = (compute[((yy + 12))] + (pad_temp_shared[((((rc_inner * 385) + (yy * 110)) + (((int)threadIdx.x) * 2)))] * placeholder_shared[((((((int)threadIdx.z) * 3) + rc_inner) + 72))]));
          compute[((yy + 2))] = (compute[((yy + 2))] + (pad_temp_shared[(((((rc_inner * 385) + (yy * 110)) + (((int)threadIdx.x) * 2)) + 220))] * placeholder_shared[(((((int)threadIdx.z) * 3) + rc_inner))]));
          compute[((yy + 6))] = (compute[((yy + 6))] + (pad_temp_shared[(((((rc_inner * 385) + (yy * 110)) + (((int)threadIdx.x) * 2)) + 220))] * placeholder_shared[((((((int)threadIdx.z) * 3) + rc_inner) + 24))]));
          compute[((yy + 10))] = (compute[((yy + 10))] + (pad_temp_shared[(((((rc_inner * 385) + (yy * 110)) + (((int)threadIdx.x) * 2)) + 220))] * placeholder_shared[((((((int)threadIdx.z) * 3) + rc_inner) + 48))]));
          compute[((yy + 14))] = (compute[((yy + 14))] + (pad_temp_shared[(((((rc_inner * 385) + (yy * 110)) + (((int)threadIdx.x) * 2)) + 220))] * placeholder_shared[((((((int)threadIdx.z) * 3) + rc_inner) + 72))]));
        }
      }
    }
  }
  for (int ax2_inner_inner_inner = 0; ax2_inner_inner_inner < 2; ++ax2_inner_inner_inner) {
    T_relu[(((((((((int)vz) * 401408) + (((int)threadIdx.z) * 12544)) + (((int)vy) * 448)) + (ax2_inner_inner_inner * 112)) + (((int)vx) * 28)) + ((int)threadIdx.x)))] = max((compute[(ax2_inner_inner_inner)] + placeholder2[((((((int)vz) & 1) * 32) + ((int)threadIdx.z)))]), 0.000000e+00f);
    T_relu[((((((((((int)vz) * 401408) + (((int)threadIdx.z) * 12544)) + (((int)vy) * 448)) + (ax2_inner_inner_inner * 112)) + (((int)vx) * 28)) + ((int)threadIdx.x)) + 100352))] = max((compute[((ax2_inner_inner_inner + 4))] + placeholder2[(((((((int)vz) & 1) * 32) + ((int)threadIdx.z)) + 8))]), 0.000000e+00f);
    T_relu[((((((((((int)vz) * 401408) + (((int)threadIdx.z) * 12544)) + (((int)vy) * 448)) + (ax2_inner_inner_inner * 112)) + (((int)vx) * 28)) + ((int)threadIdx.x)) + 200704))] = max((compute[((ax2_inner_inner_inner + 8))] + placeholder2[(((((((int)vz) & 1) * 32) + ((int)threadIdx.z)) + 16))]), 0.000000e+00f);
    T_relu[((((((((((int)vz) * 401408) + (((int)threadIdx.z) * 12544)) + (((int)vy) * 448)) + (ax2_inner_inner_inner * 112)) + (((int)vx) * 28)) + ((int)threadIdx.x)) + 301056))] = max((compute[((ax2_inner_inner_inner + 12))] + placeholder2[(((((((int)vz) & 1) * 32) + ((int)threadIdx.z)) + 24))]), 0.000000e+00f);
    T_relu[((((((((((int)vz) * 401408) + (((int)threadIdx.z) * 12544)) + (((int)vy) * 448)) + (ax2_inner_inner_inner * 112)) + (((int)vx) * 28)) + ((int)threadIdx.x)) + 224))] = max((compute[((ax2_inner_inner_inner + 2))] + placeholder2[((((((int)vz) & 1) * 32) + ((int)threadIdx.z)))]), 0.000000e+00f);
    T_relu[((((((((((int)vz) * 401408) + (((int)threadIdx.z) * 12544)) + (((int)vy) * 448)) + (ax2_inner_inner_inner * 112)) + (((int)vx) * 28)) + ((int)threadIdx.x)) + 100576))] = max((compute[((ax2_inner_inner_inner + 6))] + placeholder2[(((((((int)vz) & 1) * 32) + ((int)threadIdx.z)) + 8))]), 0.000000e+00f);
    T_relu[((((((((((int)vz) * 401408) + (((int)threadIdx.z) * 12544)) + (((int)vy) * 448)) + (ax2_inner_inner_inner * 112)) + (((int)vx) * 28)) + ((int)threadIdx.x)) + 200928))] = max((compute[((ax2_inner_inner_inner + 10))] + placeholder2[(((((((int)vz) & 1) * 32) + ((int)threadIdx.z)) + 16))]), 0.000000e+00f);
    T_relu[((((((((((int)vz) * 401408) + (((int)threadIdx.z) * 12544)) + (((int)vy) * 448)) + (ax2_inner_inner_inner * 112)) + (((int)vx) * 28)) + ((int)threadIdx.x)) + 301280))] = max((compute[((ax2_inner_inner_inner + 14))] + placeholder2[(((((((int)vz) & 1) * 32) + ((int)threadIdx.z)) + 24))]), 0.000000e+00f);
  }
    offset+=blocksize[0];
    }
  }
  else
  {
    float compute[16];
  __shared__ float pad_temp_shared[1155];
  __shared__ float placeholder_shared[96];
  for (int yy_init = 0; yy_init < 2; ++yy_init) {
    compute[(yy_init)] = 0.000000e+00f;
    compute[((yy_init + 4))] = 0.000000e+00f;
    compute[((yy_init + 8))] = 0.000000e+00f;
    compute[((yy_init + 12))] = 0.000000e+00f;
    compute[((yy_init + 2))] = 0.000000e+00f;
    compute[((yy_init + 6))] = 0.000000e+00f;
    compute[((yy_init + 10))] = 0.000000e+00f;
    compute[((yy_init + 14))] = 0.000000e+00f;
  }
  for (int ry_outer = 0; ry_outer < 7; ++ry_outer) {
    for (int rx_outer = 0; rx_outer < 7; ++rx_outer) {
      __syncthreads();
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 6; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
        if ((((((int)threadIdx.z) * 145) + (((int)threadIdx.x) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 1155) {
          if (((((int)threadIdx.x) * 6) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 145) {
            pad_temp_shared[((((((int)threadIdx.z) * 145) + (((int)threadIdx.x) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((3 <= (((((int)vy) * 8) + (((((((int)threadIdx.z) * 145) + (((int)threadIdx.x) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 385) / 55)) + ry_outer)) && ((((((int)vy) * 8) + (((((((int)threadIdx.z) * 145) + (((int)threadIdx.x) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 385) / 55)) + ry_outer) < 227)) && (3 <= (((((int)vx) * 56) + rx_outer) + ((((((int)threadIdx.z) * 145) + (((int)threadIdx.x) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 55)))) && ((((((int)vx) * 56) + rx_outer) + ((((((int)threadIdx.z) * 145) + (((int)threadIdx.x) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 55)) < 227)) ? placeholder[(((((((((((((int)vz) >> 1) * 150528) + (((((((int)threadIdx.z) * 145) + (((int)threadIdx.x) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 385) * 50176)) + (((int)vy) * 1792)) + ((((((((int)threadIdx.z) * 145) + (((int)threadIdx.x) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 385) / 55) * 224)) + (ry_outer * 224)) + (((int)vx) * 56)) + rx_outer) + ((((((int)threadIdx.z) * 145) + (((int)threadIdx.x) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 55)) - 675))] : 0.000000e+00f);
          }
        }
      }
      if (((((int)threadIdx.z) * 4) + (((int)threadIdx.x) / 3)) < 32) {
        if (((((int)threadIdx.z) * 12) + ((int)threadIdx.x)) < 96) {
          if (((int)threadIdx.x) < 12) {
            placeholder_shared[(((((int)threadIdx.z) * 12) + ((int)threadIdx.x)))] = placeholder1[(((((((((int)vz) & 1) * 4704) + (((int)threadIdx.z) * 588)) + (((int)threadIdx.x) * 49)) + (ry_outer * 7)) + rx_outer))];
          }
        }
      }
      __syncthreads();
      for (int rc_inner = 0; rc_inner < 3; ++rc_inner) {
        for (int yy = 0; yy < 2; ++yy) {
          compute[(yy)] = (compute[(yy)] + (pad_temp_shared[((((rc_inner * 385) + (yy * 110)) + (((int)threadIdx.x) * 2)))] * placeholder_shared[(((((int)threadIdx.z) * 3) + rc_inner))]));
          compute[((yy + 4))] = (compute[((yy + 4))] + (pad_temp_shared[((((rc_inner * 385) + (yy * 110)) + (((int)threadIdx.x) * 2)))] * placeholder_shared[((((((int)threadIdx.z) * 3) + rc_inner) + 24))]));
          compute[((yy + 8))] = (compute[((yy + 8))] + (pad_temp_shared[((((rc_inner * 385) + (yy * 110)) + (((int)threadIdx.x) * 2)))] * placeholder_shared[((((((int)threadIdx.z) * 3) + rc_inner) + 48))]));
          compute[((yy + 12))] = (compute[((yy + 12))] + (pad_temp_shared[((((rc_inner * 385) + (yy * 110)) + (((int)threadIdx.x) * 2)))] * placeholder_shared[((((((int)threadIdx.z) * 3) + rc_inner) + 72))]));
          compute[((yy + 2))] = (compute[((yy + 2))] + (pad_temp_shared[(((((rc_inner * 385) + (yy * 110)) + (((int)threadIdx.x) * 2)) + 220))] * placeholder_shared[(((((int)threadIdx.z) * 3) + rc_inner))]));
          compute[((yy + 6))] = (compute[((yy + 6))] + (pad_temp_shared[(((((rc_inner * 385) + (yy * 110)) + (((int)threadIdx.x) * 2)) + 220))] * placeholder_shared[((((((int)threadIdx.z) * 3) + rc_inner) + 24))]));
          compute[((yy + 10))] = (compute[((yy + 10))] + (pad_temp_shared[(((((rc_inner * 385) + (yy * 110)) + (((int)threadIdx.x) * 2)) + 220))] * placeholder_shared[((((((int)threadIdx.z) * 3) + rc_inner) + 48))]));
          compute[((yy + 14))] = (compute[((yy + 14))] + (pad_temp_shared[(((((rc_inner * 385) + (yy * 110)) + (((int)threadIdx.x) * 2)) + 220))] * placeholder_shared[((((((int)threadIdx.z) * 3) + rc_inner) + 72))]));
        }
      }
    }
  }
  for (int ax2_inner_inner_inner = 0; ax2_inner_inner_inner < 2; ++ax2_inner_inner_inner) {
    T_relu[(((((((((int)vz) * 401408) + (((int)threadIdx.z) * 12544)) + (((int)vy) * 448)) + (ax2_inner_inner_inner * 112)) + (((int)vx) * 28)) + ((int)threadIdx.x)))] = max((compute[(ax2_inner_inner_inner)] + placeholder2[((((((int)vz) & 1) * 32) + ((int)threadIdx.z)))]), 0.000000e+00f);
    T_relu[((((((((((int)vz) * 401408) + (((int)threadIdx.z) * 12544)) + (((int)vy) * 448)) + (ax2_inner_inner_inner * 112)) + (((int)vx) * 28)) + ((int)threadIdx.x)) + 100352))] = max((compute[((ax2_inner_inner_inner + 4))] + placeholder2[(((((((int)vz) & 1) * 32) + ((int)threadIdx.z)) + 8))]), 0.000000e+00f);
    T_relu[((((((((((int)vz) * 401408) + (((int)threadIdx.z) * 12544)) + (((int)vy) * 448)) + (ax2_inner_inner_inner * 112)) + (((int)vx) * 28)) + ((int)threadIdx.x)) + 200704))] = max((compute[((ax2_inner_inner_inner + 8))] + placeholder2[(((((((int)vz) & 1) * 32) + ((int)threadIdx.z)) + 16))]), 0.000000e+00f);
    T_relu[((((((((((int)vz) * 401408) + (((int)threadIdx.z) * 12544)) + (((int)vy) * 448)) + (ax2_inner_inner_inner * 112)) + (((int)vx) * 28)) + ((int)threadIdx.x)) + 301056))] = max((compute[((ax2_inner_inner_inner + 12))] + placeholder2[(((((((int)vz) & 1) * 32) + ((int)threadIdx.z)) + 24))]), 0.000000e+00f);
    T_relu[((((((((((int)vz) * 401408) + (((int)threadIdx.z) * 12544)) + (((int)vy) * 448)) + (ax2_inner_inner_inner * 112)) + (((int)vx) * 28)) + ((int)threadIdx.x)) + 224))] = max((compute[((ax2_inner_inner_inner + 2))] + placeholder2[((((((int)vz) & 1) * 32) + ((int)threadIdx.z)))]), 0.000000e+00f);
    T_relu[((((((((((int)vz) * 401408) + (((int)threadIdx.z) * 12544)) + (((int)vy) * 448)) + (ax2_inner_inner_inner * 112)) + (((int)vx) * 28)) + ((int)threadIdx.x)) + 100576))] = max((compute[((ax2_inner_inner_inner + 6))] + placeholder2[(((((((int)vz) & 1) * 32) + ((int)threadIdx.z)) + 8))]), 0.000000e+00f);
    T_relu[((((((((((int)vz) * 401408) + (((int)threadIdx.z) * 12544)) + (((int)vy) * 448)) + (ax2_inner_inner_inner * 112)) + (((int)vx) * 28)) + ((int)threadIdx.x)) + 200928))] = max((compute[((ax2_inner_inner_inner + 10))] + placeholder2[(((((((int)vz) & 1) * 32) + ((int)threadIdx.z)) + 16))]), 0.000000e+00f);
    T_relu[((((((((((int)vz) * 401408) + (((int)threadIdx.z) * 12544)) + (((int)vy) * 448)) + (ax2_inner_inner_inner * 112)) + (((int)vx) * 28)) + ((int)threadIdx.x)) + 301280))] = max((compute[((ax2_inner_inner_inner + 14))] + placeholder2[(((((((int)vz) & 1) * 32) + ((int)threadIdx.z)) + 24))]), 0.000000e+00f);
  }
  }
}
extern "C" __global__ void fused_nn_max_pool2d_add_nn_relu_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ T_relu, float* __restrict__ placeholder1) {
    int vx=blockIdx.x;
  int vy=blockIdx.y;
  int vz=blockIdx.z;
  int offset=0;

  if((blocknum[0]*blocknum[1]*blocknum[2])>blocksize[0])
  {
    offset=vx;
    while(offset<(blocknum[0]*blocknum[1]*blocknum[2]))
    {
    vz=(offset)/(blocknum[0]*blocknum[1]);
    vy= (offset-(vz*blocknum[0]*blocknum[1]))/blocknum[0];
    vx=offset - (vz*blocknum[0]*blocknum[1])-vy*blocknum[0];
    float tensor[1];
  tensor[(0)] = -3.402823e+38f;
  for (int dh = 0; dh < 3; ++dh) {
    for (int dw = 0; dw < 3; ++dw) {
      tensor[(0)] = max(tensor[(0)], (((1 <= ((((((((int)vx) * 1024) + ((int)threadIdx.x)) % 3136) / 56) * 2) + dh)) && (1 <= (((((((int)vx) * 1024) + ((int)threadIdx.x)) % 56) * 2) + dw))) ? placeholder[(((((((((((int)vx) * 1024) + ((int)threadIdx.x)) / 56) * 224) + (dh * 112)) + ((((((int)vx) * 1024) + ((int)threadIdx.x)) % 56) * 2)) + dw) - 113))] : -3.402823e+38f));
    }
  }
  T_relu[(((((int)vx) * 1024) + ((int)threadIdx.x)))] = max((tensor[(0)] + placeholder1[(((((((int)vx) * 1024) + ((int)threadIdx.x)) % 200704) / 3136))]), 0.000000e+00f);
    offset+=blocksize[0];
    }
  }
  else
  {
    float tensor[1];
  tensor[(0)] = -3.402823e+38f;
  for (int dh = 0; dh < 3; ++dh) {
    for (int dw = 0; dw < 3; ++dw) {
      tensor[(0)] = max(tensor[(0)], (((1 <= ((((((((int)vx) * 1024) + ((int)threadIdx.x)) % 3136) / 56) * 2) + dh)) && (1 <= (((((((int)vx) * 1024) + ((int)threadIdx.x)) % 56) * 2) + dw))) ? placeholder[(((((((((((int)vx) * 1024) + ((int)threadIdx.x)) / 56) * 224) + (dh * 112)) + ((((((int)vx) * 1024) + ((int)threadIdx.x)) % 56) * 2)) + dw) - 113))] : -3.402823e+38f));
    }
  }
  T_relu[(((((int)vx) * 1024) + ((int)threadIdx.x)))] = max((tensor[(0)] + placeholder1[(((((((int)vx) * 1024) + ((int)threadIdx.x)) % 200704) / 3136))]), 0.000000e+00f);
  }
}

extern "C" __global__ void fused_nn_conv2d_add_add_nn_relu_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {

}
extern "C" __global__ void fused_add_nn_relu_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ T_relu, float* __restrict__ placeholder, float* __restrict__ placeholder1) {

}
extern "C" __global__ void fused_nn_softmax_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ T_softmax_norm) {

}
extern "C" __global__ void fused_add_nn_relu_2_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ T_relu, float* __restrict__ placeholder, float* __restrict__ placeholder1) {

}
extern "C" __global__ void fused_nn_conv2d_1_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {

}

extern "C" __global__ void fused_nn_conv2d_add_add_nn_relu_1_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {

}
extern "C" __global__ void fused_nn_conv2d_add_nn_relu_2_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {

}
extern "C" __global__ void fused_nn_dense_add_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {

}
extern "C" __global__ void fused_add_10_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ T_add, float* __restrict__ placeholder, float* __restrict__ placeholder1) {

}
extern "C" __global__ void fused_nn_conv2d_add_nn_relu_1_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {

}
extern "C" __global__ void fused_nn_conv2d_add_nn_relu_4_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {

}
extern "C" __global__ void fused_nn_conv2d_add_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {

}
extern "C" __global__ void fused_nn_conv2d_add_add_nn_relu_2_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {

}
extern "C" __global__ void fused_nn_conv2d_add_2_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {

}
extern "C" __global__ void fused_nn_conv2d_3_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {

}
extern "C" __global__ void fused_nn_conv2d_add_nn_relu_6_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {

}
extern "C" __global__ void fused_nn_conv2d_add_3_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {

}
extern "C" __global__ void fused_nn_conv2d_add_nn_relu_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {

}
extern "C" __global__ void fused_nn_conv2d_2_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {

}
extern "C" __global__ void fused_nn_conv2d_add_multiply_add_nn_relu_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3, float* __restrict__ placeholder4) {

}
extern "C" __global__ void fused_nn_global_avg_pool2d_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ tensor) {

}
extern "C" __global__ void fused_nn_batch_flatten_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ tensor, float* __restrict__ placeholder) {

}
extern "C" __global__ void fused_nn_conv2d_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {

}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_5_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {

}
extern "C" __global__ void fused_add_nn_relu_3_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ T_relu, float* __restrict__ placeholder, float* __restrict__ placeholder1) {

}
extern "C" __global__ void fused_nn_conv2d_add_nn_relu_3_kernel0(int* flag,int* blocknum,int* blocksize,float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {

}

























