#include <stdio.h>
#include<hip/hip_runtime.h>


#define BLOCK_NUM 80   //块数量
#define THREAD_NUM 256 // 每个块中的线程数
#define R_SIZE BLOCK_NUM * THREAD_NUM
#define M_SIZE R_SIZE * R_SIZE

#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)

#define GPU_RETURN_STATUS(cmd) \
{ \
    hipError_t result = cmd; \
    if (result != hipSuccess) { \
        std::cout << #cmd " error, return code:" << result << " | " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); \
    } \
}

// These are the inline versions for all of the SDK helper functions
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    const char *errorStr = NULL;
    errorStr = hipGetErrorString(err);
    fprintf(stderr,
            "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, "
            "line %i.\n",
            err, errorStr, file, line);
    exit(EXIT_FAILURE);
  }
}


__device__ uint get_smid(void) {

    uint ret;
  
    asm("mov.u32 %0, %smid;" : "=r"(ret) );
  
    return ret;
  
}

__global__ void mat_mul(int *mat1, int *mat2, int *result, int *sm) {
    sm[blockIdx.x] = get_smid();
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    // 每个线程计算一行
    const int row = bid * THREAD_NUM + tid;
    for (int c = 0; c < R_SIZE; c++) {
        for (int n = 0; n < R_SIZE; n++) {
            result[row*R_SIZE+c] += mat1[row*R_SIZE+n] * mat2[n*R_SIZE+c];
        }
    }
}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        printf("args num error! argc:%d", argc);
        exit(1);
    }
    int gpu_no = atoi(argv[1]);
    checkCudaErrors(hipSetDevice(gpu_no));
    int nstreams = 2;
        // allocate and initialize an array of stream handles
    hipStream_t *streams =
        (hipStream_t *)malloc(nstreams * sizeof(hipStream_t));
    for (int i = 0; i < nstreams; i++) {
        checkCudaErrors(hipStreamCreate(&(streams[i])));
    }

    int *mat1[nstreams], *mat2[nstreams], *result[nstreams], *sm[nstreams];
    int *g_mat1[nstreams], *g_mat2[nstreams], *g_mat_result[nstreams], *g_sm[nstreams];


    for (int times = 0; times < nstreams; ++times) {
        
        // 用一位数组表示二维矩阵
        mat1[times] = (int*) malloc(M_SIZE * sizeof(int));
        mat2[times] = (int*) malloc(M_SIZE * sizeof(int));
        result[times] = (int*) malloc(M_SIZE * sizeof(int));
        sm[times] = (int*) malloc(BLOCK_NUM * sizeof(int));

        // initialize
        for (int i = 0; i < M_SIZE; i++) {
            mat1[times][i] = rand()/1000000;
            mat2[times][i] = rand()/1000000;
            result[times][i] = 0;
        }
        for (int i = 0; i < BLOCK_NUM; ++i) {
            sm[times][i] = -1;
        }

        hipMalloc((void **)&(g_mat1[times]), sizeof(int) * M_SIZE);
        hipMalloc((void **)&(g_mat2[times]), sizeof(int) * M_SIZE);
        hipMalloc((void **)&(g_mat_result[times]), sizeof(int) * M_SIZE);
        hipMalloc((void **)&(g_sm[times]), sizeof(int) * BLOCK_NUM);

        hipMemcpy(g_mat1[times], mat1[times], sizeof(int) * M_SIZE, hipMemcpyHostToDevice);
        hipMemcpy(g_mat2[times], mat2[times], sizeof(int) * M_SIZE, hipMemcpyHostToDevice);
        hipMemcpy(g_sm[times], sm[times], sizeof(int) * BLOCK_NUM, hipMemcpyHostToDevice);
    }

    hipEvent_t start_event[nstreams], stop_event[nstreams], all_start_event, all_end_event;

    for (int times = 0; times < nstreams; ++times) {
        checkCudaErrors(hipEventCreate(&start_event[times]));
        checkCudaErrors(hipEventCreate(&stop_event[times]));
    }
    checkCudaErrors(hipEventCreate(&all_start_event));
    checkCudaErrors(hipEventCreate(&all_end_event));
    checkCudaErrors(hipEventRecord(all_start_event, 0));
    float elapsed_time[nstreams];

    for (int i = 0; i < nstreams; ++i) {
        hipStream_t stream = streams[i];
        hipEventRecord(start_event[i], stream);
        mat_mul<<<BLOCK_NUM, THREAD_NUM>>>(g_mat1[i], g_mat2[i], g_mat_result[i], g_sm[i]);
        checkCudaErrors(hipEventRecord(stop_event[i], stream));
    }

    checkCudaErrors(hipEventRecord(all_end_event, 0));
    for (int i = 0; i < nstreams; ++i) {
        checkCudaErrors(hipEventSynchronize(stop_event[i])); // Waits until the completion of all work currently captured in event
    }
    checkCudaErrors(hipEventSynchronize(all_end_event));

    for (int ii = 0; ii < nstreams; ++ii) {
        hipMemcpy(result[ii], g_mat_result[ii], sizeof(int) * M_SIZE, hipMemcpyDeviceToHost);
        hipMemcpy(sm[ii], g_sm[ii], sizeof(int) * BLOCK_NUM, hipMemcpyDeviceToHost);
        printf("-----stream %d-----\n", ii);
        for (int i = 0; i < BLOCK_NUM; ++i) {
            printf("block %d -- sm %d\n", i, sm[ii][i]);
        }
        printf("\n");
    }

    for (int i = 0; i < nstreams; ++i) {
        checkCudaErrors(hipEventElapsedTime(&elapsed_time[i], start_event[i], stop_event[i]));
        printf("Stream%d Measured time for sample = %.3fms\n", i, elapsed_time[i]);
    }
    float elapsed;
    checkCudaErrors(hipEventElapsedTime(&elapsed, all_start_event, all_end_event));
    printf("Total GPU Measured time for sample = %.3fms\n", elapsed); 
    return 0;
}