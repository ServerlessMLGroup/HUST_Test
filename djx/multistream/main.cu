#include <stdio.h>
#include<hip/hip_runtime.h>


#define BLOCK_NUM 80   //块数量
#define THREAD_NUM 256 // 每个块中的线程数
#define R_SIZE BLOCK_NUM * THREAD_NUM
#define M_SIZE R_SIZE * R_SIZE

#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)

#define GPU_RETURN_STATUS(cmd) \
{ \
    hipError_t result = cmd; \
    if (result != hipSuccess) { \
        std::cout << #cmd " error, return code:" << result << " | " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); \
    } \
}

// These are the inline versions for all of the SDK helper functions
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    const char *errorStr = NULL;
    errorStr = hipGetErrorString(err);
    fprintf(stderr,
            "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, "
            "line %i.\n",
            err, errorStr, file, line);
    exit(EXIT_FAILURE);
  }
}


__device__ uint get_smid(void) {

    uint ret;
  
    asm("mov.u32 %0, %smid;" : "=r"(ret) );
  
    return ret;
  
}

__global__ void mat_mul(int *mat1, int *mat2, int *result, int *sm) {
    sm[blockIdx.x] = get_smid();
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    // 每个线程计算一行
    const int row = bid * THREAD_NUM + tid;
    for (int c = 0; c < R_SIZE; c++) {
        for (int n = 0; n < R_SIZE; n++) {
            result[row*R_SIZE+c] += mat1[row*R_SIZE+n] * mat2[n*R_SIZE+c];
        }
    }
}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        printf("args num error! argc:%d", argc);
        exit(1);
    }
    int gpu_no = atoi(argv[1]);
    checkCudaErrors(hipSetDevice(gpu_no));
    int *mat1, *mat2, *result, *sm;
    int *g_mat1, *g_mat2, *g_mat_result, *g_sm;
    
    // 用一位数组表示二维矩阵
    mat1 = (int*) malloc(M_SIZE * sizeof(int));
    mat2 = (int*) malloc(M_SIZE * sizeof(int));
    result = (int*) malloc(M_SIZE * sizeof(int));
    sm = (int*) malloc(BLOCK_NUM * sizeof(int));

    // initialize
    for (int i = 0; i < M_SIZE; i++) {
        mat1[i] = rand()/1000000;
        mat2[i] = rand()/1000000;
        result[i] = 0;
    }
    for (int i = 0; i < BLOCK_NUM; ++i) {
        sm[i] = -1;
    }

    hipMalloc((void **)&g_mat1, sizeof(int) * M_SIZE);
    hipMalloc((void **)&g_mat2, sizeof(int) * M_SIZE);
    hipMalloc((void **)&g_mat_result, sizeof(int) * M_SIZE);
    hipMalloc((void **)&g_sm, sizeof(int) * BLOCK_NUM);

    hipMemcpy(g_mat1, mat1, sizeof(int) * M_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(g_mat2, mat2, sizeof(int) * M_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(g_sm, sm, sizeof(int) * BLOCK_NUM, hipMemcpyHostToDevice);

    mat_mul<<<BLOCK_NUM, THREAD_NUM>>>(g_mat1, g_mat2, g_mat_result, g_sm);

    hipMemcpy(result, g_mat_result, sizeof(int) * M_SIZE, hipMemcpyDeviceToHost);
    hipMemcpy(sm, g_sm, sizeof(int) * BLOCK_NUM, hipMemcpyDeviceToHost);
    for (int i = 0; i < BLOCK_NUM; ++i) {
        printf("block %d -- sm %d\n", i, sm[i]);
    }
    return 0;
}