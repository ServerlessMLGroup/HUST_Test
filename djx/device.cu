#include <iostream>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <thread>
#include <random>
#include <ctime>
using namespace std;

enum class Unit{
    Byte, KB, MB, GB, TB, PB, EB
};


double convert(double size, Unit unit)
{
    double result = size;
    switch (unit)
    {
    case Unit::EB:
        result /= 1024;     // flow through
    case Unit::PB:
        result /= 1024;     // flow through
    case Unit::TB:
        result /= 1024;     // flow through
    case Unit::GB:
        result /= 1024;     // flow through
    case Unit::MB:
        result /= 1024;     // flow through
    case Unit::KB:
        result /= 1024;     // flow through
    case Unit::Byte:
        result /= 1;
    default:
        break;
    }
    return result;
}

void getMem() {
    size_t free, total;
    hipMemGetInfo(&free, &total);
    printf("Free mem = %.4f MB, Total = %.4f MB \n", convert(free, Unit::MB), convert(total, Unit::MB));
}

void getMembycu() {
    size_t free, total;
    int err = hipMemGetInfo(&free, &total);
    if (err) {
        cout<<"getMembycu error:"<<err<<endl;
    }
    else {
        printf("Free mem = %.4f MB, Total = %.4f MB \n", convert(free, Unit::MB), convert(total, Unit::MB));
    }
}

void getLimit() {
    size_t value;
    int err = hipDeviceGetLimit(&value, hipLimitStackSize);
    if(err) {
        printf("[getLimit]:[hipLimitStackSize]:%d", err);
        exit(1);
    }
    //printf("getLimit:stack_size = %.4f MB\n", convert(value, Unit::MB));

    err = hipDeviceGetLimit(&value, hipLimitPrintfFifoSize);
    if(err) {
        printf("[getLimit]:[hipLimitPrintfFifoSize]:%d", err);
        exit(1);
    }
    //printf("getLimit:printf_fifo_size = %.4f MB\n", convert(value, Unit::MB));

    err = hipDeviceGetLimit(&value, hipLimitMallocHeapSize);
    if(err) {
        printf("[getLimit]:[hipLimitMallocHeapSize]:%d", err);
        exit(1);
    }
    //printf("getLimit:malloc_heap_size = %.4f MB\n", convert(value, Unit::MB));
}


int main(int argc，char** argv)
{
    if (argc < 2)
        printf("args num error! argc:%d", argc);
    hipSetDevice(argv[1]);
    while(1) {
        getMem();
        std::this_thread::sleep_for(std::chrono:: milliseconds (50));
    }

}