#include <iostream>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <thread>
#include <random>
#include <ctime>
using namespace std;

enum class Unit{
    Byte, KB, MB, GB, TB, PB, EB
};


double convert(double size, Unit unit)
{
    double result = size;
    switch (unit)
    {
    case Unit::EB:
        result /= 1024;     // flow through
    case Unit::PB:
        result /= 1024;     // flow through
    case Unit::TB:
        result /= 1024;     // flow through
    case Unit::GB:
        result /= 1024;     // flow through
    case Unit::MB:
        result /= 1024;     // flow through
    case Unit::KB:
        result /= 1024;     // flow through
    case Unit::Byte:
        result /= 1;
    default:
        break;
    }
    return result;
}

void getMem() {
    size_t free, total;
    int err=hipMemGetInfo(&free, &total);
    if(err){
       cout<<"hipMemGetInfo error:"<<err<<endl;
       return;
    }
    printf("Free mem = %.4f MB, Total = %.4f MB \n", convert(free, Unit::MB), convert(total, Unit::MB));
}

int main(int argc，char** argv) {
    if (argc < 2) {
        printf("args num error! argc:%d", argc);
    }
    hipSetDevice(argv[1]);
    while(1) {
        getMem();
        std::this_thread::sleep_for(std::chrono:: milliseconds (50));
    }
}